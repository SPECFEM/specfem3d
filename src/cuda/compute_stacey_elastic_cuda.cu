#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 1
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and CNRS / INRIA / University of Pau
 ! (c) Princeton University / California Institute of Technology and CNRS / INRIA / University of Pau
 !                             July 2012
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_stacey_elastic_kernel(realw* veloc,
                                              realw* accel,
                                              int* abs_boundary_ispec,
                                              int* abs_boundary_ijk,
                                              realw* abs_boundary_normal,
                                              realw* abs_boundary_jacobian2Dw,
                                              int* ibool,
                                              realw* rho_vp,
                                              realw* rho_vs,
                                              int* ispec_is_inner,
                                              int* ispec_is_elastic,
                                              int phase_is_inner,
                                              int SIMULATION_TYPE,
                                              int SAVE_FORWARD,
                                              int num_abs_boundary_faces,
                                              realw* b_accel,
                                              realw* b_absorb_field) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int i,j,k,iglob,ispec;
  realw vx,vy,vz,vn;
  realw nx,ny,nz;
  realw rho_vp_temp,rho_vs_temp;
  realw tx,ty,tz;
  realw jacobianw;

  // don't compute points outside NGLLSQUARE==NGLL2==25
  // way 2: no further check needed since blocksize = 25
  if( iface < num_abs_boundary_faces){

  //if(igll < NGLL2 && iface < num_abs_boundary_faces) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    if(ispec_is_inner[ispec] == phase_is_inner && ispec_is_elastic[ispec] ) {

      i = abs_boundary_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)]-1;
      j = abs_boundary_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)]-1;
      k = abs_boundary_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)]-1;
      iglob = ibool[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

      // gets associated velocity

      vx = veloc[iglob*3+0];
      vy = veloc[iglob*3+1];
      vz = veloc[iglob*3+2];

      // gets associated normal
      nx = abs_boundary_normal[INDEX3(NDIM,NGLL2,0,igll,iface)];
      ny = abs_boundary_normal[INDEX3(NDIM,NGLL2,1,igll,iface)];
      nz = abs_boundary_normal[INDEX3(NDIM,NGLL2,2,igll,iface)];

      // // velocity component in normal direction (normal points out of element)
      vn = vx*nx + vy*ny + vz*nz;

      rho_vp_temp = rho_vp[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];
      rho_vs_temp = rho_vs[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];

      tx = rho_vp_temp*vn*nx + rho_vs_temp*(vx-vn*nx);
      ty = rho_vp_temp*vn*ny + rho_vs_temp*(vy-vn*ny);
      tz = rho_vp_temp*vn*nz + rho_vs_temp*(vz-vn*nz);

      jacobianw = abs_boundary_jacobian2Dw[INDEX2(NGLL2,igll,iface)];

      atomicAdd(&accel[iglob*3],-tx*jacobianw);
      atomicAdd(&accel[iglob*3+1],-ty*jacobianw);
      atomicAdd(&accel[iglob*3+2],-tz*jacobianw);

      if(SIMULATION_TYPE == 3) {
        atomicAdd(&b_accel[iglob*3  ],-b_absorb_field[INDEX3(NDIM,NGLL2,0,igll,iface)]);
        atomicAdd(&b_accel[iglob*3+1],-b_absorb_field[INDEX3(NDIM,NGLL2,1,igll,iface)]);
        atomicAdd(&b_accel[iglob*3+2],-b_absorb_field[INDEX3(NDIM,NGLL2,2,igll,iface)]);
      }
      else if(SAVE_FORWARD && SIMULATION_TYPE == 1) {
        b_absorb_field[INDEX3(NDIM,NGLL2,0,igll,iface)] = tx*jacobianw;
        b_absorb_field[INDEX3(NDIM,NGLL2,1,igll,iface)] = ty*jacobianw;
        b_absorb_field[INDEX3(NDIM,NGLL2,2,igll,iface)] = tz*jacobianw;
      } // SIMULATION_TYPE
    }
  } // num_abs_boundary_faces

}

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(compute_stacey_elastic_cuda,
              COMPUTE_STACEY_ELASTIC_CUDA)(long* Mesh_pointer_f,
                                           int* phase_is_innerf,
                                           int* SIMULATION_TYPEf,
                                           int* SAVE_FORWARDf,
                                           realw* h_b_absorb_field) {

TRACE("compute_stacey_elastic_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  // check
  if( mp->d_num_abs_boundary_faces == 0 ) return;

  int phase_is_inner    = *phase_is_innerf;
  int SIMULATION_TYPE   = *SIMULATION_TYPEf;
  int SAVE_FORWARD      = *SAVE_FORWARDf;

  // way 1
  // > NGLLSQUARE==NGLL2==25, but we handle this inside kernel
  //int blocksize = 32;

  // way 2: seems sligthly faster
  // > NGLLSQUARE==NGLL2==25, no further check inside kernel
  int blocksize = NGLL2;

  int num_blocks_x = mp->d_num_abs_boundary_faces;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  if(SIMULATION_TYPE == 3 && mp->d_num_abs_boundary_faces > 0) {
    // The read is done in fortran
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_field,h_b_absorb_field,
                                       mp->d_b_reclen_field,hipMemcpyHostToDevice),7700);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("between cudamemcpy and compute_stacey_elastic_kernel");
#endif

  compute_stacey_elastic_kernel<<<grid,threads>>>(mp->d_veloc,
                                                  mp->d_accel,
                                                  mp->d_abs_boundary_ispec,
                                                  mp->d_abs_boundary_ijk,
                                                  mp->d_abs_boundary_normal,
                                                  mp->d_abs_boundary_jacobian2Dw,
                                                  mp->d_ibool,
                                                  mp->d_rho_vp,
                                                  mp->d_rho_vs,
                                                  mp->d_ispec_is_inner,
                                                  mp->d_ispec_is_elastic,
                                                  phase_is_inner,
                                                  SIMULATION_TYPE,SAVE_FORWARD,
                                                  mp->d_num_abs_boundary_faces,
                                                  mp->d_b_accel,
                                                  mp->d_b_absorb_field);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_stacey_elastic_kernel");
#endif

  // ! adjoint simulations: stores absorbed wavefield part
  // if (SIMULATION_TYPE == 1 .and. SAVE_FORWARD .and. num_abs_boundary_faces > 0 ) &
  //   write(IOABS,rec=it) b_reclen_field,b_absorb_field,b_reclen_field

  if(SIMULATION_TYPE == 1 && SAVE_FORWARD && mp->d_num_abs_boundary_faces > 0 ) {
    print_CUDA_error_if_any(hipMemcpy(h_b_absorb_field,mp->d_b_absorb_field,
                                       mp->d_b_reclen_field,hipMemcpyDeviceToHost),7701);
    // The write is done in fortran
    // write_abs_(&fid,(char*)b_absorb_field,&b_reclen_field,&it);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after compute_stacey_elastic after cudamemcpy");
#endif
}

