#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  3 . 0
 !               ---------------------------------------
 !
 !     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
 !                        Princeton University, USA
 !                and CNRS / University of Marseille, France
 !                 (there are currently many more authors!)
 ! (c) Princeton University and CNRS / University of Marseille, July 2012
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_stacey_acoustic_kernel(realw* potential_dot_acoustic,
                                               realw* potential_dot_dot_acoustic,
                                               int* abs_boundary_ispec,
                                               int* abs_boundary_ijk,
                                               realw* abs_boundary_jacobian2Dw,
                                               int* d_ibool,
                                               realw* rhostore,
                                               realw* kappastore,
                                               int* ispec_is_acoustic,
                                               int SIMULATION_TYPE,
                                               int SAVE_FORWARD,
                                               int num_abs_boundary_faces,
                                               realw* b_potential_dot_acoustic,
                                               realw* b_potential_dot_dot_acoustic,
                                               realw* b_absorb_potential,
                                               int gravity) {

  int igll = threadIdx.x;
  int iface = blockIdx.x + gridDim.x*blockIdx.y;

  int i,j,k,iglob,ispec;
  realw rhol,kappal,cpl;
  realw jacobianw;
  realw vel;

  // don't compute points outside NGLLSQUARE==NGLL2==25
  // way 2: no further check needed since blocksize = 25
  if (iface < num_abs_boundary_faces){

  //  if (igll<NGLL2 && iface < num_abs_boundary_faces) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    if (ispec_is_acoustic[ispec]) {

      i = abs_boundary_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)]-1;
      j = abs_boundary_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)]-1;
      k = abs_boundary_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)]-1;

      iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

      // determines bulk sound speed
      rhol = rhostore[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];

      kappal = kappastore[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];

      cpl = sqrt( kappal / rhol );

      // velocity
      if (gravity ){
        // daniel: TODO - check gravity and stacey condition here...
        // uses a potential definition of: s = grad(chi)
        vel = potential_dot_acoustic[iglob] / rhol ;
      }else{
        // uses a potential definition of: s = 1/rho grad(chi)
        vel = potential_dot_acoustic[iglob] / rhol;
      }

      // gets associated, weighted jacobian
      jacobianw = abs_boundary_jacobian2Dw[INDEX2(NGLL2,igll,iface)];

      // Sommerfeld condition
      atomicAdd(&potential_dot_dot_acoustic[iglob],-vel*jacobianw/cpl);

      // adjoint simulations
      if (SIMULATION_TYPE == 3){
        // Sommerfeld condition
        atomicAdd(&b_potential_dot_dot_acoustic[iglob],-b_absorb_potential[INDEX2(NGLL2,igll,iface)]);
      }else if (SIMULATION_TYPE == 1 && SAVE_FORWARD ){
        // saves boundary values
        b_absorb_potential[INDEX2(NGLL2,igll,iface)] = vel*jacobianw/cpl;
      }
    }
//  }
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_stacey_acoustic_cuda,
              COMPUTE_STACEY_ACOUSTIC_CUDA)(long* Mesh_pointer,
                                            int* iphasef,
                                            realw* h_b_absorb_potential) {
TRACE("compute_stacey_acoustic_cuda");
  //double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks if anything to do
  if (mp->d_num_abs_boundary_faces == 0) return;

  int iphase          = *iphasef;

  // only add these contributions in first pass
  if (iphase != 1) return;

  // way 1: Elapsed time: 4.385948e-03
  // > NGLLSQUARE==NGLL2==25, but we handle this inside kernel
  //  int blocksize = 32;

  // way 2: Elapsed time: 4.379034e-03
  // > NGLLSQUARE==NGLL2==25, no further check inside kernel
  int blocksize = NGLL2;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->d_num_abs_boundary_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  //  adjoint simulations: reads in absorbing boundary
  if (mp->simulation_type == 3){
    // copies array to GPU
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_potential,h_b_absorb_potential,
                                       mp->d_b_reclen_potential,hipMemcpyHostToDevice),7700);
  }

  compute_stacey_acoustic_kernel<<<grid,threads>>>(mp->d_potential_dot_acoustic,
                                                   mp->d_potential_dot_dot_acoustic,
                                                   mp->d_abs_boundary_ispec,
                                                   mp->d_abs_boundary_ijk,
                                                   mp->d_abs_boundary_jacobian2Dw,
                                                   mp->d_ibool,
                                                   mp->d_rhostore,
                                                   mp->d_kappastore,
                                                   mp->d_ispec_is_acoustic,
                                                   mp->simulation_type,
                                                   mp->save_forward,
                                                   mp->d_num_abs_boundary_faces,
                                                   mp->d_b_potential_dot_acoustic,
                                                   mp->d_b_potential_dot_dot_acoustic,
                                                   mp->d_b_absorb_potential,
                                                   mp->gravity);

  //  adjoint simulations: stores absorbed wavefield part
  if (mp->simulation_type == 1 && mp->save_forward ){
    // (hipMemcpy implicitly synchronizes all other cuda operations)
    // copies array to CPU
    print_CUDA_error_if_any(hipMemcpy(h_b_absorb_potential,mp->d_b_absorb_potential,
                                       mp->d_b_reclen_potential,hipMemcpyDeviceToHost),7701);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_stacey_acoustic_kernel");
#endif
}

