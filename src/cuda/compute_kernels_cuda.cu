#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 1
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and University of Pau / CNRS / INRIA
 ! (c) Princeton University / California Institute of Technology and University of Pau / CNRS / INRIA
 !                            April 2011
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/types.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// ELASTIC SIMULATIONS

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_kernels_cudakernel(int* ispec_is_elastic,
                                           int* ibool,
                                           realw* accel,
                                           realw* b_displ,
                                           realw* epsilondev_xx,
                                           realw* epsilondev_yy,
                                           realw* epsilondev_xy,
                                           realw* epsilondev_xz,
                                           realw* epsilondev_yz,
                                           realw* b_epsilondev_xx,
                                           realw* b_epsilondev_yy,
                                           realw* b_epsilondev_xy,
                                           realw* b_epsilondev_xz,
                                           realw* b_epsilondev_yz,
                                           realw* rho_kl,
                                           realw deltat,
                                           realw* mu_kl,
                                           realw* kappa_kl,
                                           realw* epsilon_trace_over_3,
                                           realw* b_epsilon_trace_over_3,
                                           int NSPEC_AB) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;

  // handles case when there is 1 extra block (due to rectangular grid)
  if(ispec < NSPEC_AB) {

    // elastic elements only
    if( ispec_is_elastic[ispec] ) {

      int ijk = threadIdx.x;
      int ijk_ispec = ijk + NGLL3*ispec;
      int iglob = ibool[ijk_ispec] - 1 ;

      // isotropic kernels:
      // density kernel
      rho_kl[ijk_ispec] += deltat * (accel[3*iglob]*b_displ[3*iglob]+
                                     accel[3*iglob+1]*b_displ[3*iglob+1]+
                                     accel[3*iglob+2]*b_displ[3*iglob+2]);


      // shear modulus kernel
      mu_kl[ijk_ispec] += deltat * (epsilondev_xx[ijk_ispec]*b_epsilondev_xx[ijk_ispec]+
                                    epsilondev_yy[ijk_ispec]*b_epsilondev_yy[ijk_ispec]+
                                    (epsilondev_xx[ijk_ispec]+epsilondev_yy[ijk_ispec])*
                                    (b_epsilondev_xx[ijk_ispec]+b_epsilondev_yy[ijk_ispec])+
                                    2*(epsilondev_xy[ijk_ispec]*b_epsilondev_xy[ijk_ispec]+
                                       epsilondev_xz[ijk_ispec]*b_epsilondev_xz[ijk_ispec]+
                                       epsilondev_yz[ijk_ispec]*b_epsilondev_yz[ijk_ispec]));

      // bulk modulus kernel
      kappa_kl[ijk_ispec] += deltat*(9*epsilon_trace_over_3[ijk_ispec]*
                                     b_epsilon_trace_over_3[ijk_ispec]);

    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_kernels_elastic_cuda,
              COMPUTE_KERNELS_ELASTIC_CUDA)(long* Mesh_pointer,
                                            realw* deltat_f) {
TRACE("compute_kernels_elastic_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  int blocksize = NGLL3; // NGLLX*NGLLY*NGLLZ
  realw deltat = *deltat_f;

  int num_blocks_x = mp->NSPEC_AB;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  compute_kernels_cudakernel<<<grid,threads>>>(mp->d_ispec_is_elastic,mp->d_ibool,
                                               mp->d_accel, mp->d_b_displ,
                                               mp->d_epsilondev_xx,
                                               mp->d_epsilondev_yy,
                                               mp->d_epsilondev_xy,
                                               mp->d_epsilondev_xz,
                                               mp->d_epsilondev_yz,
                                               mp->d_b_epsilondev_xx,
                                               mp->d_b_epsilondev_yy,
                                               mp->d_b_epsilondev_xy,
                                               mp->d_b_epsilondev_xz,
                                               mp->d_b_epsilondev_yz,
                                               mp->d_rho_kl,
                                               deltat,
                                               mp->d_mu_kl,
                                               mp->d_kappa_kl,
                                               mp->d_epsilon_trace_over_3,
                                               mp->d_b_epsilon_trace_over_3,
                                               mp->NSPEC_AB);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_elastic_cuda");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// NOISE SIMULATIONS

/* ----------------------------------------------------------------------------------------------- */


__global__ void compute_kernels_strength_noise_cuda_kernel(realw* displ,
                                                           int* free_surface_ispec,
                                                           int* free_surface_ijk,
                                                           int* ibool,
                                                           realw* noise_surface_movie,
                                                           realw* normal_x_noise,
                                                           realw* normal_y_noise,
                                                           realw* normal_z_noise,
                                                           realw* Sigma_kl,
                                                           realw deltat,
                                                           int num_free_surface_faces) {
  int iface = blockIdx.x + blockIdx.y*gridDim.x;

  if(iface < num_free_surface_faces) {

    int ispec = free_surface_ispec[iface]-1;
    int igll = threadIdx.x;
    int ipoin = igll + NGLL2*iface;
    int i = free_surface_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)] - 1 ;
    int j = free_surface_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)] - 1;
    int k = free_surface_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)] - 1;

    int iglob = ibool[INDEX4(5,5,5,i,j,k,ispec)] - 1 ;

    realw eta = ( noise_surface_movie[INDEX3(NDIM,NGLL2,0,igll,iface)]*normal_x_noise[ipoin]+
                 noise_surface_movie[INDEX3(NDIM,NGLL2,1,igll,iface)]*normal_y_noise[ipoin]+
                 noise_surface_movie[INDEX3(NDIM,NGLL2,2,igll,iface)]*normal_z_noise[ipoin]);

    Sigma_kl[INDEX4(5,5,5,i,j,k,ispec)] += deltat*eta*(normal_x_noise[ipoin]*displ[3*iglob]+
                                                       normal_y_noise[ipoin]*displ[1+3*iglob]+
                                                       normal_z_noise[ipoin]*displ[2+3*iglob]);
  }

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_kernels_strgth_noise_cu,
              COMPUTE_KERNELS_STRGTH_NOISE_CU)(long* Mesh_pointer,
                                                    realw* h_noise_surface_movie,
                                                    realw* deltat) {

TRACE("compute_kernels_strgth_noise_cu");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  hipMemcpy(mp->d_noise_surface_movie,h_noise_surface_movie,
             3*NGLL2*(mp->num_free_surface_faces)*sizeof(realw),hipMemcpyHostToDevice);


  int num_blocks_x = mp->num_free_surface_faces;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(NGLL2,1,1);

  compute_kernels_strength_noise_cuda_kernel<<<grid,threads>>>(mp->d_displ,
                                                               mp->d_free_surface_ispec,
                                                               mp->d_free_surface_ijk,
                                                               mp->d_ibool,
                                                               mp->d_noise_surface_movie,
                                                               mp->d_normal_x_noise,
                                                               mp->d_normal_y_noise,
                                                               mp->d_normal_z_noise,
                                                               mp->d_Sigma_kl,*deltat,
                                                               mp->num_free_surface_faces);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_strength_noise_cuda_kernel");
#endif
}



/* ----------------------------------------------------------------------------------------------- */

// ACOUSTIC SIMULATIONS

/* ----------------------------------------------------------------------------------------------- */


__device__ void compute_gradient_kernel(int ijk,
                                        int ispec,
                                        realw* scalar_field,
                                        realw* vector_field_element,
                                        realw* hprime_xx,
                                        realw* hprime_yy,
                                        realw* hprime_zz,
                                        realw* d_xix,
                                        realw* d_xiy,
                                        realw* d_xiz,
                                        realw* d_etax,
                                        realw* d_etay,
                                        realw* d_etaz,
                                        realw* d_gammax,
                                        realw* d_gammay,
                                        realw* d_gammaz,
                                        realw rhol,
                                        int gravity) {

  realw temp1l,temp2l,temp3l;
  realw hp1,hp2,hp3;
  realw xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl;
  realw rho_invl;
  int l,offset,offset1,offset2,offset3;

  //const int NGLLX = 5;
  const int NGLL3_ALIGN = NGLL3_PADDED;

  int K = (ijk/NGLL2);
  int J = ((ijk-K*NGLL2)/NGLLX);
  int I = (ijk-K*NGLL2-J*NGLLX);

  // derivative along x
  temp1l = 0.f;
  for( l=0; l<NGLLX;l++){
    hp1 = hprime_xx[l*NGLLX+I];
    offset1 = K*NGLL2+J*NGLLX+l;
    temp1l += scalar_field[offset1]*hp1;
  }

  // derivative along y
  temp2l = 0.f;
  for( l=0; l<NGLLX;l++){
    hp2 = hprime_yy[l*NGLLX+J];
    offset2 = K*NGLL2+l*NGLLX+I;
    temp2l += scalar_field[offset2]*hp2;
  }

  // derivative along z
  temp3l = 0.f;
  for( l=0; l<NGLLX;l++){
    hp3 = hprime_zz[l*NGLLX+K];
    offset3 = l*NGLL2+J*NGLLX+I;
    temp3l += scalar_field[offset3]*hp3;

  }

  offset = ispec*NGLL3_ALIGN + ijk;

  xixl = d_xix[offset];
  xiyl = d_xiy[offset];
  xizl = d_xiz[offset];
  etaxl = d_etax[offset];
  etayl = d_etay[offset];
  etazl = d_etaz[offset];
  gammaxl = d_gammax[offset];
  gammayl = d_gammay[offset];
  gammazl = d_gammaz[offset];

  if( gravity ){
    // daniel: TODO - check gravity case here
    rho_invl = 1.0f / rhol;
  }else{
    rho_invl = 1.0f / rhol;
  }
  // derivatives of acoustic scalar potential field on GLL points
  vector_field_element[0] = (temp1l*xixl + temp2l*etaxl + temp3l*gammaxl) * rho_invl;
  vector_field_element[1] = (temp1l*xiyl + temp2l*etayl + temp3l*gammayl) * rho_invl;
  vector_field_element[2] = (temp1l*xizl + temp2l*etazl + temp3l*gammazl) * rho_invl;

}

/* ----------------------------------------------------------------------------------------------- */


__global__ void compute_kernels_acoustic_kernel(int* ispec_is_acoustic,
                                                int* ibool,
                                                realw* rhostore,
                                                realw* kappastore,
                                                realw* hprime_xx,
                                                realw* hprime_yy,
                                                realw* hprime_zz,
                                                realw* d_xix,
                                                realw* d_xiy,
                                                realw* d_xiz,
                                                realw* d_etax,
                                                realw* d_etay,
                                                realw* d_etaz,
                                                realw* d_gammax,
                                                realw* d_gammay,
                                                realw* d_gammaz,
                                                realw* potential_dot_dot_acoustic,
                                                realw* b_potential_acoustic,
                                                realw* b_potential_dot_dot_acoustic,
                                                realw* rho_ac_kl,
                                                realw* kappa_ac_kl,
                                                realw deltat,
                                                int NSPEC_AB,
                                                int gravity) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;

  // handles case when there is 1 extra block (due to rectangular grid)
  if( ispec < NSPEC_AB ){

    // acoustic elements only
    if( ispec_is_acoustic[ispec] ) {

      int ijk = threadIdx.x;

      // local and global indices
      int ijk_ispec = ijk + NGLL3*ispec;
      int ijk_ispec_padded = ijk + NGLL3_PADDED*ispec;
      int iglob = ibool[ijk_ispec] - 1;

      realw accel_elm[3];
      realw b_displ_elm[3];
      realw rhol,kappal;

      // shared memory between all threads within this block
      __shared__ realw scalar_field_displ[NGLL3];
      __shared__ realw scalar_field_accel[NGLL3];

      // copy field values
      scalar_field_displ[ijk] = b_potential_acoustic[iglob];
      scalar_field_accel[ijk] = potential_dot_dot_acoustic[iglob];
      __syncthreads();

      // gets material parameter
      rhol = rhostore[ijk_ispec_padded];

      // displacement vector from backward field
      compute_gradient_kernel(ijk,ispec,scalar_field_displ,b_displ_elm,
                              hprime_xx,hprime_yy,hprime_zz,
                              d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                              rhol,gravity);

      // acceleration vector
      compute_gradient_kernel(ijk,ispec,scalar_field_accel,accel_elm,
                              hprime_xx,hprime_yy,hprime_zz,
                              d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                              rhol,gravity);

      // density kernel
      rho_ac_kl[ijk_ispec] -= deltat * rhol * (accel_elm[0]*b_displ_elm[0] +
                                               accel_elm[1]*b_displ_elm[1] +
                                               accel_elm[2]*b_displ_elm[2]);

      // bulk modulus kernel
      kappal = kappastore[ijk_ispec];
      kappa_ac_kl[ijk_ispec] -= deltat / kappal * potential_dot_dot_acoustic[iglob]
                                                * b_potential_dot_dot_acoustic[iglob];
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(compute_kernels_acoustic_cuda,
              COMPUTE_KERNELS_ACOUSTIC_CUDA)(
                                             long* Mesh_pointer,
                                             realw* deltat_f) {

TRACE("compute_kernels_acoustic_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  int blocksize = NGLL3; // NGLLX*NGLLY*NGLLZ
  realw deltat = *deltat_f;

  int num_blocks_x = mp->NSPEC_AB;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  compute_kernels_acoustic_kernel<<<grid,threads>>>(mp->d_ispec_is_acoustic,
                                                    mp->d_ibool,
                                                    mp->d_rhostore,
                                                    mp->d_kappastore,
                                                    mp->d_hprime_xx,
                                                    mp->d_hprime_yy,
                                                    mp->d_hprime_zz,
                                                    mp->d_xix,
                                                    mp->d_xiy,
                                                    mp->d_xiz,
                                                    mp->d_etax,
                                                    mp->d_etay,
                                                    mp->d_etaz,
                                                    mp->d_gammax,
                                                    mp->d_gammay,
                                                    mp->d_gammaz,
                                                    mp->d_potential_dot_dot_acoustic,
                                                    mp->d_b_potential_acoustic,
                                                    mp->d_b_potential_dot_dot_acoustic,
                                                    mp->d_rho_ac_kl,
                                                    mp->d_kappa_ac_kl,
                                                    deltat,
                                                    mp->NSPEC_AB,
                                                    mp->gravity);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_acoustic_kernel");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// preconditioner (approximate Hessian kernel)

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_kernels_hess_el_cudakernel(int* ispec_is_elastic,
                                                   int* ibool,
                                                   realw* accel,
                                                   realw* b_accel,
                                                   realw* hess_kl,
                                                   realw deltat,
                                                   int NSPEC_AB) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;

  // handles case when there is 1 extra block (due to rectangular grid)
  if(ispec < NSPEC_AB) {

    // elastic elements only
    if( ispec_is_elastic[ispec] ) {

      int ijk = threadIdx.x;
      int ijk_ispec = ijk + NGLL3*ispec;
      int iglob = ibool[ijk_ispec] - 1 ;

      // approximate hessian
      hess_kl[ijk_ispec] += deltat * (accel[3*iglob]*b_accel[3*iglob]+
                                      accel[3*iglob+1]*b_accel[3*iglob+1]+
                                      accel[3*iglob+2]*b_accel[3*iglob+2]);
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_kernels_hess_ac_cudakernel(int* ispec_is_acoustic,
                                                   int* ibool,
                                                   realw* potential_dot_dot_acoustic,
                                                   realw* b_potential_dot_dot_acoustic,
                                                   realw* rhostore,
                                                   realw* hprime_xx,
                                                   realw* hprime_yy,
                                                   realw* hprime_zz,
                                                   realw* d_xix,
                                                   realw* d_xiy,
                                                   realw* d_xiz,
                                                   realw* d_etax,
                                                   realw* d_etay,
                                                   realw* d_etaz,
                                                   realw* d_gammax,
                                                   realw* d_gammay,
                                                   realw* d_gammaz,
                                                   realw* hess_kl,
                                                   realw deltat,
                                                   int NSPEC_AB,
                                                   int gravity) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;

  // handles case when there is 1 extra block (due to rectangular grid)
  if(ispec < NSPEC_AB) {

    // acoustic elements only
    if( ispec_is_acoustic[ispec] ){

      // local and global indices
      int ijk = threadIdx.x;
      int ijk_ispec = ijk + NGLL3*ispec;
      int iglob = ibool[ijk_ispec] - 1 ;

      int ijk_ispec_padded = ijk + NGLL3_PADDED*ispec;

      realw accel_elm[3];
      realw b_accel_elm[3];
      realw rhol;

      // shared memory between all threads within this block
      __shared__ realw scalar_field_accel[NGLL3];
      __shared__ realw scalar_field_b_accel[NGLL3];

      // copy field values
      scalar_field_accel[ijk] = potential_dot_dot_acoustic[iglob];
      scalar_field_b_accel[ijk] = b_potential_dot_dot_acoustic[iglob];
      __syncthreads();

      // gets material parameter
      rhol = rhostore[ijk_ispec_padded];

      // acceleration vector
      compute_gradient_kernel(ijk,ispec,
                              scalar_field_accel,accel_elm,
                              hprime_xx,hprime_yy,hprime_zz,
                              d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                              rhol,gravity);

      // acceleration vector from backward field
      compute_gradient_kernel(ijk,ispec,
                              scalar_field_b_accel,b_accel_elm,
                              hprime_xx,hprime_yy,hprime_zz,
                              d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                              rhol,gravity);
      // approximates hessian
      hess_kl[ijk_ispec] += deltat * (accel_elm[0]*b_accel_elm[0] +
                                      accel_elm[1]*b_accel_elm[1] +
                                      accel_elm[2]*b_accel_elm[2]);

    } // ispec_is_acoustic

  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_kernels_hess_cuda,
              COMPUTE_KERNELS_HESS_CUDA)(long* Mesh_pointer,
                                         realw* deltat_f,
                                         int* ELASTIC_SIMULATION,
                                         int* ACOUSTIC_SIMULATION) {
  TRACE("compute_kernels_hess_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  int blocksize = NGLL3; // NGLLX*NGLLY*NGLLZ
  realw deltat = *deltat_f;

  int num_blocks_x = mp->NSPEC_AB;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  if( *ELASTIC_SIMULATION ) {
    compute_kernels_hess_el_cudakernel<<<grid,threads>>>(mp->d_ispec_is_elastic,
                                                         mp->d_ibool,
                                                         mp->d_accel,
                                                         mp->d_b_accel,
                                                         mp->d_hess_el_kl,
                                                         deltat,
                                                         mp->NSPEC_AB);
  }

  if( *ACOUSTIC_SIMULATION ) {
    compute_kernels_hess_ac_cudakernel<<<grid,threads>>>(mp->d_ispec_is_acoustic,
                                                         mp->d_ibool,
                                                         mp->d_potential_dot_dot_acoustic,
                                                         mp->d_b_potential_dot_dot_acoustic,
                                                         mp->d_rhostore,
                                                         mp->d_hprime_xx,
                                                         mp->d_hprime_yy,
                                                         mp->d_hprime_zz,
                                                         mp->d_xix,
                                                         mp->d_xiy,
                                                         mp->d_xiz,
                                                         mp->d_etax,
                                                         mp->d_etay,
                                                         mp->d_etaz,
                                                         mp->d_gammax,
                                                         mp->d_gammay,
                                                         mp->d_gammaz,
                                                         mp->d_hess_ac_kl,
                                                         deltat,
                                                         mp->NSPEC_AB,
                                                         mp->gravity);
  }


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_hess_cuda");
#endif
}

