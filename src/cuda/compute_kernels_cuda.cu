#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  3 . 0
 !               ---------------------------------------
 !
 !     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
 !                        Princeton University, USA
 !                and CNRS / University of Marseille, France
 !                 (there are currently many more authors!)
 ! (c) Princeton University and CNRS / University of Marseille, July 2012
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

// ELASTIC SIMULATIONS

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_kernels_ani_cudakernel(int* ispec_is_elastic,
                                           int* d_ibool,
                                           realw* accel,
                                           realw* b_displ,
                                           realw* epsilondev_xx,realw* epsilondev_yy,realw* epsilondev_xy,
                                           realw* epsilondev_xz,realw* epsilondev_yz,
                                           realw* b_epsilondev_xx,realw* b_epsilondev_yy,realw* b_epsilondev_xy,
                                           realw* b_epsilondev_xz,realw* b_epsilondev_yz,
                                           realw* rho_kl,
                                           realw deltat,
                                           realw* cijkl_kl,
                                           realw* epsilon_trace_over_3,
                                           realw* b_epsilon_trace_over_3,
                                           int NSPEC_AB) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;
  int ijk = threadIdx.x;
  int ijk_ispec = ijk + NGLL3*ispec;
  int ijk21_ispec = ijk + 21*NGLL3*ispec;

  realw prod[21];
  realw eps[6];
  realw b_eps[6];
  realw epsdev[6];
  realw b_epsdev[6];
  realw eps_trace_over_3,b_eps_trace_over_3;
  int i,j;

  // handles case when there is 1 extra block (due to rectangular grid)
  if (ispec < NSPEC_AB) {

    // elastic elements only
    if (ispec_is_elastic[ispec]) {
      int iglob = d_ibool[ijk + NGLL3_PADDED*ispec] - 1;

      // anisotropic kernels:
      // density kernel
      rho_kl[ijk_ispec] += deltat * (accel[3*iglob]*b_displ[3*iglob]+
                                     accel[3*iglob+1]*b_displ[3*iglob+1]+
                                     accel[3*iglob+2]*b_displ[3*iglob+2]);


      // anisotropic kernel
      epsdev[0] = epsilondev_xx[ijk_ispec];
      epsdev[1] = epsilondev_yy[ijk_ispec];
      epsdev[2] = epsilondev_xy[ijk_ispec];
      epsdev[3] = epsilondev_xz[ijk_ispec];
      epsdev[4] = epsilondev_yz[ijk_ispec];

      b_epsdev[0] = b_epsilondev_xx[ijk_ispec];
      b_epsdev[1] = b_epsilondev_yy[ijk_ispec];
      b_epsdev[2] = b_epsilondev_xy[ijk_ispec];
      b_epsdev[3] = b_epsilondev_xz[ijk_ispec];
      b_epsdev[4] = b_epsilondev_yz[ijk_ispec];

      eps_trace_over_3 = epsilon_trace_over_3[ijk_ispec];
      b_eps_trace_over_3 = b_epsilon_trace_over_3[ijk_ispec];

      //! Building of the local matrix of the strain tensor
      //! for the adjoint field and the regular backward field
      //!eps11 et eps22
      eps[0] = epsdev[0] + eps_trace_over_3;
      eps[1] = epsdev[1] + eps_trace_over_3;
      //!eps33
      eps[2] = -(eps[0]+eps[1])+3*eps_trace_over_3;
      //!eps23
      eps[3] = epsdev[4];
      //!eps13
      eps[4] = epsdev[3];
      //!eps12
      eps[5] = epsdev[2];

      // backward arrays
      b_eps[0] = b_epsdev[0] + b_eps_trace_over_3;
      b_eps[1] = b_epsdev[1] + b_eps_trace_over_3;
      b_eps[2] = -(b_eps[0]+b_eps[1])+3*b_eps_trace_over_3;
      b_eps[3] = b_epsdev[4];
      b_eps[4] = b_epsdev[3];
      b_eps[5] = b_epsdev[2];

      //! Computing the 21 strain products without assuming eps(i)*b_eps(j) = eps(j)*b_eps(i)
      int p = 0;
      for( i=0; i<6; i++){
        for( j=i; j<6; j++){
          prod[p] = eps[i] * b_eps[j];
          if (j > i ){
            prod[p] = prod[p] + eps[j]*b_eps[i];
            if (j > 2 && i < 3){ prod[p] = prod[p]*2; }
          }
          if (i > 2){ prod[p] = prod[p]*4; }
          p++;
        }
      }

      // all 21 anisotropic coefficients
      for( i=0; i<21; i++){
        cijkl_kl[i+ijk21_ispec] += deltat * prod[i];
      }

    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_kernels_cudakernel(int* ispec_is_elastic,
                                           int* d_ibool,
                                           realw* accel,
                                           realw* b_displ,
                                           realw* epsilondev_xx,realw* epsilondev_yy,realw* epsilondev_xy,
                                           realw* epsilondev_xz,realw* epsilondev_yz,
                                           realw* b_epsilondev_xx,realw* b_epsilondev_yy,realw* b_epsilondev_xy,
                                           realw* b_epsilondev_xz,realw* b_epsilondev_yz,
                                           realw* rho_kl,
                                           realw deltat,
                                           realw* mu_kl,
                                           realw* kappa_kl,
                                           realw* epsilon_trace_over_3,
                                           realw* b_epsilon_trace_over_3,
                                           int NSPEC_AB) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;
  int ijk = threadIdx.x;
  int ijk_ispec = ijk + NGLL3*ispec;

  // handles case when there is 1 extra block (due to rectangular grid)
  if (ispec < NSPEC_AB) {

    // elastic elements only
    if (ispec_is_elastic[ispec]) {
      int iglob = d_ibool[ijk + NGLL3_PADDED*ispec] - 1 ;

      // isotropic kernels:
      // density kernel
      rho_kl[ijk_ispec] += deltat * (accel[3*iglob]*b_displ[3*iglob]+
                                     accel[3*iglob+1]*b_displ[3*iglob+1]+
                                     accel[3*iglob+2]*b_displ[3*iglob+2]);


      // shear modulus kernel
      mu_kl[ijk_ispec] += deltat * (epsilondev_xx[ijk_ispec]*b_epsilondev_xx[ijk_ispec]+
                                    epsilondev_yy[ijk_ispec]*b_epsilondev_yy[ijk_ispec]+
                                    (epsilondev_xx[ijk_ispec]+epsilondev_yy[ijk_ispec])*
                                    (b_epsilondev_xx[ijk_ispec]+b_epsilondev_yy[ijk_ispec])+
                                    2*(epsilondev_xy[ijk_ispec]*b_epsilondev_xy[ijk_ispec]+
                                       epsilondev_xz[ijk_ispec]*b_epsilondev_xz[ijk_ispec]+
                                       epsilondev_yz[ijk_ispec]*b_epsilondev_yz[ijk_ispec]));

      // bulk modulus kernel
      kappa_kl[ijk_ispec] += deltat*(9*epsilon_trace_over_3[ijk_ispec]*
                                     b_epsilon_trace_over_3[ijk_ispec]);

    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_kernels_elastic_cuda,
              COMPUTE_KERNELS_ELASTIC_CUDA)(long* Mesh_pointer,
                                            realw* deltat_f) {

  TRACE("compute_kernels_elastic_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  int blocksize = NGLL3; // NGLLX*NGLLY*NGLLZ
  realw deltat = *deltat_f;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->NSPEC_AB,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  if (mp->anisotropic_kl ){
    compute_kernels_ani_cudakernel<<<grid,threads>>>(mp->d_ispec_is_elastic,mp->d_ibool,
                                                     mp->d_accel, mp->d_b_displ,
                                                     mp->d_epsilondev_xx,
                                                     mp->d_epsilondev_yy,
                                                     mp->d_epsilondev_xy,
                                                     mp->d_epsilondev_xz,
                                                     mp->d_epsilondev_yz,
                                                     mp->d_b_epsilondev_xx,
                                                     mp->d_b_epsilondev_yy,
                                                     mp->d_b_epsilondev_xy,
                                                     mp->d_b_epsilondev_xz,
                                                     mp->d_b_epsilondev_yz,
                                                     mp->d_rho_kl,
                                                     deltat,
                                                     mp->d_cijkl_kl,
                                                     mp->d_epsilon_trace_over_3,
                                                     mp->d_b_epsilon_trace_over_3,
                                                     mp->NSPEC_AB);

  }else{
    compute_kernels_cudakernel<<<grid,threads>>>(mp->d_ispec_is_elastic,mp->d_ibool,
                                                 mp->d_accel, mp->d_b_displ,
                                                 mp->d_epsilondev_xx,
                                                 mp->d_epsilondev_yy,
                                                 mp->d_epsilondev_xy,
                                                 mp->d_epsilondev_xz,
                                                 mp->d_epsilondev_yz,
                                                 mp->d_b_epsilondev_xx,
                                                 mp->d_b_epsilondev_yy,
                                                 mp->d_b_epsilondev_xy,
                                                 mp->d_b_epsilondev_xz,
                                                 mp->d_b_epsilondev_yz,
                                                 mp->d_rho_kl,
                                                 deltat,
                                                 mp->d_mu_kl,
                                                 mp->d_kappa_kl,
                                                 mp->d_epsilon_trace_over_3,
                                                 mp->d_b_epsilon_trace_over_3,
                                                 mp->NSPEC_AB);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_elastic_cuda");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// NOISE SIMULATIONS

/* ----------------------------------------------------------------------------------------------- */


__global__ void compute_kernels_strength_noise_cuda_kernel(realw* displ,
                                                           int* free_surface_ispec,
                                                           int* free_surface_ijk,
                                                           int* d_ibool,
                                                           realw* noise_surface_movie,
                                                           realw* normal_x_noise,
                                                           realw* normal_y_noise,
                                                           realw* normal_z_noise,
                                                           realw* sigma_kl,
                                                           realw deltat,
                                                           int num_free_surface_faces) {
  int iface = blockIdx.x + blockIdx.y*gridDim.x;
  int igll = threadIdx.x;
  int ipoin = igll + NGLL2*iface;

  if (iface < num_free_surface_faces) {

    int ispec = free_surface_ispec[iface]-1;

    int i = free_surface_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)] - 1;
    int j = free_surface_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)] - 1;
    int k = free_surface_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)] - 1;

    int iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)] - 1;

    realw eta = ( noise_surface_movie[INDEX3(NDIM,NGLL2,0,igll,iface)]*normal_x_noise[ipoin]+
                 noise_surface_movie[INDEX3(NDIM,NGLL2,1,igll,iface)]*normal_y_noise[ipoin]+
                 noise_surface_movie[INDEX3(NDIM,NGLL2,2,igll,iface)]*normal_z_noise[ipoin]);

    sigma_kl[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)] += deltat*eta*(normal_x_noise[ipoin]*displ[3*iglob]+
                                                       normal_y_noise[ipoin]*displ[1+3*iglob]+
                                                       normal_z_noise[ipoin]*displ[2+3*iglob]);
  }

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_kernels_strgth_noise_cu,
              COMPUTE_KERNELS_STRGTH_NOISE_CU)(long* Mesh_pointer,
                                                    realw* h_noise_surface_movie,
                                                    realw* deltat) {

TRACE("compute_kernels_strgth_noise_cu");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks if anything to do
  if (mp->num_free_surface_faces == 0) return;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->num_free_surface_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(NGLL2,1,1);

  print_CUDA_error_if_any(hipMemcpy(mp->d_noise_surface_movie,h_noise_surface_movie,
                          NDIM*NGLL2*(mp->num_free_surface_faces)*sizeof(realw),hipMemcpyHostToDevice),81000);

  compute_kernels_strength_noise_cuda_kernel<<<grid,threads>>>(mp->d_displ,
                                                               mp->d_free_surface_ispec,
                                                               mp->d_free_surface_ijk,
                                                               mp->d_ibool,
                                                               mp->d_noise_surface_movie,
                                                               mp->d_normal_x_noise,
                                                               mp->d_normal_y_noise,
                                                               mp->d_normal_z_noise,
                                                               mp->d_sigma_kl,*deltat,
                                                               mp->num_free_surface_faces);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_strength_noise_cuda_kernel");
#endif
}



/* ----------------------------------------------------------------------------------------------- */

// ACOUSTIC SIMULATIONS

/* ----------------------------------------------------------------------------------------------- */


__device__ void compute_gradient_kernel(int ijk,
                                        int ispec,
                                        realw* scalar_field,
                                        realw* vector_field_element,
                                        realw* d_hprime_xx,
                                        realw* d_xix,realw* d_xiy,realw* d_xiz,
                                        realw* d_etax,realw* d_etay,realw* d_etaz,
                                        realw* d_gammax,realw* d_gammay,realw* d_gammaz,
                                        realw rhol,
                                        int gravity) {

  realw temp1l,temp2l,temp3l;
  realw hp1,hp2,hp3;
  realw xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl;
  realw rho_invl;
  int l,offset,offset1,offset2,offset3;

  const int NGLL3_ALIGN = NGLL3_PADDED;

  int K = (ijk/NGLL2);
  int J = ((ijk-K*NGLL2)/NGLLX);
  int I = (ijk-K*NGLL2-J*NGLLX);

  // derivative along x
  temp1l = 0.f;
  for( l=0; l<NGLLX;l++){
    hp1 = d_hprime_xx[l*NGLLX+I];
    offset1 = K*NGLL2+J*NGLLX+l;
    temp1l += scalar_field[offset1]*hp1;
  }

  // derivative along y
  temp2l = 0.f;
  for( l=0; l<NGLLX;l++){
    // assumes hprime_xx == hprime_yy
    hp2 = d_hprime_xx[l*NGLLX+J];
    offset2 = K*NGLL2+l*NGLLX+I;
    temp2l += scalar_field[offset2]*hp2;
  }

  // derivative along z
  temp3l = 0.f;
  for( l=0; l<NGLLX;l++){
    // assumes hprime_xx == hprime_zz
    hp3 = d_hprime_xx[l*NGLLX+K];
    offset3 = l*NGLL2+J*NGLLX+I;
    temp3l += scalar_field[offset3]*hp3;
  }

  offset = ispec*NGLL3_ALIGN + ijk;

  xixl = d_xix[offset];
  xiyl = d_xiy[offset];
  xizl = d_xiz[offset];
  etaxl = d_etax[offset];
  etayl = d_etay[offset];
  etazl = d_etaz[offset];
  gammaxl = d_gammax[offset];
  gammayl = d_gammay[offset];
  gammazl = d_gammaz[offset];

  if (gravity ){
    // daniel: TODO - check gravity case here
    rho_invl = 1.0f / rhol;
  }else{
    rho_invl = 1.0f / rhol;
  }
  // derivatives of acoustic scalar potential field on GLL points
  vector_field_element[0] = (temp1l*xixl + temp2l*etaxl + temp3l*gammaxl) * rho_invl;
  vector_field_element[1] = (temp1l*xiyl + temp2l*etayl + temp3l*gammayl) * rho_invl;
  vector_field_element[2] = (temp1l*xizl + temp2l*etazl + temp3l*gammazl) * rho_invl;

}

/* ----------------------------------------------------------------------------------------------- */


__global__ void compute_kernels_acoustic_kernel(int* ispec_is_acoustic,
                                                int* d_ibool,
                                                realw* rhostore,
                                                realw* kappastore,
                                                realw* d_hprime_xx,
                                                realw* d_xix,realw* d_xiy,realw* d_xiz,
                                                realw* d_etax,realw* d_etay,realw* d_etaz,
                                                realw* d_gammax,realw* d_gammay,realw* d_gammaz,
                                                realw* potential_dot_dot_acoustic,
                                                realw* b_potential_acoustic,
                                                realw* b_potential_dot_dot_acoustic,
                                                realw* rho_ac_kl,
                                                realw* kappa_ac_kl,
                                                realw deltat,
                                                int NSPEC_AB,
                                                int gravity) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;
  int ijk = threadIdx.x;

  // local and global indices
  int ijk_ispec = ijk + NGLL3*ispec;
  int ijk_ispec_padded = ijk + NGLL3_PADDED*ispec;
  int iglob;

  // shared memory between all threads within this block
  __shared__ realw scalar_field_displ[NGLL3];
  __shared__ realw scalar_field_accel[NGLL3];

  int active = 0;

  // handles case when there is 1 extra block (due to rectangular grid)
  if (ispec < NSPEC_AB ){
    // acoustic elements only
    if (ispec_is_acoustic[ispec] ){
      active = 1;

      // copy field values
      iglob = d_ibool[ijk_ispec_padded] - 1;
      scalar_field_displ[ijk] = b_potential_acoustic[iglob];
      scalar_field_accel[ijk] = potential_dot_dot_acoustic[iglob];
    }
  }

  // synchronizes threads
  __syncthreads();

  if (active ){
    realw accel_elm[3];
    realw b_displ_elm[3];
    realw rhol,kappal;

    // gets material parameter
    rhol = rhostore[ijk_ispec_padded];

    // displacement vector from backward field
    compute_gradient_kernel(ijk,ispec,scalar_field_displ,b_displ_elm,
                            d_hprime_xx,
                            d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                            rhol,gravity);

    // acceleration vector
    compute_gradient_kernel(ijk,ispec,scalar_field_accel,accel_elm,
                            d_hprime_xx,
                            d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                            rhol,gravity);

    // density kernel
    rho_ac_kl[ijk_ispec] -= deltat * rhol * (accel_elm[0]*b_displ_elm[0] +
                                             accel_elm[1]*b_displ_elm[1] +
                                             accel_elm[2]*b_displ_elm[2]);

    // bulk modulus kernel
    kappal = kappastore[ijk_ispec];
    kappa_ac_kl[ijk_ispec] -= deltat / kappal * potential_dot_dot_acoustic[iglob]
                                              * b_potential_dot_dot_acoustic[iglob];
  } // active
}

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(compute_kernels_acoustic_cuda,
              COMPUTE_KERNELS_ACOUSTIC_CUDA)(long* Mesh_pointer,
                                             realw* deltat_f) {

TRACE("compute_kernels_acoustic_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  int blocksize = NGLL3; // NGLLX*NGLLY*NGLLZ
  realw deltat = *deltat_f;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->NSPEC_AB,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  compute_kernels_acoustic_kernel<<<grid,threads>>>(mp->d_ispec_is_acoustic,
                                                    mp->d_ibool,
                                                    mp->d_rhostore,
                                                    mp->d_kappastore,
                                                    mp->d_hprime_xx,
                                                    mp->d_xix,mp->d_xiy,mp->d_xiz,
                                                    mp->d_etax,mp->d_etay,mp->d_etaz,
                                                    mp->d_gammax,mp->d_gammay,mp->d_gammaz,
                                                    mp->d_potential_dot_dot_acoustic,
                                                    mp->d_b_potential_acoustic,
                                                    mp->d_b_potential_dot_dot_acoustic,
                                                    mp->d_rho_ac_kl,
                                                    mp->d_kappa_ac_kl,
                                                    deltat,
                                                    mp->NSPEC_AB,
                                                    mp->gravity);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_acoustic_kernel");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// preconditioner (approximate Hessian kernel)

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_kernels_hess_el_cudakernel(int* ispec_is_elastic,
                                                   int* d_ibool,
                                                   realw* accel,
                                                   realw* b_accel,
                                                   realw* hess_kl,
                                                   realw deltat,
                                                   int NSPEC_AB) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;
  int ijk = threadIdx.x;

  // handles case when there is 1 extra block (due to rectangular grid)
  if (ispec < NSPEC_AB) {

    // elastic elements only
    if (ispec_is_elastic[ispec]) {
      int iglob = d_ibool[ijk + NGLL3_PADDED*ispec] - 1;

      // approximate hessian
      hess_kl[ijk + NGLL3*ispec] += deltat * (accel[3*iglob]*b_accel[3*iglob]+
                                              accel[3*iglob+1]*b_accel[3*iglob+1]+
                                              accel[3*iglob+2]*b_accel[3*iglob+2]);
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_kernels_hess_ac_cudakernel(int* ispec_is_acoustic,
                                                   int* d_ibool,
                                                   realw* potential_dot_dot_acoustic,
                                                   realw* b_potential_dot_dot_acoustic,
                                                   realw* rhostore,
                                                   realw* d_hprime_xx,
                                                   realw* d_xix,realw* d_xiy,realw* d_xiz,
                                                   realw* d_etax,realw* d_etay,realw* d_etaz,
                                                   realw* d_gammax,realw* d_gammay,realw* d_gammaz,
                                                   realw* hess_kl,
                                                   realw deltat,
                                                   int NSPEC_AB,
                                                   int gravity) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;
  int ijk = threadIdx.x;
  int ijk_ispec_padded = ijk + NGLL3_PADDED*ispec;
  int iglob;

  // shared memory between all threads within this block
  __shared__ realw scalar_field_accel[NGLL3];
  __shared__ realw scalar_field_b_accel[NGLL3];

  int active = 0;

  // handles case when there is 1 extra block (due to rectangular grid)
  if (ispec < NSPEC_AB) {

    // acoustic elements only
    if (ispec_is_acoustic[ispec] ){
      active = 1;

      // global indices
      iglob = d_ibool[ijk_ispec_padded] - 1;

      // copy field values
      scalar_field_accel[ijk] = potential_dot_dot_acoustic[iglob];
      scalar_field_b_accel[ijk] = b_potential_dot_dot_acoustic[iglob];
    }
  }

  // synchronizes threads
  __syncthreads();

  if (active ){
    realw accel_elm[3];
    realw b_accel_elm[3];
    realw rhol;

    // gets material parameter
    rhol = rhostore[ijk_ispec_padded];

    // acceleration vector
    compute_gradient_kernel(ijk,ispec,
                            scalar_field_accel,accel_elm,
                            d_hprime_xx,
                            d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                            rhol,gravity);

    // acceleration vector from backward field
    compute_gradient_kernel(ijk,ispec,
                            scalar_field_b_accel,b_accel_elm,
                            d_hprime_xx,
                            d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                            rhol,gravity);
    // approximates hessian
    hess_kl[ijk + NGLL3*ispec] += deltat * (accel_elm[0]*b_accel_elm[0] +
                                            accel_elm[1]*b_accel_elm[1] +
                                            accel_elm[2]*b_accel_elm[2]);

  } // active
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_kernels_hess_cuda,
              COMPUTE_KERNELS_HESS_CUDA)(long* Mesh_pointer,
                                         realw* deltat_f,
                                         int* ELASTIC_SIMULATION,
                                         int* ACOUSTIC_SIMULATION) {
  TRACE("compute_kernels_hess_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  int blocksize = NGLL3; // NGLLX*NGLLY*NGLLZ
  realw deltat = *deltat_f;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->NSPEC_AB,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  if (*ELASTIC_SIMULATION) {
    compute_kernels_hess_el_cudakernel<<<grid,threads>>>(mp->d_ispec_is_elastic,
                                                         mp->d_ibool,
                                                         mp->d_accel,
                                                         mp->d_b_accel,
                                                         mp->d_hess_el_kl,
                                                         deltat,
                                                         mp->NSPEC_AB);
  }

  if (*ACOUSTIC_SIMULATION) {
    compute_kernels_hess_ac_cudakernel<<<grid,threads>>>(mp->d_ispec_is_acoustic,
                                                         mp->d_ibool,
                                                         mp->d_potential_dot_dot_acoustic,
                                                         mp->d_b_potential_dot_dot_acoustic,
                                                         mp->d_rhostore,
                                                         mp->d_hprime_xx,
                                                         mp->d_xix,mp->d_xiy,mp->d_xiz,
                                                         mp->d_etax,mp->d_etay,mp->d_etaz,
                                                         mp->d_gammax,mp->d_gammay,mp->d_gammaz,
                                                         mp->d_hess_ac_kl,
                                                         deltat,
                                                         mp->NSPEC_AB,
                                                         mp->gravity);
  }


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_kernels_hess_cuda");
#endif
}

