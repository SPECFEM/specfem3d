#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  3 . 0
 !               ---------------------------------------
 !
 !     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
 !                              CNRS, France
 !                       and Princeton University, USA
 !                 (there are currently many more authors!)
 !                           (c) October 2017
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 3 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

// acoustic sources

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_add_sources_acoustic_kernel(field* potential_dot_dot_acoustic,
                                                    int* d_ibool,
                                                    realw* sourcearrays,
                                                    field* stf_pre_compute,
                                                    int myrank,
                                                    int* islice_selected_source,
                                                    int* ispec_selected_source,
                                                    int* ispec_is_acoustic,
                                                    realw* kappastore,
                                                    int NSOURCES) {
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;

  int isource  = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int ispec,iglob;
  field stf;
  realw kappal;

  if (isource < NSOURCES){

    if (myrank == islice_selected_source[isource]) {

      ispec = ispec_selected_source[isource]-1;

      if (ispec_is_acoustic[ispec]) {

        iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)] - 1;

        stf = stf_pre_compute[isource];
        kappal = kappastore[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];

        atomicAdd(&potential_dot_dot_acoustic[iglob],
                  -sourcearrays[INDEX5(NSOURCES,NDIM,NGLLX,NGLLX,isource, 0,i,j,k)]*stf/kappal);

        // debug: without atomic operation
        //      potential_dot_dot_acoustic[iglob] +=
        //              -sourcearrays[INDEX5(NSOURCES, 3, NGLLX,NGLLX,isource, 0, i,j,k)]*stf/kappal;
      }
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

// Converts source time function to the correct GPU precision, and adapts format for NB_RUNS_ON_ACOUSTIC_GPU option
void get_stf_for_gpu(field* stf_pre_compute, double* h_stf_pre_compute, int * run_number_of_the_source, int NSOURCES) {

  TRACE("get_stf_for_gpu");
  realw realw_to_field[NB_RUNS_ACOUSTIC_GPU];

  //Conversion to GPU precision
  //Converts source time function to the field format. The stf value is saved only into its corresponding run. For other runs, a zero will be added

  for (int i_source=0;i_source < NSOURCES;i_source++){
    for (int i_run=0;i_run < NB_RUNS_ACOUSTIC_GPU;i_run++)
      if (run_number_of_the_source[i_source] == i_run){
        realw_to_field[i_run]= (realw)h_stf_pre_compute[i_source];
      }
      else{
        realw_to_field[i_run] = 0.0f;
      }
      //function Make_field is overloaded to convert array of realw into field structure
      stf_pre_compute[i_source] = Make_field(realw_to_field);
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern EXTERN_LANG
void FC_FUNC_(compute_add_sources_ac_cuda,
              COMPUTE_ADD_SOURCES_AC_CUDA)(long* Mesh_pointer,
                                           int* NSOURCESf,
                                           double* h_stf_pre_compute,int* run_number_of_the_source) {

  TRACE("compute_add_sources_ac_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // check if anything to do
  if (mp->nsources_local == 0) return;

  int NSOURCES = *NSOURCESf;

  field* stf_pre_compute = (field*)malloc(NSOURCES * sizeof(field));
  get_stf_for_gpu(stf_pre_compute,h_stf_pre_compute,run_number_of_the_source,NSOURCES);

  // copies pre-computed source time factors onto GPU
  print_CUDA_error_if_any(hipMemcpy(mp->d_stf_pre_compute,stf_pre_compute,
                                     NSOURCES*sizeof(field),hipMemcpyHostToDevice),1877);
  free(stf_pre_compute);

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(NSOURCES,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(NGLLX,NGLLY,NGLLZ);

  compute_add_sources_acoustic_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_potential_dot_dot_acoustic,
                                                                              mp->d_ibool,
                                                                              mp->d_sourcearrays,
                                                                              mp->d_stf_pre_compute,
                                                                              mp->myrank,
                                                                              mp->d_islice_selected_source,
                                                                              mp->d_ispec_selected_source,
                                                                              mp->d_ispec_is_acoustic,
                                                                              mp->d_kappastore,
                                                                              NSOURCES);


  GPU_ERROR_CHECKING("compute_add_sources_ac_cuda");
}

/* ----------------------------------------------------------------------------------------------- */

extern EXTERN_LANG
void FC_FUNC_(compute_add_sources_ac_s3_cuda,
              COMPUTE_ADD_SOURCES_AC_s3_CUDA)(long* Mesh_pointer,
                                              int* NSOURCESf,
                                              double* h_stf_pre_compute,int* run_number_of_the_source) {

  TRACE("compute_add_sources_ac_s3_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // check if anything to do
  if (mp->nsources_local == 0) return;

  int NSOURCES = *NSOURCESf;

  field* stf_pre_compute = (field*)malloc(NSOURCES * sizeof(field));
  get_stf_for_gpu(stf_pre_compute,h_stf_pre_compute,run_number_of_the_source,NSOURCES);

  // copies source time factors onto GPU
  print_CUDA_error_if_any(hipMemcpy(mp->d_stf_pre_compute,stf_pre_compute,
                                     NSOURCES*sizeof(field),hipMemcpyHostToDevice),55);

  free(stf_pre_compute);

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(NSOURCES,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(NGLLX,NGLLY,NGLLZ);

  compute_add_sources_acoustic_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_potential_dot_dot_acoustic,
                                                                              mp->d_ibool,
                                                                              mp->d_sourcearrays,
                                                                              mp->d_stf_pre_compute,
                                                                              mp->myrank,
                                                                              mp->d_islice_selected_source,
                                                                              mp->d_ispec_selected_source,
                                                                              mp->d_ispec_is_acoustic,
                                                                              mp->d_kappastore,
                                                                              NSOURCES);

  GPU_ERROR_CHECKING("compute_add_sources_ac_s3_cuda");
}


/* ----------------------------------------------------------------------------------------------- */

// acoustic adjoint sources

/* ----------------------------------------------------------------------------------------------- */

__global__ void add_sources_ac_SIM_TYPE_2_OR_3_kernel(field* potential_dot_dot_acoustic,
                                                      int nrec,
                                                      int it,
                                                      int NSTEP_BETWEEN_ADJSRC,
                                                      field* source_adjoint,
                                                      realw* xir_store,
                                                      realw* etar_store,
                                                      realw* gammar_store,
                                                      int* d_ibool,
                                                      int* ispec_is_acoustic,
                                                      int* ispec_selected_recloc,
                                                      int nadj_rec_local,
                                                      realw* kappastore) {

  int irec_local = blockIdx.x + gridDim.x*blockIdx.y;

  // because of grid shape, irec_local can be too big
  if (irec_local < nadj_rec_local) {

    int ispec = ispec_selected_recloc[irec_local]-1;
    if (ispec_is_acoustic[ispec]){
      int i = threadIdx.x;
      int j = threadIdx.y;
      int k = threadIdx.z;

      int iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

      realw hxir    = xir_store[INDEX2(NGLLX,i,irec_local)];
      realw hetar   = etar_store[INDEX2(NGLLX,j,irec_local)];
      realw hgammar = gammar_store[INDEX2(NGLLX,k,irec_local)];

      field source_adj = source_adjoint[INDEX3(NDIM,nadj_rec_local,0,irec_local,it)];
      //realw kappal = kappastore[INDEX4(NGLLX,NGLLY,NGLLZ,i,j,k,ispec)];

      //potential_dot_dot_acoustic[iglob] += adj_sourcearrays[INDEX6(nadj_rec_local,NTSTEP_BETWEEN_ADJSRC,3,5,5,
      //                                            pre_computed_irec_local_index[irec],
      //                                            pre_computed_index,
      //                                            0,
      //                                            i,j,k)]/kappal;

      // beware, for acoustic medium, a pressure source would be taking the negative
      // and divide by Kappa of the fluid;
      //
      // note: we take the first component of the adj_sourcearrays

      //realw stf = - source_adj * hxir * hetar * hgammar / kappal;

      // VM VM : change the adjoint source to be consistent with CPU code
      field stf = source_adj * hxir * hetar * hgammar;
      atomicAdd(&potential_dot_dot_acoustic[iglob],stf);

                //+adj_sourcearrays[INDEX6(nadj_rec_local,NTSTEP_BETWEEN_ADJSRC,3,5,5,
                //                         pre_computed_irec_local_index[irec],pre_computed_index-1,
                //                         0,i,j,k)] // / kappal
                //                         );
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */


extern EXTERN_LANG
void FC_FUNC_(add_sources_ac_sim_2_or_3_cuda,
              ADD_SOURCES_AC_SIM_2_OR_3_CUDA)(long* Mesh_pointer,
                                              realw* h_source_adjoint,
                                              int* nrec,
                                              int* nadj_rec_local,
                                              int* NTSTEP_BETWEEN_READ_ADJSRC,
                                              int* it) {

  TRACE("add_sources_ac_sim_2_or_3_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks
  if (*nadj_rec_local != mp->nadj_rec_local) exit_on_cuda_error("add_sources_ac_sim_type_2_or_3: nadj_rec_local not equal\n");

  // note: for acoustic simulations with fused wavefields, NB_RUNS_ACOUSTIC_GPU > 1
  //       and thus the number of adjoint sources might become different in future
  //       todo: not implemented yet for adjoint/kernel simulation
  //if (*nadj_rec_local/NB_RUNS_ACOUSTIC_GPU != mp->nadj_rec_local)
  //  exit_on_cuda_error("add_sources_ac_sim_type_2_or_3: nadj_rec_local not equal\n");

  // checks if anything to do
  if (mp->nadj_rec_local == 0) return;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->nadj_rec_local,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y,1);
  dim3 threads(NGLLX,NGLLY,NGLLZ);

  int it_index = *NTSTEP_BETWEEN_READ_ADJSRC - (*it-1) % *NTSTEP_BETWEEN_READ_ADJSRC - 1 ;

  // copies extracted array values onto GPU
  if ( (*it-1) % *NTSTEP_BETWEEN_READ_ADJSRC==0){
    print_CUDA_error_if_any(hipMemcpy(mp->d_source_adjoint,h_source_adjoint,
                                       mp->nadj_rec_local*NDIM*sizeof(field)*(*NTSTEP_BETWEEN_READ_ADJSRC),hipMemcpyHostToDevice),99099);
  }

  // launches cuda kernel for acoustic adjoint sources
  add_sources_ac_SIM_TYPE_2_OR_3_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_potential_dot_dot_acoustic,
                                                                                *nrec,it_index,*NTSTEP_BETWEEN_READ_ADJSRC,
                                                                                mp->d_source_adjoint,
                                                                                mp->d_hxir_adj,
                                                                                mp->d_hetar_adj,
                                                                                mp->d_hgammar_adj,
                                                                                mp->d_ibool,
                                                                                mp->d_ispec_is_acoustic,
                                                                                mp->d_ispec_selected_adjrec_loc,
                                                                                mp->nadj_rec_local,
                                                                                mp->d_kappastore);


  GPU_ERROR_CHECKING("add_sources_acoustic_SIM_TYPE_2_OR_3_kernel");
}
