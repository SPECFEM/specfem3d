#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 0
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and University of Pau / CNRS / INRIA
 ! (c) Princeton University / California Institute of Technology and University of Pau / CNRS / INRIA
 !                            April 2011
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/types.h>
#include <unistd.h>
#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"
// #include "epik_user.h"


/* ----------------------------------------------------------------------------------------------- */

// elastic domain sources

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_add_sources_kernel(realw* accel,
                                           int* ibool,
                                           int* ispec_is_inner,
                                           int phase_is_inner,
                                           realw* sourcearrays,
                                           double* stf_pre_compute,
                                           int myrank,
                                           int* islice_selected_source,
                                           int* ispec_selected_source,
                                           int* ispec_is_elastic,
                                           int NSOURCES) {
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;

  int isource  = blockIdx.x + gridDim.x*blockIdx.y; // bx
  int ispec;
  int iglob;
  realw stf;

  if(isource < NSOURCES) { // when NSOURCES > 65535, but mod(nspec_top,2) > 0, we end up with an extra block.

    if(myrank == islice_selected_source[isource]) {

      ispec = ispec_selected_source[isource]-1;

      if(ispec_is_inner[ispec] == phase_is_inner && ispec_is_elastic[ispec] ) {

        stf = (realw) stf_pre_compute[isource];
        iglob = ibool[INDEX4(5,5,5,i,j,k,ispec)]-1;

        atomicAdd(&accel[iglob*3],
                  sourcearrays[INDEX5(NSOURCES, 3, 5, 5,isource, 0, i,j,k)]*stf);
        atomicAdd(&accel[iglob*3+1],
                  sourcearrays[INDEX5(NSOURCES, 3, 5, 5,isource, 1, i,j,k)]*stf);
        atomicAdd(&accel[iglob*3+2],
                  sourcearrays[INDEX5(NSOURCES, 3, 5, 5,isource, 2, i,j,k)]*stf);
      }
    }
  }

}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_add_sources_el_cuda,
              COMPUTE_ADD_SOURCES_EL_CUDA)(long* Mesh_pointer_f,
                                            int* phase_is_innerf,
                                            int* NSOURCESf,
                                            double* h_stf_pre_compute,
                                            int* myrankf) {

TRACE("compute_add_sources_el_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  // check if anything to do
  if( mp->nsources_local == 0 ) return;

  int phase_is_inner = *phase_is_innerf;
  int NSOURCES = *NSOURCESf;
  int myrank = *myrankf;

  int num_blocks_x = NSOURCES;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  //double* d_stf_pre_compute;
  print_CUDA_error_if_any(hipMemcpy(mp->d_stf_pre_compute,h_stf_pre_compute,
                                     NSOURCES*sizeof(double),hipMemcpyHostToDevice),18);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(5,5,5);

  compute_add_sources_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel,
                                               mp->d_ibool,
                                               mp->d_ispec_is_inner,
                                               phase_is_inner,
                                               mp->d_sourcearrays,
                                               mp->d_stf_pre_compute,
                                               myrank,
                                               mp->d_islice_selected_source,
                                               mp->d_ispec_selected_source,
                                               mp->d_ispec_is_elastic,
                                               NSOURCES);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_add_sources_kernel");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_add_sources_el_s3_cuda,
              COMPUTE_ADD_SOURCES_EL_S3_CUDA)(long* Mesh_pointer,
                                              double* h_stf_pre_compute,
                                              int* NSOURCESf,
                                              int* phase_is_inner,
                                              int* myrank) {
  TRACE("compute_add_sources_el_s3_cuda");
  // EPIK_TRACER("compute_add_sources_el_s3_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  int NSOURCES = *NSOURCESf;

  print_CUDA_error_if_any(hipMemcpy(mp->d_stf_pre_compute,h_stf_pre_compute,
                                     NSOURCES*sizeof(double),hipMemcpyHostToDevice),18);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_add_sources_el_s3_cuda");
#endif

  int num_blocks_x = NSOURCES;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(5,5,5);

  compute_add_sources_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_accel,mp->d_ibool,
                                               mp->d_ispec_is_inner, *phase_is_inner,
                                               mp->d_sourcearrays,
                                               mp->d_stf_pre_compute,
                                               *myrank,
                                               mp->d_islice_selected_source,mp->d_ispec_selected_source,
                                               mp->d_ispec_is_elastic,
                                               NSOURCES);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_add_sources_el_s3_cuda");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// NOISE sources

/* ----------------------------------------------------------------------------------------------- */

__global__ void add_source_master_rec_noise_cuda_kernel(int* ibool,
                                                        int* ispec_selected_rec,
                                                        int irec_master_noise,
                                                        realw* accel,
                                                        realw* noise_sourcearray,
                                                        int it) {
  int tx = threadIdx.x;
  int iglob = ibool[tx + NGLL3*(ispec_selected_rec[irec_master_noise-1]-1)]-1;

  // not sure if we need atomic operations but just in case...
  // accel[3*iglob] += noise_sourcearray[3*tx + 3*125*it];
  // accel[1+3*iglob] += noise_sourcearray[1+3*tx + 3*125*it];
  // accel[2+3*iglob] += noise_sourcearray[2+3*tx + 3*125*it];

  atomicAdd(&accel[iglob*3],noise_sourcearray[3*tx + 3*NGLL3*it]);
  atomicAdd(&accel[iglob*3+1],noise_sourcearray[1+3*tx + 3*NGLL3*it]);
  atomicAdd(&accel[iglob*3+2],noise_sourcearray[2+3*tx + 3*NGLL3*it]);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(add_source_master_rec_noise_cu,
              ADD_SOURCE_MASTER_REC_NOISE_CU)(long* Mesh_pointer_f,
                                                int* myrank_f,
                                                int* it_f,
                                                int* irec_master_noise_f,
                                                int* islice_selected_rec) {

TRACE("add_source_master_rec_noise_cu");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  int it = *it_f-1; // -1 for Fortran -> C indexing differences
  int irec_master_noise = *irec_master_noise_f;
  int myrank = *myrank_f;

  dim3 grid(1,1,1);
  dim3 threads(NGLL3,1,1);

  if(myrank == islice_selected_rec[irec_master_noise-1]) {
    add_source_master_rec_noise_cuda_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_ibool,
                                                              mp->d_ispec_selected_rec,
                                                              irec_master_noise,
                                                              mp->d_accel,
                                                              mp->d_noise_sourcearray,
                                                              it);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("add_source_master_rec_noise_cuda_kernel");
#endif
  }
}

/* ----------------------------------------------------------------------------------------------- */

// ADJOINT sources

/* ----------------------------------------------------------------------------------------------- */

__global__ void add_sources_el_SIM_TYPE_2_OR_3_kernel(realw* accel,
                                                     int nrec,
                                                     realw* adj_sourcearrays,
                                                     int* ibool,
                                                     int* ispec_is_inner,
                                                     int* ispec_is_elastic,
                                                     int* ispec_selected_rec,
                                                     int phase_is_inner,
                                                     int* pre_computed_irec,
                                                     int nadj_rec_local) {

  int irec_local = blockIdx.x + gridDim.x*blockIdx.y;

  if(irec_local < nadj_rec_local) { // when nrec > 65535, but mod(nspec_top,2) > 0, we end up with an extra block.

    int irec = pre_computed_irec[irec_local];

    int ispec = ispec_selected_rec[irec]-1;
    if( ispec_is_elastic[ispec] ){

      if(ispec_is_inner[ispec] == phase_is_inner) {
        int i = threadIdx.x;
        int j = threadIdx.y;
        int k = threadIdx.z;
        int iglob = ibool[INDEX4(5,5,5,i,j,k,ispec)]-1;

        // atomic operations are absolutely necessary for correctness!
        atomicAdd(&accel[3*iglob],adj_sourcearrays[INDEX5(5,5,5,3,
                                                              i,j,k,
                                                              0,
                                                              irec_local)]);

        atomicAdd(&accel[1+3*iglob], adj_sourcearrays[INDEX5(5,5,5,3,
                                                             i,j,k,
                                                             1,
                                                             irec_local)]);

        atomicAdd(&accel[2+3*iglob],adj_sourcearrays[INDEX5(5,5,5,3,
                                                            i,j,k,
                                                            2,
                                                            irec_local)]);
      }
    } // ispec_is_elastic
  }

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(add_sources_el_sim_type_2_or_3,
              ADD_SOURCES_EL_SIM_TYPE_2_OR_3)(long* Mesh_pointer,
                                               realw* h_adj_sourcearrays,
                                               int* phase_is_inner,
                                               int* h_ispec_is_inner,
                                               int* h_ispec_is_elastic,
                                               int* h_ispec_selected_rec,
                                               int* myrank,
                                               int* nrec,
                                               int* time_index,
                                               int* h_islice_selected_rec,
                                               int* nadj_rec_local,
                                               int* NTSTEP_BETWEEN_READ_ADJSRC) {

TRACE("add_sources_el_sim_type_2_or_3");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks
  if( *nadj_rec_local != mp->nadj_rec_local) exit_on_error("add_sources_el_sim_type_2_or_3: nadj_rec_local not equal\n");

  // make sure grid dimension is less than 65535 in x dimension
  int num_blocks_x = mp->nadj_rec_local;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y,1);
  dim3 threads(5,5,5);

  // build slice of adj_sourcearrays because full array is *very* large.
  // note: this extracts array values for local adjoint sources at given time step "time_index"
  //          from large adj_sourcearrays array into h_adj_sourcearrays_slice
  int ispec,i,j,k;
  int irec_local = 0;
  for(int irec = 0; irec < *nrec; irec++) {
    if(*myrank == h_islice_selected_rec[irec]) {
      irec_local++;

      // takes only elastic sources
      ispec = h_ispec_selected_rec[irec]-1;
      if( h_ispec_is_elastic[ispec] ){

        if( h_ispec_is_inner[ispec] == *phase_is_inner) {
          for(k=0;k<5;k++) {
            for(j=0;j<5;j++) {
              for(i=0;i<5;i++) {

                mp->h_adj_sourcearrays_slice[INDEX5(5,5,5,3,
                                                i,j,k,0,
                                                irec_local-1)]
                        = h_adj_sourcearrays[INDEX6(*nadj_rec_local,
                                                    *NTSTEP_BETWEEN_READ_ADJSRC,
                                                    3,5,5,
                                                    irec_local-1,
                                                    *time_index-1,
                                                    0,i,j,k)];

                mp->h_adj_sourcearrays_slice[INDEX5(5,5,5,3,
                                                i,j,k,1,
                                                irec_local-1)]
                        = h_adj_sourcearrays[INDEX6(*nadj_rec_local,
                                                    *NTSTEP_BETWEEN_READ_ADJSRC,
                                                    3,5,5,
                                                    irec_local-1,
                                                    *time_index-1,
                                                    1,i,j,k)];

                mp->h_adj_sourcearrays_slice[INDEX5(5,5,5,3,
                                                i,j,k,2,
                                                irec_local-1)]
                        = h_adj_sourcearrays[INDEX6(*nadj_rec_local,
                                                    *NTSTEP_BETWEEN_READ_ADJSRC,
                                                    3,5,5,
                                                    irec_local-1,
                                                    *time_index-1,
                                                    2,i,j,k)];
              }
            }
          }
        } // phase_is_inner
      } // h_ispec_is_elastic
    }
  }
  // check all local sources were added
  if( irec_local != mp->nadj_rec_local) exit_on_error("irec_local not equal to nadj_rec_local\n");

  // copies extracted array values onto GPU
  hipMemcpy(mp->d_adj_sourcearrays, mp->h_adj_sourcearrays_slice,
             (mp->nadj_rec_local)*3*NGLL3*sizeof(realw),hipMemcpyHostToDevice);


  // the irec_local variable needs to be precomputed (as
  // h_pre_comp..), because normally it is in the loop updating accel,
  // and due to how it's incremented, it cannot be parallelized

  add_sources_el_SIM_TYPE_2_OR_3_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel,
                                                         *nrec,
                                                         mp->d_adj_sourcearrays,
                                                         mp->d_ibool,
                                                         mp->d_ispec_is_inner,
                                                         mp->d_ispec_is_elastic,
                                                         mp->d_ispec_selected_rec,
                                                         *phase_is_inner,
                                                         mp->d_pre_computed_irec,
                                                         mp->nadj_rec_local);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("add_sources_SIM_TYPE_2_OR_3_kernel");
#endif
}

