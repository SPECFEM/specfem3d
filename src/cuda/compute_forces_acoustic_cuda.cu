#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 1
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and CNRS / INRIA / University of Pau
 ! (c) Princeton University / California Institute of Technology and CNRS / INRIA / University of Pau
 !                             July 2012
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// prepares a device array with with all inter-element edge-nodes -- this
// is followed by a memcpy and MPI operations
__global__ void prepare_boundary_potential_on_device(realw* d_potential_dot_dot_acoustic,
                                                     realw* d_send_potential_dot_dot_buffer,
                                                     int num_interfaces_ext_mesh,
                                                     int max_nibool_interfaces_ext_mesh,
                                                     int* d_nibool_interfaces_ext_mesh,
                                                     int* d_ibool_interfaces_ext_mesh) {

  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
  int iinterface=0;

  for( iinterface=0; iinterface < num_interfaces_ext_mesh; iinterface++) {
    if(id<d_nibool_interfaces_ext_mesh[iinterface]) {
      d_send_potential_dot_dot_buffer[(id + max_nibool_interfaces_ext_mesh*iinterface)] =
        d_potential_dot_dot_acoustic[(d_ibool_interfaces_ext_mesh[id+max_nibool_interfaces_ext_mesh*iinterface]-1)];
    }
  }

}


/* ----------------------------------------------------------------------------------------------- */

// prepares and transfers the inter-element edge-nodes to the host to be MPI'd
extern "C"
void FC_FUNC_(transfer_boun_pot_from_device,
              TRANSFER_BOUN_POT_FROM_DEVICE)(
                                              int* size,
                                              long* Mesh_pointer_f,
                                              realw* potential_dot_dot_acoustic,
                                              realw* send_potential_dot_dot_buffer,
                                              int* num_interfaces_ext_mesh,
                                              int* max_nibool_interfaces_ext_mesh,
                                              int* nibool_interfaces_ext_mesh,
                                              int* ibool_interfaces_ext_mesh,
                                              int* FORWARD_OR_ADJOINT){

TRACE("transfer_boun_pot_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  if( *num_interfaces_ext_mesh == 0 ) return;

  int blocksize = BLOCKSIZE_TRANSFER;
  int size_padded = ((int)ceil(((double)(mp->max_nibool_interfaces_ext_mesh))/((double)blocksize)))*blocksize;
  int num_blocks_x = size_padded/blocksize;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  if(*FORWARD_OR_ADJOINT == 1) {
    prepare_boundary_potential_on_device<<<grid,threads>>>(mp->d_potential_dot_dot_acoustic,
                                                           mp->d_send_potential_dot_dot_buffer,
                                                           mp->num_interfaces_ext_mesh,
                                                           mp->max_nibool_interfaces_ext_mesh,
                                                           mp->d_nibool_interfaces_ext_mesh,
                                                           mp->d_ibool_interfaces_ext_mesh);
  }
  else if(*FORWARD_OR_ADJOINT == 3) {
    prepare_boundary_potential_on_device<<<grid,threads>>>(mp->d_b_potential_dot_dot_acoustic,
                                                           mp->d_send_potential_dot_dot_buffer,
                                                           mp->num_interfaces_ext_mesh,
                                                           mp->max_nibool_interfaces_ext_mesh,
                                                           mp->d_nibool_interfaces_ext_mesh,
                                                           mp->d_ibool_interfaces_ext_mesh);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after prepare_boundary_potential_on_device");
#endif

  print_CUDA_error_if_any(hipMemcpy(send_potential_dot_dot_buffer,mp->d_send_potential_dot_dot_buffer,
      (mp->max_nibool_interfaces_ext_mesh)*(mp->num_interfaces_ext_mesh)*sizeof(realw),hipMemcpyDeviceToHost),98000);

  // finish timing of kernel+memcpy
  // hipEventRecord( stop, 0 );
  // hipEventSynchronize( stop );
  // hipEventElapsedTime( &time, start, stop );
  // hipEventDestroy( start );
  // hipEventDestroy( stop );
  // printf("boundary xfer d->h Time: %f ms\n",time);
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("transfer_boun_pot_from_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */


__global__ void assemble_boundary_potential_on_device(realw* d_potential_dot_dot_acoustic,
                                                      realw* d_send_potential_dot_dot_buffer,
                                                      int num_interfaces_ext_mesh,
                                                      int max_nibool_interfaces_ext_mesh,
                                                      int* d_nibool_interfaces_ext_mesh,
                                                      int* d_ibool_interfaces_ext_mesh) {

  int id = threadIdx.x + blockIdx.x*blockDim.x + blockIdx.y*gridDim.x*blockDim.x;
  int iinterface=0;

  for( iinterface=0; iinterface < num_interfaces_ext_mesh; iinterface++) {
    if(id<d_nibool_interfaces_ext_mesh[iinterface]) {

      // for testing atomic operations against not atomic operations (0.1ms vs. 0.04 ms)
      // d_potential_dot_dot_acoustic[3*(d_ibool_interfaces_ext_mesh[id+max_nibool_interfaces_ext_mesh*iinterface]-1)] +=
      // d_send_potential_dot_dot_buffer[3*(id + max_nibool_interfaces_ext_mesh*iinterface)];

      atomicAdd(&d_potential_dot_dot_acoustic[(d_ibool_interfaces_ext_mesh[id+max_nibool_interfaces_ext_mesh*iinterface]-1)],
                d_send_potential_dot_dot_buffer[(id + max_nibool_interfaces_ext_mesh*iinterface)]);
    }
  }
  // ! This step is done via previous function transfer_and_assemble...
  // ! do iinterface = 1, num_interfaces_ext_mesh
  // !   do ipoin = 1, nibool_interfaces_ext_mesh(iinterface)
  // !     array_val(:,ibool_interfaces_ext_mesh(ipoin,iinterface)) = &
  // !          array_val(:,ibool_interfaces_ext_mesh(ipoin,iinterface)) + buffer_recv_vector_ext_mesh(:,ipoin,iinterface)
  // !   enddo
  // ! enddo
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_asmbl_pot_to_device,
              TRANSFER_ASMBL_POT_TO_DEVICE)(
                                                long* Mesh_pointer,
                                                realw* potential_dot_dot_acoustic,
                                                realw* buffer_recv_scalar_ext_mesh,
                                                int* num_interfaces_ext_mesh,
                                                int* max_nibool_interfaces_ext_mesh,
                                                int* nibool_interfaces_ext_mesh,
                                                int* ibool_interfaces_ext_mesh,
                                                int* FORWARD_OR_ADJOINT) {

TRACE("transfer_asmbl_pot_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  //double start_time = get_time();
  // hipEvent_t start, stop;
  // realw time;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord( start, 0 );

  if( *num_interfaces_ext_mesh == 0 ) return;

  // copies buffer onto GPU
  hipMemcpy(mp->d_send_potential_dot_dot_buffer, buffer_recv_scalar_ext_mesh,
             (mp->max_nibool_interfaces_ext_mesh)*(mp->num_interfaces_ext_mesh)*sizeof(realw), hipMemcpyHostToDevice);

  // assembles on GPU
  int blocksize = BLOCKSIZE_TRANSFER;
  int size_padded = ((int)ceil(((double)mp->max_nibool_interfaces_ext_mesh)/((double)blocksize)))*blocksize;
  int num_blocks_x = size_padded/blocksize;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  if(*FORWARD_OR_ADJOINT == 1) {
    //assemble forward field
    assemble_boundary_potential_on_device<<<grid,threads>>>(mp->d_potential_dot_dot_acoustic,
                                                          mp->d_send_potential_dot_dot_buffer,
                                                          mp->num_interfaces_ext_mesh,
                                                          mp->max_nibool_interfaces_ext_mesh,
                                                          mp->d_nibool_interfaces_ext_mesh,
                                                          mp->d_ibool_interfaces_ext_mesh);
  }
  else if(*FORWARD_OR_ADJOINT == 3) {
    //assemble reconstructed/backward field
    assemble_boundary_potential_on_device<<<grid,threads>>>(mp->d_b_potential_dot_dot_acoustic,
                                                            mp->d_send_potential_dot_dot_buffer,
                                                            mp->num_interfaces_ext_mesh,
                                                            mp->max_nibool_interfaces_ext_mesh,
                                                            mp->d_nibool_interfaces_ext_mesh,
                                                            mp->d_ibool_interfaces_ext_mesh);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  // hipEventRecord( stop, 0 );
  // hipEventSynchronize( stop );
  // hipEventElapsedTime( &time, start, stop );
  // hipEventDestroy( start );
  // hipEventDestroy( stop );
  // printf("Boundary Assemble Kernel Execution Time: %f ms\n",time);
  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);
  exit_on_cuda_error("transfer_asmbl_pot_to_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

/* KERNEL 2 */

/* ----------------------------------------------------------------------------------------------- */


__global__ void Kernel_2_acoustic_impl(int nb_blocks_to_compute,
                                       int NGLOB, int* d_ibool,
                                       int* d_phase_ispec_inner_acoustic,
                                       int num_phase_ispec_acoustic,
                                       int d_iphase,
                                       int use_mesh_coloring_gpu,
                                       realw* d_potential_acoustic, realw* d_potential_dot_dot_acoustic,
                                       realw* d_xix, realw* d_xiy, realw* d_xiz,
                                       realw* d_etax, realw* d_etay, realw* d_etaz,
                                       realw* d_gammax, realw* d_gammay, realw* d_gammaz,
                                       realw* d_hprime_xx,
                                       realw* hprimewgll_xx,
                                       realw* wgllwgll_xy,realw* wgllwgll_xz,realw* wgllwgll_yz,
                                       realw* d_rhostore,
                                       int gravity,
                                       realw* minus_g,
                                       realw* d_kappastore,
                                       realw* wgll_cube){

  int bx = blockIdx.y*gridDim.x+blockIdx.x;
  int tx = threadIdx.x;

  //const int NGLL3 = NGLL3;
  const int NGLL3_ALIGN = NGLL3_PADDED;

  int K = (tx/NGLL2);
  int J = ((tx-K*NGLL2)/NGLLX);
  int I = (tx-K*NGLL2-J*NGLLX);

  int active,offset;
  int iglob = 0;
  int working_element;
  realw temp1l,temp2l,temp3l;
  realw xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl,jacobianl;
  realw dpotentialdxl,dpotentialdyl,dpotentialdzl;
  realw fac1,fac2,fac3;
  realw rho_invl,kappa_invl;
  realw sum_terms;
  realw gravity_term;

#ifndef MANUALLY_UNROLLED_LOOPS
  int l;
  int offset1,offset2,offset3;
  realw hp1,hp2,hp3;
#endif

  __shared__ realw s_dummy_loc[NGLL3];

  __shared__ realw s_temp1[NGLL3];
  __shared__ realw s_temp2[NGLL3];
  __shared__ realw s_temp3[NGLL3];

// use only NGLL^3 = 125 active threads, plus 3 inactive/ghost threads,
// because we used memory padding from NGLL^3 = 125 to 128 to get coalescent memory accesses
  active = (tx < NGLL3 && bx < nb_blocks_to_compute) ? 1:0;

// copy from global memory to shared memory
// each thread writes one of the NGLL^3 = 125 data points
  if (active) {

#ifdef USE_MESH_COLORING_GPU
    working_element = bx;
#else
    //mesh coloring
    if( use_mesh_coloring_gpu ){
      working_element = bx;
    }else{
      // iphase-1 and working_element-1 for Fortran->C array conventions
      working_element = d_phase_ispec_inner_acoustic[bx + num_phase_ispec_acoustic*(d_iphase-1)]-1;
    }
#endif

    // iglob = d_ibool[working_element*NGLL3_ALIGN + tx]-1;
    iglob = d_ibool[working_element*NGLL3 + tx]-1;

#ifdef USE_TEXTURES
    s_dummy_loc[tx] = tex1Dfetch(tex_potential_acoustic, iglob);
#else
    // changing iglob indexing to match fortran row changes fast style
    s_dummy_loc[tx] = d_potential_acoustic[iglob];
#endif
  }

// synchronize all the threads (one thread for each of the NGLL grid points of the
// current spectral element) because we need the whole element to be ready in order
// to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

#ifndef MAKE_KERNEL2_BECOME_STUPID_FOR_TESTS

  if (active) {

#ifndef MANUALLY_UNROLLED_LOOPS

    temp1l = 0.f;
    temp2l = 0.f;
    temp3l = 0.f;

    for (l=0;l<NGLLX;l++) {
        hp1 = d_hprime_xx[l*NGLLX+I];
        offset1 = K*NGLL2+J*NGLLX+l;
        temp1l += s_dummy_loc[offset1]*hp1;

        //assumes that hprime_xx = hprime_yy = hprime_zz
        hp2 = d_hprime_xx[l*NGLLX+J];
        offset2 = K*NGLL2+l*NGLLX+I;
        temp2l += s_dummy_loc[offset2]*hp2;

        hp3 = d_hprime_xx[l*NGLLX+K];
        offset3 = l*NGLL2+J*NGLLX+I;
        temp3l += s_dummy_loc[offset3]*hp3;
    }
#else

    temp1l = s_dummy_loc[K*NGLL2+J*NGLLX]*d_hprime_xx[I]
            + s_dummy_loc[K*NGLL2+J*NGLLX+1]*d_hprime_xx[NGLLX+I]
            + s_dummy_loc[K*NGLL2+J*NGLLX+2]*d_hprime_xx[2*NGLLX+I]
            + s_dummy_loc[K*NGLL2+J*NGLLX+3]*d_hprime_xx[3*NGLLX+I]
            + s_dummy_loc[K*NGLL2+J*NGLLX+4]*d_hprime_xx[4*NGLLX+I];

    temp2l = s_dummy_loc[K*NGLL2+I]*d_hprime_xx[J]
            + s_dummy_loc[K*NGLL2+NGLLX+I]*d_hprime_xx[NGLLX+J]
            + s_dummy_loc[K*NGLL2+2*NGLLX+I]*d_hprime_xx[2*NGLLX+J]
            + s_dummy_loc[K*NGLL2+3*NGLLX+I]*d_hprime_xx[3*NGLLX+J]
            + s_dummy_loc[K*NGLL2+4*NGLLX+I]*d_hprime_xx[4*NGLLX+J];

    temp3l = s_dummy_loc[J*NGLLX+I]*d_hprime_xx[K]
            + s_dummy_loc[NGLL2+J*NGLLX+I]*d_hprime_xx[NGLLX+K]
            + s_dummy_loc[2*NGLL2+J*NGLLX+I]*d_hprime_xx[2*NGLLX+K]
            + s_dummy_loc[3*NGLL2+J*NGLLX+I]*d_hprime_xx[3*NGLLX+K]
            + s_dummy_loc[4*NGLL2+J*NGLLX+I]*d_hprime_xx[4*NGLLX+K];

#endif

    // compute derivatives of ux, uy and uz with respect to x, y and z
    offset = working_element*NGLL3_ALIGN + tx;

    xixl = d_xix[offset];
    xiyl = d_xiy[offset];
    xizl = d_xiz[offset];
    etaxl = d_etax[offset];
    etayl = d_etay[offset];
    etazl = d_etaz[offset];
    gammaxl = d_gammax[offset];
    gammayl = d_gammay[offset];
    gammazl = d_gammaz[offset];

    jacobianl = 1.f / (xixl*(etayl*gammazl-etazl*gammayl)
                      -xiyl*(etaxl*gammazl-etazl*gammaxl)
                      +xizl*(etaxl*gammayl-etayl*gammaxl));

    // derivatives of potential
    dpotentialdxl = xixl*temp1l + etaxl*temp2l + gammaxl*temp3l;
    dpotentialdyl = xiyl*temp1l + etayl*temp2l + gammayl*temp3l;
    dpotentialdzl = xizl*temp1l + etazl*temp2l + gammazl*temp3l;

    // pre-computes gravity sum term
    if( gravity ){
      // uses potential definition: s = grad(chi)

      // gravity term: 1/kappa grad(chi) * g
      // assumes that g only acts in (negative) z-direction
      kappa_invl = 1.f / d_kappastore[working_element*NGLL3 + tx];
      iglob = d_ibool[working_element*NGLL3 + tx]-1;

      // daniel: TODO - check gravity
//      if( kappa_invl <= 0.0f ){
//        printf("kappa error: %f %f\n",kappa_invl,d_kappastore[working_element*NGLL3 + tx]);
//        printf("kappa error: thread %d %d \n",tx,working_element);
//        asm("trap;");
//      }
//      if( iglob <= 0 ){
//        printf("iglob error: %d %d %d \n",iglob,tx,working_element);
//        asm("trap;");
//      }

      gravity_term = minus_g[iglob] * kappa_invl * jacobianl * wgll_cube[tx] * dpotentialdzl;

      // daniel: TODO - check gravity
      //gravity_term = 0.f;
      //if( iglob == 5 ){
      //  printf("iglob infos: %f %f %f %f %f \n",minus_g[iglob],kappa_invl,jacobianl,wgll_cube[tx],dpotentialdzl);
      //}
    }

    // density (reciproc)
    rho_invl = 1.f / d_rhostore[offset];

    // form the dot product with the test vector
    s_temp1[tx] = jacobianl * rho_invl * (dpotentialdxl*xixl + dpotentialdyl*xiyl + dpotentialdzl*xizl);
    s_temp2[tx] = jacobianl * rho_invl * (dpotentialdxl*etaxl + dpotentialdyl*etayl + dpotentialdzl*etazl);
    s_temp3[tx] = jacobianl * rho_invl * (dpotentialdxl*gammaxl + dpotentialdyl*gammayl + dpotentialdzl*gammazl);
  }

// synchronize all the threads (one thread for each of the NGLL grid points of the
// current spectral element) because we need the whole element to be ready in order
// to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  if (active) {

#ifndef MANUALLY_UNROLLED_LOOPS

    temp1l = 0.f;
    temp2l = 0.f;
    temp3l = 0.f;

    for (l=0;l<NGLLX;l++) {
        fac1 = hprimewgll_xx[I*NGLLX+l];
        offset1 = K*NGLL2+J*NGLLX+l;
        temp1l += s_temp1[offset1]*fac1;

        //assumes hprimewgll_xx = hprimewgll_yy = hprimewgll_zz
        fac2 = hprimewgll_xx[J*NGLLX+l];
        offset2 = K*NGLL2+l*NGLLX+I;
        temp2l += s_temp2[offset2]*fac2;

        fac3 = hprimewgll_xx[K*NGLLX+l];
        offset3 = l*NGLL2+J*NGLLX+I;
        temp3l += s_temp3[offset3]*fac3;
    }
#else

    temp1l = s_temp1[K*NGLL2+J*NGLLX]*hprimewgll_xx[I*NGLLX]
            + s_temp1[K*NGLL2+J*NGLLX+1]*hprimewgll_xx[I*NGLLX+1]
            + s_temp1[K*NGLL2+J*NGLLX+2]*hprimewgll_xx[I*NGLLX+2]
            + s_temp1[K*NGLL2+J*NGLLX+3]*hprimewgll_xx[I*NGLLX+3]
            + s_temp1[K*NGLL2+J*NGLLX+4]*hprimewgll_xx[I*NGLLX+4];


    temp2l = s_temp2[K*NGLL2+I]*hprimewgll_xx[J*NGLLX]
            + s_temp2[K*NGLL2+NGLLX+I]*hprimewgll_xx[J*NGLLX+1]
            + s_temp2[K*NGLL2+2*NGLLX+I]*hprimewgll_xx[J*NGLLX+2]
            + s_temp2[K*NGLL2+3*NGLLX+I]*hprimewgll_xx[J*NGLLX+3]
            + s_temp2[K*NGLL2+4*NGLLX+I]*hprimewgll_xx[J*NGLLX+4];


    temp3l = s_temp3[J*NGLLX+I]*hprimewgll_xx[K*NGLLX]
            + s_temp3[NGLL2+J*NGLLX+I]*hprimewgll_xx[K*NGLLX+1]
            + s_temp3[2*NGLL2+J*NGLLX+I]*hprimewgll_xx[K*NGLLX+2]
            + s_temp3[3*NGLL2+J*NGLLX+I]*hprimewgll_xx[K*NGLLX+3]
            + s_temp3[4*NGLL2+J*NGLLX+I]*hprimewgll_xx[K*NGLLX+4];


#endif

    fac1 = wgllwgll_yz[K*NGLLX+J];
    fac2 = wgllwgll_xz[K*NGLLX+I];
    fac3 = wgllwgll_xy[J*NGLLX+I];

    sum_terms = -(fac1*temp1l + fac2*temp2l + fac3*temp3l);
    if( gravity ) sum_terms += gravity_term;

    iglob = d_ibool[working_element*NGLL3 + tx]-1;

#ifdef USE_TEXTURES
    d_potential_dot_dot_acoustic[iglob] = tex1Dfetch(tex_potential_dot_dot_acoustic, iglob)
                                            + sum_terms;
#else

#ifdef USE_MESH_COLORING_GPU
    // no atomic operation needed, colors don't share global points between elements
    d_potential_dot_dot_acoustic[iglob] += sum_terms;
#else
    //mesh coloring
    if( use_mesh_coloring_gpu ){

      // no atomic operation needed, colors don't share global points between elements
      d_potential_dot_dot_acoustic[iglob] += sum_terms;

    }else{

      atomicAdd(&d_potential_dot_dot_acoustic[iglob],sum_terms);

    }
#endif

#endif
  }

#else  // of #ifndef MAKE_KERNEL2_BECOME_STUPID_FOR_TESTS
  d_potential_dot_dot_acoustic[iglob] = 123.123f;
#endif // of #ifndef MAKE_KERNEL2_BECOME_STUPID_FOR_TESTS
}


/* ----------------------------------------------------------------------------------------------- */

void Kernel_2_acoustic(int nb_blocks_to_compute, Mesh* mp, int d_iphase,
                       int* d_ibool,
                       realw* d_xix,
                       realw* d_xiy,
                       realw* d_xiz,
                       realw* d_etax,
                       realw* d_etay,
                       realw* d_etaz,
                       realw* d_gammax,
                       realw* d_gammay,
                       realw* d_gammaz,
                       realw* d_rhostore,
                       realw* d_kappastore)
{

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("before acoustic kernel Kernel 2");
#endif

  /* if the grid can handle the number of blocks, we let it be 1D */
  /* grid_2_x = nb_elem_color; */
  /* nb_elem_color is just how many blocks we are computing now */

  int num_blocks_x = nb_blocks_to_compute;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  int threads_2 = NGLL3_PADDED;//BLOCK_SIZE_K2;
  dim3 grid_2(num_blocks_x,num_blocks_y);


  // Cuda timing
  // hipEvent_t start, stop;
  // realw time;
  // hipEventCreate(&start);
  // hipEventCreate(&stop);
  // hipEventRecord( start, 0 );

  Kernel_2_acoustic_impl<<< grid_2, threads_2, 0, 0 >>>(nb_blocks_to_compute,
                                                        mp->NGLOB_AB,
                                                        d_ibool,
                                                        mp->d_phase_ispec_inner_acoustic,
                                                        mp->num_phase_ispec_acoustic,
                                                        d_iphase,
                                                        mp->use_mesh_coloring_gpu,
                                                        mp->d_potential_acoustic, mp->d_potential_dot_dot_acoustic,
                                                        d_xix, d_xiy, d_xiz,
                                                        d_etax, d_etay, d_etaz,
                                                        d_gammax, d_gammay, d_gammaz,
                                                        mp->d_hprime_xx,
                                                        mp->d_hprimewgll_xx,
                                                        mp->d_wgllwgll_xy, mp->d_wgllwgll_xz, mp->d_wgllwgll_yz,
                                                        d_rhostore,
                                                        mp->gravity,
                                                        mp->d_minus_g,
                                                        d_kappastore,
                                                        mp->d_wgll_cube);

  if(mp->simulation_type == 3) {
    Kernel_2_acoustic_impl<<< grid_2, threads_2, 0, 0 >>>(nb_blocks_to_compute,
                                                          mp->NGLOB_AB,
                                                          d_ibool,
                                                          mp->d_phase_ispec_inner_acoustic,
                                                          mp->num_phase_ispec_acoustic,
                                                          d_iphase,
                                                          mp->use_mesh_coloring_gpu,
                                                          mp->d_b_potential_acoustic, mp->d_b_potential_dot_dot_acoustic,
                                                          d_xix, d_xiy, d_xiz,
                                                          d_etax, d_etay, d_etaz,
                                                          d_gammax, d_gammay, d_gammaz,
                                                          mp->d_hprime_xx,
                                                          mp->d_hprimewgll_xx,
                                                          mp->d_wgllwgll_xy, mp->d_wgllwgll_xz, mp->d_wgllwgll_yz,
                                                          d_rhostore,
                                                          mp->gravity,
                                                          mp->d_minus_g,
                                                          d_kappastore,
                                                          mp->d_wgll_cube);
  }

  // hipEventRecord( stop, 0 );
  // hipEventSynchronize( stop );
  // hipEventElapsedTime( &time, start, stop );
  // hipEventDestroy( start );
  // hipEventDestroy( stop );
  // printf("Kernel2 Execution Time: %f ms\n",time);

  /* hipDeviceSynchronize(); */
  /* TRACE("Kernel 2 finished"); */
#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //printf("Tried to start with %dx1 blocks\n",nb_blocks_to_compute);
  exit_on_cuda_error("kernel Kernel_2");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// main compute_forces_acoustic CUDA routine

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_forces_acoustic_cuda,
              COMPUTE_FORCES_ACOUSTIC_CUDA)(long* Mesh_pointer_f,
                                            int* iphase,
                                            int* nspec_outer_acoustic,
                                            int* nspec_inner_acoustic) {

  TRACE("compute_forces_acoustic_cuda");
  //double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); // get Mesh from fortran integer wrapper

  int num_elements;

  if( *iphase == 1 )
    num_elements = *nspec_outer_acoustic;
  else
    num_elements = *nspec_inner_acoustic;

  if( num_elements == 0 ) return;

  // mesh coloring
  if( mp->use_mesh_coloring_gpu ){

    // note: array offsets require sorted arrays, such that e.g. ibool starts with elastic elements
    //         and followed by acoustic ones.
    //         acoustic elements also start with outer than inner element ordering

    int nb_colors,nb_blocks_to_compute;
    int istart;
    int color_offset,color_offset_nonpadded;

    // sets up color loop
    if( *iphase == 1 ){
      // outer elements
      nb_colors = mp->num_colors_outer_acoustic;
      istart = 0;

      // array offsets (acoustic elements start after elastic ones)
      color_offset = mp->nspec_elastic * NGLL3_PADDED;
      color_offset_nonpadded = mp->nspec_elastic * NGLL3;
    }else{
      // inner element colors (start after outer elements)
      nb_colors = mp->num_colors_outer_acoustic + mp->num_colors_inner_acoustic;
      istart = mp->num_colors_outer_acoustic;

      // array offsets (inner elements start after outer ones)
      color_offset = ( mp->nspec_elastic + (*nspec_outer_acoustic) ) * NGLL3_PADDED;
      color_offset_nonpadded = ( mp->nspec_elastic + (*nspec_outer_acoustic) ) * NGLL3;
    }

    // loops over colors
    for(int icolor = istart; icolor < nb_colors; icolor++){

      nb_blocks_to_compute = mp->h_num_elem_colors_acoustic[icolor];

      Kernel_2_acoustic(nb_blocks_to_compute,mp,*iphase,
                         mp->d_ibool + color_offset_nonpadded,
                         mp->d_xix + color_offset,
                         mp->d_xiy + color_offset,
                         mp->d_xiz + color_offset,
                         mp->d_etax + color_offset,
                         mp->d_etay + color_offset,
                         mp->d_etaz + color_offset,
                         mp->d_gammax + color_offset,
                         mp->d_gammay + color_offset,
                         mp->d_gammaz + color_offset,
                         mp->d_rhostore + color_offset,
                         mp->d_kappastore + color_offset_nonpadded);

      // for padded and aligned arrays
      color_offset += nb_blocks_to_compute * NGLL3_PADDED;
      // for no-aligned arrays
      color_offset_nonpadded += nb_blocks_to_compute * NGLL3;
    }

  }else{

    // no mesh coloring: uses atomic updates
    Kernel_2_acoustic(num_elements, mp, *iphase,
                      mp->d_ibool,
                      mp->d_xix,
                      mp->d_xiy,
                      mp->d_xiz,
                      mp->d_etax,
                      mp->d_etay,
                      mp->d_etaz,
                      mp->d_gammax,
                      mp->d_gammay,
                      mp->d_gammaz,
                      mp->d_rhostore,
                      mp->d_kappastore);

  }
}



/* ----------------------------------------------------------------------------------------------- */

/* KERNEL for enforce free surface */

/* ----------------------------------------------------------------------------------------------- */


__global__ void enforce_free_surface_cuda_kernel(
                                       realw* potential_acoustic,
                                       realw* potential_dot_acoustic,
                                       realw* potential_dot_dot_acoustic,
                                       int num_free_surface_faces,
                                       int* free_surface_ispec,
                                       int* free_surface_ijk,
                                       int* ibool,
                                       int* ispec_is_acoustic) {
  // gets spectral element face id
  int iface = blockIdx.x + gridDim.x*blockIdx.y;

  // for all faces on free surface
  if( iface < num_free_surface_faces ){

    int ispec = free_surface_ispec[iface]-1;

    // checks if element is in acoustic domain
    if( ispec_is_acoustic[ispec] ){

      // gets global point index
      int igll = threadIdx.x + threadIdx.y*blockDim.x;

      int i = free_surface_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)] - 1; // (1,igll,iface)
      int j = free_surface_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)] - 1;
      int k = free_surface_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)] - 1;

      int iglob = ibool[INDEX4(5,5,5,i,j,k,ispec)] - 1;

      // sets potentials to zero at free surface
      potential_acoustic[iglob] = 0;
      potential_dot_acoustic[iglob] = 0;
      potential_dot_dot_acoustic[iglob] = 0;
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(acoustic_enforce_free_surf_cuda,
              ACOUSTIC_ENFORCE_FREE_SURF_CUDA)(long* Mesh_pointer_f,
                                               int* ABSORB_FREE_SURFACE) {

TRACE("acoustic_enforce_free_surf_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  // checks if anything to do
  if( *ABSORB_FREE_SURFACE == 0 ){

    // does not absorb free surface, thus we enforce the potential to be zero at surface

    // block sizes
    int num_blocks_x = mp->num_free_surface_faces;
    int num_blocks_y = 1;
    while(num_blocks_x > 65535) {
      num_blocks_x = (int) ceil(num_blocks_x*0.5f);
      num_blocks_y = num_blocks_y*2;
    }
    dim3 grid(num_blocks_x,num_blocks_y,1);
    dim3 threads(NGLL2,1,1);

    // sets potentials to zero at free surface
    enforce_free_surface_cuda_kernel<<<grid,threads>>>(mp->d_potential_acoustic,
                                                       mp->d_potential_dot_acoustic,
                                                       mp->d_potential_dot_dot_acoustic,
                                                       mp->num_free_surface_faces,
                                                       mp->d_free_surface_ispec,
                                                       mp->d_free_surface_ijk,
                                                       mp->d_ibool,
                                                       mp->d_ispec_is_acoustic);
    // for backward/reconstructed potentials
    if(mp->simulation_type == 3) {
      enforce_free_surface_cuda_kernel<<<grid,threads>>>(mp->d_b_potential_acoustic,
                                                         mp->d_b_potential_dot_acoustic,
                                                         mp->d_b_potential_dot_dot_acoustic,
                                                         mp->num_free_surface_faces,
                                                         mp->d_free_surface_ispec,
                                                         mp->d_free_surface_ijk,
                                                         mp->d_ibool,
                                                         mp->d_ispec_is_acoustic);

    }
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("enforce_free_surface_cuda");
#endif
}

