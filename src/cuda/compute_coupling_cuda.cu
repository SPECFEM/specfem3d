#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  3 . 0
 !               ---------------------------------------
 !
 !     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
 !                        Princeton University, USA
 !                and CNRS / University of Marseille, France
 !                 (there are currently many more authors!)
 ! (c) Princeton University and CNRS / University of Marseille, July 2012
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

// ACOUSTIC - ELASTIC coupling

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_coupling_acoustic_el_kernel(realw* displ,
                                                    realw* potential_dot_dot_acoustic,
                                                    int num_coupling_ac_el_faces,
                                                    int* coupling_ac_el_ispec,
                                                    int* coupling_ac_el_ijk,
                                                    realw* coupling_ac_el_normal,
                                                    realw* coupling_ac_el_jacobian2Dw,
                                                    int* d_ibool) {

  int igll = threadIdx.x;
  int iface = blockIdx.x + gridDim.x*blockIdx.y;

  int i,j,k,iglob,ispec;
  realw displ_x,displ_y,displ_z,displ_n;
  realw nx,ny,nz;
  realw jacobianw;

  if (iface < num_coupling_ac_el_faces){

    // don't compute points outside NGLLSQUARE==NGLL2==25
    // way 2: no further check needed since blocksize = 25
    //  if (igll<NGLL2) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = coupling_ac_el_ispec[iface] - 1;

    i = coupling_ac_el_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)] - 1;
    j = coupling_ac_el_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)] - 1;
    k = coupling_ac_el_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)] - 1;

    iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)] - 1;

    // elastic displacement on global point
    displ_x = displ[iglob*3] ; // (1,iglob)
    displ_y = displ[iglob*3+1] ; // (2,iglob)
    displ_z = displ[iglob*3+2] ; // (3,iglob)

    // gets associated normal on GLL point
    nx = coupling_ac_el_normal[INDEX3(NDIM,NGLL2,0,igll,iface)]; // (1,igll,iface)
    ny = coupling_ac_el_normal[INDEX3(NDIM,NGLL2,1,igll,iface)]; // (2,igll,iface)
    nz = coupling_ac_el_normal[INDEX3(NDIM,NGLL2,2,igll,iface)]; // (3,igll,iface)

    // calculates displacement component along normal
    // (normal points outwards of acoustic element)
    displ_n = displ_x*nx + displ_y*ny + displ_z*nz;

    // gets associated, weighted jacobian
    jacobianw = coupling_ac_el_jacobian2Dw[INDEX2(NGLL2,igll,iface)];

    // continuity of pressure and normal displacement on global point

    // note: Newmark time scheme together with definition of scalar potential:
    //          pressure = - chi_dot_dot
    //          requires that this coupling term uses the updated displacement at time step [t+delta_t],
    //          which is done at the very beginning of the time loop
    //          (see e.g. Chaljub & Vilotte, Nissen-Meyer thesis...)
    //          it also means you have to calculate and update this here first before
    //          calculating the coupling on the elastic side for the acceleration...
    atomicAdd(&potential_dot_dot_acoustic[iglob],+ jacobianw*displ_n);

  //  }
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_coupling_ac_el_cuda,
              COMPUTE_COUPLING_AC_EL_CUDA)(long* Mesh_pointer,
                                           int* iphasef,
                                           int* num_coupling_ac_el_facesf) {
  TRACE("compute_coupling_ac_el_cuda");
  //double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  int iphase            = *iphasef;
  int num_coupling_ac_el_faces  = *num_coupling_ac_el_facesf;

  // only add these contributions in first pass
  if (iphase != 1) return;

  // way 1: exact blocksize to match NGLLSQUARE
  int blocksize = NGLL2;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(num_coupling_ac_el_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // launches GPU kernel
  compute_coupling_acoustic_el_kernel<<<grid,threads>>>(mp->d_displ,
                                                       mp->d_potential_dot_dot_acoustic,
                                                       num_coupling_ac_el_faces,
                                                       mp->d_coupling_ac_el_ispec,
                                                       mp->d_coupling_ac_el_ijk,
                                                       mp->d_coupling_ac_el_normal,
                                                       mp->d_coupling_ac_el_jacobian2Dw,
                                                       mp->d_ibool);

  //  adjoint simulations
  if (mp->simulation_type == 3){
    compute_coupling_acoustic_el_kernel<<<grid,threads>>>(mp->d_b_displ,
                                                          mp->d_b_potential_dot_dot_acoustic,
                                                          num_coupling_ac_el_faces,
                                                          mp->d_coupling_ac_el_ispec,
                                                          mp->d_coupling_ac_el_ijk,
                                                          mp->d_coupling_ac_el_normal,
                                                          mp->d_coupling_ac_el_jacobian2Dw,
                                                          mp->d_ibool);

  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);
  exit_on_cuda_error("compute_coupling_acoustic_el_kernel");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// ELASTIC - ACOUSTIC coupling

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_coupling_elastic_ac_kernel(realw* potential_dot_dot_acoustic,
                                                    realw* accel,
                                                    int num_coupling_ac_el_faces,
                                                    int* coupling_ac_el_ispec,
                                                    int* coupling_ac_el_ijk,
                                                    realw* coupling_ac_el_normal,
                                                    realw* coupling_ac_el_jacobian2Dw,
                                                    int* d_ibool,
                                                    int gravity,
                                                    realw* minus_g,
                                                    realw* rhostore,
                                                    realw* displ) {

  int igll = threadIdx.x;
  int iface = blockIdx.x + gridDim.x*blockIdx.y;

  int i,j,k,iglob,ispec;
  realw pressure;
  realw nx,ny,nz;
  realw jacobianw;
  realw rhol;

  if (iface < num_coupling_ac_el_faces){

    // don't compute points outside NGLLSQUARE==NGLL2==25
    // way 2: no further check needed since blocksize = 25
    //  if (igll<NGLL2) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = coupling_ac_el_ispec[iface] - 1;

    i = coupling_ac_el_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)] - 1;
    j = coupling_ac_el_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)] - 1;
    k = coupling_ac_el_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)] - 1;

    iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)] - 1;

    // gets associated normal on GLL point
    // note: normal points away from acoustic element
    nx = coupling_ac_el_normal[INDEX3(NDIM,NGLL2,0,igll,iface)]; // (1,igll,iface)
    ny = coupling_ac_el_normal[INDEX3(NDIM,NGLL2,1,igll,iface)]; // (2,igll,iface)
    nz = coupling_ac_el_normal[INDEX3(NDIM,NGLL2,2,igll,iface)]; // (3,igll,iface)

    // gets associated, weighted jacobian
    jacobianw = coupling_ac_el_jacobian2Dw[INDEX2(NGLL2,igll,iface)];

    // acoustic pressure on global point
    if (gravity ){
      // takes density (from acoustic? element)
      rhol = rhostore[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];

      // note: uses potential chi such that displacement s = grad(chi),
      //         pressure becomes: p = - kappa ( div( s ) ) = rho ( - dot_dot_chi + g * s )
      //  g only acting in negative z-direction

      // daniel: TODO - check gravity and coupling would be displ * nz  correct?
      pressure = rhol*( - potential_dot_dot_acoustic[iglob]
                       + minus_g[iglob] * displ[iglob*3+2] );

      //daniel: TODO - check gravity and coupling
      //pressure = - potential_dot_dot_acoustic[iglob] ;
      //if (iface == 128 && igll == 5){
      //  printf("coupling acoustic: %f %f \n",potential_dot_dot_acoustic[iglob],
      //             minus_g[iglob] * displ[iglob*3+2]);
      //}

    }else{
      // no gravity: uses potential chi such that displacement s = 1/rho grad(chi)
      //                  pressure p = - kappa ( div( s )) then becomes: p = - dot_dot_chi
      //                  ( multiplied with factor 1/kappa due to setup of equation of motion )
      pressure = - potential_dot_dot_acoustic[iglob];
    }

    // continuity of displacement and pressure on global point
    //
    // note: Newmark time scheme together with definition of scalar potential:
    //          pressure = - chi_dot_dot
    //          requires that this coupling term uses the *UPDATED* pressure (chi_dot_dot), i.e.
    //          pressure at time step [t + delta_t]
    //          (see e.g. Chaljub & Vilotte, Nissen-Meyer thesis...)
    //          it means you have to calculate and update the acoustic pressure first before
    //          calculating this term...
    atomicAdd(&accel[iglob*3],+ jacobianw*nx*pressure);
    atomicAdd(&accel[iglob*3+1],+ jacobianw*ny*pressure);
    atomicAdd(&accel[iglob*3+2],+ jacobianw*nz*pressure);

    //  }
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_coupling_el_ac_cuda,
              COMPUTE_COUPLING_EL_AC_CUDA)(long* Mesh_pointer,
                                           int* iphasef,
                                           int* num_coupling_ac_el_facesf) {
  TRACE("compute_coupling_el_ac_cuda");
  //double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  int iphase            = *iphasef;
  int num_coupling_ac_el_faces  = *num_coupling_ac_el_facesf;

  // only add these contributions in first pass
  if (iphase != 1) return;

  // way 1: exact blocksize to match NGLLSQUARE
  int blocksize = 25;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(num_coupling_ac_el_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // launches GPU kernel
  compute_coupling_elastic_ac_kernel<<<grid,threads>>>(mp->d_potential_dot_dot_acoustic,
                                                       mp->d_accel,
                                                       num_coupling_ac_el_faces,
                                                       mp->d_coupling_ac_el_ispec,
                                                       mp->d_coupling_ac_el_ijk,
                                                       mp->d_coupling_ac_el_normal,
                                                       mp->d_coupling_ac_el_jacobian2Dw,
                                                       mp->d_ibool,
                                                       mp->gravity,
                                                       mp->d_minus_g,
                                                       mp->d_rhostore,
                                                       mp->d_displ);

  //  adjoint simulations
  if (mp->simulation_type == 3){
    compute_coupling_elastic_ac_kernel<<<grid,threads>>>(mp->d_b_potential_dot_dot_acoustic,
                                                         mp->d_b_accel,
                                                         num_coupling_ac_el_faces,
                                                         mp->d_coupling_ac_el_ispec,
                                                         mp->d_coupling_ac_el_ijk,
                                                         mp->d_coupling_ac_el_normal,
                                                         mp->d_coupling_ac_el_jacobian2Dw,
                                                         mp->d_ibool,
                                                         mp->gravity,
                                                         mp->d_minus_g,
                                                         mp->d_rhostore,
                                                         mp->d_b_displ);

  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);
  exit_on_cuda_error("compute_coupling_el_ac_cuda");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

/* APPROXIMATE_OCEAN_LOAD load on free surface */

/* ----------------------------------------------------------------------------------------------- */


__global__ void compute_coupling_ocean_cuda_kernel(realw* accel,
                                               realw* rmassx,realw* rmassy,realw* rmassz,
                                               realw* rmass_ocean_load,
                                               int num_free_surface_faces,
                                               int* free_surface_ispec,
                                               int* free_surface_ijk,
                                               realw* free_surface_normal,
                                               int* d_ibool,
                                               int* updated_dof_ocean_load) {
  // gets spectral element face id
  int igll = threadIdx.x ;  //  threadIdx.y*blockDim.x will be always = 0 for thread block (25,1,1)
  int iface = blockIdx.x + gridDim.x*blockIdx.y;

  realw nx,ny,nz;
  realw force_normal_comp;

  // for all faces on free surface
  if (iface < num_free_surface_faces ){

    int ispec = free_surface_ispec[iface]-1;

    // gets global point index
    int i = free_surface_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)] - 1; // (1,igll,iface)
    int j = free_surface_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)] - 1;
    int k = free_surface_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)] - 1;

    int iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)] - 1;

    //if (igll == 0) printf("igll %d %d %d %d\n",igll,i,j,k,iglob);

    // only update this global point once

    // daniel: TODO - there might be better ways to implement a mutex like below,
    //            and find a workaround to not use the temporary update array.
    //            atomicExch: returns the old value, i.e. 0 indicates that we still have to do this point

    if (atomicExch(&updated_dof_ocean_load[iglob],1) == 0){

      // get normal
      nx = free_surface_normal[INDEX3(NDIM,NGLL2,0,igll,iface)]; //(1,igll,iface)
      ny = free_surface_normal[INDEX3(NDIM,NGLL2,1,igll,iface)];
      nz = free_surface_normal[INDEX3(NDIM,NGLL2,2,igll,iface)];

      // make updated component of right-hand side
      // we divide by rmass() which is 1 / M
      // we use the total force which includes the Coriolis term above
      force_normal_comp = accel[iglob*3]*nx / rmassx[iglob]
                          + accel[iglob*3+1]*ny / rmassy[iglob]
                          + accel[iglob*3+2]*nz / rmassz[iglob];

      // probably wouldn't need atomicAdd anymore, but just to be sure...
      atomicAdd(&accel[iglob*3],   + (rmass_ocean_load[iglob] - rmassx[iglob]) * force_normal_comp * nx);
      atomicAdd(&accel[iglob*3+1], + (rmass_ocean_load[iglob] - rmassy[iglob]) * force_normal_comp * ny);
      atomicAdd(&accel[iglob*3+2], + (rmass_ocean_load[iglob] - rmassz[iglob]) * force_normal_comp * nz);
    }
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_coupling_ocean_cuda,
              COMPUTE_COUPLING_OCEAN_CUDA)(long* Mesh_pointer) {

  TRACE("\tcompute_coupling_ocean_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks if anything to do
  if (mp->num_free_surface_faces == 0) return;

  // block sizes: exact blocksize to match NGLLSQUARE
  int blocksize = NGLL2;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->num_free_surface_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);


  // initializes temporary array to zero
  print_CUDA_error_if_any(hipMemset(mp->d_updated_dof_ocean_load,0,
                                     sizeof(int)*mp->NGLOB_AB),88501);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("before kernel compute_coupling_ocean_cuda");
#endif

  compute_coupling_ocean_cuda_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_accel,
                                                                           mp->d_rmassx,mp->d_rmassy,mp->d_rmassz,
                                                                           mp->d_rmass_ocean_load,
                                                                           mp->num_free_surface_faces,
                                                                           mp->d_free_surface_ispec,
                                                                           mp->d_free_surface_ijk,
                                                                           mp->d_free_surface_normal,
                                                                           mp->d_ibool,
                                                                           mp->d_updated_dof_ocean_load);
  // for backward/reconstructed potentials
  if (mp->simulation_type == 3) {
    // re-initializes array
    print_CUDA_error_if_any(hipMemset(mp->d_updated_dof_ocean_load,0,
                                       sizeof(int)*mp->NGLOB_AB),88502);

    compute_coupling_ocean_cuda_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_b_accel,
                                                                             mp->d_rmassx,mp->d_rmassy,mp->d_rmassz,
                                                                             mp->d_rmass_ocean_load,
                                                                             mp->num_free_surface_faces,
                                                                             mp->d_free_surface_ispec,
                                                                             mp->d_free_surface_ijk,
                                                                             mp->d_free_surface_normal,
                                                                             mp->d_ibool,
                                                                             mp->d_updated_dof_ocean_load);

  }


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_coupling_ocean_cuda");
#endif
}

