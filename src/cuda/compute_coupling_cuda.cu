#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 1
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and University of Pau / CNRS / INRIA
 ! (c) Princeton University / California Institute of Technology and University of Pau / CNRS / INRIA
 !                            April 2011
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// ACOUSTIC - ELASTIC coupling

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_coupling_acoustic_el_kernel(realw* displ,
                                                    realw* potential_dot_dot_acoustic,
                                                    int num_coupling_ac_el_faces,
                                                    int* coupling_ac_el_ispec,
                                                    int* coupling_ac_el_ijk,
                                                    realw* coupling_ac_el_normal,
                                                    realw* coupling_ac_el_jacobian2Dw,
                                                    int* ibool,
                                                    int* ispec_is_inner,
                                                    int phase_is_inner) {

  int igll = threadIdx.x;
  int iface = blockIdx.x + gridDim.x*blockIdx.y;

  int i,j,k,iglob,ispec;
  realw displ_x,displ_y,displ_z,displ_n;
  realw nx,ny,nz;
  realw jacobianw;

  if( iface < num_coupling_ac_el_faces){

    // don't compute points outside NGLLSQUARE==NGLL2==25
    // way 2: no further check needed since blocksize = 25
    //  if(igll<NGLL2) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = coupling_ac_el_ispec[iface] - 1;

    if(ispec_is_inner[ispec] == phase_is_inner ) {

      i = coupling_ac_el_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)] - 1;
      j = coupling_ac_el_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)] - 1;
      k = coupling_ac_el_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)] - 1;
      iglob = ibool[INDEX4(5,5,5,i,j,k,ispec)] - 1;

      // elastic displacement on global point
      displ_x = displ[iglob*3] ; // (1,iglob)
      displ_y = displ[iglob*3+1] ; // (2,iglob)
      displ_z = displ[iglob*3+2] ; // (3,iglob)

      // gets associated normal on GLL point
      nx = coupling_ac_el_normal[INDEX3(NDIM,NGLL2,0,igll,iface)]; // (1,igll,iface)
      ny = coupling_ac_el_normal[INDEX3(NDIM,NGLL2,1,igll,iface)]; // (2,igll,iface)
      nz = coupling_ac_el_normal[INDEX3(NDIM,NGLL2,2,igll,iface)]; // (3,igll,iface)

      // calculates displacement component along normal
      // (normal points outwards of acoustic element)
      displ_n = displ_x*nx + displ_y*ny + displ_z*nz;

      // gets associated, weighted jacobian
      jacobianw = coupling_ac_el_jacobian2Dw[INDEX2(NGLL2,igll,iface)];

      // continuity of pressure and normal displacement on global point

      // note: Newmark time scheme together with definition of scalar potential:
      //          pressure = - chi_dot_dot
      //          requires that this coupling term uses the updated displacement at time step [t+delta_t],
      //          which is done at the very beginning of the time loop
      //          (see e.g. Chaljub & Vilotte, Nissen-Meyer thesis...)
      //          it also means you have to calculate and update this here first before
      //          calculating the coupling on the elastic side for the acceleration...
      atomicAdd(&potential_dot_dot_acoustic[iglob],+ jacobianw*displ_n);

    }
  //  }
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_coupling_ac_el_cuda,
              COMPUTE_COUPLING_AC_EL_CUDA)(long* Mesh_pointer_f,
                                           int* phase_is_innerf,
                                           int* num_coupling_ac_el_facesf,
                                           int* SIMULATION_TYPEf) {
  TRACE("compute_coupling_ac_el_cuda");
  //double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container
  int phase_is_inner            = *phase_is_innerf;
  int num_coupling_ac_el_faces  = *num_coupling_ac_el_facesf;
  int SIMULATION_TYPE           = *SIMULATION_TYPEf;

  // way 1: exact blocksize to match NGLLSQUARE
  int blocksize = NGLL2;
  int num_blocks_x = num_coupling_ac_el_faces;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // launches GPU kernel
  compute_coupling_acoustic_el_kernel<<<grid,threads>>>(mp->d_displ,
                                                       mp->d_potential_dot_dot_acoustic,
                                                       num_coupling_ac_el_faces,
                                                       mp->d_coupling_ac_el_ispec,
                                                       mp->d_coupling_ac_el_ijk,
                                                       mp->d_coupling_ac_el_normal,
                                                       mp->d_coupling_ac_el_jacobian2Dw,
                                                       mp->d_ibool,
                                                       mp->d_ispec_is_inner,
                                                       phase_is_inner);

  //  adjoint simulations
  if (SIMULATION_TYPE == 3 ){
    compute_coupling_acoustic_el_kernel<<<grid,threads>>>(mp->d_b_displ,
                                                          mp->d_b_potential_dot_dot_acoustic,
                                                          num_coupling_ac_el_faces,
                                                          mp->d_coupling_ac_el_ispec,
                                                          mp->d_coupling_ac_el_ijk,
                                                          mp->d_coupling_ac_el_normal,
                                                          mp->d_coupling_ac_el_jacobian2Dw,
                                                          mp->d_ibool,
                                                          mp->d_ispec_is_inner,
                                                          phase_is_inner);

  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);
  exit_on_cuda_error("compute_coupling_acoustic_el_kernel");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// ELASTIC - ACOUSTIC coupling

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_coupling_elastic_ac_kernel(realw* potential_dot_dot_acoustic,
                                                    realw* accel,
                                                    int num_coupling_ac_el_faces,
                                                    int* coupling_ac_el_ispec,
                                                    int* coupling_ac_el_ijk,
                                                    realw* coupling_ac_el_normal,
                                                    realw* coupling_ac_el_jacobian2Dw,
                                                    int* ibool,
                                                    int* ispec_is_inner,
                                                    int phase_is_inner,
                                                    int gravity,
                                                    realw* minus_g,
                                                    realw* rhostore,
                                                    realw* displ) {

  int igll = threadIdx.x;
  int iface = blockIdx.x + gridDim.x*blockIdx.y;

  int i,j,k,iglob,ispec;
  realw pressure;
  realw nx,ny,nz;
  realw jacobianw;
  realw rhol;

  if( iface < num_coupling_ac_el_faces){

    // don't compute points outside NGLLSQUARE==NGLL2==25
    // way 2: no further check needed since blocksize = 25
    //  if(igll<NGLL2) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = coupling_ac_el_ispec[iface] - 1;

    if(ispec_is_inner[ispec] == phase_is_inner ) {

      i = coupling_ac_el_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)] - 1;
      j = coupling_ac_el_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)] - 1;
      k = coupling_ac_el_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)] - 1;
      iglob = ibool[INDEX4(5,5,5,i,j,k,ispec)] - 1;

      // gets associated normal on GLL point
      // note: normal points away from acoustic element
      nx = coupling_ac_el_normal[INDEX3(NDIM,NGLL2,0,igll,iface)]; // (1,igll,iface)
      ny = coupling_ac_el_normal[INDEX3(NDIM,NGLL2,1,igll,iface)]; // (2,igll,iface)
      nz = coupling_ac_el_normal[INDEX3(NDIM,NGLL2,2,igll,iface)]; // (3,igll,iface)

      // gets associated, weighted jacobian
      jacobianw = coupling_ac_el_jacobian2Dw[INDEX2(NGLL2,igll,iface)];

      // acoustic pressure on global point
      if( gravity ){
        // takes density (from acoustic? element)
        rhol = rhostore[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];

        // note: uses potential chi such that displacement s = grad(chi),
        //         pressure becomes: p = - kappa ( div( s ) ) = rho ( - dot_dot_chi + g * s )
        //  g only acting in negative z-direction

        // daniel: TODO - check gravity and coupling would be displ * nz  correct?
        pressure = rhol*( - potential_dot_dot_acoustic[iglob]
                         + minus_g[iglob] * displ[iglob*3+2] );

        //daniel: TODO - check gravity and coupling
        //pressure = - potential_dot_dot_acoustic[iglob] ;
        //if( iface == 128 && igll == 5 ){
        //  printf("coupling acoustic: %f %f \n",potential_dot_dot_acoustic[iglob],
        //             minus_g[iglob] * displ[iglob*3+2]);
        //}

      }else{
        // no gravity: uses potential chi such that displacement s = 1/rho grad(chi)
        //                  pressure p = - kappa ( div( s ) ) then becomes: p = - dot_dot_chi
        //                  ( multiplied with factor 1/kappa due to setup of equation of motion )
        pressure = - potential_dot_dot_acoustic[iglob];
      }

      // continuity of displacement and pressure on global point
      //
      // note: Newmark time scheme together with definition of scalar potential:
      //          pressure = - chi_dot_dot
      //          requires that this coupling term uses the *UPDATED* pressure (chi_dot_dot), i.e.
      //          pressure at time step [t + delta_t]
      //          (see e.g. Chaljub & Vilotte, Nissen-Meyer thesis...)
      //          it means you have to calculate and update the acoustic pressure first before
      //          calculating this term...
      atomicAdd(&accel[iglob*3],+ jacobianw*nx*pressure);
      atomicAdd(&accel[iglob*3+1],+ jacobianw*ny*pressure);
      atomicAdd(&accel[iglob*3+2],+ jacobianw*nz*pressure);
    }
    //  }
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(compute_coupling_el_ac_cuda,
              COMPUTE_COUPLING_EL_AC_CUDA)(long* Mesh_pointer_f,
                                           int* phase_is_innerf,
                                           int* num_coupling_ac_el_facesf,
                                           int* SIMULATION_TYPEf) {
  TRACE("compute_coupling_el_ac_cuda");
  //double start_time = get_time();

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container
  int phase_is_inner            = *phase_is_innerf;
  int num_coupling_ac_el_faces  = *num_coupling_ac_el_facesf;
  int SIMULATION_TYPE           = *SIMULATION_TYPEf;

  // way 1: exact blocksize to match NGLLSQUARE
  int blocksize = 25;

  int num_blocks_x = num_coupling_ac_el_faces;
  int num_blocks_y = 1;
  while(num_blocks_x > 65535) {
    num_blocks_x = (int) ceil(num_blocks_x*0.5f);
    num_blocks_y = num_blocks_y*2;
  }

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // launches GPU kernel
  compute_coupling_elastic_ac_kernel<<<grid,threads>>>(mp->d_potential_dot_dot_acoustic,
                                                       mp->d_accel,
                                                       num_coupling_ac_el_faces,
                                                       mp->d_coupling_ac_el_ispec,
                                                       mp->d_coupling_ac_el_ijk,
                                                       mp->d_coupling_ac_el_normal,
                                                       mp->d_coupling_ac_el_jacobian2Dw,
                                                       mp->d_ibool,
                                                       mp->d_ispec_is_inner,
                                                       phase_is_inner,
                                                       mp->gravity,
                                                       mp->d_minus_g,
                                                       mp->d_rhostore,
                                                       mp->d_displ);

  //  adjoint simulations
  if (SIMULATION_TYPE == 3 ){
    compute_coupling_elastic_ac_kernel<<<grid,threads>>>(mp->d_b_potential_dot_dot_acoustic,
                                                         mp->d_b_accel,
                                                         num_coupling_ac_el_faces,
                                                         mp->d_coupling_ac_el_ispec,
                                                         mp->d_coupling_ac_el_ijk,
                                                         mp->d_coupling_ac_el_normal,
                                                         mp->d_coupling_ac_el_jacobian2Dw,
                                                         mp->d_ibool,
                                                         mp->d_ispec_is_inner,
                                                         phase_is_inner,
                                                         mp->gravity,
                                                         mp->d_minus_g,
                                                         mp->d_rhostore,
                                                         mp->d_b_displ);

  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //double end_time = get_time();
  //printf("Elapsed time: %e\n",end_time-start_time);
  exit_on_cuda_error("compute_coupling_el_ac_cuda");
#endif
}
