#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  3 . 0
 !               ---------------------------------------
 !
 !     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
 !                        Princeton University, USA
 !                and CNRS / University of Marseille, France
 !                 (there are currently many more authors!)
 ! (c) Princeton University and CNRS / University of Marseille, July 2012
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_stacey_elastic_kernel(realw* veloc,
                                              realw* accel,
                                              int* abs_boundary_ispec,
                                              int* abs_boundary_ijk,
                                              realw* abs_boundary_normal,
                                              realw* abs_boundary_jacobian2Dw,
                                              int* d_ibool,
                                              realw* rho_vp,
                                              realw* rho_vs,
                                              int* ispec_is_elastic,
                                              int SIMULATION_TYPE,
                                              int SAVE_FORWARD,
                                              int num_abs_boundary_faces,
                                              realw* b_absorb_field) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int i,j,k,iglob,ispec;
  realw vx,vy,vz,vn;
  realw nx,ny,nz;
  realw rho_vp_temp,rho_vs_temp;
  realw tx,ty,tz;
  realw jacobianw;

  // don't compute points outside NGLLSQUARE==NGLL2==25
  // way 2: no further check needed since blocksize = 25
  if (iface < num_abs_boundary_faces){

  //if (igll < NGLL2 && iface < num_abs_boundary_faces) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    if (ispec_is_elastic[ispec]) {

      i = abs_boundary_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)]-1;
      j = abs_boundary_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)]-1;
      k = abs_boundary_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)]-1;

      iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

      // gets associated velocity

      vx = veloc[iglob*3+0];
      vy = veloc[iglob*3+1];
      vz = veloc[iglob*3+2];

      // gets associated normal
      nx = abs_boundary_normal[INDEX3(NDIM,NGLL2,0,igll,iface)];
      ny = abs_boundary_normal[INDEX3(NDIM,NGLL2,1,igll,iface)];
      nz = abs_boundary_normal[INDEX3(NDIM,NGLL2,2,igll,iface)];

      // // velocity component in normal direction (normal points out of element)
      vn = vx*nx + vy*ny + vz*nz;

      rho_vp_temp = rho_vp[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];
      rho_vs_temp = rho_vs[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];

      tx = rho_vp_temp*vn*nx + rho_vs_temp*(vx-vn*nx);
      ty = rho_vp_temp*vn*ny + rho_vs_temp*(vy-vn*ny);
      tz = rho_vp_temp*vn*nz + rho_vs_temp*(vz-vn*nz);

      jacobianw = abs_boundary_jacobian2Dw[INDEX2(NGLL2,igll,iface)];

      atomicAdd(&accel[iglob*3],-tx*jacobianw);
      atomicAdd(&accel[iglob*3+1],-ty*jacobianw);
      atomicAdd(&accel[iglob*3+2],-tz*jacobianw);

      if (SAVE_FORWARD && SIMULATION_TYPE == 1) {
        b_absorb_field[INDEX3(NDIM,NGLL2,0,igll,iface)] = tx*jacobianw;
        b_absorb_field[INDEX3(NDIM,NGLL2,1,igll,iface)] = ty*jacobianw;
        b_absorb_field[INDEX3(NDIM,NGLL2,2,igll,iface)] = tz*jacobianw;
      } // SIMULATION_TYPE
    }
  } // num_abs_boundary_faces

}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_stacey_elastic_sim3_kernel(int* abs_boundary_ispec,
                                                   int* abs_boundary_ijk,
                                                   int* d_ibool,
                                                   int* ispec_is_elastic,
                                                   int num_abs_boundary_faces,
                                                   realw* b_accel,
                                                   realw* b_absorb_field) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int i,j,k,iglob,ispec;

  // don't compute points outside NGLLSQUARE==NGLL2==25
  // way 2: no further check needed since blocksize = 25
  if (iface < num_abs_boundary_faces){

  //if (igll < NGLL2 && iface < num_abs_boundary_faces) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    if (ispec_is_elastic[ispec]) {

      i = abs_boundary_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)]-1;
      j = abs_boundary_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)]-1;
      k = abs_boundary_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)]-1;

      iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

      atomicAdd(&b_accel[iglob*3  ],-b_absorb_field[INDEX3(NDIM,NGLL2,0,igll,iface)]);
      atomicAdd(&b_accel[iglob*3+1],-b_absorb_field[INDEX3(NDIM,NGLL2,1,igll,iface)]);
      atomicAdd(&b_accel[iglob*3+2],-b_absorb_field[INDEX3(NDIM,NGLL2,2,igll,iface)]);
    }
  } // num_abs_boundary_faces

}

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(compute_stacey_viscoelastic_cuda,
              COMPUTE_STACEY_VISCOELASTIC_CUDA)(long* Mesh_pointer,
                                           int* iphasef,
                                           realw* b_absorb_field) {

  TRACE("\tcompute_stacey_viscoelastic_cuda");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  // checks if anything to do
  if (mp->d_num_abs_boundary_faces == 0) return;

  int iphase    = *iphasef;

  // only add these contributions in first pass
  if (iphase != 1) return;

  // way 1
  // > NGLLSQUARE==NGLL2==25, but we handle this inside kernel
  //int blocksize = 32;

  // way 2: seems sligthly faster
  // > NGLLSQUARE==NGLL2==25, no further check inside kernel
  int blocksize = NGLL2;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->d_num_abs_boundary_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  if (mp->simulation_type == 3) {
    // reading is done in fortran routine
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_field,b_absorb_field,
                                       mp->d_b_reclen_field,hipMemcpyHostToDevice),7700);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("between cudamemcpy and compute_stacey_elastic_kernel");
#endif

  compute_stacey_elastic_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_veloc,
                                                  mp->d_accel,
                                                  mp->d_abs_boundary_ispec,
                                                  mp->d_abs_boundary_ijk,
                                                  mp->d_abs_boundary_normal,
                                                  mp->d_abs_boundary_jacobian2Dw,
                                                  mp->d_ibool,
                                                  mp->d_rho_vp,
                                                  mp->d_rho_vs,
                                                  mp->d_ispec_is_elastic,
                                                  mp->simulation_type,
                                                  mp->save_forward,
                                                  mp->d_num_abs_boundary_faces,
                                                  mp->d_b_absorb_field);

  // adjoint simulations
  if (mp->simulation_type == 3){
    compute_stacey_elastic_sim3_kernel<<<grid,threads,0,mp->compute_stream>>>(mp->d_abs_boundary_ispec,
                                                         mp->d_abs_boundary_ijk,
                                                         mp->d_ibool,
                                                         mp->d_ispec_is_elastic,
                                                         mp->d_num_abs_boundary_faces,
                                                         mp->d_b_accel,
                                                         mp->d_b_absorb_field);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("compute_stacey_elastic_kernel");
#endif

  if (mp->simulation_type == 1 && mp->save_forward) {
    // explicitly wait until compute stream is done
    // (hipMemcpy implicitly synchronizes all other cuda operations)
    hipStreamSynchronize(mp->compute_stream);

    // copies absorb_field values to CPU
    print_CUDA_error_if_any(hipMemcpy(b_absorb_field,mp->d_b_absorb_field,
                                       mp->d_b_reclen_field,hipMemcpyDeviceToHost),7701);
    // writing is done in fortran routine
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after compute_stacey_elastic after cudamemcpy");
#endif
}

