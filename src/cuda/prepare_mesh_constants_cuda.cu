/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  3 . 0
 !               ---------------------------------------
 !
 !     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
 !                              CNRS, France
 !                       and Princeton University, USA
 !                 (there are currently many more authors!)
 !                           (c) October 2017
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 3 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include "mesh_constants_cuda.h"
#include "prepare_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

#ifdef USE_OLDER_CUDA4_GPU
#else
  #ifdef USE_TEXTURES_FIELDS
    // elastic
    extern realw_texture d_displ_tex;
    extern realw_texture d_veloc_tex;
    extern realw_texture d_accel_tex;
    // backward/reconstructed
    extern realw_texture d_b_displ_tex;
    extern realw_texture d_b_veloc_tex;
    extern realw_texture d_b_accel_tex;
    // acoustic
    extern realw_texture d_potential_tex;
    extern realw_texture d_potential_dot_dot_tex;
    // backward/reconstructed
    extern realw_texture d_b_potential_tex;
    extern realw_texture d_b_potential_dot_dot_tex;
  #endif
  #ifdef USE_TEXTURES_CONSTANTS
    extern realw_texture d_hprime_xx_tex;
  #endif
#endif


/* ----------------------------------------------------------------------------------------------- */

// helper functions

/* ----------------------------------------------------------------------------------------------- */


// copies integer array from CPU host to GPU device
void copy_todevice_int(void** d_array_addr_ptr,int* h_array,int size){
  TRACE("  copy_todevice_int");

  // allocates memory on GPU
  //
  // note: hipMalloc uses a double-pointer, such that it can return an error code in case it fails
  //          we thus pass the address to the pointer above (as void double-pointer) to have it
  //          pointing to the correct pointer of the array here
  print_CUDA_error_if_any(hipMalloc((void**)d_array_addr_ptr,size*sizeof(int)),
                          12001);

  // copies values onto GPU
  //
  // note: hipMemcpy uses the pointer to the array, we thus re-cast the value of
  //          the double-pointer above to have the correct pointer to the array
  print_CUDA_error_if_any(hipMemcpy((int*) *d_array_addr_ptr,h_array,size*sizeof(int),hipMemcpyHostToDevice),
                          12002);
}

/* ----------------------------------------------------------------------------------------------- */

// copies integer array from CPU host to GPU device
void copy_todevice_realw(void** d_array_addr_ptr,realw* h_array,int size){
  TRACE("  copy_todevice_realw");

  // allocates memory on GPU
  print_CUDA_error_if_any(hipMalloc((void**)d_array_addr_ptr,size*sizeof(realw)),
                          22001);
  // copies values onto GPU
  print_CUDA_error_if_any(hipMemcpy((realw*) *d_array_addr_ptr,h_array,size*sizeof(realw),hipMemcpyHostToDevice),
                          22002);
}


/* ----------------------------------------------------------------------------------------------- */

// GPU preparation

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_constants_device,
              PREPARE_CONSTANTS_DEVICE)(long* Mesh_pointer,
                                        int* h_NGLLX, int* NSPEC_AB, int* NGLOB_AB,
                                        int* NSPEC_IRREGULAR,int* h_irregular_element_number,
                                        realw* h_xix, realw* h_xiy, realw* h_xiz,
                                        realw* h_etax, realw* h_etay, realw* h_etaz,
                                        realw* h_gammax, realw* h_gammay, realw* h_gammaz,
                                        realw* xix_regular, realw* jacobian_regular,
                                        int* h_ibool,
                                        int* num_interfaces_ext_mesh, int* max_nibool_interfaces_ext_mesh,
                                        int* h_nibool_interfaces_ext_mesh, int* h_ibool_interfaces_ext_mesh,
                                        realw* h_hprime_xx, realw* h_hprimewgll_xx,
                                        realw* h_wgllwgll_xy,realw* h_wgllwgll_xz,realw* h_wgllwgll_yz,
                                        int* ABSORBING_CONDITIONS,
                                        int* h_abs_boundary_ispec, int* h_abs_boundary_ijk,
                                        realw* h_abs_boundary_normal,
                                        realw* h_abs_boundary_jacobian2Dw,
                                        int* h_num_abs_boundary_faces,
                                        int* h_ispec_is_inner,
                                        int* NSOURCES, int* nsources_local_f,
                                        realw* h_sourcearrays,
                                        int* h_islice_selected_source, int* h_ispec_selected_source,
                                        int* h_ispec_selected_rec,
                                        int* nrec,int* nrec_local,
                                        int* SIMULATION_TYPE,
                                        int* USE_MESH_COLORING_GPU_f,
                                        int* nspec_acoustic,int* nspec_elastic,
                                        int* h_myrank,
                                        int* SAVE_FORWARD,
                                        realw* h_xir,realw* h_etar, realw* h_gammar,double * nu,
                                        int* islice_selected_rec,
                                        int* NTSTEP_BETWEEN_OUTPUT_SEISMOS,
                                        int* SAVE_SEISMOGRAMS_DISPLACEMENT,int* SAVE_SEISMOGRAMS_VELOCITY,
                                        int* SAVE_SEISMOGRAMS_ACCELERATION,int* SAVE_SEISMOGRAMS_PRESSURE) {

  TRACE("prepare_constants_device");

  // allocates mesh parameter structure
  Mesh* mp = (Mesh*) malloc( sizeof(Mesh) );
  if (mp == NULL) exit_on_error("error allocating mesh pointer");
  *Mesh_pointer = (long)mp;

  // sets processes mpi rank
  mp->myrank = *h_myrank;

  // sets global parameters
  mp->NSPEC_AB = *NSPEC_AB;
  mp->NGLOB_AB = *NGLOB_AB;

  // constants
  mp->simulation_type = *SIMULATION_TYPE;
  mp->absorbing_conditions = *ABSORBING_CONDITIONS;
  mp->save_forward = *SAVE_FORWARD;

  // sets constant arrays
  setConst_hprime_xx(h_hprime_xx,mp);
  // setConst_hprime_yy(h_hprime_yy,mp); // only needed if NGLLX != NGLLY != NGLLZ
  // setConst_hprime_zz(h_hprime_zz,mp); // only needed if NGLLX != NGLLY != NGLLZ

  setConst_hprimewgll_xx(h_hprimewgll_xx,mp);
  //setConst_hprimewgll_yy(h_hprimewgll_yy,mp); // only needed if NGLLX != NGLLY != NGLLZ
  //setConst_hprimewgll_zz(h_hprimewgll_zz,mp); // only needed if NGLLX != NGLLY != NGLLZ

  setConst_wgllwgll_xy(h_wgllwgll_xy,mp);
  setConst_wgllwgll_xz(h_wgllwgll_xz,mp);
  setConst_wgllwgll_yz(h_wgllwgll_yz,mp);

  // Using texture memory for the hprime-style constants is slower on
  // Fermi generation hardware, but *may* be faster on Kepler
  // generation. We will reevaluate this again, so might as well leave
  // in the code with with #USE_TEXTURES_FIELDS not-defined.
  #ifdef USE_TEXTURES_CONSTANTS
  {
    #ifdef USE_OLDER_CUDA4_GPU
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      const textureReference* d_hprime_xx_tex_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_hprime_xx_tex_ptr, HIP_SYMBOL("d_hprime_xx_tex")), 4101);
      print_CUDA_error_if_any(hipBindTexture(0, d_hprime_xx_tex_ptr, mp->d_hprime_xx, &channelDesc, sizeof(realw)*(NGLL2)), 4001);
   #else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      print_CUDA_error_if_any(hipBindTexture(0, &d_hprime_xx_tex, mp->d_hprime_xx, &channelDesc, sizeof(realw)*(NGLL2)), 4001);
   #endif
  }
  #endif

  copy_todevice_int((void**)&mp->d_irregular_element_number,h_irregular_element_number,mp->NSPEC_AB);
  mp->xix_regular = *xix_regular;
  mp->jacobian_regular = *jacobian_regular;

  // mesh
  // Assuming NGLLX=5. Padded is then 128 (5^3+3)
  int size_padded = NGLL3_PADDED * (*NSPEC_IRREGULAR > 0 ? *NSPEC_IRREGULAR : 1);

// DK DK August 2018: adding this test, following a suggestion by Etienne Bachmann
  if (*h_NGLLX != NGLLX) exit_on_error("make sure that the NGLL constants are equal in the two files setup/constants.h and src/cuda/mesh_constants_cuda.h and then please re-compile; also make sure that the value of NGLL3_PADDED is consistent with the value of NGLL\n");

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xix, size_padded*sizeof(realw)),1001);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiy, size_padded*sizeof(realw)),1002);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiz, size_padded*sizeof(realw)),1003);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etax, size_padded*sizeof(realw)),1004);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etay, size_padded*sizeof(realw)),1005);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etaz, size_padded*sizeof(realw)),1006);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammax, size_padded*sizeof(realw)),1007);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammay, size_padded*sizeof(realw)),1008);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammaz, size_padded*sizeof(realw)),1009);

  // transfer constant element data with padding
  /*
  // way 1: slow...
  for(int i=0;i < mp->NSPEC_AB;i++) {
    print_CUDA_error_if_any(hipMemcpy(mp->d_xix + i*NGLL3_PADDED, &h_xix[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1501);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xiy+i*NGLL3_PADDED,   &h_xiy[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1502);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xiz+i*NGLL3_PADDED,   &h_xiz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1503);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etax+i*NGLL3_PADDED,  &h_etax[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1504);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etay+i*NGLL3_PADDED,  &h_etay[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1505);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etaz+i*NGLL3_PADDED,  &h_etaz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1506);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammax+i*NGLL3_PADDED,&h_gammax[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1507);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammay+i*NGLL3_PADDED,&h_gammay[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1508);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammaz+i*NGLL3_PADDED,&h_gammaz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1509);
    print_CUDA_error_if_any(hipMemcpy(mp->d_kappav+i*NGLL3_PADDED,&h_kappav[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1510);
    print_CUDA_error_if_any(hipMemcpy(mp->d_muv+i*NGLL3_PADDED,   &h_muv[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1511);
  }
  */
  // way 2: faster ....
  if (*NSPEC_IRREGULAR > 0 ){
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_xix, NGLL3_PADDED*sizeof(realw),
                                         h_xix, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),1501);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_xiy, NGLL3_PADDED*sizeof(realw),
                                         h_xiy, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),1502);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_xiz, NGLL3_PADDED*sizeof(realw),
                                         h_xiz, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),1503);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_etax, NGLL3_PADDED*sizeof(realw),
                                         h_etax, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),1504);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_etay, NGLL3_PADDED*sizeof(realw),
                                         h_etay, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),1505);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_etaz, NGLL3_PADDED*sizeof(realw),
                                         h_etaz, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),1506);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_gammax, NGLL3_PADDED*sizeof(realw),
                                         h_gammax, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),1507);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_gammay, NGLL3_PADDED*sizeof(realw),
                                         h_gammay, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),1508);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_gammaz, NGLL3_PADDED*sizeof(realw),
                                         h_gammaz, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),1509);
  }
  size_padded = NGLL3_PADDED * (mp->NSPEC_AB);


  // global indexing (padded)
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool, size_padded*sizeof(int)),1600);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_ibool, NGLL3_PADDED*sizeof(int),
                                       h_ibool, NGLL3*sizeof(int), NGLL3*sizeof(int),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1601);


  // prepare interprocess-edge exchange information
  mp->num_interfaces_ext_mesh = *num_interfaces_ext_mesh;
  mp->max_nibool_interfaces_ext_mesh = *max_nibool_interfaces_ext_mesh;
  if (mp->num_interfaces_ext_mesh > 0){
    copy_todevice_int((void**)&mp->d_nibool_interfaces_ext_mesh,h_nibool_interfaces_ext_mesh,
                      mp->num_interfaces_ext_mesh);
    copy_todevice_int((void**)&mp->d_ibool_interfaces_ext_mesh,h_ibool_interfaces_ext_mesh,
                      (mp->num_interfaces_ext_mesh)*(mp->max_nibool_interfaces_ext_mesh));
  }

  // setup two streams, one for compute and one for host<->device memory copies
  // compute stream
  hipStreamCreate(&mp->compute_stream);
  // copy stream (needed to transfer mpi buffers)
  if (mp->num_interfaces_ext_mesh * mp->max_nibool_interfaces_ext_mesh > 0){
    hipStreamCreate(&mp->copy_stream);
  }

  // inner elements
  copy_todevice_int((void**)&mp->d_ispec_is_inner,h_ispec_is_inner,mp->NSPEC_AB);

  // absorbing boundaries
  mp->d_num_abs_boundary_faces = *h_num_abs_boundary_faces;
  if (mp->absorbing_conditions && mp->d_num_abs_boundary_faces > 0){
    copy_todevice_int((void**)&mp->d_abs_boundary_ispec,h_abs_boundary_ispec,mp->d_num_abs_boundary_faces);
    copy_todevice_int((void**)&mp->d_abs_boundary_ijk,h_abs_boundary_ijk,
                      3*NGLL2*(mp->d_num_abs_boundary_faces));
    copy_todevice_realw((void**)&mp->d_abs_boundary_normal,h_abs_boundary_normal,
                        NDIM*NGLL2*(mp->d_num_abs_boundary_faces));
    copy_todevice_realw((void**)&mp->d_abs_boundary_jacobian2Dw,h_abs_boundary_jacobian2Dw,
                        NGLL2*(mp->d_num_abs_boundary_faces));
  }

  // sources
  mp->nsources_local = *nsources_local_f;
  if (mp->simulation_type == 1  || mp->simulation_type == 3){
    // not needed in case of pure adjoint simulations (SIMULATION_TYPE == 2)
    copy_todevice_realw((void**)&mp->d_sourcearrays,h_sourcearrays,(*NSOURCES)*NDIM*NGLL3);

    // buffer for source time function values
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_stf_pre_compute,(*NSOURCES)*sizeof(field)),1303);
  }
  copy_todevice_int((void**)&mp->d_islice_selected_source,h_islice_selected_source,(*NSOURCES));
  copy_todevice_int((void**)&mp->d_ispec_selected_source,h_ispec_selected_source,(*NSOURCES));


  // receiver stations
  mp->save_seismograms_d = *SAVE_SEISMOGRAMS_DISPLACEMENT;
  mp->save_seismograms_v = *SAVE_SEISMOGRAMS_VELOCITY;
  mp->save_seismograms_a = *SAVE_SEISMOGRAMS_ACCELERATION;
  mp->save_seismograms_p = *SAVE_SEISMOGRAMS_PRESSURE;

  mp->nrec_local = *nrec_local; // number of receiver located in this partition
  // note that:
  // size(ispec_selected_rec) = nrec
  if (mp->nrec_local > 0){
    copy_todevice_realw((void**)&mp->d_hxir,h_xir,NGLLX*mp->nrec_local);
    copy_todevice_realw((void**)&mp->d_hetar,h_etar,NGLLY*mp->nrec_local);
    copy_todevice_realw((void**)&mp->d_hgammar,h_gammar,NGLLZ*mp->nrec_local);

    realw* h_nu;
    h_nu = (realw*)malloc(NDIM * NDIM * mp->nrec_local * sizeof(realw));
    int irec_loc = 0;
    for (int i=0;i < (*nrec);i++){
      if (mp->myrank == islice_selected_rec[i]){
         for (int j = 0; j < 9; j++) h_nu[j + NDIM * NDIM * irec_loc] = (realw)nu[j + NDIM * NDIM * i];
         irec_loc = irec_loc + 1;
      }
    }
    copy_todevice_realw((void**)&mp->d_nu,h_nu,NDIM * NDIM * (*nrec_local));
    free(h_nu);

    // seismograms
    int size =  (*NTSTEP_BETWEEN_OUTPUT_SEISMOS) * (*nrec_local);

    if (mp->save_seismograms_d)
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_seismograms_d,NDIM*size * sizeof(realw)),1801);
    if (mp->save_seismograms_v)
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_seismograms_v,NDIM*size * sizeof(realw)),1802);
    if (mp->save_seismograms_a)
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_seismograms_a,NDIM*size * sizeof(realw)),1803);
    if (mp->save_seismograms_p)
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_seismograms_p,size * sizeof(field)),1804);

    int *ispec_selected_rec_loc;
    ispec_selected_rec_loc = (int*)malloc(mp->nrec_local * sizeof(int));
    irec_loc = 0;
    for(int i=0;i<*nrec;i++) {
      if ( mp->myrank == islice_selected_rec[i]){
        ispec_selected_rec_loc[irec_loc] = h_ispec_selected_rec[i];
        irec_loc = irec_loc+1;
      }
    }
    copy_todevice_int((void**)&mp->d_ispec_selected_rec_loc,ispec_selected_rec_loc,mp->nrec_local);
    free(ispec_selected_rec_loc);
  }
  copy_todevice_int((void**)&mp->d_ispec_selected_rec,h_ispec_selected_rec,(*nrec));

#ifdef USE_MESH_COLORING_GPUX
  mp->use_mesh_coloring_gpu = 1;
  if (! *USE_MESH_COLORING_GPU_f) exit_on_error("error with USE_MESH_COLORING_GPU constant; please re-compile\n");
#else
  // mesh coloring
  // note: this here passes the coloring as an option to the kernel routines
  //          the performance seems to be the same if one uses the pre-processing directives above or not
  mp->use_mesh_coloring_gpu = *USE_MESH_COLORING_GPU_f;
#endif

  // number of elements per domain
  mp->nspec_acoustic = *nspec_acoustic;
  mp->nspec_elastic = *nspec_elastic;

  // gravity flag initialization
  mp->gravity = 0;
  // Kelvin_voigt initialization
  mp->Kelvin_Voigt_damping = 0;
  // JC JC here we will need to add GPU support for the new C-PML routines

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_constants_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// for ACOUSTIC simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_acoustic_device,
              PREPARE_FIELDS_ACOUSTIC_DEVICE)(long* Mesh_pointer,
                                              realw* rmass_acoustic, realw* rhostore, realw* kappastore,
                                              int* num_phase_ispec_acoustic, int* phase_ispec_inner_acoustic,
                                              int* ispec_is_acoustic,
                                              int* NOISE_TOMOGRAPHY,
                                              int* num_free_surface_faces,
                                              int* free_surface_ispec,
                                              int* free_surface_ijk,
                                              int* b_reclen_potential, realw* b_absorb_potential,
                                              int* ELASTIC_SIMULATION,
                                              int* num_coupling_ac_el_faces,
                                              int* coupling_ac_el_ispec,
                                              int* coupling_ac_el_ijk,
                                              realw* coupling_ac_el_normal,
                                              realw* coupling_ac_el_jacobian2Dw,
                                              int* num_colors_outer_acoustic,
                                              int* num_colors_inner_acoustic,
                                              int* num_elem_colors_acoustic) {

  TRACE("prepare_fields_acoustic_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  // allocates arrays on device (GPU)
  int size = mp->NGLOB_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_acoustic),sizeof(field)*size),2001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_dot_acoustic),sizeof(field)*size),2002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_dot_dot_acoustic),sizeof(field)*size),2003);
  // initializes values to zero
  //print_CUDA_error_if_any(hipMemset(mp->d_potential_acoustic,0,sizeof(field)*size),2007);
  //print_CUDA_error_if_any(hipMemset(mp->d_potential_dot_acoustic,0,sizeof(field)*size),2007);
  //print_CUDA_error_if_any(hipMemset(mp->d_potential_dot_dot_acoustic,0,sizeof(field)*size),2007);

  #ifdef USE_TEXTURES_FIELDS
  {
    #ifdef USE_OLDER_CUDA4_GPU
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      const textureReference* d_potential_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_potential_tex_ref_ptr, HIP_SYMBOL("d_potential_tex")), 2001);
      print_CUDA_error_if_any(hipBindTexture(0, d_potential_tex_ref_ptr, mp->d_potential_acoustic, &channelDesc, sizeof(realw)*size), 2001);

      const textureReference* d_potential_dot_dot_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_potential_dot_dot_tex_ref_ptr, HIP_SYMBOL("d_potential_dot_dot_tex")), 2003);
      print_CUDA_error_if_any(hipBindTexture(0, d_potential_dot_dot_tex_ref_ptr, mp->d_potential_dot_dot_acoustic, &channelDesc, sizeof(realw)*size), 2003);
    #else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      print_CUDA_error_if_any(hipBindTexture(0, &d_potential_tex, mp->d_potential_acoustic, &channelDesc, sizeof(realw)*size), 2001);
      print_CUDA_error_if_any(hipBindTexture(0, &d_potential_dot_dot_tex, mp->d_potential_dot_dot_acoustic, &channelDesc, sizeof(realw)*size), 2003);
    #endif
  }
  #endif

  // mpi buffer
  mp->size_mpi_buffer_potential = (mp->num_interfaces_ext_mesh) * (mp->max_nibool_interfaces_ext_mesh);
  if (mp->size_mpi_buffer_potential > 0){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_potential_dot_dot_buffer),mp->size_mpi_buffer_potential *sizeof(field)),2004);
  }

  // mass matrix
  copy_todevice_realw((void**)&mp->d_rmass_acoustic,rmass_acoustic,mp->NGLOB_AB);

  // density
  // padded array
  // Assuming NGLLX==5. Padded is then 128 (5^3+3)
  int size_padded = NGLL3_PADDED * mp->NSPEC_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rhostore),size_padded*sizeof(realw)),2006);
  // transfer constant element data with padding
  /*
  // way 1: slow...
  for(int i=0; i < mp->NSPEC_AB; i++) {
    print_CUDA_error_if_any(hipMemcpy(mp->d_rhostore+i*NGLL3_PADDED, &rhostore[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),2106);
  }
  */
  // way 2: faster ...
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_rhostore, NGLL3_PADDED*sizeof(realw),
                                       rhostore, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),2106);

  // non-padded array
  copy_todevice_realw((void**)&mp->d_kappastore,kappastore,NGLL3*mp->NSPEC_AB);

  // phase elements
  mp->num_phase_ispec_acoustic = *num_phase_ispec_acoustic;
  copy_todevice_int((void**)&mp->d_phase_ispec_inner_acoustic,phase_ispec_inner_acoustic,
                    2*mp->num_phase_ispec_acoustic);
  copy_todevice_int((void**)&mp->d_ispec_is_acoustic,ispec_is_acoustic,mp->NSPEC_AB);

  // free surface
  if (*NOISE_TOMOGRAPHY == 0){
    // allocate surface arrays
    mp->num_free_surface_faces = *num_free_surface_faces;
    if (mp->num_free_surface_faces > 0){
      copy_todevice_int((void**)&mp->d_free_surface_ispec,free_surface_ispec,mp->num_free_surface_faces);
      copy_todevice_int((void**)&mp->d_free_surface_ijk,free_surface_ijk,
                        3*NGLL2*mp->num_free_surface_faces);
    }
  }

  // absorbing boundaries
  if (mp->absorbing_conditions && mp->d_num_abs_boundary_faces > 0){
    // absorb_field array used for file i/o
    if (mp->simulation_type == 3 || ( mp->simulation_type == 1 && mp->save_forward )){
      // note: b_reclen_potential is record length in bytes ( CUSTOM_REAL * NGLLSQUARE * num_abs_boundary_faces )
      mp->d_b_reclen_potential = *b_reclen_potential;
      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_potential,mp->d_b_reclen_potential),2201);
      print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_potential,b_absorb_potential,mp->d_b_reclen_potential,hipMemcpyHostToDevice),2202);
    }
  }

  // coupling with elastic parts
  if (*ELASTIC_SIMULATION && *num_coupling_ac_el_faces > 0){
    copy_todevice_int((void**)&mp->d_coupling_ac_el_ispec,coupling_ac_el_ispec,(*num_coupling_ac_el_faces));
    copy_todevice_int((void**)&mp->d_coupling_ac_el_ijk,coupling_ac_el_ijk,3*NGLL2*(*num_coupling_ac_el_faces));
    copy_todevice_realw((void**)&mp->d_coupling_ac_el_normal,coupling_ac_el_normal,
                        3*NGLL2*(*num_coupling_ac_el_faces));
    copy_todevice_realw((void**)&mp->d_coupling_ac_el_jacobian2Dw,coupling_ac_el_jacobian2Dw,
                        NGLL2*(*num_coupling_ac_el_faces));
  }

  // mesh coloring
  if (mp->use_mesh_coloring_gpu ){
    mp->num_colors_outer_acoustic = *num_colors_outer_acoustic;
    mp->num_colors_inner_acoustic = *num_colors_inner_acoustic;
    mp->h_num_elem_colors_acoustic = (int*) num_elem_colors_acoustic;
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_acoustic_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_acoustic_adj_dev,
              PREPARE_FIELDS_ACOUSTIC_ADJ_DEV)(long* Mesh_pointer,
                                              int* APPROXIMATE_HESS_KL) {

  TRACE("prepare_fields_acoustic_adj_dev");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  // kernel simulations
  if (mp->simulation_type != 3) return;

  // allocates backward/reconstructed arrays on device (GPU)
  int size = mp->NGLOB_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_acoustic),sizeof(field)*size),3014);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_dot_acoustic),sizeof(field)*size),3015);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_dot_dot_acoustic),sizeof(field)*size),3016);
  // initializes values to zero
  //print_CUDA_error_if_any(hipMemset(mp->d_b_potential_acoustic,0,sizeof(realw)*size),3007);
  //print_CUDA_error_if_any(hipMemset(mp->d_b_potential_dot_acoustic,0,sizeof(realw)*size),3007);
  //print_CUDA_error_if_any(hipMemset(mp->d_b_potential_dot_dot_acoustic,0,sizeof(realw)*size),3007);

  #ifdef USE_TEXTURES_FIELDS
  {
    #ifdef USE_OLDER_CUDA4_GPU
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      const textureReference* d_b_potential_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_b_potential_tex_ref_ptr, HIP_SYMBOL("d_b_potential_tex")), 3001);
      print_CUDA_error_if_any(hipBindTexture(0, d_b_potential_tex_ref_ptr, mp->d_b_potential_acoustic, &channelDesc, sizeof(realw)*size), 3001);

      const textureReference* d_b_potential_dot_dot_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_b_potential_dot_dot_tex_ref_ptr, HIP_SYMBOL("d_b_potential_dot_dot_tex")),3003);
      print_CUDA_error_if_any(hipBindTexture(0, d_b_potential_dot_dot_tex_ref_ptr, mp->d_b_potential_dot_dot_acoustic, &channelDesc, sizeof(realw)*size), 3003);
    #else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      print_CUDA_error_if_any(hipBindTexture(0, &d_b_potential_tex, mp->d_b_potential_acoustic, &channelDesc, sizeof(realw)*size), 3001);
      print_CUDA_error_if_any(hipBindTexture(0, &d_b_potential_dot_dot_tex, mp->d_b_potential_dot_dot_acoustic, &channelDesc, sizeof(realw)*size), 3003);
    #endif
  }
  #endif

  // allocates kernels
  size = NGLL3*mp->NSPEC_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_ac_kl),size*sizeof(realw)),3017);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_kappa_ac_kl),size*sizeof(realw)),3018);
  // initializes kernel values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_rho_ac_kl,0,size*sizeof(realw)),3019);
  print_CUDA_error_if_any(hipMemset(mp->d_kappa_ac_kl,0,size*sizeof(realw)),3020);

  // preconditioner
  if (*APPROXIMATE_HESS_KL ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_ac_kl),size*sizeof(realw)),3030);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_rho_ac_kl),size*sizeof(realw)),3032);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_kappa_ac_kl),size*sizeof(realw)),3033);

    // initializes with zeros
    print_CUDA_error_if_any(hipMemset(mp->d_hess_ac_kl,0,size*sizeof(realw)),3031);
    print_CUDA_error_if_any(hipMemset(mp->d_hess_rho_ac_kl,0,size*sizeof(realw)),3034);
    print_CUDA_error_if_any(hipMemset(mp->d_hess_kappa_ac_kl,0,size*sizeof(realw)),3035);

  }

  // mpi buffer
  if (mp->size_mpi_buffer_potential > 0){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_send_potential_dot_dot_buffer),mp->size_mpi_buffer_potential*sizeof(field)),3014);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_acoustic_adj_dev");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// for ELASTIC simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_elastic_device,
              PREPARE_FIELDS_ELASTIC_DEVICE)(long* Mesh_pointer,
                                             realw* rmassx, realw* rmassy, realw* rmassz,
                                             realw* rho_vp, realw* rho_vs,
                                             realw* h_kappav, realw* h_muv,
                                             int* num_phase_ispec_elastic,
                                             int* phase_ispec_inner_elastic,
                                             int* ispec_is_elastic,
                                             realw* b_absorb_field, int* b_reclen_field,
                                             int* COMPUTE_AND_STORE_STRAIN,
                                             realw* epsilondev_xx,realw* epsilondev_yy,realw* epsilondev_xy,
                                             realw* epsilondev_xz,realw* epsilondev_yz,
                                             int* ATTENUATION,
                                             int* R_size,
                                             realw* R_xx,realw* R_yy,realw* R_xy,realw* R_xz,realw* R_yz,
                                             realw* factor_common,
                                             realw* alphaval,realw* betaval,realw* gammaval,
                                             int* APPROXIMATE_OCEAN_LOAD,
                                             realw* rmass_ocean_load,
                                             int* NOISE_TOMOGRAPHY,
                                             realw* free_surface_normal,
                                             int* free_surface_ispec,
                                             int* free_surface_ijk,
                                             int* num_free_surface_faces,
                                             int* ACOUSTIC_SIMULATION,
                                             int* num_colors_outer_elastic,
                                             int* num_colors_inner_elastic,
                                             int* num_elem_colors_elastic,
                                             int* ANISOTROPY,
                                             realw *c11store,realw *c12store,realw *c13store,
                                             realw *c14store,realw *c15store,realw *c16store,
                                             realw *c22store,realw *c23store,realw *c24store,
                                             realw *c25store,realw *c26store,realw *c33store,
                                             realw *c34store,realw *c35store,realw *c36store,
                                             realw *c44store,realw *c45store,realw *c46store,
                                             realw *c55store,realw *c56store,realw *c66store ){

  TRACE("prepare_fields_elastic_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);
  int size;

  // debug
  //printf("prepare_fields_elastic_device: rank %d - wavefield setup\n",mp->myrank);
  //synchronize_mpi();

  // elastic wavefields
  size = NDIM * mp->NGLOB_AB;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_displ),sizeof(realw)*size),4001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_veloc),sizeof(realw)*size),4002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_accel),sizeof(realw)*size),4003);
  // initializes values to zero
  //print_CUDA_error_if_any(hipMemset(mp->d_displ,0,sizeof(realw)*size),4007);
  //print_CUDA_error_if_any(hipMemset(mp->d_veloc,0,sizeof(realw)*size),4007);
  //print_CUDA_error_if_any(hipMemset(mp->d_accel,0,sizeof(realw)*size),4007);

  #ifdef USE_TEXTURES_FIELDS
  {
    #ifdef USE_OLDER_CUDA4_GPU
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      const textureReference* d_displ_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_displ_tex_ref_ptr, HIP_SYMBOL("d_displ_tex")), 4001);
      print_CUDA_error_if_any(hipBindTexture(0, d_displ_tex_ref_ptr, mp->d_displ, &channelDesc, sizeof(realw)*size), 4001);

      const textureReference* d_veloc_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_veloc_tex_ref_ptr, HIP_SYMBOL("d_veloc_tex")), 4002);
      print_CUDA_error_if_any(hipBindTexture(0, d_veloc_tex_ref_ptr, mp->d_veloc, &channelDesc, sizeof(realw)*size), 4002);

      const textureReference* d_accel_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_accel_tex_ref_ptr, HIP_SYMBOL("d_accel_tex")), 4003);
      print_CUDA_error_if_any(hipBindTexture(0, d_accel_tex_ref_ptr, mp->d_accel, &channelDesc, sizeof(realw)*size), 4003);
    #else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      print_CUDA_error_if_any(hipBindTexture(0, &d_displ_tex, mp->d_displ, &channelDesc, sizeof(realw)*size), 4001);
      print_CUDA_error_if_any(hipBindTexture(0, &d_veloc_tex, mp->d_veloc, &channelDesc, sizeof(realw)*size), 4002);
      print_CUDA_error_if_any(hipBindTexture(0, &d_accel_tex, mp->d_accel, &channelDesc, sizeof(realw)*size), 4003);
    #endif
  }
  #endif

  // debug
  //synchronize_mpi();

  // MPI buffer
  mp->size_mpi_buffer = NDIM * (mp->num_interfaces_ext_mesh) * (mp->max_nibool_interfaces_ext_mesh);
  if (mp->size_mpi_buffer > 0){
    // note: Allocate pinned mpi-buffers.
    //       MPI buffers use pinned memory allocated by hipHostMalloc, which
    //       enables the use of asynchronous memory copies from host <-> device
    // send buffer
    print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_send_accel_buffer),sizeof(float)*(mp->size_mpi_buffer)),8004);
    //mp->send_buffer = (float*)malloc((mp->size_mpi_buffer)*sizeof(float));
    // adjoint
    //print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_send_b_accel_buffer),sizeof(float)*(mp->size_mpi_buffer)),8004);
    // mp->b_send_buffer = (float*)malloc((size_mpi_buffer)*sizeof(float));
    // receive buffer
    print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_recv_accel_buffer),sizeof(float)*(mp->size_mpi_buffer)),8004);
    mp->recv_buffer = (float*)malloc((mp->size_mpi_buffer)*sizeof(float));

    // non-pinned buffer
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_accel_buffer),mp->size_mpi_buffer*sizeof(realw)),4004);
    // adjoint
    if (mp->simulation_type == 3){
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_send_accel_buffer),mp->size_mpi_buffer*sizeof(realw)),4004);
    }
  }

  // debug
  //printf("prepare_fields_elastic_device: rank %d - mass matrix\n",mp->myrank);
  //synchronize_mpi();

  // mass matrix
  copy_todevice_realw((void**)&mp->d_rmassx,rmassx,mp->NGLOB_AB);
  copy_todevice_realw((void**)&mp->d_rmassy,rmassy,mp->NGLOB_AB);
  copy_todevice_realw((void**)&mp->d_rmassz,rmassz,mp->NGLOB_AB);

  // element indices
  copy_todevice_int((void**)&mp->d_ispec_is_elastic,ispec_is_elastic,mp->NSPEC_AB);

  // phase elements
  mp->num_phase_ispec_elastic = *num_phase_ispec_elastic;

  copy_todevice_int((void**)&mp->d_phase_ispec_inner_elastic,phase_ispec_inner_elastic,2*mp->num_phase_ispec_elastic);

  // debug
  //synchronize_mpi();

  // absorbing conditions
  if (mp->absorbing_conditions && mp->d_num_abs_boundary_faces > 0){

    // debug
    //printf("prepare_fields_elastic_device: rank %d - absorbing boundary setup\n",mp->myrank);

    // non-padded arrays
    // rho_vp, rho_vs non-padded; they are needed for stacey boundary condition
    copy_todevice_realw((void**)&mp->d_rho_vp,rho_vp,NGLL3*mp->NSPEC_AB);
    copy_todevice_realw((void**)&mp->d_rho_vs,rho_vs,NGLL3*mp->NSPEC_AB);

    // absorb_field array used for file i/o
    if (mp->simulation_type == 3 || ( mp->simulation_type == 1 && mp->save_forward )){
      // note: b_reclen_field is length in bytes already (CUSTOM_REAL * NDIM * NGLLSQUARE * num_abs_boundary_faces )
      mp->d_b_reclen_field = *b_reclen_field;

      // debug
      //printf("prepare_fields_elastic_device: rank %d - absorbing boundary i/o %d\n",mp->myrank,mp->d_b_reclen_field);

      print_CUDA_error_if_any(hipMalloc((void**)&mp->d_b_absorb_field,mp->d_b_reclen_field),4101);
      print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_field,b_absorb_field,mp->d_b_reclen_field,hipMemcpyHostToDevice),4102);

    }
  }
  int size_padded = NGLL3_PADDED*mp->NSPEC_AB;
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_kappav, size_padded*sizeof(realw)),1010);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_muv, size_padded*sizeof(realw)),1011);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_kappav, NGLL3_PADDED*sizeof(realw),
                                       h_kappav, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1510);
  print_CUDA_error_if_any(hipMemcpy2D(mp->d_muv, NGLL3_PADDED*sizeof(realw),
                                       h_muv, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                       mp->NSPEC_AB, hipMemcpyHostToDevice),1511);

  // debug
  //synchronize_mpi();

  // strains used for attenuation and kernel simulations
  if (*COMPUTE_AND_STORE_STRAIN ){
    // debug
    //printf("prepare_fields_elastic_device: rank %d - strain setup\n",mp->myrank);
    //synchronize_mpi();

    // strains
    size = NGLL3 * mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing
    copy_todevice_realw((void**)&mp->d_epsilondev_xx,epsilondev_xx,size);
    copy_todevice_realw((void**)&mp->d_epsilondev_yy,epsilondev_yy,size);
    copy_todevice_realw((void**)&mp->d_epsilondev_xy,epsilondev_xy,size);
    copy_todevice_realw((void**)&mp->d_epsilondev_xz,epsilondev_xz,size);
    copy_todevice_realw((void**)&mp->d_epsilondev_yz,epsilondev_yz,size);
  }

  // attenuation memory variables
  if (*ATTENUATION ){
    // debug
    //printf("prepare_fields_elastic_device: rank %d - attenuation setup\n",mp->myrank);
    //synchronize_mpi();

    // memory arrays
    size = *R_size;
    copy_todevice_realw((void**)&mp->d_R_xx,R_xx,size);
    copy_todevice_realw((void**)&mp->d_R_yy,R_yy,size);
    copy_todevice_realw((void**)&mp->d_R_xy,R_xy,size);
    copy_todevice_realw((void**)&mp->d_R_xz,R_xz,size);
    copy_todevice_realw((void**)&mp->d_R_yz,R_yz,size);
    // attenuation factors
    copy_todevice_realw((void**)&mp->d_factor_common,factor_common,N_SLS*NGLL3*mp->NSPEC_AB);
    // alpha,beta,gamma factors
    copy_todevice_realw((void**)&mp->d_alphaval,alphaval,N_SLS);
    copy_todevice_realw((void**)&mp->d_betaval,betaval,N_SLS);
    copy_todevice_realw((void**)&mp->d_gammaval,gammaval,N_SLS);
  }

  // anisotropy
  if (*ANISOTROPY ){
    // debug
    //printf("prepare_fields_elastic_device: rank %d - attenuation setup\n",mp->myrank);
    //synchronize_mpi();

    // Assuming NGLLX==5. Padded is then 128 (5^3+3)
    size_padded = NGLL3_PADDED * (mp->NSPEC_AB);

    // allocates memory on GPU
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c11store),size_padded*sizeof(realw)),4700);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c12store),size_padded*sizeof(realw)),4701);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c13store),size_padded*sizeof(realw)),4702);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c14store),size_padded*sizeof(realw)),4703);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c15store),size_padded*sizeof(realw)),4704);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c16store),size_padded*sizeof(realw)),4705);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c22store),size_padded*sizeof(realw)),4706);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c23store),size_padded*sizeof(realw)),4707);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c24store),size_padded*sizeof(realw)),4708);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c25store),size_padded*sizeof(realw)),4709);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c26store),size_padded*sizeof(realw)),4710);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c33store),size_padded*sizeof(realw)),4711);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c34store),size_padded*sizeof(realw)),4712);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c35store),size_padded*sizeof(realw)),4713);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c36store),size_padded*sizeof(realw)),4714);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c44store),size_padded*sizeof(realw)),4715);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c45store),size_padded*sizeof(realw)),4716);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c46store),size_padded*sizeof(realw)),4717);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c55store),size_padded*sizeof(realw)),4718);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c56store),size_padded*sizeof(realw)),4719);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c66store),size_padded*sizeof(realw)),4720);

    // transfer constant element data with padding
    /*
    // way 1: slower ...
    for(int i=0;i < mp->NSPEC_AB;i++) {
      print_CUDA_error_if_any(hipMemcpy(mp->d_c11store + i*NGLL3_PADDED, &c11store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4800);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c12store + i*NGLL3_PADDED, &c12store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4801);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c13store + i*NGLL3_PADDED, &c13store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4802);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c14store + i*NGLL3_PADDED, &c14store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4803);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c15store + i*NGLL3_PADDED, &c15store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4804);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c16store + i*NGLL3_PADDED, &c16store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4805);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c22store + i*NGLL3_PADDED, &c22store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4806);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c23store + i*NGLL3_PADDED, &c23store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4807);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c24store + i*NGLL3_PADDED, &c24store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4808);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c25store + i*NGLL3_PADDED, &c25store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4809);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c26store + i*NGLL3_PADDED, &c26store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4810);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c33store + i*NGLL3_PADDED, &c33store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4811);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c34store + i*NGLL3_PADDED, &c34store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4812);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c35store + i*NGLL3_PADDED, &c35store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4813);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c36store + i*NGLL3_PADDED, &c36store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4814);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c44store + i*NGLL3_PADDED, &c44store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4815);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c45store + i*NGLL3_PADDED, &c45store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4816);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c46store + i*NGLL3_PADDED, &c46store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4817);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c55store + i*NGLL3_PADDED, &c55store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4818);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c56store + i*NGLL3_PADDED, &c56store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4819);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c66store + i*NGLL3_PADDED, &c66store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4820);
    }
    */
    // way 2: faster ...
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c11store, NGLL3_PADDED*sizeof(realw),
                                         c11store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c12store, NGLL3_PADDED*sizeof(realw),
                                         c12store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c13store, NGLL3_PADDED*sizeof(realw),
                                         c13store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c14store, NGLL3_PADDED*sizeof(realw),
                                         c14store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c15store, NGLL3_PADDED*sizeof(realw),
                                         c15store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c16store, NGLL3_PADDED*sizeof(realw),
                                         c16store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c22store, NGLL3_PADDED*sizeof(realw),
                                         c22store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c23store, NGLL3_PADDED*sizeof(realw),
                                         c23store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c24store, NGLL3_PADDED*sizeof(realw),
                                         c24store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c25store, NGLL3_PADDED*sizeof(realw),
                                         c25store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c26store, NGLL3_PADDED*sizeof(realw),
                                         c26store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c33store, NGLL3_PADDED*sizeof(realw),
                                         c33store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c34store, NGLL3_PADDED*sizeof(realw),
                                         c34store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c35store, NGLL3_PADDED*sizeof(realw),
                                         c35store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c36store, NGLL3_PADDED*sizeof(realw),
                                         c36store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c44store, NGLL3_PADDED*sizeof(realw),
                                         c44store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c45store, NGLL3_PADDED*sizeof(realw),
                                         c45store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c46store, NGLL3_PADDED*sizeof(realw),
                                         c46store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c55store, NGLL3_PADDED*sizeof(realw),
                                         c55store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c56store, NGLL3_PADDED*sizeof(realw),
                                         c56store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    print_CUDA_error_if_any(hipMemcpy2D(mp->d_c66store, NGLL3_PADDED*sizeof(realw),
                                         c66store, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                         mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
  }

  // ocean load approximation
  if (*APPROXIMATE_OCEAN_LOAD ){
    // debug
    //printf("prepare_fields_elastic_device: rank %d - ocean load setup\n",mp->myrank);
    //synchronize_mpi();

    // oceans needs a free surface
    mp->num_free_surface_faces = *num_free_surface_faces;
    if (mp->num_free_surface_faces > 0){
      // mass matrix
      copy_todevice_realw((void**)&mp->d_rmass_ocean_load,rmass_ocean_load,mp->NGLOB_AB);
      // surface normal
      copy_todevice_realw((void**)&mp->d_free_surface_normal,free_surface_normal,
                          3*NGLL2*(mp->num_free_surface_faces));
      // temporary global array: used to synchronize updates on global accel array
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_updated_dof_ocean_load),
                                         sizeof(int)*mp->NGLOB_AB),4505);

      if (*NOISE_TOMOGRAPHY == 0 && *ACOUSTIC_SIMULATION == 0){
        copy_todevice_int((void**)&mp->d_free_surface_ispec,free_surface_ispec,mp->num_free_surface_faces);
        copy_todevice_int((void**)&mp->d_free_surface_ijk,free_surface_ijk,
                          3*NGLL2*mp->num_free_surface_faces);
      }
    }
  }

  // mesh coloring
  if (mp->use_mesh_coloring_gpu ){
    mp->num_colors_outer_elastic = *num_colors_outer_elastic;
    mp->num_colors_inner_elastic = *num_colors_inner_elastic;
    mp->h_num_elem_colors_elastic = (int*) num_elem_colors_elastic;
  }

  // JC JC here we will need to add GPU support for the new C-PML routines

  // debug
  //printf("prepare_fields_elastic_device: rank %d - done\n",mp->myrank);
  //synchronize_mpi();

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_elastic_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_elastic_adj_dev,
              PREPARE_FIELDS_ELASTIC_ADJ_DEV)(long* Mesh_pointer,
                                             int* size_f,
                                             int* COMPUTE_AND_STORE_STRAIN,
                                             realw* epsilon_trace_over_3,
                                             realw* b_epsilondev_xx,realw* b_epsilondev_yy,realw* b_epsilondev_xy,
                                             realw* b_epsilondev_xz,realw* b_epsilondev_yz,
                                             realw* b_epsilon_trace_over_3,
                                             int* ATTENUATION,
                                             int* R_size,
                                             realw* b_R_xx,realw* b_R_yy,realw* b_R_xy,realw* b_R_xz,realw* b_R_yz,
                                             realw* b_alphaval,realw* b_betaval,realw* b_gammaval,
                                             int* ANISOTROPIC_KL,
                                             int* APPROXIMATE_HESS_KL){

  TRACE("prepare_fields_elastic_adj_dev");

  Mesh* mp = (Mesh*)(*Mesh_pointer);
  int size;

  // checks if kernel simulation
  if (mp->simulation_type != 3) return;

  // kernel simulations
  // debug
  //printf("prepare_fields_elastic_adj_dev: rank %d - kernel setup\n",mp->myrank);
  //synchronize_mpi();

  // backward/reconstructed wavefields
  // allocates backward/reconstructed arrays on device (GPU)
  size = *size_f;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_displ),sizeof(realw)*size),5201);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_veloc),sizeof(realw)*size),5202);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_accel),sizeof(realw)*size),5203);
  // initializes values to zero
  //print_CUDA_error_if_any(hipMemset(mp->d_b_displ,0,sizeof(realw)*size),5207);
  //print_CUDA_error_if_any(hipMemset(mp->d_b_veloc,0,sizeof(realw)*size),5207);
  //print_CUDA_error_if_any(hipMemset(mp->d_b_accel,0,sizeof(realw)*size),5207);

  #ifdef USE_TEXTURES_FIELDS
  {
    #ifdef USE_OLDER_CUDA4_GPU
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      const textureReference* d_b_displ_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_b_displ_tex_ref_ptr, HIP_SYMBOL("d_b_displ_tex")), 4001);
      print_CUDA_error_if_any(hipBindTexture(0, d_b_displ_tex_ref_ptr, mp->d_b_displ, &channelDesc, sizeof(realw)*size), 4001);

      const textureReference* d_b_veloc_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_b_veloc_tex_ref_ptr, HIP_SYMBOL("d_b_veloc_tex")), 4002);
      print_CUDA_error_if_any(hipBindTexture(0, d_b_veloc_tex_ref_ptr, mp->d_b_veloc, &channelDesc, sizeof(realw)*size), 4002);

      const textureReference* d_b_accel_tex_ref_ptr;
      print_CUDA_error_if_any(hipGetTextureReference(&d_b_accel_tex_ref_ptr, HIP_SYMBOL("d_b_accel_tex")), 4003);
      print_CUDA_error_if_any(hipBindTexture(0, d_b_accel_tex_ref_ptr, mp->d_b_accel, &channelDesc, sizeof(realw)*size), 4003);
    #else
      hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
      print_CUDA_error_if_any(hipBindTexture(0, &d_b_displ_tex, mp->d_b_displ, &channelDesc, sizeof(realw)*size), 4001);
      print_CUDA_error_if_any(hipBindTexture(0, &d_b_veloc_tex, mp->d_b_veloc, &channelDesc, sizeof(realw)*size), 4002);
      print_CUDA_error_if_any(hipBindTexture(0, &d_b_accel_tex, mp->d_b_accel, &channelDesc, sizeof(realw)*size), 4003);
    #endif
  }
  #endif


  // anisotropic kernel flag
  mp->anisotropic_kl = *ANISOTROPIC_KL;

  // anisotropic/isotropic kernels
  // debug
  //printf("prepare_fields_elastic_adj_dev: rank %d -  anisotropic/isotropic kernels\n",mp->myrank);
  //synchronize_mpi();

  // allocates kernels
  size = NGLL3 * mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing
  // density kernel
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_kl),size*sizeof(realw)),5204);
  // initializes kernel values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_rho_kl,0,size*sizeof(realw)),5214);

  if (mp->anisotropic_kl ){
    // anisotropic kernels
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_cijkl_kl),21*size*sizeof(realw)),5205);
    print_CUDA_error_if_any(hipMemset(mp->d_cijkl_kl,0,21*size*sizeof(realw)),5215);

  }else{
    // isotropic kernels
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_mu_kl),size*sizeof(realw)),5206);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_kappa_kl),size*sizeof(realw)),5207);
    print_CUDA_error_if_any(hipMemset(mp->d_mu_kl,0,size*sizeof(realw)),5216);
    print_CUDA_error_if_any(hipMemset(mp->d_kappa_kl,0,size*sizeof(realw)),5217);
  }

  // strains used for attenuation and kernel simulations
  if (*COMPUTE_AND_STORE_STRAIN ){
    // strains
    // debug
    //printf("prepare_fields_elastic_adj_dev: rank %d - strains\n",mp->myrank);
    //synchronize_mpi();

    size = NGLL3 * mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing

    // solid pressure
    copy_todevice_realw((void**)&mp->d_epsilon_trace_over_3,epsilon_trace_over_3,size);

    // backward solid pressure
    copy_todevice_realw((void**)&mp->d_b_epsilon_trace_over_3,b_epsilon_trace_over_3,size);

    // prepares backward strains
    copy_todevice_realw((void**)&mp->d_b_epsilondev_xx,b_epsilondev_xx,size);
    copy_todevice_realw((void**)&mp->d_b_epsilondev_yy,b_epsilondev_yy,size);
    copy_todevice_realw((void**)&mp->d_b_epsilondev_xy,b_epsilondev_xy,size);
    copy_todevice_realw((void**)&mp->d_b_epsilondev_xz,b_epsilondev_xz,size);
    copy_todevice_realw((void**)&mp->d_b_epsilondev_yz,b_epsilondev_yz,size);
  }

  // attenuation memory variables
  if (*ATTENUATION ){
    // debug
    //printf("prepare_fields_elastic_adj_dev: rank %d - attenuation\n",mp->myrank);
    //synchronize_mpi();

    size = *R_size;

    copy_todevice_realw((void**)&mp->d_b_R_xx,b_R_xx,size);
    copy_todevice_realw((void**)&mp->d_b_R_yy,b_R_yy,size);
    copy_todevice_realw((void**)&mp->d_b_R_xy,b_R_xy,size);
    copy_todevice_realw((void**)&mp->d_b_R_xz,b_R_xz,size);
    copy_todevice_realw((void**)&mp->d_b_R_yz,b_R_yz,size);

    // alpha,beta,gamma factors for backward fields
    copy_todevice_realw((void**)&mp->d_b_alphaval,b_alphaval,N_SLS);
    copy_todevice_realw((void**)&mp->d_b_betaval,b_betaval,N_SLS);
    copy_todevice_realw((void**)&mp->d_b_gammaval,b_gammaval,N_SLS);
  }

  // approximate hessian kernel
  if (*APPROXIMATE_HESS_KL ){
    // debug
    //printf("prepare_fields_elastic_adj_dev: rank %d - hessian kernel\n",mp->myrank);
    //synchronize_mpi();

    size = NGLL3 * mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_el_kl),size*sizeof(realw)),5450);
    print_CUDA_error_if_any(hipMemset(mp->d_hess_el_kl,0,size*sizeof(realw)),5451);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_rho_el_kl),size*sizeof(realw)),5452);
    print_CUDA_error_if_any(hipMemset(mp->d_hess_rho_el_kl,0,size*sizeof(realw)),5453);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_kappa_el_kl),size*sizeof(realw)),5454);
    print_CUDA_error_if_any(hipMemset(mp->d_hess_kappa_el_kl,0,size*sizeof(realw)),5455);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_mu_el_kl),size*sizeof(realw)),5456);
    print_CUDA_error_if_any(hipMemset(mp->d_hess_mu_el_kl,0,size*sizeof(realw)),5457);
  }

  // debug
  //printf("prepare_fields_elastic_adj_dev: rank %d - done\n",mp->myrank);
  //synchronize_mpi();

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_elastic_adj_dev");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// purely adjoint & kernel simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_sim2_or_3_const_device,
              PREPARE_SIM2_OR_3_CONST_DEVICE)(long* Mesh_pointer,int *nadj_rec_local, int* NTSTEP_BETWEEN_READ_ADJSRC) {

  TRACE("prepare_sim2_or_3_const_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  // adjoint source arrays
  mp->nadj_rec_local = *nadj_rec_local;
  if (mp->nadj_rec_local > 0){
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_source_adjoint,
                                       (mp->nadj_rec_local)* NDIM * sizeof(field) * (*NTSTEP_BETWEEN_READ_ADJSRC)),6005);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_sim2_or_3_const_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// for NOISE simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_noise_device,
              PREPARE_FIELDS_NOISE_DEVICE)(long* Mesh_pointer,
                                           int* NSPEC_AB, int* NGLOB_AB,
                                           int* free_surface_ispec,
                                           int* free_surface_ijk,
                                           int* num_free_surface_faces,
                                           int* NOISE_TOMOGRAPHY,
                                           int* NSTEP,
                                           realw* noise_sourcearray,
                                           realw* normal_x_noise, realw* normal_y_noise, realw* normal_z_noise,
                                           realw* mask_noise,
                                           realw* free_surface_jacobian2Dw) {

  TRACE("prepare_fields_noise_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  // free surface
  mp->num_free_surface_faces = *num_free_surface_faces;

  copy_todevice_int((void**)&mp->d_free_surface_ispec,free_surface_ispec,mp->num_free_surface_faces);
  copy_todevice_int((void**)&mp->d_free_surface_ijk,free_surface_ijk,
                    3*NGLL2*mp->num_free_surface_faces);

  // alloc storage for the surface buffer to be copied
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_noise_surface_movie,
                                     3*NGLL2*mp->num_free_surface_faces*sizeof(realw)),7005);

  // prepares noise source array
  if (*NOISE_TOMOGRAPHY == 1){
    copy_todevice_realw((void**)&mp->d_noise_sourcearray,noise_sourcearray,
                        3*NGLL3*(*NSTEP));
  }

  // prepares noise directions
  if (*NOISE_TOMOGRAPHY > 1){
    int nface_size = NGLL2*(*num_free_surface_faces);
    // allocates memory on GPU
    copy_todevice_realw((void**)&mp->d_normal_x_noise,normal_x_noise,nface_size);
    copy_todevice_realw((void**)&mp->d_normal_y_noise,normal_y_noise,nface_size);
    copy_todevice_realw((void**)&mp->d_normal_z_noise,normal_z_noise,nface_size);
    copy_todevice_realw((void**)&mp->d_mask_noise,mask_noise,nface_size);
    copy_todevice_realw((void**)&mp->d_free_surface_jacobian2Dw,free_surface_jacobian2Dw,nface_size);
  }

  // prepares noise strength kernel
  if (*NOISE_TOMOGRAPHY == 3){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_sigma_kl),NGLL3*(mp->NSPEC_AB)*sizeof(realw)),7401);
    // initializes kernel values to zero
    print_CUDA_error_if_any(hipMemset(mp->d_sigma_kl,0,NGLL3*mp->NSPEC_AB*sizeof(realw)),7403);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //printf("jacobian_size = %d\n",25*(*num_free_surface_faces));
  exit_on_cuda_error("prepare_fields_noise_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// GRAVITY simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_gravity_device,
              PREPARE_FIELDS_gravity_DEVICE)(long* Mesh_pointer,
                                             int* GRAVITY,
                                             realw* minus_deriv_gravity,
                                             realw* minus_g,
                                             realw* h_wgll_cube,
                                             int* ACOUSTIC_SIMULATION,
                                             realw* rhostore) {

  TRACE("prepare_fields_gravity_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);

  setConst_wgll_cube(h_wgll_cube,mp);

  mp->gravity = *GRAVITY;
  if (mp->gravity ){

    copy_todevice_realw((void**)&mp->d_minus_deriv_gravity,minus_deriv_gravity,mp->NGLOB_AB);
    copy_todevice_realw((void**)&mp->d_minus_g,minus_g,mp->NGLOB_AB);

    if (*ACOUSTIC_SIMULATION == 0){
      // density
      // rhostore not allocated yet
      int size_padded = NGLL3_PADDED * (mp->NSPEC_AB);
      // padded array
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rhostore),size_padded*sizeof(realw)),8006);
      // transfer constant element data with padding
      /*
      // way 1: slower ...
      for(int i=0; i < mp->NSPEC_AB; i++) {
        print_CUDA_error_if_any(hipMemcpy(mp->d_rhostore+i*NGLL3_PADDED, &rhostore[i*NGLL3],
                                           NGLL3*sizeof(realw),hipMemcpyHostToDevice),8007);
      }
      */
      // way 2: faster...
      print_CUDA_error_if_any(hipMemcpy2D(mp->d_rhostore, NGLL3_PADDED*sizeof(realw),
                                           rhostore, NGLL3*sizeof(realw), NGLL3*sizeof(realw),
                                           mp->NSPEC_AB, hipMemcpyHostToDevice),4800);
    }
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_gravity_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// unused yet...

/*
extern "C"
void FC_FUNC_(prepare_seismogram_fields,
              PREPARE_SEISMOGRAM_FIELDS)(long* Mesh_pointer,int* nrec_local, double* nu, double* hxir, double* hetar, double* hgammar) {

  TRACE("prepare_constants_device");
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_nu),3*3*(*nrec_local)*sizeof(double)),8100);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hxir),5*(*nrec_local)*sizeof(double)),8100);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hetar),5*(*nrec_local)*sizeof(double)),8100);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hgammar),5*(*nrec_local)*sizeof(double)),8100);

  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_seismograms_d,3*(*nrec_local)*sizeof(realw)),8101);
  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_seismograms_v,3*(*nrec_local)*sizeof(realw)),8101);
  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_seismograms_a,3*(*nrec_local)*sizeof(realw)),8101);

  print_CUDA_error_if_any(hipMemcpy(mp->d_nu,nu,3*3*(*nrec_local)*sizeof(double),hipMemcpyHostToDevice),8101);
  print_CUDA_error_if_any(hipMemcpy(mp->d_hxir,hxir,5*(*nrec_local)*sizeof(double),hipMemcpyHostToDevice),8101);
  print_CUDA_error_if_any(hipMemcpy(mp->d_hetar,hetar,5*(*nrec_local)*sizeof(double),hipMemcpyHostToDevice),8101);
  print_CUDA_error_if_any(hipMemcpy(mp->d_hgammar,hgammar,5*(*nrec_local)*sizeof(double),hipMemcpyHostToDevice),8101);

  hipHostMalloc((void**)&mp->h_seismograms_d_it,3**nrec_local*sizeof(realw));
  hipHostMalloc((void**)&mp->h_seismograms_v_it,3**nrec_local*sizeof(realw));
  hipHostMalloc((void**)&mp->h_seismograms_a_it,3**nrec_local*sizeof(realw));
}
*/





/* ----------------------------------------------------------------------------------------------- */

// FAULT simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fault_device,
              PREPARE_FAULT_DEVICE)(long* Mesh_pointer,
                                    int* KELVIN_VOIGT_DAMPING,
//                            int* testtrue,
                                    realw* Kelvin_Voigt_eta) {

  TRACE("prepare_fault_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer);
  mp -> Kelvin_Voigt_damping = *KELVIN_VOIGT_DAMPING ;
  if (mp-> Kelvin_Voigt_damping ){
//    if(*testtrue) printf("\ntesttrue!\n");
//    if(! (*KELVIN_VOIGT_DAMPING)) printf("\nKV test pass!\n");
//    printf("myrank = %d , size of damping = %6d, isAllocated? = %d\n",mp->myrank,sizeof(Kelvin_Voigt_eta),mp -> Kelvin_Voigt_damping);
    copy_todevice_realw((void**)&mp->d_Kelvin_Voigt_eta,Kelvin_Voigt_eta,mp-> NSPEC_AB);
  }
}


/* ----------------------------------------------------------------------------------------------- */

// cleanup

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_cleanup_device,
              PREPARE_CLEANUP_DEVICE)(long* Mesh_pointer,
                                      int* ACOUSTIC_SIMULATION,
                                      int* ELASTIC_SIMULATION,
                                      int* ABSORBING_CONDITIONS,
                                      int* NOISE_TOMOGRAPHY,
                                      int* COMPUTE_AND_STORE_STRAIN,
                                      int* ATTENUATION,
                                      int* ANISOTROPY,
                                      int* APPROXIMATE_OCEAN_LOAD,
                                      int* APPROXIMATE_HESS_KL) {

TRACE("prepare_cleanup_device");

  // frees allocated memory arrays
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  hipFree(mp->d_irregular_element_number);

  // frees memory on GPU
  // mesh
  hipFree(mp->d_xix);
  hipFree(mp->d_xiy);
  hipFree(mp->d_xiz);
  hipFree(mp->d_etax);
  hipFree(mp->d_etay);
  hipFree(mp->d_etaz);
  hipFree(mp->d_gammax);
  hipFree(mp->d_gammay);
  hipFree(mp->d_gammaz);

  // absorbing boundaries
  if (*ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0){
    hipFree(mp->d_abs_boundary_ispec);
    hipFree(mp->d_abs_boundary_ijk);
    hipFree(mp->d_abs_boundary_normal);
    hipFree(mp->d_abs_boundary_jacobian2Dw);
  }

  // interfaces
  if (mp->num_interfaces_ext_mesh > 0){
    hipFree(mp->d_nibool_interfaces_ext_mesh);
    hipFree(mp->d_ibool_interfaces_ext_mesh);
  }

  // global indexing
  hipFree(mp->d_ispec_is_inner);
  hipFree(mp->d_ibool);

  // sources
  if (mp->simulation_type == 1  || mp->simulation_type == 3){
    hipFree(mp->d_sourcearrays);
    hipFree(mp->d_stf_pre_compute);
  }

  hipFree(mp->d_islice_selected_source);
  hipFree(mp->d_ispec_selected_source);

  // receivers
  if (mp->nrec_local > 0){
    hipFree(mp->d_hxir);
    hipFree(mp->d_hetar);
    hipFree(mp->d_hgammar);
    if (mp->save_seismograms_d) hipFree(mp->d_seismograms_d);
    if (mp->save_seismograms_v) hipFree(mp->d_seismograms_v);
    if (mp->save_seismograms_a) hipFree(mp->d_seismograms_a);
    if (mp->save_seismograms_p) hipFree(mp->d_seismograms_p);
    hipFree(mp->d_nu);
    hipFree(mp->d_ispec_selected_rec_loc);
    }
    hipFree(mp->d_ispec_selected_rec);

  // ACOUSTIC arrays
  if (*ACOUSTIC_SIMULATION ){
    hipFree(mp->d_potential_acoustic);
    hipFree(mp->d_potential_dot_acoustic);
    hipFree(mp->d_potential_dot_dot_acoustic);
    hipFree(mp->d_send_potential_dot_dot_buffer);
    hipFree(mp->d_rmass_acoustic);
    hipFree(mp->d_rhostore);
    hipFree(mp->d_kappastore);
    hipFree(mp->d_phase_ispec_inner_acoustic);
    hipFree(mp->d_ispec_is_acoustic);

    if (*NOISE_TOMOGRAPHY == 0){
      hipFree(mp->d_free_surface_ispec);
      hipFree(mp->d_free_surface_ijk);
    }

    if (*ABSORBING_CONDITIONS) hipFree(mp->d_b_absorb_potential);

    if (mp->simulation_type == 3) {
      hipFree(mp->d_b_potential_acoustic);
      hipFree(mp->d_b_potential_dot_acoustic);
      hipFree(mp->d_b_potential_dot_dot_acoustic);
      hipFree(mp->d_rho_ac_kl);
      hipFree(mp->d_kappa_ac_kl);
      if (*APPROXIMATE_HESS_KL) {
        hipFree(mp->d_hess_ac_kl);
        hipFree(mp->d_hess_rho_ac_kl);
        hipFree(mp->d_hess_kappa_ac_kl);
      }
    }

  } // ACOUSTIC_SIMULATION

  // ELASTIC arrays
  if (*ELASTIC_SIMULATION ){
    hipFree(mp->d_displ);
    hipFree(mp->d_veloc);
    hipFree(mp->d_accel);

    hipFree(mp->d_send_accel_buffer);
    if (mp->simulation_type == 3) hipFree(mp->d_b_send_accel_buffer);

    hipFree(mp->d_rmassx);
    hipFree(mp->d_rmassy);
    hipFree(mp->d_rmassz);

    hipFree(mp->d_phase_ispec_inner_elastic);
    hipFree(mp->d_ispec_is_elastic);

    if (*ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0){
      hipFree(mp->d_rho_vp);
      hipFree(mp->d_rho_vs);

      if (mp->simulation_type == 3 || ( mp->simulation_type == 1 && mp->save_forward ))
          hipFree(mp->d_b_absorb_field);
    }

    hipFree(mp->d_kappav);
    hipFree(mp->d_muv);

    if (mp->simulation_type == 3) {
      hipFree(mp->d_b_displ);
      hipFree(mp->d_b_veloc);
      hipFree(mp->d_b_accel);
      hipFree(mp->d_rho_kl);
      if (mp->anisotropic_kl ){
        hipFree(mp->d_cijkl_kl);
      }else{
        hipFree(mp->d_mu_kl);
        hipFree(mp->d_kappa_kl);
      }
      if (*APPROXIMATE_HESS_KL) {
        hipFree(mp->d_hess_el_kl);
        hipFree(mp->d_hess_rho_el_kl);
        hipFree(mp->d_hess_kappa_el_kl);
        hipFree(mp->d_hess_mu_el_kl);
      }
    }

    if (*COMPUTE_AND_STORE_STRAIN ){
      hipFree(mp->d_epsilondev_xx);
      hipFree(mp->d_epsilondev_yy);
      hipFree(mp->d_epsilondev_xy);
      hipFree(mp->d_epsilondev_xz);
      hipFree(mp->d_epsilondev_yz);
      if (mp->simulation_type == 3){
        hipFree(mp->d_epsilon_trace_over_3);
        hipFree(mp->d_b_epsilon_trace_over_3);
        hipFree(mp->d_b_epsilondev_xx);
        hipFree(mp->d_b_epsilondev_yy);
        hipFree(mp->d_b_epsilondev_xy);
        hipFree(mp->d_b_epsilondev_xz);
        hipFree(mp->d_b_epsilondev_yz);
      }
    }

    if (*ATTENUATION ){
      hipFree(mp->d_factor_common);
      hipFree(mp->d_alphaval);
      hipFree(mp->d_betaval);
      hipFree(mp->d_gammaval);
      hipFree(mp->d_R_xx);
      hipFree(mp->d_R_yy);
      hipFree(mp->d_R_xy);
      hipFree(mp->d_R_xz);
      hipFree(mp->d_R_yz);
      if (mp->simulation_type == 3){
        hipFree(mp->d_b_R_xx);
        hipFree(mp->d_b_R_yy);
        hipFree(mp->d_b_R_xy);
        hipFree(mp->d_b_R_xz);
        hipFree(mp->d_b_R_yz);
        hipFree(mp->d_b_alphaval);
        hipFree(mp->d_b_betaval);
        hipFree(mp->d_b_gammaval);
      }
    }

    if (*ANISOTROPY ){
      hipFree(mp->d_c11store);
      hipFree(mp->d_c12store);
      hipFree(mp->d_c13store);
      hipFree(mp->d_c14store);
      hipFree(mp->d_c15store);
      hipFree(mp->d_c16store);
      hipFree(mp->d_c22store);
      hipFree(mp->d_c23store);
      hipFree(mp->d_c24store);
      hipFree(mp->d_c25store);
      hipFree(mp->d_c26store);
      hipFree(mp->d_c33store);
      hipFree(mp->d_c34store);
      hipFree(mp->d_c35store);
      hipFree(mp->d_c36store);
      hipFree(mp->d_c44store);
      hipFree(mp->d_c45store);
      hipFree(mp->d_c46store);
      hipFree(mp->d_c55store);
      hipFree(mp->d_c56store);
      hipFree(mp->d_c66store);
    }

    if (*APPROXIMATE_OCEAN_LOAD ){
      if (mp->num_free_surface_faces > 0){
        hipFree(mp->d_rmass_ocean_load);
        hipFree(mp->d_free_surface_normal);
        hipFree(mp->d_updated_dof_ocean_load);
        if (*NOISE_TOMOGRAPHY == 0){
          hipFree(mp->d_free_surface_ispec);
          hipFree(mp->d_free_surface_ijk);
        }
      }
    }
  } // ELASTIC_SIMULATION

  // purely adjoint & kernel array
  if (mp->simulation_type == 2 || mp->simulation_type == 3){
    if (mp->nadj_rec_local > 0){
      hipFree(mp->d_source_adjoint);
    }
  }

  // NOISE arrays
  if (*NOISE_TOMOGRAPHY > 0){
    hipFree(mp->d_free_surface_ispec);
    hipFree(mp->d_free_surface_ijk);
    hipFree(mp->d_noise_surface_movie);
    if (*NOISE_TOMOGRAPHY == 1) hipFree(mp->d_noise_sourcearray);
    if (*NOISE_TOMOGRAPHY > 1){
      hipFree(mp->d_normal_x_noise);
      hipFree(mp->d_normal_y_noise);
      hipFree(mp->d_normal_z_noise);
      hipFree(mp->d_mask_noise);
      hipFree(mp->d_free_surface_jacobian2Dw);
    }
    if (*NOISE_TOMOGRAPHY == 3) hipFree(mp->d_sigma_kl);
  }

  // mesh pointer - not needed anymore
  free(mp);
}


