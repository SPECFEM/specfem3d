#include "hip/hip_runtime.h"
/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  2 . 0
 !               ---------------------------------------
 !
 !          Main authors: Dimitri Komatitsch and Jeroen Tromp
 !    Princeton University, USA and University of Pau / CNRS / INRIA
 ! (c) Princeton University / California Institute of Technology and University of Pau / CNRS / INRIA
 !                            April 2011
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#ifdef WITH_MPI
#include <mpi.h>
#endif

#include <sys/time.h>
#include <sys/resource.h>

#include "config.h"
#include "mesh_constants_cuda.h"
#include "prepare_constants_cuda.h"


/* ----------------------------------------------------------------------------------------------- */

// Helper functions

/* ----------------------------------------------------------------------------------------------- */

double get_time()
{
  struct timeval t;
  struct timezone tzp;
  gettimeofday(&t, &tzp);
  return t.tv_sec + t.tv_usec*1e-6;
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(pause_for_debug,PAUSE_FOR_DEBUG)() {
TRACE("pause_for_debug");

  pause_for_debugger(1);
}

/* ----------------------------------------------------------------------------------------------- */

void pause_for_debugger(int pause) {
  if(pause) {
    int myrank;
#ifdef WITH_MPI
    MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
#else
    myrank = 0;
#endif
    printf("I'm rank %d\n",myrank);
    int i = 0;
    char hostname[256];
    gethostname(hostname, sizeof(hostname));
    printf("PID %d on %s:%d ready for attach\n", getpid(), hostname,myrank);
    FILE *file = fopen("/scratch/eiger/rietmann/attach_gdb.txt","w+");
    if (file != NULL){
      fprintf(file,"PID %d on %s:%d ready for attach\n", getpid(), hostname,myrank);
      fclose(file);
    }
    fflush(stdout);
    while (0 == i)
      sleep(5);
  }
}

/* ----------------------------------------------------------------------------------------------- */

void exit_on_cuda_error(char* kernel_name) {
  // sync and check to catch errors from previous async operations
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess)
    {
      fprintf(stderr,"Error after %s: %s\n", kernel_name, hipGetErrorString(err));
      pause_for_debugger(0);
      //free(kernel_name);
#ifdef WITH_MPI
      MPI_Abort(MPI_COMM_WORLD,1);
#endif
      exit(EXIT_FAILURE);
    }
}

/* ----------------------------------------------------------------------------------------------- */

void exit_on_error(char* info)
{
  printf("\nERROR: %s\n",info);
  fflush(stdout);
#ifdef WITH_MPI
  MPI_Abort(MPI_COMM_WORLD,1);
#endif
  //free(info);
  exit(EXIT_FAILURE);
  return;
}

/* ----------------------------------------------------------------------------------------------- */

void print_CUDA_error_if_any(hipError_t err, int num)
{
  if (hipSuccess != err)
  {
    printf("\nCUDA error !!!!! <%s> !!!!! \nat CUDA call error code: # %d\n",hipGetErrorString(err),num);
    fflush(stdout);
#ifdef WITH_MPI
    MPI_Abort(MPI_COMM_WORLD,1);
#endif
    exit(EXIT_FAILURE);
  }
  return;
}

/* ----------------------------------------------------------------------------------------------- */

void get_free_memory(double* free_db, double* used_db, double* total_db) {

  // gets memory usage in byte
  size_t free_byte ;
  size_t total_byte ;
  hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
  if ( hipSuccess != cuda_status ){
    printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
    exit(EXIT_FAILURE);
  }

  *free_db = (double)free_byte ;
  *total_db = (double)total_byte ;
  *used_db = *total_db - *free_db ;
  return;
}

/* ----------------------------------------------------------------------------------------------- */

// Saves GPU memory usage to file
void output_free_memory(int myrank,char* info_str) {

  FILE* fp;
  char filename[BUFSIZ];
  double free_db,used_db,total_db;

  get_free_memory(&free_db,&used_db,&total_db);

  sprintf(filename,"../in_out_files/OUTPUT_FILES/gpu_device_mem_usage_proc_%06d.txt",myrank);
  fp = fopen(filename,"a+");
  if (fp != NULL){
    fprintf(fp,"%d: @%s GPU memory usage: used = %f MB, free = %f MB, total = %f MB\n", myrank, info_str,
            used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
    fclose(fp);
  }
}

/* ----------------------------------------------------------------------------------------------- */

// Fortran-callable version of above method
extern "C"
void FC_FUNC_(output_free_device_memory,
              OUTPUT_FREE_DEVICE_MEMORY)(int* myrank) {
TRACE("output_free_device_memory");

  char info[6];
  sprintf(info,"f %d:",*myrank);
  output_free_memory(*myrank,info);
}

/* ----------------------------------------------------------------------------------------------- */

/*
void show_free_memory(char* info_str) {

  // show memory usage of GPU
  int myrank;
#ifdef WITH_MPI
  MPI_Comm_rank(MPI_COMM_WORLD,&myrank);
#else
  myrank = 0;
#endif
  double free_db,used_db,total_db;

  get_free_memory(&free_db,&used_db,&total_db);

  printf("%d: @%s GPU memory usage: used = %f MB, free = %f MB, total = %f MB\n", myrank, info_str,
   used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);

}
*/

/*
extern "C"
void FC_FUNC_(show_free_device_memory,
              SHOW_FREE_DEVICE_MEMORY)() {
 TRACE("show_free_device_memory");

 show_free_memory("from fortran");
}
*/

/* ----------------------------------------------------------------------------------------------- */


extern "C"
void FC_FUNC_(get_free_device_memory,
              get_FREE_DEVICE_MEMORY)(realw* free, realw* used, realw* total ) {
TRACE("get_free_device_memory");

  double free_db,used_db,total_db;

  get_free_memory(&free_db,&used_db,&total_db);

  // converts to MB
  *free = (realw) free_db/1024.0/1024.0;
  *used = (realw) used_db/1024.0/1024.0;
  *total = (realw) total_db/1024.0/1024.0;
  return;
}


/* ----------------------------------------------------------------------------------------------- */
//daniel: helper function
/*
__global__ void check_phase_ispec_kernel(int num_phase_ispec,
                                         int* phase_ispec,
                                         int NSPEC_AB,
                                         int* ier) {

  int i,ispec,iphase,count0,count1;
  *ier = 0;

  for(iphase=0; iphase < 2; iphase++){
    count0 = 0;
    count1 = 0;

    for(i=0; i < num_phase_ispec; i++){
      ispec = phase_ispec[iphase*num_phase_ispec + i] - 1;
      if( ispec < -1 || ispec >= NSPEC_AB ){
        printf("Error in d_phase_ispec_inner_elastic %d %d\n",i,ispec);
        *ier = 1;
        return;
      }
      if( ispec >= 0 ){ count0++;}
      if( ispec < 0 ){ count1++;}
    }

    printf("check_phase_ispec done: phase %d, count = %d %d \n",iphase,count0,count1);

  }
}

void check_phase_ispec(long* Mesh_pointer_f,int type){

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  printf("check phase_ispec for type=%d\n",type);

  dim3 grid(1,1);
  dim3 threads(1,1,1);

  int* h_debug = (int*) calloc(1,sizeof(int));
  int* d_debug;
  hipMalloc((void**)&d_debug,sizeof(int));

  if( type == 1 ){
    check_phase_ispec_kernel<<<grid,threads>>>(mp->num_phase_ispec_elastic,
                                             mp->d_phase_ispec_inner_elastic,
                                             mp->NSPEC_AB,
                                             d_debug);
  }else if( type == 2 ){
    check_phase_ispec_kernel<<<grid,threads>>>(mp->num_phase_ispec_acoustic,
                                               mp->d_phase_ispec_inner_acoustic,
                                               mp->NSPEC_AB,
                                               d_debug);
  }

  hipMemcpy(h_debug,d_debug,1*sizeof(int),hipMemcpyDeviceToHost);
  hipFree(d_debug);
  if( *h_debug != 0 ){printf("error for type=%d\n",type); exit(1);}
  free(h_debug);
  fflush(stdout);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("check_phase_ispec");
#endif

}
*/

/* ----------------------------------------------------------------------------------------------- */
//daniel: helper function
/*
__global__ void check_ispec_is_kernel(int NSPEC_AB,
                                      int* ispec_is,
                                      int* ier) {

  int ispec,count0,count1;

  *ier = 0;
  count0 = 0;
  count1 = 0;
  for(ispec=0; ispec < NSPEC_AB; ispec++){
    if( ispec_is[ispec] < -1 || ispec_is[ispec] > 1 ){
      printf("Error in ispec_is %d %d\n",ispec,ispec_is[ispec]);
      *ier = 1;
      return;
      //exit(1);
    }
    if( ispec_is[ispec] == 0 ){count0++;}
    if( ispec_is[ispec] != 0 ){count1++;}
  }
  printf("check_ispec_is done: count = %d %d\n",count0,count1);
}

void check_ispec_is(long* Mesh_pointer_f,int type){

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  printf("check ispec_is for type=%d\n",type);

  dim3 grid(1,1);
  dim3 threads(1,1,1);

  int* h_debug = (int*) calloc(1,sizeof(int));
  int* d_debug;
  hipMalloc((void**)&d_debug,sizeof(int));

  if( type == 0 ){
    check_ispec_is_kernel<<<grid,threads>>>(mp->NSPEC_AB,
                                            mp->d_ispec_is_inner,
                                            d_debug);
  }else if( type == 1 ){
    check_ispec_is_kernel<<<grid,threads>>>(mp->NSPEC_AB,
                                            mp->d_ispec_is_elastic,
                                            d_debug);
  }else if( type == 2 ){
    check_ispec_is_kernel<<<grid,threads>>>(mp->NSPEC_AB,
                                            mp->d_ispec_is_acoustic,
                                            d_debug);
  }

  hipMemcpy(h_debug,d_debug,1*sizeof(int),hipMemcpyDeviceToHost);
  hipFree(d_debug);
  if( *h_debug != 0 ){printf("error for type=%d\n",type); exit(1);}
  free(h_debug);
  fflush(stdout);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("check_ispec_is");
#endif
}
*/
/* ----------------------------------------------------------------------------------------------- */
//daniel: helper function
/*
__global__ void check_array_ispec_kernel(int num_array_ispec,
                                         int* array_ispec,
                                         int NSPEC_AB,
                                         int* ier) {

  int i,ispec,count0,count1;

  *ier = 0;
  count0 = 0;
  count1 = 0;

  for(i=0; i < num_array_ispec; i++){
    ispec = array_ispec[i] - 1;
    if( ispec < -1 || ispec >= NSPEC_AB ){
      printf("Error in d_array_ispec %d %d\n",i,ispec);
      *ier = 1;
      return;
    }
    if( ispec >= 0 ){ count0++;}
    if( ispec < 0 ){ count1++;}
  }

  printf("check_array_ispec done: count = %d %d \n",count0,count1);
}

void check_array_ispec(long* Mesh_pointer_f,int type){

  Mesh* mp = (Mesh*)(*Mesh_pointer_f); //get mesh pointer out of fortran integer container

  printf("check array_ispec for type=%d\n",type);

  dim3 grid(1,1);
  dim3 threads(1,1,1);

  int* h_debug = (int*) calloc(1,sizeof(int));
  int* d_debug;
  hipMalloc((void**)&d_debug,sizeof(int));

  if( type == 1 ){
    check_array_ispec_kernel<<<grid,threads>>>(mp->d_num_abs_boundary_faces,
                                               mp->d_abs_boundary_ispec,
                                               mp->NSPEC_AB,
                                               d_debug);
  }

  hipMemcpy(h_debug,d_debug,1*sizeof(int),hipMemcpyDeviceToHost);
  hipFree(d_debug);
  if( *h_debug != 0 ){printf("error for type=%d\n",type); exit(1);}
  free(h_debug);
  fflush(stdout);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("check_array_ispec");
#endif

}
*/

/* ----------------------------------------------------------------------------------------------- */

// GPU preparation

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_cuda_device,
              PREPARE_CUDA_DEVICE)(int* myrank_f,int* ncuda_devices) {
  TRACE("prepare_cuda_device");

  // Gets rank number of MPI process
  int myrank = *myrank_f;

/*
  // cuda initialization (needs -lcuda library)
  // note:   hipInit initializes the driver API.
  //             it is needed for any following CUDA driver API function call (format cuFUNCTION(..) )
  //             however, for the CUDA runtime API functions (format cudaFUNCTION(..) )
  //             the initialization is implicit, thus hipInit() here would not be needed...
  hipError_t status = hipInit(0);
  if ( hipSuccess != status ) exit_on_error("CUDA driver API device initialization failed\n");

  // returns a handle to the first cuda compute device
  hipDevice_t dev;
  status = hipDeviceGet(&dev, 0);
  if ( hipSuccess != status ) exit_on_error("CUDA device not found\n");

  // gets device properties
  int major,minor;
  status = hipDeviceComputeCapability(&major,&minor,dev);
  if ( hipSuccess != status ) exit_on_error("CUDA device information not found\n");

  // make sure that the device has compute capability >= 1.3
  if (major < 1){
    fprintf(stderr,"Compute capability major number should be at least 1, got: %d \nexiting...\n",major);
    exit_on_error("CUDA Compute capability major number should be at least 1\n");
  }
  if (major == 1 && minor < 3){
    fprintf(stderr,"Compute capability should be at least 1.3, got: %d.%d \nexiting...\n",major,minor);
    exit_on_error("CUDA Compute capability major number should be at least 1.3\n");
  }
*/

  // note: from here on we use the runtime API  ...

  // Gets number of GPU devices
  int device_count = 0;
  hipGetDeviceCount(&device_count);
  exit_on_cuda_error("CUDA runtime error: hipGetDeviceCount failed\ncheck if driver and runtime libraries work together\nexiting...\n");

  // returns device count to fortran
  if (device_count == 0) exit_on_error("CUDA runtime error: there is no device supporting CUDA\n");
  *ncuda_devices = device_count;


  // Sets the active device
  if(device_count > 1) {
    // generalized for more GPUs per node
    // note: without previous context release, hipSetDevice will complain with the cuda error
    //         "setting the device when a process is active is not allowed"
    // releases previous contexts
    hipDeviceReset();

    //printf("rank %d: cuda device count = %d sets device = %d \n",myrank,device_count,myrank % device_count);
    //MPI_Barrier(MPI_COMM_WORLD);

    // sets active device
    hipSetDevice( myrank % device_count );
    exit_on_cuda_error("hipSetDevice");
  }

  // returns a handle to the active device
  int device;
  hipGetDevice(&device);

  // get device properties
  struct hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp,device);

  // exit if the machine has no CUDA-enabled device
  if (deviceProp.major == 9999 && deviceProp.minor == 9999){
    fprintf(stderr,"No CUDA-enabled device found, exiting...\n\n");
    exit_on_error("CUDA runtime error: there is no CUDA-enabled device found\n");
  }

  // outputs device infos to file
  char filename[BUFSIZ];
  FILE* fp;
  sprintf(filename,"../in_out_files/OUTPUT_FILES/gpu_device_info_proc_%06d.txt",myrank);
  fp = fopen(filename,"a+");
  if (fp != NULL){
    // display device properties
    fprintf(fp,"Device Name = %s\n",deviceProp.name);
    fprintf(fp,"multiProcessorCount: %d\n",deviceProp.multiProcessorCount);
    fprintf(fp,"totalGlobalMem (in MB): %f\n",(unsigned long) deviceProp.totalGlobalMem / (1024.f * 1024.f));
    fprintf(fp,"totalGlobalMem (in GB): %f\n",(unsigned long) deviceProp.totalGlobalMem / (1024.f * 1024.f * 1024.f));
    fprintf(fp,"sharedMemPerBlock (in bytes): %lu\n",(unsigned long) deviceProp.sharedMemPerBlock);
    fprintf(fp,"Maximum number of threads per block: %d\n",deviceProp.maxThreadsPerBlock);
    fprintf(fp,"Maximum size of each dimension of a block: %d x %d x %d\n",
            deviceProp.maxThreadsDim[0],deviceProp.maxThreadsDim[1],deviceProp.maxThreadsDim[2]);
    fprintf(fp,"Maximum sizes of each dimension of a grid: %d x %d x %d\n",
            deviceProp.maxGridSize[0],deviceProp.maxGridSize[1],deviceProp.maxGridSize[2]);
    fprintf(fp,"Compute capability of the device = %d.%d\n", deviceProp.major, deviceProp.minor);
    if(deviceProp.canMapHostMemory){
      fprintf(fp,"canMapHostMemory: TRUE\n");
    }else{
      fprintf(fp,"canMapHostMemory: FALSE\n");
    }
    if(deviceProp.deviceOverlap){
      fprintf(fp,"deviceOverlap: TRUE\n");
    }else{
      fprintf(fp,"deviceOverlap: FALSE\n");
    }

    // outputs initial memory infos via hipMemGetInfo()
    double free_db,used_db,total_db;
    get_free_memory(&free_db,&used_db,&total_db);
    fprintf(fp,"%d: GPU memory usage: used = %f MB, free = %f MB, total = %f MB\n", myrank,
            used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);

    fclose(fp);
  }

  // make sure that the device has compute capability >= 1.3
  if (deviceProp.major < 1){
    fprintf(stderr,"Compute capability major number should be at least 1, exiting...\n\n");
    exit_on_error("CUDA Compute capability major number should be at least 1\n");
  }
  if (deviceProp.major == 1 && deviceProp.minor < 3){
    fprintf(stderr,"Compute capability should be at least 1.3, exiting...\n");
    exit_on_error("CUDA Compute capability major number should be at least 1.3\n");
  }
  // we use pinned memory for asynchronous copy
  if( ! deviceProp.canMapHostMemory){
    fprintf(stderr,"Device capability should allow to map host memory, exiting...\n");
    exit_on_error("CUDA Device capability canMapHostMemory should be TRUE\n");
  }
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_constants_device,
              PREPARE_CONSTANTS_DEVICE)(long* Mesh_pointer,
                                        int* h_NGLLX,
                                        int* NSPEC_AB, int* NGLOB_AB,
                                        realw* h_xix, realw* h_xiy, realw* h_xiz,
                                        realw* h_etax, realw* h_etay, realw* h_etaz,
                                        realw* h_gammax, realw* h_gammay, realw* h_gammaz,
                                        realw* h_kappav, realw* h_muv,
                                        int* h_ibool,
                                        int* num_interfaces_ext_mesh,
                                        int* max_nibool_interfaces_ext_mesh,
                                        int* h_nibool_interfaces_ext_mesh,
                                        int* h_ibool_interfaces_ext_mesh,
                                        realw* h_hprime_xx,realw* h_hprime_yy,realw* h_hprime_zz,
                                        realw* h_hprimewgll_xx,realw* h_hprimewgll_yy,realw* h_hprimewgll_zz,
                                        realw* h_wgllwgll_xy,realw* h_wgllwgll_xz,realw* h_wgllwgll_yz,
                                        int* ABSORBING_CONDITIONS,
                                        int* h_abs_boundary_ispec, int* h_abs_boundary_ijk,
                                        realw* h_abs_boundary_normal,
                                        realw* h_abs_boundary_jacobian2Dw,
                                        int* h_num_abs_boundary_faces,
                                        int* h_ispec_is_inner,
                                        int* NSOURCES,
                                        int* nsources_local_f,
                                        realw* h_sourcearrays,
                                        int* h_islice_selected_source,
                                        int* h_ispec_selected_source,
                                        int* h_number_receiver_global,
                                        int* h_ispec_selected_rec,
                                        int* nrec_f,
                                        int* nrec_local_f,
                                        int* SIMULATION_TYPE,
                                        int* USE_MESH_COLORING_GPU_f,
                                        int* nspec_acoustic,int* nspec_elastic,
                                        int* my_neighbours_ext_mesh,
                                        int* request_send_vector_ext_mesh,
                                        int* request_recv_vector_ext_mesh,
                                        realw* buffer_recv_vector_ext_mesh
                                        ) {

TRACE("prepare_constants_device");

  // allocates mesh parameter structure
  Mesh* mp = (Mesh*) malloc( sizeof(Mesh) );
  if (mp == NULL) exit_on_error("error allocating mesh pointer");
  *Mesh_pointer = (long)mp;

  // checks if NGLLX == 5
  if( *h_NGLLX != NGLLX ){
    exit_on_error("NGLLX must be 5 for CUDA devices");
  }


#ifdef WITH_MPI
  int nproc;
  MPI_Comm_size(MPI_COMM_WORLD,&nproc);
  mp->NPROCS=nproc;
#else
  mp->NPROCS = 1;
#endif


  // sets global parameters
  mp->NSPEC_AB = *NSPEC_AB;
  mp->NGLOB_AB = *NGLOB_AB;

  // sets constant arrays
  setConst_hprime_xx(h_hprime_xx,mp);
  // only needed if NGLLX != NGLLY != NGLLZ
  // setConst_hprime_yy(h_hprime_yy,mp);
  // setConst_hprime_zz(h_hprime_zz,mp);
  setConst_hprimewgll_xx(h_hprimewgll_xx,mp);
  setConst_hprimewgll_yy(h_hprimewgll_yy,mp);
  setConst_hprimewgll_zz(h_hprimewgll_zz,mp);
  setConst_wgllwgll_xy(h_wgllwgll_xy,mp);
  setConst_wgllwgll_xz(h_wgllwgll_xz,mp);
  setConst_wgllwgll_yz(h_wgllwgll_yz,mp);

  // Using texture memory for the hprime-style constants is slower on
  // Fermi generation hardware, but *may* be faster on Kepler
  // generation. We will reevaluate this again, so might as well leave
  // in the code with with #USE_TEXTURES_FIELDS not-defined.
  #ifdef USE_TEXTURES_CONSTANTS
  {
    const textureReference* d_hprime_xx_tex_ptr;
    print_CUDA_error_if_any(hipGetTextureReference(&d_hprime_xx_tex_ptr, HIP_SYMBOL("d_hprime_xx_tex")), 4101);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    print_CUDA_error_if_any(hipBindTexture(0, d_hprime_xx_tex_ptr, mp->d_hprime_xx, &channelDesc, sizeof(realw)*(NGLL2)), 4001);
  }
  #endif


  // Allocate pinned mpi-buffers.
  // MPI buffers use pinned memory allocated by hipHostMalloc, which
  // enables the use of asynchronous memory copies from host <->
  // device
  int size_mpi_buffer = 3 * (*num_interfaces_ext_mesh) * (*max_nibool_interfaces_ext_mesh);
  print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_send_accel_buffer),sizeof(float)*(size_mpi_buffer)),8004);
  mp->send_buffer = (float*)malloc((size_mpi_buffer)*sizeof(float));
  mp->size_mpi_send_buffer = size_mpi_buffer;

  print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_recv_accel_buffer),sizeof(float)*(size_mpi_buffer)),8004);
  mp->recv_buffer = (float*)malloc((size_mpi_buffer)*sizeof(float));
  mp->size_mpi_recv_buffer = size_mpi_buffer;

  print_CUDA_error_if_any(hipHostMalloc((void**)&(mp->h_send_b_accel_buffer),sizeof(float)*(size_mpi_buffer)),8004);
  // mp->b_send_buffer = (float*)malloc((size_mpi_buffer)*sizeof(float));

  mp->num_interfaces_ext_mesh = *num_interfaces_ext_mesh;
  mp->max_nibool_interfaces_ext_mesh = *max_nibool_interfaces_ext_mesh;
  mp->nibool_interfaces_ext_mesh = h_nibool_interfaces_ext_mesh;
  mp->my_neighbours_ext_mesh = my_neighbours_ext_mesh;
  mp->request_send_vector_ext_mesh = request_send_vector_ext_mesh;
  mp->request_recv_vector_ext_mesh = request_recv_vector_ext_mesh;
  mp->buffer_recv_vector_ext_mesh = buffer_recv_vector_ext_mesh;

  // setup two streams, one for compute and one for host<->device memory copies
  hipStreamCreate(&mp->compute_stream);
  hipStreamCreate(&mp->copy_stream);
  hipStreamCreate(&mp->b_copy_stream);

  /* Assuming NGLLX=5. Padded is then 128 (5^3+3) */
  int size_padded = NGLL3_PADDED * (mp->NSPEC_AB);

  // mesh
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xix, size_padded*sizeof(realw)),1001);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiy, size_padded*sizeof(realw)),1002);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_xiz, size_padded*sizeof(realw)),1003);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etax, size_padded*sizeof(realw)),1004);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etay, size_padded*sizeof(realw)),1005);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_etaz, size_padded*sizeof(realw)),1006);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammax, size_padded*sizeof(realw)),1007);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammay, size_padded*sizeof(realw)),1008);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_gammaz, size_padded*sizeof(realw)),1009);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_kappav, size_padded*sizeof(realw)),1010);
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_muv, size_padded*sizeof(realw)),1011);

  // transfer constant element data with padding
  for(int i=0;i < mp->NSPEC_AB;i++) {
    print_CUDA_error_if_any(hipMemcpy(mp->d_xix + i*NGLL3_PADDED, &h_xix[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1501);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xiy+i*NGLL3_PADDED,   &h_xiy[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1502);
    print_CUDA_error_if_any(hipMemcpy(mp->d_xiz+i*NGLL3_PADDED,   &h_xiz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1503);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etax+i*NGLL3_PADDED,  &h_etax[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1504);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etay+i*NGLL3_PADDED,  &h_etay[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1505);
    print_CUDA_error_if_any(hipMemcpy(mp->d_etaz+i*NGLL3_PADDED,  &h_etaz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1506);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammax+i*NGLL3_PADDED,&h_gammax[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1507);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammay+i*NGLL3_PADDED,&h_gammay[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1508);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammaz+i*NGLL3_PADDED,&h_gammaz[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1509);
    print_CUDA_error_if_any(hipMemcpy(mp->d_kappav+i*NGLL3_PADDED,&h_kappav[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1510);
    print_CUDA_error_if_any(hipMemcpy(mp->d_muv+i*NGLL3_PADDED,   &h_muv[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),1511);
  }

  // global indexing
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool,size_padded*sizeof(int)),1021);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ibool, h_ibool,
                                     NGLL3*(mp->NSPEC_AB)*sizeof(int),hipMemcpyHostToDevice),1022);


  // prepare interprocess-edge exchange information
  mp->num_interfaces_ext_mesh = *num_interfaces_ext_mesh;
  mp->max_nibool_interfaces_ext_mesh = *max_nibool_interfaces_ext_mesh;
  if( mp->num_interfaces_ext_mesh > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_nibool_interfaces_ext_mesh,
                                       (mp->num_interfaces_ext_mesh)*sizeof(int)),1201);
    print_CUDA_error_if_any(hipMemcpy(mp->d_nibool_interfaces_ext_mesh,h_nibool_interfaces_ext_mesh,
                                       (mp->num_interfaces_ext_mesh)*sizeof(int),hipMemcpyHostToDevice),1202);

    print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ibool_interfaces_ext_mesh,
                                       (mp->num_interfaces_ext_mesh)*(mp->max_nibool_interfaces_ext_mesh)*sizeof(int)),1203);
    print_CUDA_error_if_any(hipMemcpy(mp->d_ibool_interfaces_ext_mesh,h_ibool_interfaces_ext_mesh,
                                       (mp->num_interfaces_ext_mesh)*(mp->max_nibool_interfaces_ext_mesh)*sizeof(int),
                                       hipMemcpyHostToDevice),1204);
  }

  // inner elements
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_ispec_is_inner,mp->NSPEC_AB*sizeof(int)),1205);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_is_inner, h_ispec_is_inner,
                                     mp->NSPEC_AB*sizeof(int),hipMemcpyHostToDevice),1206);

  // absorbing boundaries
  mp->d_num_abs_boundary_faces = *h_num_abs_boundary_faces;
  if( *ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_abs_boundary_ispec),
                                       (mp->d_num_abs_boundary_faces)*sizeof(int)),1101);
    print_CUDA_error_if_any(hipMemcpy(mp->d_abs_boundary_ispec, h_abs_boundary_ispec,
                                       (mp->d_num_abs_boundary_faces)*sizeof(int),
                                       hipMemcpyHostToDevice),1102);

    print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_abs_boundary_ijk),
                                       3*NGLL2*(mp->d_num_abs_boundary_faces)*sizeof(int)),1103);
    print_CUDA_error_if_any(hipMemcpy(mp->d_abs_boundary_ijk, h_abs_boundary_ijk,
                                       3*NGLL2*(mp->d_num_abs_boundary_faces)*sizeof(int),
                                       hipMemcpyHostToDevice),1104);

    print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_abs_boundary_normal),
                                       3*NGLL2*(mp->d_num_abs_boundary_faces)*sizeof(realw)),1105);
    print_CUDA_error_if_any(hipMemcpy(mp->d_abs_boundary_normal, h_abs_boundary_normal,
                                       3*NGLL2*(mp->d_num_abs_boundary_faces)*sizeof(realw),
                                       hipMemcpyHostToDevice),1106);

    print_CUDA_error_if_any(hipMalloc((void**) &(mp->d_abs_boundary_jacobian2Dw),
                                       NGLL2*(mp->d_num_abs_boundary_faces)*sizeof(realw)),1107);
    print_CUDA_error_if_any(hipMemcpy(mp->d_abs_boundary_jacobian2Dw, h_abs_boundary_jacobian2Dw,
                                       NGLL2*(mp->d_num_abs_boundary_faces)*sizeof(realw),
                                       hipMemcpyHostToDevice),1108);
  }

  // sources
  mp->nsources_local = *nsources_local_f;
  if (*SIMULATION_TYPE == 1  || *SIMULATION_TYPE == 3){
    // not needed in case of pure adjoint simulations (SIMULATION_TYPE == 2)
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_sourcearrays,
                                       sizeof(realw)* *NSOURCES*3*NGLL3),1301);
    print_CUDA_error_if_any(hipMemcpy(mp->d_sourcearrays, h_sourcearrays,
                                       sizeof(realw)* *NSOURCES*3*NGLL3,hipMemcpyHostToDevice),1302);

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_stf_pre_compute,
                                       *NSOURCES*sizeof(double)),1303);
  }

  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_islice_selected_source,
                                     sizeof(int) * *NSOURCES),1401);
  print_CUDA_error_if_any(hipMemcpy(mp->d_islice_selected_source, h_islice_selected_source,
                                     sizeof(int)* *NSOURCES,hipMemcpyHostToDevice),1402);

  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_ispec_selected_source,
                                     sizeof(int)* *NSOURCES),1403);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_selected_source, h_ispec_selected_source,
                                     sizeof(int)* *NSOURCES,hipMemcpyHostToDevice),1404);


  // receiver stations
  int nrec = *nrec_f; // total number of receivers
  mp->nrec_local = *nrec_local_f; // number of receiver located in this partition
  //int nrec_local = *nrec_local_f;
  // note that:
  // size(number_receiver_global) = nrec_local
  // size(ispec_selected_rec) = nrec
  if( mp->nrec_local > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_number_receiver_global),mp->nrec_local*sizeof(int)),1);
    print_CUDA_error_if_any(hipMemcpy(mp->d_number_receiver_global,h_number_receiver_global,
                                     mp->nrec_local*sizeof(int),hipMemcpyHostToDevice),1512);
  }
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ispec_selected_rec),nrec*sizeof(int)),1513);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_selected_rec,h_ispec_selected_rec,
                                     nrec*sizeof(int),hipMemcpyHostToDevice),1514);

#ifdef USE_MESH_COLORING_GPU
  mp->use_mesh_coloring_gpu = 1;
  if( ! *USE_MESH_COLORING_GPU_f ) exit_on_error("error with USE_MESH_COLORING_GPU constant; please re-compile\n");
#else
  // mesh coloring
  // note: this here passes the coloring as an option to the kernel routines
  //          the performance seems to be the same if one uses the pre-processing directives above or not
  mp->use_mesh_coloring_gpu = *USE_MESH_COLORING_GPU_f;
#endif

  // number of elements per domain
  mp->nspec_acoustic = *nspec_acoustic;
  mp->nspec_elastic = *nspec_elastic;

  // gravity flag initialization
  mp->gravity = 0;

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_constants_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// for ACOUSTIC simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_acoustic_device,
              PREPARE_FIELDS_ACOUSTIC_DEVICE)(long* Mesh_pointer_f,
                                              realw* rmass_acoustic,
                                              realw* rhostore,
                                              realw* kappastore,
                                              int* num_phase_ispec_acoustic,
                                              int* phase_ispec_inner_acoustic,
                                              int* ispec_is_acoustic,
                                              int* NOISE_TOMOGRAPHY,
                                              int* num_free_surface_faces,
                                              int* free_surface_ispec,
                                              int* free_surface_ijk,
                                              int* ABSORBING_CONDITIONS,
                                              int* b_reclen_potential,
                                              realw* b_absorb_potential,
                                              int* ELASTIC_SIMULATION,
                                              int* num_coupling_ac_el_faces,
                                              int* coupling_ac_el_ispec,
                                              int* coupling_ac_el_ijk,
                                              realw* coupling_ac_el_normal,
                                              realw* coupling_ac_el_jacobian2Dw,
                                              int* num_colors_outer_acoustic,
                                              int* num_colors_inner_acoustic,
                                              int* num_elem_colors_acoustic) {

  TRACE("prepare_fields_acoustic_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);
  /* Assuming NGLLX==5. Padded is then 128 (5^3+3) */
  int size_padded = NGLL3_PADDED * mp->NSPEC_AB;
  int size_nonpadded = NGLL3 * mp->NSPEC_AB;
  int size_glob = mp->NGLOB_AB;

  // allocates arrays on device (GPU)
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_acoustic),sizeof(realw)*size_glob),2001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_dot_acoustic),sizeof(realw)*size_glob),2002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_potential_dot_dot_acoustic),sizeof(realw)*size_glob),2003);

  // mpi buffer
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_potential_dot_dot_buffer),
                      (mp->max_nibool_interfaces_ext_mesh)*(mp->num_interfaces_ext_mesh)*sizeof(realw)),2004);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmass_acoustic),sizeof(realw)*size_glob),2005);
  print_CUDA_error_if_any(hipMemcpy(mp->d_rmass_acoustic,rmass_acoustic,
                                     sizeof(realw)*size_glob,hipMemcpyHostToDevice),2100);

  // padded array
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rhostore),size_padded*sizeof(realw)),2006);
  // transfer constant element data with padding
  for(int i=0; i < mp->NSPEC_AB; i++) {
    print_CUDA_error_if_any(hipMemcpy(mp->d_rhostore+i*NGLL3_PADDED, &rhostore[i*NGLL3],
                                       NGLL3*sizeof(realw),hipMemcpyHostToDevice),2106);
  }

  // non-padded array
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_kappastore),size_nonpadded*sizeof(realw)),2007);
  print_CUDA_error_if_any(hipMemcpy(mp->d_kappastore,kappastore,
                                     NGLL3*mp->NSPEC_AB*sizeof(realw),hipMemcpyHostToDevice),2105);

  // phase elements
  mp->num_phase_ispec_acoustic = *num_phase_ispec_acoustic;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_phase_ispec_inner_acoustic),
                                      mp->num_phase_ispec_acoustic*2*sizeof(int)),2008);
  print_CUDA_error_if_any(hipMemcpy(mp->d_phase_ispec_inner_acoustic,phase_ispec_inner_acoustic,
                                     mp->num_phase_ispec_acoustic*2*sizeof(int),hipMemcpyHostToDevice),2101);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ispec_is_acoustic),
                                     mp->NSPEC_AB*sizeof(int)),2009);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_is_acoustic,ispec_is_acoustic,
                                     mp->NSPEC_AB*sizeof(int),hipMemcpyHostToDevice),2102);

  // free surface
  if( *NOISE_TOMOGRAPHY == 0 ){
    // allocate surface arrays
    mp->num_free_surface_faces = *num_free_surface_faces;
    if( mp->num_free_surface_faces > 0 ){
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_ispec),
                                       mp->num_free_surface_faces*sizeof(int)),2201);
      print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ispec,free_surface_ispec,
                                       mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),2203);

      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_ijk),
                                       3*NGLL2*mp->num_free_surface_faces*sizeof(int)),2202);
      print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ijk,free_surface_ijk,
                                       3*NGLL2*mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),2204);
    }
  }

  // absorbing boundaries
  if( *ABSORBING_CONDITIONS ){
    mp->d_b_reclen_potential = *b_reclen_potential;
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_absorb_potential),mp->d_b_reclen_potential),2301);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_potential,b_absorb_potential,
                                       mp->d_b_reclen_potential,hipMemcpyHostToDevice),2302);
  }


  // for seismograms
  if( mp->nrec_local > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_station_seismo_potential),
                                       mp->nrec_local*NGLL3*sizeof(realw)),2107);

    mp->h_station_seismo_potential = (realw*) malloc( mp->nrec_local*NGLL3*sizeof(realw) );
    if( mp->h_station_seismo_potential == NULL) exit_on_error("error allocating h_station_seismo_potential");
  }


  // coupling with elastic parts
  if( *ELASTIC_SIMULATION && *num_coupling_ac_el_faces > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_coupling_ac_el_ispec),
                                       (*num_coupling_ac_el_faces)*sizeof(int)),2601);
    print_CUDA_error_if_any(hipMemcpy(mp->d_coupling_ac_el_ispec,coupling_ac_el_ispec,
                                       (*num_coupling_ac_el_faces)*sizeof(int),hipMemcpyHostToDevice),2602);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_coupling_ac_el_ijk),
                                       3*NGLL2*(*num_coupling_ac_el_faces)*sizeof(int)),2603);
    print_CUDA_error_if_any(hipMemcpy(mp->d_coupling_ac_el_ijk,coupling_ac_el_ijk,
                                       3*NGLL2*(*num_coupling_ac_el_faces)*sizeof(int),hipMemcpyHostToDevice),2604);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_coupling_ac_el_normal),
                                        3*NGLL2*(*num_coupling_ac_el_faces)*sizeof(realw)),2605);
    print_CUDA_error_if_any(hipMemcpy(mp->d_coupling_ac_el_normal,coupling_ac_el_normal,
                                        3*NGLL2*(*num_coupling_ac_el_faces)*sizeof(realw),hipMemcpyHostToDevice),2606);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_coupling_ac_el_jacobian2Dw),
                                        NGLL2*(*num_coupling_ac_el_faces)*sizeof(realw)),2607);
    print_CUDA_error_if_any(hipMemcpy(mp->d_coupling_ac_el_jacobian2Dw,coupling_ac_el_jacobian2Dw,
                                        NGLL2*(*num_coupling_ac_el_faces)*sizeof(realw),hipMemcpyHostToDevice),2608);

  }

  // mesh coloring
  if( mp->use_mesh_coloring_gpu ){
    mp->num_colors_outer_acoustic = *num_colors_outer_acoustic;
    mp->num_colors_inner_acoustic = *num_colors_inner_acoustic;
    mp->h_num_elem_colors_acoustic = (int*) num_elem_colors_acoustic;
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_acoustic_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_acoustic_adj_dev,
              PREPARE_FIELDS_ACOUSTIC_ADJ_DEV)(long* Mesh_pointer_f,
                                              int* SIMULATION_TYPE,
                                              int* APPROXIMATE_HESS_KL) {

  TRACE("prepare_fields_acoustic_adj_dev");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  int size_glob = mp->NGLOB_AB;

  // kernel simulations
  if( *SIMULATION_TYPE != 3 ) return;

  // allocates backward/reconstructed arrays on device (GPU)
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_acoustic),sizeof(realw)*size_glob),3014);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_dot_acoustic),sizeof(realw)*size_glob),3015);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_potential_dot_dot_acoustic),sizeof(realw)*size_glob),3016);

  // allocates kernels
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_ac_kl),NGLL3*mp->NSPEC_AB*sizeof(realw)),3017);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_kappa_ac_kl),NGLL3*mp->NSPEC_AB*sizeof(realw)),3018);

  // initializes kernel values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_rho_ac_kl,0,
                                     NGLL3*mp->NSPEC_AB*sizeof(realw)),3019);
  print_CUDA_error_if_any(hipMemset(mp->d_kappa_ac_kl,0,
                                     NGLL3*mp->NSPEC_AB*sizeof(realw)),3020);

  // preconditioner
  if( *APPROXIMATE_HESS_KL ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_ac_kl),NGLL3*mp->NSPEC_AB*sizeof(realw)),3030);
    // initializes with zeros
    print_CUDA_error_if_any(hipMemset(mp->d_hess_ac_kl,0,
                                       NGLL3*mp->NSPEC_AB*sizeof(realw)),3031);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_acoustic_adj_dev");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// for ELASTIC simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_elastic_device,
              PREPARE_FIELDS_ELASTIC_DEVICE)(long* Mesh_pointer_f,
                                             int* size,
                                             realw* rmass,
                                             realw* rho_vp,
                                             realw* rho_vs,
                                             int* num_phase_ispec_elastic,
                                             int* phase_ispec_inner_elastic,
                                             int* ispec_is_elastic,
                                             int* ABSORBING_CONDITIONS,
                                             realw* h_b_absorb_field,
                                             int* h_b_reclen_field,
                                             int* SIMULATION_TYPE,int* SAVE_FORWARD,
                                             int* COMPUTE_AND_STORE_STRAIN,
                                             realw* epsilondev_xx,realw* epsilondev_yy,realw* epsilondev_xy,
                                             realw* epsilondev_xz,realw* epsilondev_yz,
                                             int* ATTENUATION,
                                             int* R_size,
                                             realw* R_xx,realw* R_yy,realw* R_xy,realw* R_xz,realw* R_yz,
                                             realw* one_minus_sum_beta,realw* factor_common,
                                             realw* alphaval,realw* betaval,realw* gammaval,
                                             int* OCEANS,
                                             realw* rmass_ocean_load,
                                             int* NOISE_TOMOGRAPHY,
                                             realw* free_surface_normal,
                                             int* free_surface_ispec,
                                             int* free_surface_ijk,
                                             int* num_free_surface_faces,
                                             int* ACOUSTIC_SIMULATION,
                                             int* num_colors_outer_elastic,
                                             int* num_colors_inner_elastic,
                                             int* num_elem_colors_elastic,
                                             int* ANISOTROPY,
                                             realw *c11store,
                                             realw *c12store,
                                             realw *c13store,
                                             realw *c14store,
                                             realw *c15store,
                                             realw *c16store,
                                             realw *c22store,
                                             realw *c23store,
                                             realw *c24store,
                                             realw *c25store,
                                             realw *c26store,
                                             realw *c33store,
                                             realw *c34store,
                                             realw *c35store,
                                             realw *c36store,
                                             realw *c44store,
                                             realw *c45store,
                                             realw *c46store,
                                             realw *c55store,
                                             realw *c56store,
                                             realw *c66store){

TRACE("prepare_fields_elastic_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);
  /* Assuming NGLLX==5. Padded is then 128 (5^3+3) */
  int size_padded = NGLL3_PADDED * (mp->NSPEC_AB);
  int size_nonpadded = NGLL3 * (mp->NSPEC_AB);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_displ),sizeof(realw)*(*size)),4001);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_veloc),sizeof(realw)*(*size)),4002);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_accel),sizeof(realw)*(*size)),4003);

  #ifdef USE_TEXTURES_FIELDS
  {
    print_CUDA_error_if_any(hipGetTextureReference(&mp->d_displ_tex_ref_ptr, HIP_SYMBOL("d_displ_tex")), 4001);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    print_CUDA_error_if_any(hipBindTexture(0, mp->d_displ_tex_ref_ptr, mp->d_displ, &channelDesc, sizeof(realw)*(*size)), 4001);
  }

  {
    print_CUDA_error_if_any(hipGetTextureReference(&mp->d_accel_tex_ref_ptr, HIP_SYMBOL("d_accel_tex")), 4003);
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();
    print_CUDA_error_if_any(hipBindTexture(0, mp->d_accel_tex_ref_ptr, mp->d_accel, &channelDesc, sizeof(realw)*(*size)), 4003);
  }
  #endif

  // mpi buffer
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_send_accel_buffer),
                        3*(mp->max_nibool_interfaces_ext_mesh)*(mp->num_interfaces_ext_mesh)*sizeof(realw)),4004);

  // mass matrix
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmass),sizeof(realw)*mp->NGLOB_AB),4005);
  // transfer element data
  print_CUDA_error_if_any(hipMemcpy(mp->d_rmass,rmass,
                                     sizeof(realw)*mp->NGLOB_AB,hipMemcpyHostToDevice),4010);


  // element indices
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_ispec_is_elastic),mp->NSPEC_AB*sizeof(int)),4009);
  print_CUDA_error_if_any(hipMemcpy(mp->d_ispec_is_elastic,ispec_is_elastic,
                                     mp->NSPEC_AB*sizeof(int),hipMemcpyHostToDevice),4012);

  // phase elements
  mp->num_phase_ispec_elastic = *num_phase_ispec_elastic;
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_phase_ispec_inner_elastic),
                                     mp->num_phase_ispec_elastic*2*sizeof(int)),4008);
  print_CUDA_error_if_any(hipMemcpy(mp->d_phase_ispec_inner_elastic,phase_ispec_inner_elastic,
                                     mp->num_phase_ispec_elastic*2*sizeof(int),hipMemcpyHostToDevice),4011);

  // for seismograms
  if( mp->nrec_local > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_station_seismo_field),
                                     3*NGLL3*(mp->nrec_local)*sizeof(realw)),4015);

    mp->h_station_seismo_field = (realw*) malloc( 3*NGLL3*(mp->nrec_local)*sizeof(realw) );
    if( mp->h_station_seismo_field == NULL) exit_on_error("h_station_seismo_field not allocated \n");
  }

  // absorbing conditions
  if( *ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0){
    // non-padded arrays
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_vp),size_nonpadded*sizeof(realw)),4006);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_vs),size_nonpadded*sizeof(realw)),4007);

    // rho_vp, rho_vs non-padded; they are needed for stacey boundary condition
    print_CUDA_error_if_any(hipMemcpy(mp->d_rho_vp, rho_vp,
                                       NGLL3*mp->NSPEC_AB*sizeof(realw),hipMemcpyHostToDevice),4013);
    print_CUDA_error_if_any(hipMemcpy(mp->d_rho_vs, rho_vs,
                                       NGLL3*mp->NSPEC_AB*sizeof(realw),hipMemcpyHostToDevice),4014);

    // absorb_field array used for file i/o
    if(*SIMULATION_TYPE == 3 || ( *SIMULATION_TYPE == 1 && *SAVE_FORWARD )){
      mp->d_b_reclen_field = *h_b_reclen_field;
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_absorb_field),
                                       mp->d_b_reclen_field),4016);
      print_CUDA_error_if_any(hipMemcpy(mp->d_b_absorb_field, h_b_absorb_field,
                                       mp->d_b_reclen_field,hipMemcpyHostToDevice),4017);
    }
  }

  // strains used for attenuation and kernel simulations
  if( *COMPUTE_AND_STORE_STRAIN ){
    // strains
    int epsilondev_size = NGLL3*mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_xx,
                                       epsilondev_size*sizeof(realw)),4301);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xx,epsilondev_xx,epsilondev_size*sizeof(realw),
                                       hipMemcpyHostToDevice),4302);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_yy,
                                       epsilondev_size*sizeof(realw)),4302);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_yy,epsilondev_yy,epsilondev_size*sizeof(realw),
                                       hipMemcpyHostToDevice),4303);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_xy,
                                       epsilondev_size*sizeof(realw)),4304);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xy,epsilondev_xy,epsilondev_size*sizeof(realw),
                                       hipMemcpyHostToDevice),4305);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_xz,
                                       epsilondev_size*sizeof(realw)),4306);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_xz,epsilondev_xz,epsilondev_size*sizeof(realw),
                                       hipMemcpyHostToDevice),4307);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_epsilondev_yz,
                                       epsilondev_size*sizeof(realw)),4308);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilondev_yz,epsilondev_yz,epsilondev_size*sizeof(realw),
                                       hipMemcpyHostToDevice),4309);

  }

  // attenuation memory variables
  if( *ATTENUATION ){
    // memory arrays
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_xx),
                                       (*R_size)*sizeof(realw)),4401);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xx,R_xx,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),4402);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_yy),
                                       (*R_size)*sizeof(realw)),4403);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_yy,R_yy,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),4404);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_xy),
                                       (*R_size)*sizeof(realw)),4405);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xy,R_xy,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),4406);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_xz),
                                       (*R_size)*sizeof(realw)),4407);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_xz,R_xz,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),4408);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_R_yz),
                                       (*R_size)*sizeof(realw)),4409);
    print_CUDA_error_if_any(hipMemcpy(mp->d_R_yz,R_yz,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),4410);

    // attenuation factors
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_one_minus_sum_beta),
                                       NGLL3*mp->NSPEC_AB*sizeof(realw)),4430);
    print_CUDA_error_if_any(hipMemcpy(mp->d_one_minus_sum_beta ,one_minus_sum_beta,
                                       NGLL3*mp->NSPEC_AB*sizeof(realw),hipMemcpyHostToDevice),4431);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_factor_common),
                                       N_SLS*NGLL3*mp->NSPEC_AB*sizeof(realw)),4432);
    print_CUDA_error_if_any(hipMemcpy(mp->d_factor_common ,factor_common,
                                       N_SLS*NGLL3*mp->NSPEC_AB*sizeof(realw),hipMemcpyHostToDevice),4433);

    // alpha,beta,gamma factors
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_alphaval),
                                       N_SLS*sizeof(realw)),4434);
    print_CUDA_error_if_any(hipMemcpy(mp->d_alphaval ,alphaval,
                                       N_SLS*sizeof(realw),hipMemcpyHostToDevice),4435);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_betaval),
                                       N_SLS*sizeof(realw)),4436);
    print_CUDA_error_if_any(hipMemcpy(mp->d_betaval ,betaval,
                                       N_SLS*sizeof(realw),hipMemcpyHostToDevice),4437);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_gammaval),
                                       N_SLS*sizeof(realw)),4438);
    print_CUDA_error_if_any(hipMemcpy(mp->d_gammaval ,gammaval,
                                       N_SLS*sizeof(realw),hipMemcpyHostToDevice),4439);

  }

  // anisotropy
  if( *ANISOTROPY ){
    // allocates memory on GPU
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c11store),
                                       size_padded*sizeof(realw)),4700);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c12store),
                                       size_padded*sizeof(realw)),4701);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c13store),
                                       size_padded*sizeof(realw)),4702);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c14store),
                                       size_padded*sizeof(realw)),4703);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c15store),
                                       size_padded*sizeof(realw)),4704);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c16store),
                                       size_padded*sizeof(realw)),4705);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c22store),
                                       size_padded*sizeof(realw)),4706);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c23store),
                                       size_padded*sizeof(realw)),4707);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c24store),
                                       size_padded*sizeof(realw)),4708);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c25store),
                                       size_padded*sizeof(realw)),4709);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c26store),
                                       size_padded*sizeof(realw)),4710);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c33store),
                                       size_padded*sizeof(realw)),4711);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c34store),
                                       size_padded*sizeof(realw)),4712);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c35store),
                                       size_padded*sizeof(realw)),4713);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c36store),
                                       size_padded*sizeof(realw)),4714);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c44store),
                                       size_padded*sizeof(realw)),4715);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c45store),
                                       size_padded*sizeof(realw)),4716);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c46store),
                                       size_padded*sizeof(realw)),4717);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c55store),
                                       size_padded*sizeof(realw)),4718);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c56store),
                                       size_padded*sizeof(realw)),4719);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_c66store),
                                       size_padded*sizeof(realw)),4720);

    // transfer constant element data with padding
    for(int i=0;i < mp->NSPEC_AB;i++) {
      print_CUDA_error_if_any(hipMemcpy(mp->d_c11store + i*NGLL3_PADDED, &c11store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4800);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c12store + i*NGLL3_PADDED, &c12store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4801);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c13store + i*NGLL3_PADDED, &c13store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4802);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c14store + i*NGLL3_PADDED, &c14store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4803);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c15store + i*NGLL3_PADDED, &c15store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4804);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c16store + i*NGLL3_PADDED, &c16store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4805);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c22store + i*NGLL3_PADDED, &c22store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4806);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c23store + i*NGLL3_PADDED, &c23store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4807);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c24store + i*NGLL3_PADDED, &c24store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4808);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c25store + i*NGLL3_PADDED, &c25store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4809);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c26store + i*NGLL3_PADDED, &c26store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4810);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c33store + i*NGLL3_PADDED, &c33store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4811);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c34store + i*NGLL3_PADDED, &c34store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4812);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c35store + i*NGLL3_PADDED, &c35store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4813);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c36store + i*NGLL3_PADDED, &c36store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4814);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c44store + i*NGLL3_PADDED, &c44store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4815);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c45store + i*NGLL3_PADDED, &c45store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4816);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c46store + i*NGLL3_PADDED, &c46store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4817);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c55store + i*NGLL3_PADDED, &c55store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4818);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c56store + i*NGLL3_PADDED, &c56store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4819);
      print_CUDA_error_if_any(hipMemcpy(mp->d_c66store + i*NGLL3_PADDED, &c66store[i*NGLL3],
                                         NGLL3*sizeof(realw),hipMemcpyHostToDevice),4820);
    }
  }

  // ocean load approximation
  if( *OCEANS ){
    // oceans needs a free surface
    mp->num_free_surface_faces = *num_free_surface_faces;
    if( mp->num_free_surface_faces > 0 ){
      // mass matrix
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rmass_ocean_load),
                                         sizeof(realw)*mp->NGLOB_AB),4501);
      print_CUDA_error_if_any(hipMemcpy(mp->d_rmass_ocean_load,rmass_ocean_load,
                                         sizeof(realw)*mp->NGLOB_AB,hipMemcpyHostToDevice),4502);
      // surface normal
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_normal),
                                         3*NGLL2*(mp->num_free_surface_faces)*sizeof(realw)),4503);
      print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_normal,free_surface_normal,
                                         3*NGLL2*(mp->num_free_surface_faces)*sizeof(realw),hipMemcpyHostToDevice),4504);

      // temporary global array: used to synchronize updates on global accel array
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_updated_dof_ocean_load),
                                         sizeof(int)*mp->NGLOB_AB),4505);

      if( *NOISE_TOMOGRAPHY == 0 && *ACOUSTIC_SIMULATION == 0 ){
        print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_ispec),
                                          mp->num_free_surface_faces*sizeof(int)),4601);
        print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ispec,free_surface_ispec,
                                          mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),4603);
        print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_free_surface_ijk),
                                          3*NGLL2*mp->num_free_surface_faces*sizeof(int)),4602);
        print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ijk,free_surface_ijk,
                                          3*NGLL2*mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),4604);
      }
    }
  }

  // mesh coloring
  if( mp->use_mesh_coloring_gpu ){
    mp->num_colors_outer_elastic = *num_colors_outer_elastic;
    mp->num_colors_inner_elastic = *num_colors_inner_elastic;
    mp->h_num_elem_colors_elastic = (int*) num_elem_colors_elastic;
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_elastic_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_elastic_adj_dev,
              PREPARE_FIELDS_ELASTIC_ADJ_DEV)(long* Mesh_pointer_f,
                                             int* size,
                                             int* SIMULATION_TYPE,
                                             int* COMPUTE_AND_STORE_STRAIN,
                                             realw* epsilon_trace_over_3,
                                             realw* b_epsilondev_xx,realw* b_epsilondev_yy,realw* b_epsilondev_xy,
                                             realw* b_epsilondev_xz,realw* b_epsilondev_yz,
                                             realw* b_epsilon_trace_over_3,
                                             int* ATTENUATION,
                                             int* R_size,
                                             realw* b_R_xx,realw* b_R_yy,realw* b_R_xy,realw* b_R_xz,realw* b_R_yz,
                                             realw* b_alphaval,realw* b_betaval,realw* b_gammaval,
                                             int* APPROXIMATE_HESS_KL){

  TRACE("prepare_fields_elastic_adj_dev");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // checks if kernel simulation
  if( *SIMULATION_TYPE != 3 ) return;

  // kernel simulations
  // allocates backward/reconstructed arrays on device (GPU)
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_displ),sizeof(realw)*(*size)),5201);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_veloc),sizeof(realw)*(*size)),5202);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_accel),sizeof(realw)*(*size)),5203);

  // allocates kernels
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rho_kl),NGLL3*mp->NSPEC_AB*sizeof(realw)),5204);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_mu_kl),NGLL3*mp->NSPEC_AB*sizeof(realw)),5205);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_kappa_kl),NGLL3*mp->NSPEC_AB*sizeof(realw)),5206);

  // initializes kernel values to zero
  print_CUDA_error_if_any(hipMemset(mp->d_rho_kl,0,
                                     NGLL3*mp->NSPEC_AB*sizeof(realw)),5207);
  print_CUDA_error_if_any(hipMemset(mp->d_mu_kl,0,
                                     NGLL3*mp->NSPEC_AB*sizeof(realw)),5208);
  print_CUDA_error_if_any(hipMemset(mp->d_kappa_kl,0,
                                     NGLL3*mp->NSPEC_AB*sizeof(realw)),5209);

  // strains used for attenuation and kernel simulations
  if( *COMPUTE_AND_STORE_STRAIN ){
    // strains
    int epsilondev_size = NGLL3*mp->NSPEC_AB; // note: non-aligned; if align, check memcpy below and indexing

    // solid pressure
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_epsilon_trace_over_3),
                                       NGLL3*mp->NSPEC_AB*sizeof(realw)),5310);
    print_CUDA_error_if_any(hipMemcpy(mp->d_epsilon_trace_over_3,epsilon_trace_over_3,
                                       NGLL3*mp->NSPEC_AB*sizeof(realw),hipMemcpyHostToDevice),5311);
    // backward solid pressure
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_epsilon_trace_over_3),
                                       NGLL3*mp->NSPEC_AB*sizeof(realw)),5312);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilon_trace_over_3 ,b_epsilon_trace_over_3,
                                       NGLL3*mp->NSPEC_AB*sizeof(realw),hipMemcpyHostToDevice),5313);
    // prepares backward strains
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_epsilondev_xx),
                                       epsilondev_size*sizeof(realw)),5321);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_epsilondev_yy),
                                       epsilondev_size*sizeof(realw)),5322);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_epsilondev_xy),
                                       epsilondev_size*sizeof(realw)),5323);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_epsilondev_xz),
                                       epsilondev_size*sizeof(realw)),5324);
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_epsilondev_yz),
                                       epsilondev_size*sizeof(realw)),5325);

    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xx,b_epsilondev_xx,
                                       epsilondev_size*sizeof(realw),hipMemcpyHostToDevice),5326);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yy,b_epsilondev_yy,
                                       epsilondev_size*sizeof(realw),hipMemcpyHostToDevice),5327);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xy,b_epsilondev_xy,
                                       epsilondev_size*sizeof(realw),hipMemcpyHostToDevice),5328);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xz,b_epsilondev_xz,
                                       epsilondev_size*sizeof(realw),hipMemcpyHostToDevice),5329);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yz,b_epsilondev_yz,
                                       epsilondev_size*sizeof(realw),hipMemcpyHostToDevice),5330);
  }

  // attenuation memory variables
  if( *ATTENUATION ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_R_xx),
                                       (*R_size)*sizeof(realw)),5421);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xx,b_R_xx,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),5422);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_R_yy),
                                       (*R_size)*sizeof(realw)),5423);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_yy,b_R_yy,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),5424);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_R_xy),
                                       (*R_size)*sizeof(realw)),5425);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xy,b_R_xy,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),5426);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_R_xz),
                                       (*R_size)*sizeof(realw)),5427);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xz,b_R_xz,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),5428);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_R_yz),
                                       (*R_size)*sizeof(realw)),5429);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_yz,b_R_yz,(*R_size)*sizeof(realw),
                                       hipMemcpyHostToDevice),5420);

    // alpha,beta,gamma factors for backward fields
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_alphaval),
                                       N_SLS*sizeof(realw)),5434);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_alphaval ,b_alphaval,
                                       N_SLS*sizeof(realw),hipMemcpyHostToDevice),5435);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_betaval),
                                       N_SLS*sizeof(realw)),5436);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_betaval ,b_betaval,
                                       N_SLS*sizeof(realw),hipMemcpyHostToDevice),5437);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_b_gammaval),
                                       N_SLS*sizeof(realw)),5438);
    print_CUDA_error_if_any(hipMemcpy(mp->d_b_gammaval ,b_gammaval,
                                       N_SLS*sizeof(realw),hipMemcpyHostToDevice),5439);
  }

  if( *APPROXIMATE_HESS_KL ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hess_el_kl),NGLL3*mp->NSPEC_AB*sizeof(realw)),5450);
    // initializes with zeros
    print_CUDA_error_if_any(hipMemset(mp->d_hess_el_kl,0,
                                       NGLL3*mp->NSPEC_AB*sizeof(realw)),5451);
  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_fields_elastic_adj_dev");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// purely adjoint & kernel simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_sim2_or_3_const_device,
              PREPARE_SIM2_OR_3_CONST_DEVICE)(
                                              long* Mesh_pointer_f,
                                              int* islice_selected_rec,
                                              int* islice_selected_rec_size,
                                              int* nadj_rec_local,
                                              int* nrec,
                                              int* myrank) {

  TRACE("prepare_sim2_or_3_const_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // adjoint source arrays
  mp->nadj_rec_local = *nadj_rec_local;
  if( mp->nadj_rec_local > 0 ){
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_adj_sourcearrays,
                                       (mp->nadj_rec_local)*3*NGLL3*sizeof(realw)),6003);

    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_pre_computed_irec,
                                       (mp->nadj_rec_local)*sizeof(int)),6004);

    // prepares local irec array:
    // the irec_local variable needs to be precomputed (as
    // h_pre_comp..), because normally it is in the loop updating accel,
    // and due to how it's incremented, it cannot be parallelized
    int* h_pre_computed_irec = (int*) malloc( (mp->nadj_rec_local)*sizeof(int) );
    if( h_pre_computed_irec == NULL ) exit_on_error("prepare_sim2_or_3_const_device: h_pre_computed_irec not allocated\n");

    int irec_local = 0;
    for(int irec = 0; irec < *nrec; irec++) {
      if(*myrank == islice_selected_rec[irec]) {
        irec_local++;
        h_pre_computed_irec[irec_local-1] = irec;
      }
    }
    if( irec_local != mp->nadj_rec_local ) exit_on_error("prepare_sim2_or_3_const_device: irec_local not equal\n");
    // copies values onto GPU
    print_CUDA_error_if_any(hipMemcpy(mp->d_pre_computed_irec,h_pre_computed_irec,
                                       (mp->nadj_rec_local)*sizeof(int),hipMemcpyHostToDevice),6010);
    free(h_pre_computed_irec);

    // temporary array to prepare extracted source array values
    mp->h_adj_sourcearrays_slice = (realw*) malloc( (mp->nadj_rec_local)*3*NGLL3*sizeof(realw) );
    if( mp->h_adj_sourcearrays_slice == NULL ) exit_on_error("h_adj_sourcearrays_slice not allocated\n");

  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("prepare_sim2_or_3_const_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

// for NOISE simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_noise_device,
              PREPARE_FIELDS_NOISE_DEVICE)(long* Mesh_pointer_f,
                                           int* NSPEC_AB, int* NGLOB_AB,
                                           int* free_surface_ispec,
                                           int* free_surface_ijk,
                                           int* num_free_surface_faces,
                                           int* SIMULATION_TYPE,
                                           int* NOISE_TOMOGRAPHY,
                                           int* NSTEP,
                                           realw* noise_sourcearray,
                                           realw* normal_x_noise,
                                           realw* normal_y_noise,
                                           realw* normal_z_noise,
                                           realw* mask_noise,
                                           realw* free_surface_jacobian2Dw) {

  TRACE("prepare_fields_noise_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // free surface
  mp->num_free_surface_faces = *num_free_surface_faces;

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_free_surface_ispec,
                                     mp->num_free_surface_faces*sizeof(int)),7001);
  print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ispec, free_surface_ispec,
                                     mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),7002);

  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_free_surface_ijk,
                                     3*NGLL2*mp->num_free_surface_faces*sizeof(int)),7003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_ijk,free_surface_ijk,
                                     3*NGLL2*mp->num_free_surface_faces*sizeof(int),hipMemcpyHostToDevice),7004);

  // alloc storage for the surface buffer to be copied
  print_CUDA_error_if_any(hipMalloc((void**) &mp->d_noise_surface_movie,
                                     3*NGLL2*mp->num_free_surface_faces*sizeof(realw)),7005);

  // prepares noise source array
  if( *NOISE_TOMOGRAPHY == 1 ){
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_noise_sourcearray,
                                       3*NGLL3*(*NSTEP)*sizeof(realw)),7101);
    print_CUDA_error_if_any(hipMemcpy(mp->d_noise_sourcearray, noise_sourcearray,
                                       3*NGLL3*(*NSTEP)*sizeof(realw),hipMemcpyHostToDevice),7102);
  }

  // prepares noise directions
  if( *NOISE_TOMOGRAPHY > 1 ){
    int nface_size = NGLL2*(*num_free_surface_faces);
    // allocates memory on GPU
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_normal_x_noise,
                                       nface_size*sizeof(realw)),7301);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_normal_y_noise,
                                       nface_size*sizeof(realw)),7302);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_normal_z_noise,
                                       nface_size*sizeof(realw)),7303);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_mask_noise,
                                       nface_size*sizeof(realw)),7304);
    print_CUDA_error_if_any(hipMalloc((void**)&mp->d_free_surface_jacobian2Dw,
                                       nface_size*sizeof(realw)),7305);
    // transfers data onto GPU
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_x_noise, normal_x_noise,
                                       nface_size*sizeof(realw),hipMemcpyHostToDevice),7306);
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_y_noise, normal_y_noise,
                                       nface_size*sizeof(realw),hipMemcpyHostToDevice),7307);
    print_CUDA_error_if_any(hipMemcpy(mp->d_normal_z_noise, normal_z_noise,
                                       nface_size*sizeof(realw),hipMemcpyHostToDevice),7308);
    print_CUDA_error_if_any(hipMemcpy(mp->d_mask_noise, mask_noise,
                                       nface_size*sizeof(realw),hipMemcpyHostToDevice),7309);
    print_CUDA_error_if_any(hipMemcpy(mp->d_free_surface_jacobian2Dw, free_surface_jacobian2Dw,
                                       nface_size*sizeof(realw),hipMemcpyHostToDevice),7310);
  }

  // prepares noise strength kernel
  if( *NOISE_TOMOGRAPHY == 3 ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_Sigma_kl),
                                       NGLL3*(mp->NSPEC_AB)*sizeof(realw)),7401);
    // initializes kernel values to zero
    print_CUDA_error_if_any(hipMemset(mp->d_Sigma_kl,0,
                                       NGLL3*mp->NSPEC_AB*sizeof(realw)),7403);

  }

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  //printf("jacobian_size = %d\n",25*(*num_free_surface_faces));
  exit_on_cuda_error("prepare_fields_noise_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// GRAVITY simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_fields_gravity_device,
              PREPARE_FIELDS_gravity_DEVICE)(long* Mesh_pointer_f,
                                             int* GRAVITY,
                                             realw* minus_deriv_gravity,
                                             realw* minus_g,
                                             realw* h_wgll_cube,
                                             int* ACOUSTIC_SIMULATION,
                                             realw* rhostore) {

  TRACE("prepare_fields_gravity_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  setConst_wgll_cube(h_wgll_cube,mp);

  mp->gravity = *GRAVITY;
  if( mp->gravity ){
    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_minus_deriv_gravity),
                                       (mp->NGLOB_AB)*sizeof(realw)),8000);
    print_CUDA_error_if_any(hipMemcpy(mp->d_minus_deriv_gravity, minus_deriv_gravity,
                                       (mp->NGLOB_AB)*sizeof(realw),hipMemcpyHostToDevice),8001);

    print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_minus_g),
                                       (mp->NGLOB_AB)*sizeof(realw)),8002);
    print_CUDA_error_if_any(hipMemcpy(mp->d_minus_g, minus_g,
                                       (mp->NGLOB_AB)*sizeof(realw),hipMemcpyHostToDevice),8003);


    if( *ACOUSTIC_SIMULATION == 0 ){
      // rhostore not allocated yet
      int size_padded = NGLL3_PADDED * (mp->NSPEC_AB);
      // padded array
      print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_rhostore),size_padded*sizeof(realw)),8006);
      // transfer constant element data with padding
      for(int i=0; i < mp->NSPEC_AB; i++) {
        print_CUDA_error_if_any(hipMemcpy(mp->d_rhostore+i*NGLL3_PADDED, &rhostore[i*NGLL3],
                                           NGLL3*sizeof(realw),hipMemcpyHostToDevice),8007);
      }
    }
  }

}

extern "C"
void FC_FUNC_(prepare_seismogram_fields,
              PREPARE_SEISMOGRAM_FIELDS)(long* Mesh_pointer,int* nrec_local, double* nu, double* hxir, double* hetar, double* hgammar) {

  TRACE("prepare_constants_device");
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_nu),3*3* *nrec_local*sizeof(double)),8100);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hxir),5* *nrec_local*sizeof(double)),8100);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hetar),5* *nrec_local*sizeof(double)),8100);
  print_CUDA_error_if_any(hipMalloc((void**)&(mp->d_hgammar),5* *nrec_local*sizeof(double)),8100);

  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_seismograms_d,3**nrec_local*sizeof(realw)),8101);
  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_seismograms_v,3**nrec_local*sizeof(realw)),8101);
  print_CUDA_error_if_any(hipMalloc((void**)&mp->d_seismograms_a,3**nrec_local*sizeof(realw)),8101);

  print_CUDA_error_if_any(hipMemcpy(mp->d_nu,nu,3*3* *nrec_local*sizeof(double),hipMemcpyHostToDevice),8101);
  print_CUDA_error_if_any(hipMemcpy(mp->d_hxir,hxir,5* *nrec_local*sizeof(double),hipMemcpyHostToDevice),8101);
  print_CUDA_error_if_any(hipMemcpy(mp->d_hetar,hetar,5* *nrec_local*sizeof(double),hipMemcpyHostToDevice),8101);
  print_CUDA_error_if_any(hipMemcpy(mp->d_hgammar,hgammar,5* *nrec_local*sizeof(double),hipMemcpyHostToDevice),8101);

  hipHostMalloc((void**)&mp->h_seismograms_d_it,3**nrec_local*sizeof(realw));
  hipHostMalloc((void**)&mp->h_seismograms_v_it,3**nrec_local*sizeof(realw));
  hipHostMalloc((void**)&mp->h_seismograms_a_it,3**nrec_local*sizeof(realw));
}

/* ----------------------------------------------------------------------------------------------- */

// cleanup

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(prepare_cleanup_device,
              PREPARE_CLEANUP_DEVICE)(long* Mesh_pointer_f,
                                      int* SIMULATION_TYPE,
                                      int* SAVE_FORWARD,
                                      int* ACOUSTIC_SIMULATION,
                                      int* ELASTIC_SIMULATION,
                                      int* ABSORBING_CONDITIONS,
                                      int* NOISE_TOMOGRAPHY,
                                      int* COMPUTE_AND_STORE_STRAIN,
                                      int* ATTENUATION,
                                      int* ANISOTROPY,
                                      int* OCEANS,
                                      int* APPROXIMATE_HESS_KL) {

TRACE("prepare_cleanup_device");

  // frees allocated memory arrays
  Mesh* mp = (Mesh*)(*Mesh_pointer_f);

  // frees memory on GPU
  // mesh
  hipFree(mp->d_xix);
  hipFree(mp->d_xiy);
  hipFree(mp->d_xiz);
  hipFree(mp->d_etax);
  hipFree(mp->d_etay);
  hipFree(mp->d_etaz);
  hipFree(mp->d_gammax);
  hipFree(mp->d_gammay);
  hipFree(mp->d_gammaz);
  hipFree(mp->d_muv);

  // absorbing boundaries
  if( *ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0 ){
    hipFree(mp->d_abs_boundary_ispec);
    hipFree(mp->d_abs_boundary_ijk);
    hipFree(mp->d_abs_boundary_normal);
    hipFree(mp->d_abs_boundary_jacobian2Dw);
  }

  // interfaces
  hipFree(mp->d_nibool_interfaces_ext_mesh);
  hipFree(mp->d_ibool_interfaces_ext_mesh);

  // global indexing
  hipFree(mp->d_ispec_is_inner);
  hipFree(mp->d_ibool);

  // sources
  if (*SIMULATION_TYPE == 1  || *SIMULATION_TYPE == 3){
    hipFree(mp->d_sourcearrays);
    hipFree(mp->d_stf_pre_compute);
  }

  hipFree(mp->d_islice_selected_source);
  hipFree(mp->d_ispec_selected_source);

  // receivers
  if( mp->nrec_local > 0 ) hipFree(mp->d_number_receiver_global);
  hipFree(mp->d_ispec_selected_rec);

  // ACOUSTIC arrays
  if( *ACOUSTIC_SIMULATION ){
    hipFree(mp->d_potential_acoustic);
    hipFree(mp->d_potential_dot_acoustic);
    hipFree(mp->d_potential_dot_dot_acoustic);
    hipFree(mp->d_send_potential_dot_dot_buffer);
    hipFree(mp->d_rmass_acoustic);
    hipFree(mp->d_rhostore);
    hipFree(mp->d_kappastore);
    hipFree(mp->d_phase_ispec_inner_acoustic);
    hipFree(mp->d_ispec_is_acoustic);

    if( *NOISE_TOMOGRAPHY == 0 ){
      hipFree(mp->d_free_surface_ispec);
      hipFree(mp->d_free_surface_ijk);
    }

    if( *ABSORBING_CONDITIONS ) hipFree(mp->d_b_absorb_potential);

    if( *SIMULATION_TYPE == 3 ) {
      hipFree(mp->d_b_potential_acoustic);
      hipFree(mp->d_b_potential_dot_acoustic);
      hipFree(mp->d_b_potential_dot_dot_acoustic);
      hipFree(mp->d_rho_ac_kl);
      hipFree(mp->d_kappa_ac_kl);
      if( *APPROXIMATE_HESS_KL) hipFree(mp->d_hess_ac_kl);
    }


    if(mp->nrec_local > 0 ){
      hipFree(mp->d_station_seismo_potential);
      free(mp->h_station_seismo_potential);
    }

  } // ACOUSTIC_SIMULATION

  // ELASTIC arrays
  if( *ELASTIC_SIMULATION ){
    hipFree(mp->d_displ);
    hipFree(mp->d_veloc);
    hipFree(mp->d_accel);
    hipFree(mp->d_send_accel_buffer);
    hipFree(mp->d_rmass);

    hipFree(mp->d_phase_ispec_inner_elastic);
    hipFree(mp->d_ispec_is_elastic);

    if( mp->nrec_local > 0 ){
      hipFree(mp->d_station_seismo_field);
      free(mp->h_station_seismo_field);
    }

    if( *ABSORBING_CONDITIONS && mp->d_num_abs_boundary_faces > 0){
      hipFree(mp->d_rho_vp);
      hipFree(mp->d_rho_vs);

      if(*SIMULATION_TYPE == 3 || ( *SIMULATION_TYPE == 1 && *SAVE_FORWARD ))
          hipFree(mp->d_b_absorb_field);
    }

    if( *SIMULATION_TYPE == 3 ) {
      hipFree(mp->d_b_displ);
      hipFree(mp->d_b_veloc);
      hipFree(mp->d_b_accel);
      hipFree(mp->d_rho_kl);
      hipFree(mp->d_mu_kl);
      hipFree(mp->d_kappa_kl);
      if( *APPROXIMATE_HESS_KL ) hipFree(mp->d_hess_el_kl);
    }

    if( *COMPUTE_AND_STORE_STRAIN ){
      hipFree(mp->d_epsilondev_xx);
      hipFree(mp->d_epsilondev_yy);
      hipFree(mp->d_epsilondev_xy);
      hipFree(mp->d_epsilondev_xz);
      hipFree(mp->d_epsilondev_yz);
      if( *SIMULATION_TYPE == 3 ){
        hipFree(mp->d_epsilon_trace_over_3);
        hipFree(mp->d_b_epsilon_trace_over_3);
        hipFree(mp->d_b_epsilondev_xx);
        hipFree(mp->d_b_epsilondev_yy);
        hipFree(mp->d_b_epsilondev_xy);
        hipFree(mp->d_b_epsilondev_xz);
        hipFree(mp->d_b_epsilondev_yz);
      }
    }

    if( *ATTENUATION ){
      hipFree(mp->d_factor_common);
      hipFree(mp->d_one_minus_sum_beta);
      hipFree(mp->d_alphaval);
      hipFree(mp->d_betaval);
      hipFree(mp->d_gammaval);
      hipFree(mp->d_R_xx);
      hipFree(mp->d_R_yy);
      hipFree(mp->d_R_xy);
      hipFree(mp->d_R_xz);
      hipFree(mp->d_R_yz);
      if( *SIMULATION_TYPE == 3){
        hipFree(mp->d_b_R_xx);
        hipFree(mp->d_b_R_yy);
        hipFree(mp->d_b_R_xy);
        hipFree(mp->d_b_R_xz);
        hipFree(mp->d_b_R_yz);
        hipFree(mp->d_b_alphaval);
        hipFree(mp->d_b_betaval);
        hipFree(mp->d_b_gammaval);
      }
    }

    if( *ANISOTROPY ){
      hipFree(mp->d_c11store);
      hipFree(mp->d_c12store);
      hipFree(mp->d_c13store);
      hipFree(mp->d_c14store);
      hipFree(mp->d_c15store);
      hipFree(mp->d_c16store);
      hipFree(mp->d_c22store);
      hipFree(mp->d_c23store);
      hipFree(mp->d_c24store);
      hipFree(mp->d_c25store);
      hipFree(mp->d_c26store);
      hipFree(mp->d_c33store);
      hipFree(mp->d_c34store);
      hipFree(mp->d_c35store);
      hipFree(mp->d_c36store);
      hipFree(mp->d_c44store);
      hipFree(mp->d_c45store);
      hipFree(mp->d_c46store);
      hipFree(mp->d_c55store);
      hipFree(mp->d_c56store);
      hipFree(mp->d_c66store);
    }

    if( *OCEANS ){
      if( mp->num_free_surface_faces > 0 ){
        hipFree(mp->d_rmass_ocean_load);
        hipFree(mp->d_free_surface_normal);
        hipFree(mp->d_updated_dof_ocean_load);
        if( *NOISE_TOMOGRAPHY == 0){
          hipFree(mp->d_free_surface_ispec);
          hipFree(mp->d_free_surface_ijk);
        }
      }
    }
  } // ELASTIC_SIMULATION

  // purely adjoint & kernel array
  if( *SIMULATION_TYPE == 2 || *SIMULATION_TYPE == 3 ){
    if(mp->nadj_rec_local > 0 ){
      hipFree(mp->d_adj_sourcearrays);
      hipFree(mp->d_pre_computed_irec);
      free(mp->h_adj_sourcearrays_slice);
    }
  }

  // NOISE arrays
  if( *NOISE_TOMOGRAPHY > 0 ){
    hipFree(mp->d_free_surface_ispec);
    hipFree(mp->d_free_surface_ijk);
    hipFree(mp->d_noise_surface_movie);
    if( *NOISE_TOMOGRAPHY == 1 ) hipFree(mp->d_noise_sourcearray);
    if( *NOISE_TOMOGRAPHY > 1 ){
      hipFree(mp->d_normal_x_noise);
      hipFree(mp->d_normal_y_noise);
      hipFree(mp->d_normal_z_noise);
      hipFree(mp->d_mask_noise);
      hipFree(mp->d_free_surface_jacobian2Dw);
    }
    if( *NOISE_TOMOGRAPHY == 3 ) hipFree(mp->d_Sigma_kl);
  }

  // mesh pointer - not needed anymore
  free(mp);
}
