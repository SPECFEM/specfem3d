/*
 !=====================================================================
 !
 !               S p e c f e m 3 D  V e r s i o n  3 . 0
 !               ---------------------------------------
 !
 !     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
 !                        Princeton University, USA
 !                and CNRS / University of Marseille, France
 !                 (there are currently many more authors!)
 ! (c) Princeton University and CNRS / University of Marseille, July 2012
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 2 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include "mesh_constants_cuda.h"

/* ----------------------------------------------------------------------------------------------- */

// Transfer functions

/* ----------------------------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------------------------------- */

// for ELASTIC simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_fields_el_to_device,
              TRANSFER_FIELDS_EL_TO_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer) {

  TRACE("transfer_fields_el_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_displ,displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),40003);
  print_CUDA_error_if_any(hipMemcpy(mp->d_veloc,veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),40004);
  print_CUDA_error_if_any(hipMemcpy(mp->d_accel,accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40005);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_fields_el_from_device,
              TRANSFER_FIELDS_EL_FROM_DEVICE)(int* size, realw* displ, realw* veloc, realw* accel,long* Mesh_pointer) {

  TRACE("transfer_fields_el_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40006);
  print_CUDA_error_if_any(hipMemcpy(veloc,mp->d_veloc,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40007);
  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40008);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_fields_to_device,
              TRANSFER_B_FIELDS_TO_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,
                                           long* Mesh_pointer) {

  TRACE("transfer_b_fields_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_b_displ,b_displ,sizeof(realw)*(*size),hipMemcpyHostToDevice),41006);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_veloc,b_veloc,sizeof(realw)*(*size),hipMemcpyHostToDevice),41007);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_accel,b_accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),41008);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_fields_from_device,
              TRANSFER_B_FIELDS_FROM_DEVICE)(int* size, realw* b_displ, realw* b_veloc, realw* b_accel,long* Mesh_pointer) {

  TRACE("transfer_b_fields_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(b_displ,mp->d_b_displ,sizeof(realw)*(*size),hipMemcpyDeviceToHost),42006);
  print_CUDA_error_if_any(hipMemcpy(b_veloc,mp->d_b_veloc,sizeof(realw)*(*size),hipMemcpyDeviceToHost),42007);
  print_CUDA_error_if_any(hipMemcpy(b_accel,mp->d_b_accel,sizeof(realw)*(*size),hipMemcpyDeviceToHost),42008);

}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_accel_to_device,
              TRNASFER_ACCEL_TO_DEVICE)(int* size, realw* accel,long* Mesh_pointer) {

  TRACE("transfer_accel_to_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(mp->d_accel,accel,sizeof(realw)*(*size),hipMemcpyHostToDevice),40016);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_accel_from_device,
              TRANSFER_ACCEL_FROM_DEVICE)(int* size, realw* accel,long* Mesh_pointer) {

  TRACE("transfer_accel_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(accel,mp->d_accel,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40026);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_accel_from_device,
              TRNASFER_B_ACCEL_FROM_DEVICE)(int* size, realw* b_accel,long* Mesh_pointer) {

  TRACE("transfer_b_accel_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(b_accel,mp->d_b_accel,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40036);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_sigma_from_device,
              TRANSFER_SIGMA_FROM_DEVICE)(int* size, realw* sigma_kl,long* Mesh_pointer) {

  TRACE("transfer_sigma_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(sigma_kl,mp->d_sigma_kl,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40046);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_displ_from_device,
              TRANSFER_B_DISPL_FROM_DEVICE)(int* size, realw* displ,long* Mesh_pointer) {

  TRACE("transfer_b_displ_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_b_displ,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40056);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_displ_from_device,
              TRANSFER_DISPL_FROM_DEVICE)(int* size, realw* displ,long* Mesh_pointer) {

  TRACE("transfer_displ_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container

  print_CUDA_error_if_any(hipMemcpy(displ,mp->d_displ,sizeof(realw)*(*size),hipMemcpyDeviceToHost),40066);

}

/* ----------------------------------------------------------------------------------------------- */

// attenuation fields

extern "C"
void FC_FUNC_(transfer_b_fields_att_to_device,
              TRANSFER_B_FIELDS_ATT_TO_DEVICE)(long* Mesh_pointer,
                                               realw* b_R_xx,realw* b_R_yy,realw* b_R_xy,
                                               realw* b_R_xz,realw* b_R_yz,
                                               int* size_R,
                                               realw* b_epsilondev_xx,realw* b_epsilondev_yy,realw* b_epsilondev_xy,
                                               realw* b_epsilondev_xz,realw* b_epsilondev_yz,
                                               int* size_epsilondev) {

  TRACE("transfer_b_fields_att_to_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xx,b_R_xx,*size_R*sizeof(realw),hipMemcpyHostToDevice),43011);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_yy,b_R_yy,*size_R*sizeof(realw),hipMemcpyHostToDevice),43012);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xy,b_R_xy,*size_R*sizeof(realw),hipMemcpyHostToDevice),43013);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_xz,b_R_xz,*size_R*sizeof(realw),hipMemcpyHostToDevice),43014);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_R_yz,b_R_yz,*size_R*sizeof(realw),hipMemcpyHostToDevice),43015);

  print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xx,b_epsilondev_xx,*size_epsilondev*sizeof(realw),hipMemcpyHostToDevice),43016);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yy,b_epsilondev_yy,*size_epsilondev*sizeof(realw),hipMemcpyHostToDevice),43017);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xy,b_epsilondev_xy,*size_epsilondev*sizeof(realw),hipMemcpyHostToDevice),43018);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_xz,b_epsilondev_xz,*size_epsilondev*sizeof(realw),hipMemcpyHostToDevice),43019);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_epsilondev_yz,b_epsilondev_yz,*size_epsilondev*sizeof(realw),hipMemcpyHostToDevice),43020);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_fields_att_to_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

// attenuation fields

extern "C"
void FC_FUNC_(transfer_fields_att_from_device,
              TRANSFER_FIELDS_ATT_FROM_DEVICE)(long* Mesh_pointer,
                                               realw* R_xx,realw* R_yy,realw* R_xy,realw* R_xz,realw* R_yz,
                                               int* size_R,
                                               realw* epsilondev_xx,realw* epsilondev_yy,realw* epsilondev_xy,
                                               realw* epsilondev_xz,realw* epsilondev_yz,
                                               int* size_epsilondev) {
  TRACE("transfer_fields_att_from_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(R_xx,mp->d_R_xx,*size_R*sizeof(realw),hipMemcpyDeviceToHost),43021);
  print_CUDA_error_if_any(hipMemcpy(R_yy,mp->d_R_yy,*size_R*sizeof(realw),hipMemcpyDeviceToHost),43022);
  print_CUDA_error_if_any(hipMemcpy(R_xy,mp->d_R_xy,*size_R*sizeof(realw),hipMemcpyDeviceToHost),43023);
  print_CUDA_error_if_any(hipMemcpy(R_xz,mp->d_R_xz,*size_R*sizeof(realw),hipMemcpyDeviceToHost),43024);
  print_CUDA_error_if_any(hipMemcpy(R_yz,mp->d_R_yz,*size_R*sizeof(realw),hipMemcpyDeviceToHost),43025);

  print_CUDA_error_if_any(hipMemcpy(epsilondev_xx,mp->d_epsilondev_xx,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost),43026);
  print_CUDA_error_if_any(hipMemcpy(epsilondev_yy,mp->d_epsilondev_yy,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost),43027);
  print_CUDA_error_if_any(hipMemcpy(epsilondev_xy,mp->d_epsilondev_xy,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost),43028);
  print_CUDA_error_if_any(hipMemcpy(epsilondev_xz,mp->d_epsilondev_xz,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost),43029);
  print_CUDA_error_if_any(hipMemcpy(epsilondev_yz,mp->d_epsilondev_yz,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost),43030);


#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_fields_att_from_device");
#endif
}

// JC JC here we will need to add GPU support for the new C-PML routines

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_kernels_el_to_host,
              TRANSFER_KERNELS_EL_TO_HOST)(long* Mesh_pointer,
                                            realw* h_rho_kl,
                                            realw* h_mu_kl,
                                            realw* h_kappa_kl,
                                            realw* h_cijkl_kl,
                                            int* NSPEC_AB) {
  TRACE("transfer_kernels_el_to_host");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(h_rho_kl,mp->d_rho_kl,*NSPEC_AB*NGLL3*sizeof(realw),
                                     hipMemcpyDeviceToHost),40101);
  if (mp->anisotropic_kl ){
    print_CUDA_error_if_any(hipMemcpy(h_cijkl_kl,mp->d_cijkl_kl,*NSPEC_AB*21*NGLL3*sizeof(realw),
                                       hipMemcpyDeviceToHost),40102);
  }else{
    print_CUDA_error_if_any(hipMemcpy(h_mu_kl,mp->d_mu_kl,*NSPEC_AB*NGLL3*sizeof(realw),
                                       hipMemcpyDeviceToHost),40102);
    print_CUDA_error_if_any(hipMemcpy(h_kappa_kl,mp->d_kappa_kl,*NSPEC_AB*NGLL3*sizeof(realw),
                                       hipMemcpyDeviceToHost),40103);
  }
}

/* ----------------------------------------------------------------------------------------------- */

// for NOISE simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_kernels_noise_to_host,
              TRANSFER_KERNELS_NOISE_TO_HOST)(long* Mesh_pointer,
                                              realw* h_sigma_kl,
                                              int* NSPEC_AB) {
  TRACE("transfer_kernels_noise_to_host");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(h_sigma_kl,mp->d_sigma_kl,NGLL3*(*NSPEC_AB)*sizeof(realw),
                                     hipMemcpyDeviceToHost),40201);

}


/* ----------------------------------------------------------------------------------------------- */

// for ACOUSTIC simulations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_fields_ac_to_device,
              TRANSFER_FIELDS_AC_TO_DEVICE)(int* size,
                                            realw* potential_acoustic,
                                            realw* potential_dot_acoustic,
                                            realw* potential_dot_dot_acoustic,
                                            long* Mesh_pointer) {

  TRACE("transfer_fields_ac_to_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(mp->d_potential_acoustic,potential_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),50110);
  print_CUDA_error_if_any(hipMemcpy(mp->d_potential_dot_acoustic,potential_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),50120);
  print_CUDA_error_if_any(hipMemcpy(mp->d_potential_dot_dot_acoustic,potential_dot_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),50130);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_fields_ac_to_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_fields_ac_to_device,
              TRANSFER_B_FIELDS_AC_TO_DEVICE)(int* size,
                                              realw* b_potential_acoustic,
                                              realw* b_potential_dot_acoustic,
                                              realw* b_potential_dot_dot_acoustic,
                                              long* Mesh_pointer) {

  TRACE("transfer_b_fields_ac_to_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(mp->d_b_potential_acoustic,b_potential_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),51110);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_potential_dot_acoustic,b_potential_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),51120);
  print_CUDA_error_if_any(hipMemcpy(mp->d_b_potential_dot_dot_acoustic,b_potential_dot_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyHostToDevice),51130);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_fields_ac_to_device");
#endif
}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_fields_ac_from_device,
              TRANSFER_FIELDS_AC_FROM_DEVICE)(int* size,
                                              realw* potential_acoustic,
                                              realw* potential_dot_acoustic,
                                              realw* potential_dot_dot_acoustic,
                                              long* Mesh_pointer) {
  TRACE("transfer_fields_ac_from_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(potential_acoustic,mp->d_potential_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),52111);
  print_CUDA_error_if_any(hipMemcpy(potential_dot_acoustic,mp->d_potential_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),52121);
  print_CUDA_error_if_any(hipMemcpy(potential_dot_dot_acoustic,mp->d_potential_dot_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),52131);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_fields_ac_from_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_fields_ac_from_device,
              TRANSFER_B_FIELDS_AC_FROM_DEVICE)(int* size,
                                                realw* b_potential_acoustic,
                                                realw* b_potential_dot_acoustic,
                                                realw* b_potential_dot_dot_acoustic,
                                                long* Mesh_pointer) {
  TRACE("transfer_b_fields_ac_from_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(b_potential_acoustic,mp->d_b_potential_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),53111);
  print_CUDA_error_if_any(hipMemcpy(b_potential_dot_acoustic,mp->d_b_potential_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),53121);
  print_CUDA_error_if_any(hipMemcpy(b_potential_dot_dot_acoustic,mp->d_b_potential_dot_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),53131);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_b_fields_ac_from_device");
#endif
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_dot_dot_from_device,
              TRNASFER_DOT_DOT_FROM_DEVICE)(int* size, realw* potential_dot_dot_acoustic,long* Mesh_pointer) {

  TRACE("transfer_dot_dot_from_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(potential_dot_dot_acoustic,mp->d_potential_dot_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),50041);

}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_b_dot_dot_from_device,
              TRNASFER_B_DOT_DOT_FROM_DEVICE)(int* size, realw* b_potential_dot_dot_acoustic,long* Mesh_pointer) {

  TRACE("transfer_b_dot_dot_from_device");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(b_potential_dot_dot_acoustic,mp->d_b_potential_dot_dot_acoustic,
                                     sizeof(realw)*(*size),hipMemcpyDeviceToHost),50042);

}


/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_kernels_ac_to_host,
              TRANSFER_KERNELS_AC_TO_HOST)(long* Mesh_pointer,realw* h_rho_ac_kl,realw* h_kappa_ac_kl,int* NSPEC_AB) {

  TRACE("transfer_kernels_ac_to_host");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  int size = *NSPEC_AB*NGLL3;

  // copies kernel values over to CPU host
  print_CUDA_error_if_any(hipMemcpy(h_rho_ac_kl,mp->d_rho_ac_kl,size*sizeof(realw),
                                     hipMemcpyDeviceToHost),54101);
  print_CUDA_error_if_any(hipMemcpy(h_kappa_ac_kl,mp->d_kappa_ac_kl,size*sizeof(realw),
                                     hipMemcpyDeviceToHost),54102);
}

/* ----------------------------------------------------------------------------------------------- */

// for Hess kernel calculations

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_kernels_hess_el_tohost,
              TRANSFER_KERNELS_HESS_EL_TOHOST)(long* Mesh_pointer,realw* h_hess_kl,int* NSPEC_AB) {

  TRACE("transfer_kernels_hess_el_tohost");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(h_hess_kl,mp->d_hess_el_kl,NGLL3*(*NSPEC_AB)*sizeof(realw),
                                     hipMemcpyDeviceToHost),70201);
}

/* ----------------------------------------------------------------------------------------------- */

extern "C"
void FC_FUNC_(transfer_kernels_hess_ac_tohost,
              TRANSFER_KERNELS_HESS_AC_TOHOST)(long* Mesh_pointer,realw* h_hess_ac_kl,int* NSPEC_AB) {

  TRACE("transfer_kernels_hess_ac_tohost");

  //get mesh pointer out of fortran integer container
  Mesh* mp = (Mesh*)(*Mesh_pointer);

  print_CUDA_error_if_any(hipMemcpy(h_hess_ac_kl,mp->d_hess_ac_kl,NGLL3*(*NSPEC_AB)*sizeof(realw),
                                     hipMemcpyDeviceToHost),70202);
}

// unused...

/* ----------------------------------------------------------------------------------------------- */
/*
extern "C"
void FC_FUNC_(transfer_compute_kernel_answers_from_device,
              TRANSFER_COMPUTE_KERNEL_ANSWERS_FROM_DEVICE)(long* Mesh_pointer,
                                                           realw* rho_kl,int* size_rho,
                                                           realw* mu_kl, int* size_mu,
                                                           realw* kappa_kl, int* size_kappa) {
TRACE("transfer_compute_kernel_answers_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  hipMemcpy(rho_kl,mp->d_rho_kl,*size_rho*sizeof(realw),hipMemcpyDeviceToHost);
  if (! mp->anisotropic_kl ){
    hipMemcpy(mu_kl,mp->d_mu_kl,*size_mu*sizeof(realw),hipMemcpyDeviceToHost);
    hipMemcpy(kappa_kl,mp->d_kappa_kl,*size_kappa*sizeof(realw),hipMemcpyDeviceToHost);
  }
}
*/

/* ----------------------------------------------------------------------------------------------- */
/*
extern "C"
void FC_FUNC_(transfer_compute_kernel_fields_from_device,
              TRANSFER_COMPUTE_KERNEL_FIELDS_FROM_DEVICE)(long* Mesh_pointer,
                                                          realw* accel, int* size_accel,
                                                          realw* b_displ, int* size_b_displ,
                                                          realw* epsilondev_xx,realw* epsilondev_yy,realw* epsilondev_xy,
                                                          realw* epsilondev_xz,realw* epsilondev_yz,
                                                          int* size_epsilondev,
                                                          realw* b_epsilondev_xx,realw* b_epsilondev_yy,realw* b_epsilondev_xy,
                                                          realw* b_epsilondev_xz,realw* b_epsilondev_yz,
                                                          int* size_b_epsilondev,
                                                          realw* rho_kl,int* size_rho,
                                                          realw* mu_kl, int* size_mu,
                                                          realw* kappa_kl, int* size_kappa,
                                                          realw* epsilon_trace_over_3,
                                                          realw* b_epsilon_trace_over_3,
                                                          int* size_epsilon_trace_over_3) {
TRACE("transfer_compute_kernel_fields_from_device");

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  hipMemcpy(accel,mp->d_accel,*size_accel*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_displ,mp->d_b_displ,*size_b_displ*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_xx,mp->d_epsilondev_xx,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_yy,mp->d_epsilondev_yy,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_xy,mp->d_epsilondev_xy,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_xz,mp->d_epsilondev_xz,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(epsilondev_yz,mp->d_epsilondev_yz,*size_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_epsilondev_xx,mp->d_b_epsilondev_xx,*size_b_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_epsilondev_yy,mp->d_b_epsilondev_yy,*size_b_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_epsilondev_xy,mp->d_b_epsilondev_xy,*size_b_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_epsilondev_xz,mp->d_b_epsilondev_xz,*size_b_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(b_epsilondev_yz,mp->d_b_epsilondev_yz,*size_b_epsilondev*sizeof(realw),hipMemcpyDeviceToHost);
  hipMemcpy(rho_kl,mp->d_rho_kl,*size_rho*sizeof(realw),hipMemcpyDeviceToHost);

  if (! mp->anisotropic_kl ){
    hipMemcpy(mu_kl,mp->d_mu_kl,*size_mu*sizeof(realw),hipMemcpyDeviceToHost);
    hipMemcpy(kappa_kl,mp->d_kappa_kl,*size_kappa*sizeof(realw),hipMemcpyDeviceToHost);
  }

  hipMemcpy(epsilon_trace_over_3,mp->d_epsilon_trace_over_3,*size_epsilon_trace_over_3*sizeof(realw),
       hipMemcpyDeviceToHost);
  hipMemcpy(b_epsilon_trace_over_3,mp->d_b_epsilon_trace_over_3,*size_epsilon_trace_over_3*sizeof(realw),
       hipMemcpyDeviceToHost);

#ifdef ENABLE_VERY_SLOW_ERROR_CHECKING
  exit_on_cuda_error("after transfer_compute_kernel_fields_from_device");
#endif
}
*/

