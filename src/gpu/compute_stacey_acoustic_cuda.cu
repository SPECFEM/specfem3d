/*
 !=====================================================================
 !
 !                         S p e c f e m 3 D
 !                         -----------------
 !
 !     Main historical authors: Dimitri Komatitsch and Jeroen Tromp
 !                              CNRS, France
 !                       and Princeton University, USA
 !                 (there are currently many more authors!)
 !                           (c) October 2017
 !
 ! This program is free software; you can redistribute it and/or modify
 ! it under the terms of the GNU General Public License as published by
 ! the Free Software Foundation; either version 3 of the License, or
 ! (at your option) any later version.
 !
 ! This program is distributed in the hope that it will be useful,
 ! but WITHOUT ANY WARRANTY; without even the implied warranty of
 ! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 ! GNU General Public License for more details.
 !
 ! You should have received a copy of the GNU General Public License along
 ! with this program; if not, write to the Free Software Foundation, Inc.,
 ! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
 !
 !=====================================================================
 */

#include "mesh_constants_gpu.h"

/* ----------------------------------------------------------------------------------------------- */

// Stacey absorbing boundary - acoustic domains

/* ----------------------------------------------------------------------------------------------- */

extern EXTERN_LANG
void FC_FUNC_(compute_stacey_acoustic_cuda,
              COMPUTE_STACEY_ACOUSTIC_CUDA)(long* Mesh_pointer,
                                            int* iphasef,
                                            realw* h_b_absorb_potential,
                                            int* FORWARD_OR_ADJOINT_f) {

  TRACE("compute_stacey_acoustic_cuda");

  //double start_time = get_time_val();

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  int FORWARD_OR_ADJOINT = *FORWARD_OR_ADJOINT_f;

  // checks if anything to do
  if (mp->d_num_abs_boundary_faces == 0) return;

  int iphase          = *iphasef;

  // only add these contributions in first pass
  if (iphase != 1) return;

  // way 1: Elapsed time: 4.385948e-03
  // > NGLLSQUARE==NGLL2==25, but we handle this inside kernel
  //  int blocksize = 32;

  // way 2: Elapsed time: 4.379034e-03
  // > NGLLSQUARE==NGLL2==25, no further check inside kernel
  int blocksize = NGLL2;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->d_num_abs_boundary_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // safety check
  if (FORWARD_OR_ADJOINT != 0 && FORWARD_OR_ADJOINT != 1 && FORWARD_OR_ADJOINT != 3) {
    exit_on_error("Error invalid FORWARD_OR_ADJOINT in compute_stacey_acoustic_cuda() routine");
  }

  //  adjoint simulations: reads in absorbing boundary
  if (mp->simulation_type == 3 && FORWARD_OR_ADJOINT != 1){
    // copies array to GPU
    gpuMemcpy_todevice_void((void*)mp->d_b_absorb_potential,(void*)h_b_absorb_potential,mp->d_b_reclen_potential);
  }

  if (FORWARD_OR_ADJOINT == 0){
    // combined forward/backward fields
#ifdef USE_CUDA
    if (run_cuda){
      compute_stacey_acoustic_kernel<<<grid,threads>>>(mp->d_potential_dot_acoustic,
                                                       mp->d_potential_dot_dot_acoustic,
                                                       mp->d_abs_boundary_ispec,
                                                       mp->d_abs_boundary_ijk,
                                                       mp->d_abs_boundary_jacobian2Dw,
                                                       mp->d_ibool,
                                                       mp->d_rhostore,
                                                       mp->d_kappastore,
                                                       mp->d_ispec_is_acoustic,
                                                       mp->simulation_type,
                                                       mp->save_forward,
                                                       mp->d_num_abs_boundary_faces,
                                                       mp->d_b_potential_dot_acoustic,
                                                       mp->d_b_potential_dot_dot_acoustic,
                                                       mp->d_b_absorb_potential,
                                                       mp->gravity);
    }
#endif
#ifdef USE_HIP
    if (run_hip){
      hipLaunchKernelGGL(compute_stacey_acoustic_kernel, dim3(grid), dim3(threads), 0, 0,
                                                         mp->d_potential_dot_acoustic,
                                                         mp->d_potential_dot_dot_acoustic,
                                                         mp->d_abs_boundary_ispec,
                                                         mp->d_abs_boundary_ijk,
                                                         mp->d_abs_boundary_jacobian2Dw,
                                                         mp->d_ibool,
                                                         mp->d_rhostore,
                                                         mp->d_kappastore,
                                                         mp->d_ispec_is_acoustic,
                                                         mp->simulation_type,
                                                         mp->save_forward,
                                                         mp->d_num_abs_boundary_faces,
                                                         mp->d_b_potential_dot_acoustic,
                                                         mp->d_b_potential_dot_dot_acoustic,
                                                         mp->d_b_absorb_potential,
                                                         mp->gravity);
    }
#endif

  }else{
    // sets gpu arrays
    field *potential_dot, *potential_dot_dot;
    if (FORWARD_OR_ADJOINT == 1) {
      potential_dot = mp->d_potential_dot_acoustic;
      potential_dot_dot = mp->d_potential_dot_dot_acoustic;
    } else {
      // for backward/reconstructed fields
      potential_dot = mp->d_b_potential_dot_acoustic;
      potential_dot_dot = mp->d_b_potential_dot_dot_acoustic;
    }
    // single forward or backward fields
#ifdef USE_CUDA
    if (run_cuda){
      compute_stacey_acoustic_single_kernel<<<grid,threads>>>(potential_dot,
                                                              potential_dot_dot,
                                                              mp->d_abs_boundary_ispec,
                                                              mp->d_abs_boundary_ijk,
                                                              mp->d_abs_boundary_jacobian2Dw,
                                                              mp->d_ibool,
                                                              mp->d_rhostore,
                                                              mp->d_kappastore,
                                                              mp->d_ispec_is_acoustic,
                                                              FORWARD_OR_ADJOINT,
                                                              mp->simulation_type,
                                                              mp->save_forward,
                                                              mp->d_num_abs_boundary_faces,
                                                              mp->d_b_absorb_potential,
                                                              mp->gravity);
    }
#endif
#ifdef USE_HIP
    if (run_hip){
      hipLaunchKernelGGL(compute_stacey_acoustic_single_kernel, dim3(grid), dim3(threads), 0, 0,
                                                                potential_dot,
                                                                potential_dot_dot,
                                                                mp->d_abs_boundary_ispec,
                                                                mp->d_abs_boundary_ijk,
                                                                mp->d_abs_boundary_jacobian2Dw,
                                                                mp->d_ibool,
                                                                mp->d_rhostore,
                                                                mp->d_kappastore,
                                                                mp->d_ispec_is_acoustic,
                                                                FORWARD_OR_ADJOINT,
                                                                mp->simulation_type,
                                                                mp->save_forward,
                                                                mp->d_num_abs_boundary_faces,
                                                                mp->d_b_absorb_potential,
                                                                mp->gravity);
    }
#endif
  }

  //  adjoint simulations: stores absorbed wavefield part
  if (mp->simulation_type == 1 && mp->save_forward){
    // (hipMemcpy implicitly synchronizes all other cuda operations)
    // copies array to CPU
    gpuMemcpy_tohost_void((void*)h_b_absorb_potential,(void*)mp->d_b_absorb_potential,mp->d_b_reclen_potential);
  }

  GPU_ERROR_CHECKING("compute_stacey_acoustic_kernel");
}

/* ----------------------------------------------------------------------------------------------- */

extern EXTERN_LANG
void FC_FUNC_(compute_stacey_acoustic_undoatt_cuda,
              COMPUTE_STACEY_ACOUSTIC_UNDOATT_CUDA)(long* Mesh_pointer,
                                                     int* iphasef,
                                                     int* FORWARD_OR_ADJOINT_f) {

  TRACE("compute_stacey_acoustic_undoatt_cuda");

  //double start_time = get_time_val();

  Mesh* mp = (Mesh*)(*Mesh_pointer); //get mesh pointer out of fortran integer container
  int FORWARD_OR_ADJOINT = *FORWARD_OR_ADJOINT_f;

  // safety check
  if (FORWARD_OR_ADJOINT != 1 && FORWARD_OR_ADJOINT != 3) {
    exit_on_error("Error invalid FORWARD_OR_ADJOINT in compute_stacey_acoustic_undoatt_cuda() routine");
  }

  // checks if anything to do
  if (mp->d_num_abs_boundary_faces == 0) return;

  int iphase = *iphasef;

  // only add these contributions in first pass
  if (iphase != 1) return;

  // way 1: Elapsed time: 4.385948e-03
  // > NGLLSQUARE==NGLL2==25, but we handle this inside kernel
  //  int blocksize = 32;

  // way 2: Elapsed time: 4.379034e-03
  // > NGLLSQUARE==NGLL2==25, no further check inside kernel
  int blocksize = NGLL2;

  int num_blocks_x, num_blocks_y;
  get_blocks_xy(mp->d_num_abs_boundary_faces,&num_blocks_x,&num_blocks_y);

  dim3 grid(num_blocks_x,num_blocks_y);
  dim3 threads(blocksize,1,1);

  // no absorbing boundary need to be stored, only propagates forward in time
  // sets gpu arrays
  field *potential_dot, *potential_dot_dot;
  if (FORWARD_OR_ADJOINT == 1) {
    potential_dot = mp->d_potential_dot_acoustic;
    potential_dot_dot = mp->d_potential_dot_dot_acoustic;
  } else {
    // for backward/reconstructed fields
    potential_dot = mp->d_b_potential_dot_acoustic;
    potential_dot_dot = mp->d_b_potential_dot_dot_acoustic;
  }

  // undoatt: single forward or backward fields
#ifdef USE_CUDA
  if (run_cuda){
    compute_stacey_acoustic_undoatt_kernel<<<grid,threads>>>(potential_dot,
                                                             potential_dot_dot,
                                                             mp->d_abs_boundary_ispec,
                                                             mp->d_abs_boundary_ijk,
                                                             mp->d_abs_boundary_jacobian2Dw,
                                                             mp->d_ibool,
                                                             mp->d_rhostore,
                                                             mp->d_kappastore,
                                                             mp->d_ispec_is_acoustic,
                                                             mp->d_num_abs_boundary_faces,
                                                             mp->gravity);
  }
#endif
#ifdef USE_HIP
  if (run_hip){
    hipLaunchKernelGGL(compute_stacey_acoustic_undoatt_kernel, dim3(grid), dim3(threads), 0, 0,
                                                               potential_dot,
                                                               potential_dot_dot,
                                                               mp->d_abs_boundary_ispec,
                                                               mp->d_abs_boundary_ijk,
                                                               mp->d_abs_boundary_jacobian2Dw,
                                                               mp->d_ibool,
                                                               mp->d_rhostore,
                                                               mp->d_kappastore,
                                                               mp->d_ispec_is_acoustic,
                                                               mp->d_num_abs_boundary_faces,
                                                               mp->gravity);
  }
#endif

  //  no need to store absorbed wavefield part

  GPU_ERROR_CHECKING("compute_stacey_acoustic_undoatt_cuda");
}

