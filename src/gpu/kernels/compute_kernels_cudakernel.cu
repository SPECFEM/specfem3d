#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!                          S p e c f e m 3 D
!                          -----------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/


__global__ void compute_kernels_cudakernel(int* ispec_is_elastic,
                                           int* d_ibool,
                                           realw* accel,
                                           realw* b_displ,
                                           realw* epsilondev_xx,realw* epsilondev_yy,realw* epsilondev_xy,
                                           realw* epsilondev_xz,realw* epsilondev_yz,
                                           realw* b_epsilondev_xx,realw* b_epsilondev_yy,realw* b_epsilondev_xy,
                                           realw* b_epsilondev_xz,realw* b_epsilondev_yz,
                                           realw* rho_kl,
                                           realw deltat,
                                           realw* mu_kl,
                                           realw* kappa_kl,
                                           realw* epsilon_trace_over_3,
                                           realw* b_epsilon_trace_over_3,
                                           int NSPEC_AB) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;
  int ijk = threadIdx.x;
  int ijk_ispec = ijk + NGLL3*ispec;

  // handles case when there is 1 extra block (due to rectangular grid)
  if (ispec < NSPEC_AB) {

    // elastic elements only
    if (ispec_is_elastic[ispec]) {
      int iglob = d_ibool[ijk + NGLL3_PADDED*ispec] - 1 ;

      // isotropic kernels:
      // density kernel
      rho_kl[ijk_ispec] += deltat * (accel[3*iglob]*b_displ[3*iglob]+
                                     accel[3*iglob+1]*b_displ[3*iglob+1]+
                                     accel[3*iglob+2]*b_displ[3*iglob+2]);


      // shear modulus kernel
      mu_kl[ijk_ispec] += deltat * (epsilondev_xx[ijk_ispec]*b_epsilondev_xx[ijk_ispec]+
                                    epsilondev_yy[ijk_ispec]*b_epsilondev_yy[ijk_ispec]+
                                    (epsilondev_xx[ijk_ispec]+epsilondev_yy[ijk_ispec])*
                                    (b_epsilondev_xx[ijk_ispec]+b_epsilondev_yy[ijk_ispec])+
                                    2*(epsilondev_xy[ijk_ispec]*b_epsilondev_xy[ijk_ispec]+
                                       epsilondev_xz[ijk_ispec]*b_epsilondev_xz[ijk_ispec]+
                                       epsilondev_yz[ijk_ispec]*b_epsilondev_yz[ijk_ispec]));

      // bulk modulus kernel
      kappa_kl[ijk_ispec] += deltat*(9*epsilon_trace_over_3[ijk_ispec]*b_epsilon_trace_over_3[ijk_ispec]);

      /*
      if (ijk_ispec==100){
        printf(" Kernel, %e  %e \n",b_epsilondev_xx[ijk_ispec], b_epsilondev_yy[ijk_ispec]);
      }
      */
    }
  }
}


