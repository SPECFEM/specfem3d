#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!                          S p e c f e m 3 D
!                          -----------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/


__global__ void process_smooth(realw_const_p xstore_me,
                               realw_const_p ystore_me,
                               realw_const_p zstore_me,
                               realw_const_p xstore_other,
                               realw_const_p ystore_other,
                               realw_const_p zstore_other,
                               realw_const_p data_other,
                               const realw sigma_h2_inv, const realw sigma_v2_inv,
                               const int iker,
                               const int nspec_me,
                               const int nspec_other,
                               const realw v_criterion, const realw h_criterion,
                               realw_const_p integ_factor,
                               realw_p data_smooth,
                               realw_p normalisation){

  int ispec = blockIdx.x + gridDim.x*blockIdx.y;
  int igll = threadIdx.x;

  int gll_other;
  realw x_me,y_me,z_me;
  realw x_other,y_other,z_other;
  realw center_x,center_y,center_z;
  realw dist_h,dist_v;
  realw val,val_gaussian;
  realw coef,normalisation_slice;
  realw dat;

  __shared__ int sh_test[NGLL3];
  __shared__ realw sh_x_other[NGLL3];
  __shared__ realw sh_y_other[NGLL3];
  __shared__ realw sh_z_other[NGLL3];
  __shared__ realw sh_integ_factor[NGLL3];
  __shared__ realw sh_data[NGLL3];

  // for each reference GLL point, we can check a block of 125 neighbor elements
  // by convenience, the block size is set to the number of threads 125 of this kernel
  int n_loop = nspec_other/NGLL3 + 1;

  // reference GLL point position
  x_me = xstore_me[NGLL3*ispec + igll ];
  y_me = ystore_me[NGLL3*ispec + igll ];
  z_me = zstore_me[NGLL3*ispec + igll ];

  __syncthreads();

  dat = 0.f;
  normalisation_slice = 0.f;

  //We test 125 spectral elements at a time
  for (int i=0; i < n_loop; i++){
    __syncthreads();

    // each thread helps to test a different element in the other slice (using the center position)
    // number of threads == NGLL3 == 125
    // for i==0: element range [0,124]
    // for i==1: element range [125,(125+124)]
    // ..
    // for i==n_loop-1: element range [NGLL3*(nloop-1),NGLL3*(nloop-1)+124]
    //                  where NGLL3*(nloop-1)+124 is equal to nspec_other (or slightly greater)
    int ispec_other = NGLL3*i + igll;

    if (ispec_other < nspec_other){
      // center position
      center_x = (xstore_other[ispec_other * NGLL3] + xstore_other[ispec_other * NGLL3 + (NGLL3 - 1)]) * 0.5f;
      center_y = (ystore_other[ispec_other * NGLL3] + ystore_other[ispec_other * NGLL3 + (NGLL3 - 1)]) * 0.5f;
      center_z = (zstore_other[ispec_other * NGLL3] + zstore_other[ispec_other * NGLL3 + (NGLL3 - 1)]) * 0.5f;

      // note: instead of distance we use distance squared to avoid too many sqrt() operations

      // Cartesian case
      // distance horizontal = (x-x0)**2 + (y-y0)**2, and vertical = (z-z0)**2
      dist_h = (x_me - center_x)*(x_me - center_x) + (y_me - center_y)*(y_me - center_y);
      dist_v = (z_me - center_z)*(z_me - center_z);
    } else {
      // artificial high values
      dist_v = 99999999.f;
      dist_h = 99999999.f;
    }

    // tests if element is too far away
    sh_test[igll] = ( ispec_other >= nspec_other
                    || dist_h > h_criterion
                    || dist_v > v_criterion ) ? 1 : 0 ;

    __syncthreads();

    // loops over each spectral element tested
    for (int k=0; k < NGLL3; k++){
      __syncthreads();

      // skips element if test was true (too far away)
      if (sh_test[k]) continue ;

      // loads data from other slice to shared memory
      int ispec_test = i*NGLL3 + k;
      sh_x_other[igll] = xstore_other[ispec_test*NGLL3 + igll];
      sh_y_other[igll] = ystore_other[ispec_test*NGLL3 + igll];
      sh_z_other[igll] = zstore_other[ispec_test*NGLL3 + igll];

      sh_data[igll] = data_other[ispec_test*NGLL3 + igll];
      sh_integ_factor[igll] = integ_factor[ispec_test*NGLL3 + igll];

      __syncthreads();

      // loops over gll points
      for (int j=0; j < NGLL3; j++){
        gll_other = (igll + j) % NGLL3;

        x_other = sh_x_other[gll_other];
        y_other = sh_y_other[gll_other];
        z_other = sh_z_other[gll_other];

        // Cartesian case
        // distance horizontal = (x-x0)**2 + (y-y0)**2, and vertical = (z-z0)**2
        dist_h = (x_me - x_other)*(x_me - x_other) + (y_me - y_other)*(y_me - y_other);
        dist_v = (z_me - z_other)*(z_me - z_other);

        // Gaussian function
        val = - dist_h*sigma_h2_inv - dist_v*sigma_v2_inv;

        // limits to single precision
        if (val < - 86.0f){
          // smaller than numerical precision: exp(-86) < 1.e-37
          val_gaussian = 0.0f;
        } else {
          val_gaussian = expf(val);
        }

        coef = val_gaussian * sh_integ_factor[gll_other];

        normalisation_slice = normalisation_slice + coef;
        dat += sh_data[gll_other] * coef;
      } //loop on each gll_other
    } //loop on each spec_other tested
  } //loop on each serie of 125 spec_other

  data_smooth[NGLL3*nspec_me*iker + NGLL3*ispec + igll] += dat;

  // note: normalization coefficient is added nker times
  normalisation[NGLL3*ispec + igll] += normalisation_slice;
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void normalize_data(realw_p data_smooth,
                               realw_const_p normalisation,
                               int nker,
                               int nspec_me){

  int ispec = blockIdx.x + gridDim.x*blockIdx.y;
  int igll = threadIdx.x;

  // note: normalization coefficient is added nker times, thus divide by nker
  realw norm = normalisation[NGLL3*ispec + igll] / nker;

  // avoids division by zero
  if (norm < 1.e-24) norm = 1.0f;

  // normalizes smoothed kernel values
  for (int iker=0; iker<nker; iker++) data_smooth[NGLL3*nspec_me*iker + NGLL3*ispec + igll] /= norm;
}


