#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!               S p e c f e m 3 D  V e r s i o n  3 . 0
!               ---------------------------------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/


__global__ void compute_stacey_elastic_kernel(realw* veloc,
                                              realw* accel,
                                              int* abs_boundary_ispec,
                                              int* abs_boundary_ijk,
                                              realw* abs_boundary_normal,
                                              realw* abs_boundary_jacobian2Dw,
                                              int* d_ibool,
                                              realw* rho_vp,
                                              realw* rho_vs,
                                              int* ispec_is_elastic,
                                              int SIMULATION_TYPE,
                                              int SAVE_FORWARD,
                                              int num_abs_boundary_faces,
                                              realw* b_absorb_field) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int i,j,k,iglob,ispec;
  realw vx,vy,vz,vn;
  realw nx,ny,nz;
  realw rho_vp_temp,rho_vs_temp;
  realw tx,ty,tz;
  realw jacobianw;

  // don't compute points outside NGLLSQUARE==NGLL2==25
  // way 2: no further check needed since blocksize = 25
  if (iface < num_abs_boundary_faces){

  //if (igll < NGLL2 && iface < num_abs_boundary_faces) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    if (ispec_is_elastic[ispec]) {

      i = abs_boundary_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)]-1;
      j = abs_boundary_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)]-1;
      k = abs_boundary_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)]-1;

      iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

      // gets associated velocity

      vx = veloc[iglob*3+0];
      vy = veloc[iglob*3+1];
      vz = veloc[iglob*3+2];

      // gets associated normal
      nx = abs_boundary_normal[INDEX3(NDIM,NGLL2,0,igll,iface)];
      ny = abs_boundary_normal[INDEX3(NDIM,NGLL2,1,igll,iface)];
      nz = abs_boundary_normal[INDEX3(NDIM,NGLL2,2,igll,iface)];

      // // velocity component in normal direction (normal points out of element)
      vn = vx*nx + vy*ny + vz*nz;

      rho_vp_temp = rho_vp[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];
      rho_vs_temp = rho_vs[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];

      tx = rho_vp_temp*vn*nx + rho_vs_temp*(vx-vn*nx);
      ty = rho_vp_temp*vn*ny + rho_vs_temp*(vy-vn*ny);
      tz = rho_vp_temp*vn*nz + rho_vs_temp*(vz-vn*nz);

      jacobianw = abs_boundary_jacobian2Dw[INDEX2(NGLL2,igll,iface)];

      atomicAdd(&accel[iglob*3],-tx*jacobianw);
      atomicAdd(&accel[iglob*3+1],-ty*jacobianw);
      atomicAdd(&accel[iglob*3+2],-tz*jacobianw);

      if (SAVE_FORWARD && SIMULATION_TYPE == 1) {
        b_absorb_field[INDEX3(NDIM,NGLL2,0,igll,iface)] = tx*jacobianw;
        b_absorb_field[INDEX3(NDIM,NGLL2,1,igll,iface)] = ty*jacobianw;
        b_absorb_field[INDEX3(NDIM,NGLL2,2,igll,iface)] = tz*jacobianw;
      } // SIMULATION_TYPE
    }
  } // num_abs_boundary_faces
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_stacey_elastic_sim3_kernel(int* abs_boundary_ispec,
                                                   int* abs_boundary_ijk,
                                                   int* d_ibool,
                                                   int* ispec_is_elastic,
                                                   int num_abs_boundary_faces,
                                                   realw* b_accel,
                                                   realw* b_absorb_field) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int i,j,k,iglob,ispec;

  // don't compute points outside NGLLSQUARE==NGLL2==25
  // way 2: no further check needed since blocksize = 25
  if (iface < num_abs_boundary_faces){

  //if (igll < NGLL2 && iface < num_abs_boundary_faces) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    if (ispec_is_elastic[ispec]) {

      i = abs_boundary_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)]-1;
      j = abs_boundary_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)]-1;
      k = abs_boundary_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)]-1;

      iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

      atomicAdd(&b_accel[iglob*3  ],-b_absorb_field[INDEX3(NDIM,NGLL2,0,igll,iface)]);
      atomicAdd(&b_accel[iglob*3+1],-b_absorb_field[INDEX3(NDIM,NGLL2,1,igll,iface)]);
      atomicAdd(&b_accel[iglob*3+2],-b_absorb_field[INDEX3(NDIM,NGLL2,2,igll,iface)]);
    }
  } // num_abs_boundary_faces
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_stacey_elastic_single_kernel(realw* veloc,
                                                     realw* accel,
                                                     int* abs_boundary_ispec,
                                                     int* abs_boundary_ijk,
                                                     realw* abs_boundary_normal,
                                                     realw* abs_boundary_jacobian2Dw,
                                                     int* d_ibool,
                                                     realw* rho_vp,
                                                     realw* rho_vs,
                                                     int* ispec_is_elastic,
                                                     int FORWARD_OR_ADJOINT,
                                                     int SIMULATION_TYPE,
                                                     int SAVE_FORWARD,
                                                     int num_abs_boundary_faces,
                                                     realw* b_absorb_field) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int i,j,k,iglob,ispec;
  realw vx,vy,vz,vn;
  realw nx,ny,nz;
  realw rho_vp_temp,rho_vs_temp;
  realw tx,ty,tz;
  realw jacobianw;

  // don't compute points outside NGLLSQUARE==NGLL2==25
  // way 2: no further check needed since blocksize = 25
  if (iface < num_abs_boundary_faces){

  //if (igll < NGLL2 && iface < num_abs_boundary_faces) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    if (ispec_is_elastic[ispec]) {

      i = abs_boundary_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)]-1;
      j = abs_boundary_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)]-1;
      k = abs_boundary_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)]-1;

      iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

      if (FORWARD_OR_ADJOINT == 3){
        // Sommerfeld condition
        atomicAdd(&accel[iglob*3  ],-b_absorb_field[INDEX3(NDIM,NGLL2,0,igll,iface)]);
        atomicAdd(&accel[iglob*3+1],-b_absorb_field[INDEX3(NDIM,NGLL2,1,igll,iface)]);
        atomicAdd(&accel[iglob*3+2],-b_absorb_field[INDEX3(NDIM,NGLL2,2,igll,iface)]);
      }else{
        // gets associated velocity
        vx = veloc[iglob*3+0];
        vy = veloc[iglob*3+1];
        vz = veloc[iglob*3+2];

        // gets associated normal
        nx = abs_boundary_normal[INDEX3(NDIM,NGLL2,0,igll,iface)];
        ny = abs_boundary_normal[INDEX3(NDIM,NGLL2,1,igll,iface)];
        nz = abs_boundary_normal[INDEX3(NDIM,NGLL2,2,igll,iface)];

        // // velocity component in normal direction (normal points out of element)
        vn = vx*nx + vy*ny + vz*nz;

        rho_vp_temp = rho_vp[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];
        rho_vs_temp = rho_vs[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];

        tx = rho_vp_temp*vn*nx + rho_vs_temp*(vx-vn*nx);
        ty = rho_vp_temp*vn*ny + rho_vs_temp*(vy-vn*ny);
        tz = rho_vp_temp*vn*nz + rho_vs_temp*(vz-vn*nz);

        jacobianw = abs_boundary_jacobian2Dw[INDEX2(NGLL2,igll,iface)];

        atomicAdd(&accel[iglob*3],-tx*jacobianw);
        atomicAdd(&accel[iglob*3+1],-ty*jacobianw);
        atomicAdd(&accel[iglob*3+2],-tz*jacobianw);

        if (SAVE_FORWARD && SIMULATION_TYPE == 1) {
          b_absorb_field[INDEX3(NDIM,NGLL2,0,igll,iface)] = tx*jacobianw;
          b_absorb_field[INDEX3(NDIM,NGLL2,1,igll,iface)] = ty*jacobianw;
          b_absorb_field[INDEX3(NDIM,NGLL2,2,igll,iface)] = tz*jacobianw;
        } // SIMULATION_TYPE
      }
    }
  } // num_abs_boundary_faces
}

/* ----------------------------------------------------------------------------------------------- */

__global__ void compute_stacey_elastic_undoatt_kernel(realw* veloc,
                                                      realw* accel,
                                                      int* abs_boundary_ispec,
                                                      int* abs_boundary_ijk,
                                                      realw* abs_boundary_normal,
                                                      realw* abs_boundary_jacobian2Dw,
                                                      int* d_ibool,
                                                      realw* rho_vp,
                                                      realw* rho_vs,
                                                      int* ispec_is_elastic,
                                                      int num_abs_boundary_faces) {

  int igll = threadIdx.x; // tx
  int iface = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int i,j,k,iglob,ispec;
  realw vx,vy,vz,vn;
  realw nx,ny,nz;
  realw rho_vp_temp,rho_vs_temp;
  realw tx,ty,tz;
  realw jacobianw;

  // don't compute points outside NGLLSQUARE==NGLL2==25
  // way 2: no further check needed since blocksize = 25
  if (iface < num_abs_boundary_faces){

  //if (igll < NGLL2 && iface < num_abs_boundary_faces) {

    // "-1" from index values to convert from Fortran-> C indexing
    ispec = abs_boundary_ispec[iface]-1;

    if (ispec_is_elastic[ispec]) {

      i = abs_boundary_ijk[INDEX3(NDIM,NGLL2,0,igll,iface)]-1;
      j = abs_boundary_ijk[INDEX3(NDIM,NGLL2,1,igll,iface)]-1;
      k = abs_boundary_ijk[INDEX3(NDIM,NGLL2,2,igll,iface)]-1;

      iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

      // gets associated velocity
      vx = veloc[iglob*3+0];
      vy = veloc[iglob*3+1];
      vz = veloc[iglob*3+2];

      // gets associated normal
      nx = abs_boundary_normal[INDEX3(NDIM,NGLL2,0,igll,iface)];
      ny = abs_boundary_normal[INDEX3(NDIM,NGLL2,1,igll,iface)];
      nz = abs_boundary_normal[INDEX3(NDIM,NGLL2,2,igll,iface)];

      // // velocity component in normal direction (normal points out of element)
      vn = vx*nx + vy*ny + vz*nz;

      rho_vp_temp = rho_vp[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];
      rho_vs_temp = rho_vs[INDEX4(NGLLX,NGLLX,NGLLX,i,j,k,ispec)];

      tx = rho_vp_temp*vn*nx + rho_vs_temp*(vx-vn*nx);
      ty = rho_vp_temp*vn*ny + rho_vs_temp*(vy-vn*ny);
      tz = rho_vp_temp*vn*nz + rho_vs_temp*(vz-vn*nz);

      jacobianw = abs_boundary_jacobian2Dw[INDEX2(NGLL2,igll,iface)];

      atomicAdd(&accel[iglob*3],-tx*jacobianw);
      atomicAdd(&accel[iglob*3+1],-ty*jacobianw);
      atomicAdd(&accel[iglob*3+2],-tz*jacobianw);
    }
  } // num_abs_boundary_faces
}



