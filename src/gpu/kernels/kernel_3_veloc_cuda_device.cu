#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!               S p e c f e m 3 D  V e r s i o n  3 . 0
!               ---------------------------------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/


__global__ void kernel_3_veloc_cuda_device(realw_p veloc,
                                           realw_p accel,
                                           int size,
                                           realw deltatover2) {

  int id = threadIdx.x + (blockIdx.x + blockIdx.y*gridDim.x)*blockDim.x;

  // because of block and grid sizing problems, there is a small
  // amount of buffer at the end of the calculation
  if (id < size) {
    veloc[3*id]   = veloc[3*id]   + deltatover2 * accel[3*id];
    veloc[3*id+1] = veloc[3*id+1] + deltatover2 * accel[3*id+1];
    veloc[3*id+2] = veloc[3*id+2] + deltatover2 * accel[3*id+2];
  }
}

