#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!                          S p e c f e m 3 D
!                          -----------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/


#ifdef USE_TEXTURES_FIELDS
realw_texture d_potential_tex;
realw_texture d_potential_dot_dot_tex;
//backward/reconstructed
realw_texture d_b_potential_tex;
realw_texture d_b_potential_dot_dot_tex;

//note: texture variables are implicitly static, and cannot be passed as arguments to cuda kernels;
//      thus, 1) we thus use if-statements (FORWARD_OR_ADJOINT) to determine from which texture to fetch from
//            2) we use templates
//      since if-statements are a bit slower as the variable is only known at runtime, we use option 2)

// templates definitions
template<int FORWARD_OR_ADJOINT> __device__ float texfetch_potential(int x);
template<int FORWARD_OR_ADJOINT> __device__ float texfetch_potential_dot_dot(int x);

// templates for texture fetching
// FORWARD_OR_ADJOINT == 1 <- forward arrays
template<> __device__ float texfetch_potential<1>(int x) { return tex1Dfetch(d_potential_tex, x); }
template<> __device__ float texfetch_potential_dot_dot<1>(int x) { return tex1Dfetch(d_potential_dot_dot_tex, x); }
// FORWARD_OR_ADJOINT == 3 <- backward/reconstructed arrays
template<> __device__ float texfetch_potential<3>(int x) { return tex1Dfetch(d_b_potential_tex, x); }
template<> __device__ float texfetch_potential_dot_dot<3>(int x) { return tex1Dfetch(d_b_potential_dot_dot_tex, x); }

#endif

#ifdef USE_TEXTURES_CONSTANTS
extern realw_texture d_hprime_xx_tex;
#endif


// note on performance optimizations:
//
//   instead of providing spezialized kernel routines (without mesh coloring, without gravity, etc.),
//   we only provide one "general" kernel to handle all cases. this reduces code redundancy and improves code readability.
//   as tradeoff, we take a little performance hit of around ~ 3%
//
//   performance tests done:
//   - registers: we were trying to reduce the number of registers, as this is the main limiter for the
//                occupancy of the kernel. however, there is only little difference in register pressure for one "general" kernel
//                or multiple "spezialized" kernels. reducing registers is mainly achieved through the launch_bonds() directive.
//   - branching: we were trying to reduce code branches, such as the if-active check in earlier code versions.
//                reducing the branching helps the compiler to better optimize the executable.
//   - memory accesses: the global memory accesses are avoiding texture reads for coalescent arrays, as this is
//                still faster. thus we were using no __ldg() loads or __restricted__ pointer usage,
//                as those implicitly lead the compiler to use texture reads.
//   - arithmetic intensity: ratio of floating-point operations vs. memory accesses is still low for our kernels.
//                tests with using a loop over elements to re-use the constant arrays (like hprime, wgllwgll,..) and thus
//                increasing the arithmetic intensity failed because the number of registers increased as well.
//                this increased register pressure reduced the occupancy and slowed down the kernel performance.
//   - hiding memory latency: to minimize waiting times to retrieve a memory value from global memory, we put
//                some more calculations into the same code block before calling syncthreads(). this should help the
//                compiler to move independent calculations to wherever it can overlap it with memory access operations.
//                note, especially the if (gravity )-block locations are very sensitive
//                for optimal register usage and compiler optimizations
//


/* ----------------------------------------------------------------------------------------------- */


template<int FORWARD_OR_ADJOINT> __global__ void
#ifdef USE_LAUNCH_BOUNDS
// adds compiler specification
__launch_bounds__(NGLL3_PADDED,LAUNCH_MIN_BLOCKS_ACOUSTIC)
#endif
Kernel_2_acoustic_impl(const int nb_blocks_to_compute,
                       const int* d_ibool,
                       const int* d_irregular_element_number,
                       const int* d_phase_ispec_inner_acoustic,
                       const int num_phase_ispec_acoustic,
                       const int d_iphase,
                       field_const_p d_potential_acoustic,
                       field_p d_potential_dot_dot_acoustic,
                       field_const_p d_b_potential_acoustic,
                       field_p d_b_potential_dot_dot_acoustic,
                       const int nb_field,
                       realw* d_xix,realw* d_xiy,realw* d_xiz,
                       realw* d_etax,realw* d_etay,realw* d_etaz,
                       realw* d_gammax,realw* d_gammay,realw* d_gammaz,
                       const realw xix_regular, const realw jacobian_regular,
                       realw_const_p d_hprime_xx,
                       realw_const_p hprimewgll_xx,
                       realw_const_p wgllwgll_xy,realw_const_p wgllwgll_xz,realw_const_p wgllwgll_yz,
                       realw* d_rhostore,
                       const int use_mesh_coloring_gpu,
                       const int gravity,
                       realw_const_p minus_g,
                       realw* d_kappastore,
                       realw_const_p wgll_cube){

  // block-id == number of local element id in phase_ispec array
  int bx = blockIdx.y*gridDim.x+blockIdx.x;

  // thread-id == GLL node id
  // note: use only NGLL^3 = 125 active threads, plus 3 inactive/ghost threads,
  //       because we used memory padding from NGLL^3 = 125 to 128 to get coalescent memory accesses;
  //       to avoid execution branching and the need of registers to store an active state variable,
  //       the thread ids are put in valid range
  int tx = threadIdx.x;

  int I,J,K;
  int iglob,offset;
  int working_element,ispec_irreg;

  field temp1l,temp2l,temp3l;
  realw xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl;
  realw jacobianl;

  field dpotentialdxl,dpotentialdyl,dpotentialdzl;
  realw fac1,fac2,fac3;
  realw rho_invl,kappa_invl;

  field sum_terms;
  field gravity_term;

  __shared__ field s_dummy_loc[2*NGLL3];

  __shared__ field s_temp1[NGLL3];
  __shared__ field s_temp2[NGLL3];
  __shared__ field s_temp3[NGLL3];

  __shared__ realw sh_hprime_xx[NGLL2];
  __shared__ realw sh_hprimewgll_xx[NGLL2];

// arithmetic intensity: ratio of number-of-arithmetic-operations / number-of-bytes-accessed-on-DRAM
//
// hand-counts on floating-point operations: counts addition/subtraction/multiplication/division
//                                           no counts for operations on indices in for-loops (compiler will likely unrool loops)
//
//                                           counts accesses to global memory, but no shared memory or register loads/stores
//                                           float has 4 bytes

// counts: for simulations without gravity, without mesh_coloring
//         counts floating-point operations (FLOP) per thread
//         counts global memory accesses in bytes (BYTES) per block
// 2 FLOP
//
// 0 BYTES

  // checks if anything to do
  if (bx >= nb_blocks_to_compute) return;

  // limits thread ids to range [0,125-1]
  if (tx >= NGLL3) tx = NGLL3-1;

// counts:
// + 1 FLOP
//
// + 0 BYTE

  // spectral-element id
#ifdef USE_MESH_COLORING_GPU
  working_element = bx;
#else
  //mesh coloring
  if (use_mesh_coloring_gpu ){
    working_element = bx;
  }else{
    // iphase-1 and working_element-1 for Fortran->C array conventions
    working_element = d_phase_ispec_inner_acoustic[bx + num_phase_ispec_acoustic*(d_iphase-1)]-1;
  }
#endif

  // local padded index
  offset = working_element*NGLL3_PADDED + tx;
  ispec_irreg = d_irregular_element_number[working_element] - 1;
  // global index
  iglob = d_ibool[offset] - 1;

// counts:
// + 8 FLOP
//
// (1 int + 2 float) * 128 threads = 1536 BYTE

  // loads potential values into shared memory
  if (threadIdx.x < NGLL3) {
#ifdef USE_TEXTURES_FIELDS
    s_dummy_loc[tx] = texfetch_potential<FORWARD_OR_ADJOINT>(iglob);
    if (nb_field==2) s_dummy_loc[NGLL3+tx] = texfetch_potential<3>(iglob);
#else
    // changing iglob indexing to match fortran row changes fast style
    s_dummy_loc[tx] = d_potential_acoustic[iglob];
    if (nb_field==2) s_dummy_loc[NGLL3+tx] = d_b_potential_acoustic[iglob];

#endif
  }

// counts:
// + 0 FLOP
//
// + 1 float * 125 threads = 500 BYTE

  // gravity
  if (gravity ) kappa_invl = 1.f / d_kappastore[working_element*NGLL3 + tx];


  // local index
  K = (tx/NGLL2);
  J = ((tx-K*NGLL2)/NGLLX);
  I = (tx-K*NGLL2-J*NGLLX);

// counts:
// + 8 FLOP
//
// + 0 BYTES

  // note: loads mesh values here to give compiler possibility to overlap memory fetches with some computations;
  //       arguments defined as realw* instead of const realw* __restrict__ to avoid that the compiler
  //       loads all memory by texture loads (arrays accesses are coalescent, thus no need for texture reads)
  //
  // calculates laplacian
  if (ispec_irreg >= 0){ //irregular_element
    int offset = ispec_irreg*NGLL3_PADDED + tx;
    xixl = d_xix[offset];
    xiyl = d_xiy[offset];
    xizl = d_xiz[offset];
    etaxl = d_etax[offset];
    etayl = d_etay[offset];
    etazl = d_etaz[offset];
    gammaxl = d_gammax[offset];
    gammayl = d_gammay[offset];
    gammazl = d_gammaz[offset];

    jacobianl = 1.f / (xixl*(etayl*gammazl-etazl*gammayl)
                      -xiyl*(etaxl*gammazl-etazl*gammaxl)
                      +xizl*(etaxl*gammayl-etayl*gammaxl));
  }

  // density (reciproc)
  rho_invl = 1.f / d_rhostore[offset];

// counts:
// + 16 FLOP
//
// + 10 float * 128 threads = 5120 BYTE

  // loads hprime into shared memory
  if (tx < NGLL2) {
#ifdef USE_TEXTURES_CONSTANTS
    sh_hprime_xx[tx] = tex1Dfetch(d_hprime_xx_tex,tx);
#else
    sh_hprime_xx[tx] = d_hprime_xx[tx];
#endif
    // loads hprimewgll into shared memory
    sh_hprimewgll_xx[tx] = hprimewgll_xx[tx];
  }

// counts:
// + 0 FLOP
//
// + 2 * 1 float * 25 threads = 200 BYTE

  // synchronize all the threads (one thread for each of the NGLL grid points of the
  // current spectral element) because we need the whole element to be ready in order
  // to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  // summed terms with added gll weights
  fac1 = wgllwgll_yz[K*NGLLX+J];
  fac2 = wgllwgll_xz[K*NGLLX+I];
  fac3 = wgllwgll_xy[J*NGLLX+I];

  // We make a loop over direct and adjoint wavefields inside the GPU kernel to increase arithmetic intensity
  for (int k = 0 ; k < nb_field ; k++){

  // computes first matrix product
  temp1l = Make_field(0.f);
  temp2l = Make_field(0.f);
  temp3l = Make_field(0.f);

  for (int l=0;l<NGLLX;l++) {
    //assumes that hprime_xx = hprime_yy = hprime_zz
    // 1. cut-plane along xi-direction
    temp1l += s_dummy_loc[NGLL3*k+K*NGLL2+J*NGLLX+l] * sh_hprime_xx[l*NGLLX+I];
    // 2. cut-plane along eta-direction
    temp2l += s_dummy_loc[NGLL3*k+K*NGLL2+l*NGLLX+I] * sh_hprime_xx[l*NGLLX+J];
    // 3. cut-plane along gamma-direction
    temp3l += s_dummy_loc[NGLL3*k+l*NGLL2+J*NGLLX+I] * sh_hprime_xx[l*NGLLX+K];
  }

// counts:
// + NGLLX * 3 * 8 FLOP = 120 FLOP
//
// + 0 BYTE

  // compute derivatives of ux, uy and uz with respect to x, y and z
  // derivatives of potential
  if (threadIdx.x < NGLL3) {
    if (ispec_irreg >= 0){ //irregular_element

      dpotentialdxl = xixl*temp1l + etaxl*temp2l + gammaxl*temp3l;
      dpotentialdyl = xiyl*temp1l + etayl*temp2l + gammayl*temp3l;
      dpotentialdzl = xizl*temp1l + etazl*temp2l + gammazl*temp3l;

// counts:
// + 3 * 5 FLOP = 15 FLOP
//
// + 0 BYTE

      // form the dot product with the test vector
      s_temp1[tx] = jacobianl * rho_invl * (dpotentialdxl*xixl + dpotentialdyl*xiyl + dpotentialdzl*xizl);
      s_temp2[tx] = jacobianl * rho_invl * (dpotentialdxl*etaxl + dpotentialdyl*etayl + dpotentialdzl*etazl);
      s_temp3[tx] = jacobianl * rho_invl * (dpotentialdxl*gammaxl + dpotentialdyl*gammayl + dpotentialdzl*gammazl);
    }
    else{
      s_temp1[tx] = jacobian_regular * rho_invl * temp1l * xix_regular * xix_regular;
      s_temp2[tx] = jacobian_regular * rho_invl * temp2l * xix_regular * xix_regular;
      s_temp3[tx] = jacobian_regular * rho_invl * temp3l * xix_regular * xix_regular;
    }
  }
  // pre-computes gravity sum term
  if (gravity ){
    // uses potential definition: s = grad(chi)
    //
    // gravity term: 1/kappa grad(chi) * g
    // assumes that g only acts in (negative) z-direction
    gravity_term = minus_g[iglob] * kappa_invl * jacobianl * wgll_cube[tx] * dpotentialdzl;
  }

// counts:
// + 3 * 7 FLOP = 21 FLOP
//
// + 0 BYTE

  // synchronize all the threads (one thread for each of the NGLL grid points of the
  // current spectral element) because we need the whole element to be ready in order
  // to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  // computes second matrix product
  temp1l = Make_field(0.f);
  temp2l = Make_field(0.f);
  temp3l = Make_field(0.f);

  for (int l=0;l<NGLLX;l++) {
    //assumes hprimewgll_xx = hprimewgll_yy = hprimewgll_zz
    // 1. cut-plane along xi-direction
    temp1l += s_temp1[K*NGLL2+J*NGLLX+l] * sh_hprimewgll_xx[I*NGLLX+l];
    // 2. cut-plane along eta-direction
    temp2l += s_temp2[K*NGLL2+l*NGLLX+I] * sh_hprimewgll_xx[J*NGLLX+l];
    // 3. cut-plane along gamma-direction
    temp3l += s_temp3[l*NGLL2+J*NGLLX+I] * sh_hprimewgll_xx[K*NGLLX+l];
  }

// counts:
// + NGLLX * 3 * 8 FLOP = 120 FLOP
//
// + 0 BYTE

  sum_terms = -(fac1*temp1l + fac2*temp2l + fac3*temp3l);

  // adds gravity contribution
  if (gravity) sum_terms += gravity_term;

// counts:
// + 3 * 2 FLOP + 6 FLOP = 12 FLOP
//
// + 3 float * 128 threads = 1536 BYTE

  __syncthreads();
// assembles potential array
  if (threadIdx.x < NGLL3) {
#ifdef USE_MESH_COLORING_GPU
  // no atomic operation needed, colors don't share global points between elements
#ifdef USE_TEXTURES_FIELDS
    if (k==0) d_potential_dot_dot_acoustic[iglob] = texfetch_potential_dot_dot<FORWARD_OR_ADJOINT>(iglob) + sum_terms;
    if (k==1) d_b_potential_dot_dot_acoustic[iglob] = texfetch_potential_dot_dot<3>(iglob) + sum_terms;
#else
    if (k==0) d_potential_dot_dot_acoustic[iglob] += sum_terms;
    if (k==1) d_b_potential_dot_dot_acoustic[iglob] += sum_terms;
#endif // USE_TEXTURES_FIELDS
#else  // MESH_COLORING
    //mesh coloring
    if (use_mesh_coloring_gpu ){
      // no atomic operation needed, colors don't share global points between elements
#ifdef USE_TEXTURES_FIELDS
    if (k==0) d_potential_dot_dot_acoustic[iglob] = texfetch_potential_dot_dot<FORWARD_OR_ADJOINT>(iglob) + sum_terms;
    if (k==1) d_b_potential_dot_dot_acoustic[iglob] = texfetch_potential_dot_dot<3>(iglob) + sum_terms;
#else
        if (k==0) d_potential_dot_dot_acoustic[iglob] += sum_terms;
        if (k==1) d_b_potential_dot_dot_acoustic[iglob] += sum_terms;
#endif // USE_TEXTURES_FIELDS
    }else{
          if (k==0) atomicAdd(&d_potential_dot_dot_acoustic[iglob],sum_terms);
          if (k==1) atomicAdd(&d_b_potential_dot_dot_acoustic[iglob],sum_terms);
    }
#endif // MESH_COLORING
  }
} //loop over k (forward and adjoint wavefield)

// counts:
// + 1 FLOP
//
// + 1 float * 125 threads = 500 BYTE

// -----------------
// total of: 323 FLOP per thread
//           ~ 128 * 323 = 41344 FLOP per block
//
//           8880 BYTE DRAM accesses per block
//
//           -> arithmetic intensity: 41344 FLOP / 8880 BYTES ~ 4.66 FLOP/BYTE (hand-count)
//
// -----------------
//
// nvprof: nvprof --metrics flops_sp ./xspecfem3D
//         -> 322631424 FLOPS (Single) floating-point operations for 20736 elements
//         -> 15559 FLOP per block
//
//         -> arithmetic intensity: ~ 15559 / 8880 flop/byte = 1.75 flop/byte
//
// roofline model: Tesla K20x
// ---------------------------
//   for a Kepler K20x card, the peak single-precision performance is about 3.95 TFlop/s.
//   global memory access has a bandwidth of ~ 250 GB/s.
//   thus there should be about 16 flop to hide a single byte memory access (3950./250. ~ 15.8 flop/byte = arithmetic intensity).
//
//   memory bandwidth: 250 GB/s
//   single-precision peak performance: 3.95 TFlop/s -> corner arithmetic intensity = 3950 / 250 ~ 15.8 flop/byte
//
//   note:
//     using dense matrix-matrix multiplication (SGEMM) leads to "practical" peak performance of around 2.9 TFlops.
//     (http://www.nvidia.com/docs/IO/122874/K20-and-K20X-application-performance-technical-brief.pdf)
//
//   acoustic kernel has an arithmetic intensity of: hand-counts   ~ 4.66 flop/byte
//                                                   nvprof-counts ~ 1.75 flop/byte
//
//   -> we can only achieve about: (hand-counts)   29% of the peak performance
//                                 (nvprof-counts) 11% of the peak performance
//
//                              i.e.               11% x theoretical peak performance ~ 440 GFlop/s.
//                                                 11% x "pratical"  peak performance ~ 320 GFlop/s.
//
//   CUDA_TIMING: we achieve about 224 GFlop/s (1 mpi process, 20736 elements)
//                -> that is about 8% of the "practical" peak. (or 70% of the theoretical arithmetic intensity)
//
//                this might be due to the first compute code block (before first syncthreads), where
//                the partial arithmetic intensity is lower than for the total routine.
//
// roofline model: Tesla K20c (Kepler architecture: http://www.nvidia.com/content/tesla/pdf/Tesla-KSeries-Overview-LR.pdf)
// ---------------------------
//   memory bandwidth: 208 GB/s
//   single-precision peak performance: 3.52 TFlop/s -> corner arithmetic intensity = 3520 / 208 ~ 16.9 flop/byte
//
//   we can only achieve about: (hand-counts)   27% of the peak performance -> 970.6 GFlop/s
//                              (nvprof-counts) 10% of the peak performance -> 364.5 GFlop/s - measured: 229.631 GFlop/s
//
// roofline model: nVidia GT 650m  http://www.gpuzoo.com/GPU-NVIDIA/GeForce_GT_650M_DDR3.html
// ---------------------------
//   memory bandwidth: 28.8 GB/s
//   single-precision peak performance: 625.6 GFlop/s -> corner arithmetic intensity = 625.6 / 28.8 ~ 21.7 flop/byte
//
//   we can only achieve about: (hand-counts)   21% of the peak performance -> 132.6 GFlop/s
//                              (nvprof-counts)  8% of the peak performance ->  50.5 GFlop/s - measured: 52.1907 GFlop/s
//
//
//
// better performance ideas and improvements are welcome :)

}

// note: in the past, we used templating to be able to call the same kernel_2 twice for both,
//       forward and backward wavefields. that is, calling it by
//          Kernel_2_acoustic_impl<1>
//       and
//          Kernel_2_acoustic_impl<3>
//       the templating helped to use textures for forward/backward fields.
//
//       most of this has become obsolete, textures are hardly needed for speedup anymore
//       and the Kernel_2 has become more and more specialized for different cases to
//       reduce register pressure and increase occupancy for better performance.
//       thus, in future we might re-evaluate and remove this template-feature.
//
// "forced" template instantiation
// see: https://isocpp.org/wiki/faq/templates#separate-template-fn-defn-from-decl
//      https://stackoverflow.com/questions/31705764/cuda-c-using-a-template-function-which-calls-a-template-kernel
//
// for compute_forces_acoustic_cuda.cu:
// Kernel_2_acoustic_impl<1> needs an explicit instantiation here to be able to link against it from a different .cu file

template __global__ void Kernel_2_acoustic_impl<1>(const int nb_blocks_to_compute,
                                                   const int* d_ibool,
                                                   const int* d_irregular_element_number,
                                                   const int* d_phase_ispec_inner_acoustic,
                                                   const int num_phase_ispec_acoustic,
                                                   const int d_iphase,
                                                   field_const_p d_potential_acoustic,
                                                   field_p d_potential_dot_dot_acoustic,
                                                   field_const_p d_b_potential_acoustic,
                                                   field_p d_b_potential_dot_dot_acoustic,
                                                   const int nb_field,
                                                   realw* d_xix,realw* d_xiy,realw* d_xiz,
                                                   realw* d_etax,realw* d_etay,realw* d_etaz,
                                                   realw* d_gammax,realw* d_gammay,realw* d_gammaz,
                                                   const realw xix_regular, const realw jacobian_regular,
                                                   realw_const_p d_hprime_xx,
                                                   realw_const_p hprimewgll_xx,
                                                   realw_const_p wgllwgll_xy,realw_const_p wgllwgll_xz,realw_const_p wgllwgll_yz,
                                                   realw* d_rhostore,
                                                   const int use_mesh_coloring_gpu,
                                                   const int gravity,
                                                   realw_const_p minus_g,
                                                   realw* d_kappastore,
                                                   realw_const_p wgll_cube);



/* ----------------------------------------------------------------------------------------------- */

//solving a single wavefield

__global__ void
#ifdef USE_LAUNCH_BOUNDS
// adds compiler specification
__launch_bounds__(NGLL3_PADDED,LAUNCH_MIN_BLOCKS_ACOUSTIC)
#endif
Kernel_2_acoustic_single_impl(const int nb_blocks_to_compute,
                              const int* d_ibool,
                              const int* d_phase_ispec_inner_acoustic,
                              const int num_phase_ispec_acoustic,
                              const int d_iphase,
                              field_const_p d_potential_acoustic,
                              field_p d_potential_dot_dot_acoustic,
                              realw* d_xix,realw* d_xiy,realw* d_xiz,
                              realw* d_etax,realw* d_etay,realw* d_etaz,
                              realw* d_gammax,realw* d_gammay,realw* d_gammaz,
                              const int* d_irregular_element_number,
                              const realw xix_regular, const realw jacobian_regular,
                              realw_const_p d_hprime_xx,
                              realw_const_p hprimewgll_xx,
                              realw_const_p wgllwgll_xy,realw_const_p wgllwgll_xz,realw_const_p wgllwgll_yz,
                              realw* d_rhostore,
                              const int use_mesh_coloring_gpu,
                              const int gravity,
                              realw_const_p minus_g,
                              realw* d_kappastore,
                              realw_const_p wgll_cube,
                              const int FORWAR_OR_ADJOINT){

  // block-id == number of local element id in phase_ispec array
  int bx = blockIdx.y*gridDim.x+blockIdx.x;

  // thread-id == GLL node id
  // note: use only NGLL^3 = 125 active threads, plus 3 inactive/ghost threads,
  //       because we used memory padding from NGLL^3 = 125 to 128 to get coalescent memory accesses;
  //       to avoid execution branching and the need of registers to store an active state variable,
  //       the thread ids are put in valid range
  int tx = threadIdx.x;

  int I,J,K;
  int iglob,offset;
  int working_element,ispec_irreg;

  field temp1l,temp2l,temp3l;
  realw xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl;
  realw jacobianl;

  field dpotentialdxl,dpotentialdyl,dpotentialdzl;
  realw fac1,fac2,fac3;
  realw rho_invl,kappa_invl;

  field sum_terms;
  field gravity_term;

  __shared__ field s_dummy_loc[NGLL3];

  __shared__ field s_temp1[NGLL3];
  __shared__ field s_temp2[NGLL3];
  __shared__ field s_temp3[NGLL3];

  __shared__ realw sh_hprime_xx[NGLL2];
  __shared__ realw sh_hprimewgll_xx[NGLL2];

  // checks if anything to do
  if (bx >= nb_blocks_to_compute) return;

  // limits thread ids to range [0,125-1]
  if (tx >= NGLL3) tx = NGLL3-1;

  // spectral-element id
#ifdef USE_MESH_COLORING_GPU
  working_element = bx;
#else
  //mesh coloring
  if (use_mesh_coloring_gpu ){
    working_element = bx;
  }else{
    // iphase-1 and working_element-1 for Fortran->C array conventions
    working_element = d_phase_ispec_inner_acoustic[bx + num_phase_ispec_acoustic*(d_iphase-1)]-1;
  }
#endif

  // local padded index
  offset = working_element*NGLL3_PADDED + tx;
  ispec_irreg = d_irregular_element_number[working_element] - 1;
  // global index
  iglob = d_ibool[offset] - 1;

  // loads potential values into shared memory
  if (threadIdx.x < NGLL3) {
#ifdef USE_TEXTURES_FIELDS
    if (FORWARD_OR_ADJOINT == 3){
      s_dummy_loc[tx] = texfetch_potential<3>(iglob);
    }else{
      s_dummy_loc[tx] = texfetch_potential<1>(iglob);
    }
#else
    // changing iglob indexing to match fortran row changes fast style
    s_dummy_loc[tx] = d_potential_acoustic[iglob];
#endif
  }

  // gravity
  if (gravity) kappa_invl = 1.f / d_kappastore[working_element*NGLL3 + tx];

  // local index
  K = (tx/NGLL2);
  J = ((tx-K*NGLL2)/NGLLX);
  I = (tx-K*NGLL2-J*NGLLX);

  // calculates laplacian
  if (ispec_irreg >= 0){
    //irregular_element
    int offset = ispec_irreg*NGLL3_PADDED + tx;

    xixl = d_xix[offset];
    xiyl = d_xiy[offset];
    xizl = d_xiz[offset];
    etaxl = d_etax[offset];
    etayl = d_etay[offset];
    etazl = d_etaz[offset];
    gammaxl = d_gammax[offset];
    gammayl = d_gammay[offset];
    gammazl = d_gammaz[offset];

    jacobianl = 1.f / (xixl*(etayl*gammazl-etazl*gammayl)
                      -xiyl*(etaxl*gammazl-etazl*gammaxl)
                      +xizl*(etaxl*gammayl-etayl*gammaxl));
  }

  // density (reciproc)
  rho_invl = 1.f / d_rhostore[offset];

  // loads hprime into shared memory
  if (tx < NGLL2) {
#ifdef USE_TEXTURES_CONSTANTS
    sh_hprime_xx[tx] = tex1Dfetch(d_hprime_xx_tex,tx);
#else
    sh_hprime_xx[tx] = d_hprime_xx[tx];
#endif
    // loads hprimewgll into shared memory
    sh_hprimewgll_xx[tx] = hprimewgll_xx[tx];
  }

  // synchronize all the threads (one thread for each of the NGLL grid points of the
  // current spectral element) because we need the whole element to be ready in order
  // to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  // computes first matrix product
  temp1l = Make_field(0.f);
  temp2l = Make_field(0.f);
  temp3l = Make_field(0.f);

  for (int l=0;l<NGLLX;l++) {
    //assumes that hprime_xx = hprime_yy = hprime_zz
    // 1. cut-plane along xi-direction
    temp1l += s_dummy_loc[K*NGLL2+J*NGLLX+l] * sh_hprime_xx[l*NGLLX+I];
    // 2. cut-plane along eta-direction
    temp2l += s_dummy_loc[K*NGLL2+l*NGLLX+I] * sh_hprime_xx[l*NGLLX+J];
    // 3. cut-plane along gamma-direction
    temp3l += s_dummy_loc[l*NGLL2+J*NGLLX+I] * sh_hprime_xx[l*NGLLX+K];
  }

  // compute derivatives of ux, uy and uz with respect to x, y and z
  // derivatives of potential
  if (threadIdx.x < NGLL3) {
    if (ispec_irreg >= 0){
      //irregular_element
      dpotentialdxl = xixl*temp1l + etaxl*temp2l + gammaxl*temp3l;
      dpotentialdyl = xiyl*temp1l + etayl*temp2l + gammayl*temp3l;
      dpotentialdzl = xizl*temp1l + etazl*temp2l + gammazl*temp3l;

      // form the dot product with the test vector
      s_temp1[tx] = jacobianl * rho_invl * (dpotentialdxl*xixl + dpotentialdyl*xiyl + dpotentialdzl*xizl);
      s_temp2[tx] = jacobianl * rho_invl * (dpotentialdxl*etaxl + dpotentialdyl*etayl + dpotentialdzl*etazl);
      s_temp3[tx] = jacobianl * rho_invl * (dpotentialdxl*gammaxl + dpotentialdyl*gammayl + dpotentialdzl*gammazl);
    }else{
      s_temp1[tx] = jacobian_regular * rho_invl * temp1l * xix_regular * xix_regular;
      s_temp2[tx] = jacobian_regular * rho_invl * temp2l * xix_regular * xix_regular;
      s_temp3[tx] = jacobian_regular * rho_invl * temp3l * xix_regular * xix_regular;
    }
  }

  // pre-computes gravity sum term
  if (gravity ){
    // uses potential definition: s = grad(chi)
    //
    // gravity term: 1/kappa grad(chi) * g
    // assumes that g only acts in (negative) z-direction
    gravity_term = minus_g[iglob] * kappa_invl * jacobianl * wgll_cube[tx] * dpotentialdzl;
  }

  // synchronize all the threads (one thread for each of the NGLL grid points of the
  // current spectral element) because we need the whole element to be ready in order
  // to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  // computes second matrix product
  temp1l = Make_field(0.f);
  temp2l = Make_field(0.f);
  temp3l = Make_field(0.f);

  for (int l=0;l<NGLLX;l++) {
    //assumes hprimewgll_xx = hprimewgll_yy = hprimewgll_zz
    // 1. cut-plane along xi-direction
    temp1l += s_temp1[K*NGLL2+J*NGLLX+l] * sh_hprimewgll_xx[I*NGLLX+l];
    // 2. cut-plane along eta-direction
    temp2l += s_temp2[K*NGLL2+l*NGLLX+I] * sh_hprimewgll_xx[J*NGLLX+l];
    // 3. cut-plane along gamma-direction
    temp3l += s_temp3[l*NGLL2+J*NGLLX+I] * sh_hprimewgll_xx[K*NGLLX+l];
  }

  // summed terms with added gll weights
  fac1 = wgllwgll_yz[K*NGLLX+J];
  fac2 = wgllwgll_xz[K*NGLLX+I];
  fac3 = wgllwgll_xy[J*NGLLX+I];

  sum_terms = -(fac1*temp1l + fac2*temp2l + fac3*temp3l);

  // adds gravity contribution
  if (gravity) sum_terms += gravity_term;

  // assembles potential array
  if (threadIdx.x < NGLL3) {
#ifdef USE_MESH_COLORING_GPU
  // no atomic operation needed, colors don't share global points between elements
#ifdef USE_TEXTURES_FIELDS
    if (FORWARD_OR_ADJOINT == 3){
      d_potential_dot_dot_acoustic[iglob] = texfetch_potential_dot_dot<3>(iglob) + sum_terms;
    }else{
      d_potential_dot_dot_acoustic[iglob] = texfetch_potential_dot_dot<1>(iglob) + sum_terms;
    }
#else
    d_potential_dot_dot_acoustic[iglob] += sum_terms;
#endif // USE_TEXTURES_FIELDS
#else  // MESH_COLORING
    //mesh coloring
    if (use_mesh_coloring_gpu ){
      // no atomic operation needed, colors don't share global points between elements
#ifdef USE_TEXTURES_FIELDS
      if (FORWARD_OR_ADJOINT == 3){
        d_potential_dot_dot_acoustic[iglob] = texfetch_potential_dot_dot<3>(iglob) + sum_terms;
      }else{
        d_potential_dot_dot_acoustic[iglob] = texfetch_potential_dot_dot<1>(iglob) + sum_terms;
      }
#else
      d_potential_dot_dot_acoustic[iglob] += sum_terms;
#endif // USE_TEXTURES_FIELDS
    }else{
      atomicAdd(&d_potential_dot_dot_acoustic[iglob],sum_terms);
    }
#endif // MESH_COLORING
  }
}

/* ----------------------------------------------------------------------------------------------- */

/*
// kernel useful for optimization: stripped-down version
//                                 acoustic kernel without gravity and without mesh coloring

//template<int FORWARD_OR_ADJOINT> __global__ void
#ifdef USE_LAUNCH_BOUNDS
// adds compiler specification
__launch_bounds__(NGLL3_PADDED,LAUNCH_MIN_BLOCKS_ACOUSTIC)
#endif
Kernel_2_acoustic_perf_impl(const int nb_blocks_to_compute,
                            const int* d_ibool,
                            const int* d_phase_ispec_inner_acoustic,
                            const int num_phase_ispec_acoustic,
                            const int d_iphase,
                            realw_const_p d_potential_acoustic,
                            realw_p d_potential_dot_dot_acoustic,
                            realw* d_xix,realw* d_xiy,realw* d_xiz,
                            realw* d_etax,realw* d_etay,realw* d_etaz,
                            realw* d_gammax,realw* d_gammay,realw* d_gammaz,
                            realw_const_p d_hprime_xx,
                            realw_const_p hprimewgll_xx,
                            realw_const_p wgllwgll_xy,realw_const_p wgllwgll_xz,realw_const_p wgllwgll_yz,
                            realw* d_rhostore,
                            const int use_mesh_coloring_gpu,
                            const int gravity,
                            realw_const_p minus_g,
                            realw* d_kappastore,
                            realw_const_p wgll_cube){

// note: this routine is using only 12 active blocks instead of full occupancy (16 active blocks)
//       due to small register spilling which slows down performance
//       timing: ~ 1.41 ms (Kepler: Tesla K20c)

  // block-id == number of local element id in phase_ispec array
  int bx = blockIdx.y*gridDim.x+blockIdx.x;

  // thread-id == GLL node id
  // note: use only NGLL^3 = 125 active threads, plus 3 inactive/ghost threads,
  //       because we used memory padding from NGLL^3 = 125 to 128 to get coalescent memory accesses;
  //       to avoid execution branching and the need of registers to store an active state variable,
  //       the thread ids are put in valid range
  int tx = threadIdx.x;

  int I,J,K;
  int iglob,offset;
  int working_element;

  realw temp1l,temp2l,temp3l;
  realw xixl,xiyl,xizl,etaxl,etayl,etazl,gammaxl,gammayl,gammazl,jacobianl;

  realw dpotentialdxl,dpotentialdyl,dpotentialdzl;
  realw fac1,fac2,fac3;
  realw rho_invl;

  realw sum_terms;

  __shared__ realw s_dummy_loc[NGLL3];

  __shared__ realw s_temp1[NGLL3];
  __shared__ realw s_temp2[NGLL3];
  __shared__ realw s_temp3[NGLL3];

  __shared__ realw sh_hprime_xx[NGLL2];
  __shared__ realw sh_hprimewgll_xx[NGLL2];

  // checks if anything to do
  if (bx >= nb_blocks_to_compute) return;

  // limits thread ids to range [0,125-1]
  if (tx >= NGLL3) tx = NGLL3 - 1;

  // spectral-element id
  // iphase-1 and working_element-1 for Fortran->C array conventions
  working_element = d_phase_ispec_inner_acoustic[bx + num_phase_ispec_acoustic*(d_iphase-1)] - 1;

  // local padded index
  offset = working_element*NGLL3_PADDED + tx;

  // global index
  iglob = d_ibool[offset] - 1;

  // loads potential values into shared memory
  if (threadIdx.x < NGLL3) {
    // loads potentials
#ifdef USE_TEXTURES_FIELDS
    s_dummy_loc[tx] = texfetch_potential<FORWARD_OR_ADJOINT>(iglob);
#else
    // changing iglob indexing to match fortran row changes fast style
    s_dummy_loc[tx] = d_potential_acoustic[iglob];
#endif
  }

  // local index
  K = (tx/NGLL2);
  J = ((tx-K*NGLL2)/NGLLX);
  I = (tx-K*NGLL2-J*NGLLX);

  // loads mesh values here to give compiler possibility to overlap memory fetches with some computations
  // note: arguments defined as realw* instead of const realw* __restrict__ to avoid that the compiler
  //       loads all memory by texture loads
  //       we only use the first loads explicitly by texture loads, all subsequent without. this should lead/trick
  //       the compiler to use global memory loads for all the subsequent accesses.
  //
  // calculates laplacian
  //xixl = get_global_cr( &d_xix[offset] ); // first array with texture load
  xixl = d_xix[offset];
  xiyl = d_xiy[offset]; // all subsequent without to avoid over-use of texture for coalescent access
  xizl = d_xiz[offset];
  etaxl = d_etax[offset];
  etayl = d_etay[offset];
  etazl = d_etaz[offset];
  gammaxl = d_gammax[offset];
  gammayl = d_gammay[offset];
  gammazl = d_gammaz[offset];

  jacobianl = 1.f / (xixl*(etayl*gammazl-etazl*gammayl)
                    -xiyl*(etaxl*gammazl-etazl*gammaxl)
                    +xizl*(etaxl*gammayl-etayl*gammaxl));

  // density (reciproc)
  rho_invl = 1.f / d_rhostore[offset];

  // loads hprime into shared memory
  if (tx < NGLL2) {
#ifdef USE_TEXTURES_CONSTANTS
    sh_hprime_xx[tx] = tex1Dfetch(d_hprime_xx_tex,tx);
#else
    sh_hprime_xx[tx] = d_hprime_xx[tx];
#endif
    // loads hprimewgll into shared memory
    sh_hprimewgll_xx[tx] = hprimewgll_xx[tx];
  }

  // synchronize all the threads (one thread for each of the NGLL grid points of the
  // current spectral element) because we need the whole element to be ready in order
  // to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  // computes first matrix product
  temp1l = 0.f;
  temp2l = 0.f;
  temp3l = 0.f;

  for (int l=0;l<NGLLX;l++) {
    //assumes that hprime_xx = hprime_yy = hprime_zz
    // 1. cut-plane along xi-direction
    temp1l += s_dummy_loc[K*NGLL2+J*NGLLX+l] * sh_hprime_xx[l*NGLLX+I];
    // 2. cut-plane along eta-direction
    temp2l += s_dummy_loc[K*NGLL2+l*NGLLX+I] * sh_hprime_xx[l*NGLLX+J];
    // 3. cut-plane along gamma-direction
    temp3l += s_dummy_loc[l*NGLL2+J*NGLLX+I] * sh_hprime_xx[l*NGLLX+K];
  }

  // compute derivatives of ux, uy and uz with respect to x, y and z
  // derivatives of potential
  dpotentialdxl = xixl*temp1l + etaxl*temp2l + gammaxl*temp3l;
  dpotentialdyl = xiyl*temp1l + etayl*temp2l + gammayl*temp3l;
  dpotentialdzl = xizl*temp1l + etazl*temp2l + gammazl*temp3l;

  // form the dot product with the test vector
  if (threadIdx.x < NGLL3) {
    s_temp1[tx] = jacobianl * rho_invl * (dpotentialdxl*xixl + dpotentialdyl*xiyl + dpotentialdzl*xizl);
    s_temp2[tx] = jacobianl * rho_invl * (dpotentialdxl*etaxl + dpotentialdyl*etayl + dpotentialdzl*etazl);
    s_temp3[tx] = jacobianl * rho_invl * (dpotentialdxl*gammaxl + dpotentialdyl*gammayl + dpotentialdzl*gammazl);
  }

  // synchronize all the threads (one thread for each of the NGLL grid points of the
  // current spectral element) because we need the whole element to be ready in order
  // to be able to compute the matrix products along cut planes of the 3D element below
  __syncthreads();

  // computes second matrix product
  temp1l = 0.f;
  temp2l = 0.f;
  temp3l = 0.f;

  for (int l=0;l<NGLLX;l++) {
    //assumes hprimewgll_xx = hprimewgll_yy = hprimewgll_zz
    // 1. cut-plane along xi-direction
    temp1l += s_temp1[K*NGLL2+J*NGLLX+l] * sh_hprimewgll_xx[I*NGLLX+l];
    // 2. cut-plane along eta-direction
    temp2l += s_temp2[K*NGLL2+l*NGLLX+I] * sh_hprimewgll_xx[J*NGLLX+l];
    // 3. cut-plane along gamma-direction
    temp3l += s_temp3[l*NGLL2+J*NGLLX+I] * sh_hprimewgll_xx[K*NGLLX+l];
  }

  // summed terms with added gll weights
  fac1 = wgllwgll_yz[K*NGLLX+J];
  fac2 = wgllwgll_xz[K*NGLLX+I];
  fac3 = wgllwgll_xy[J*NGLLX+I];

  sum_terms = -(fac1*temp1l + fac2*temp2l + fac3*temp3l);

  // assembles potential array
  if (threadIdx.x < NGLL3) {
      atomicAdd(&d_potential_dot_dot_acoustic[iglob],sum_terms);
  }
}
*/


