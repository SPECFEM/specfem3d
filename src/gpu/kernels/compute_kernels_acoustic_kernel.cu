#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!                          S p e c f e m 3 D
!                          -----------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/

// includes device function compute_gradient_kernel()
#include "compute_gradient_kernel.h"


__global__ void compute_kernels_acoustic_kernel(int* ispec_is_acoustic,
                                                int* d_ibool,
                                                realw* rhostore,
                                                realw* d_hprime_xx,
                                                int* d_irregular_element_number,
                                                realw* d_xix,realw* d_xiy,realw* d_xiz,
                                                realw* d_etax,realw* d_etay,realw* d_etaz,
                                                realw* d_gammax,realw* d_gammay,realw* d_gammaz,
                                                realw xix_regular,
                                                field* potential_acoustic,
                                                field* potential_dot_dot_acoustic,
                                                field* b_potential_acoustic,
                                                field* b_potential_dot_dot_acoustic,
                                                realw* rho_ac_kl,
                                                realw* kappa_ac_kl,
                                                realw deltat,
                                                int NSPEC_AB,
                                                int gravity) {

  int ispec = blockIdx.x + blockIdx.y*gridDim.x;
  int ijk = threadIdx.x;

  // local and global indices
  int iglob;

  int ijk_ispec = ijk + NGLL3*ispec;
  int ijk_ispec_padded = ijk + NGLL3_PADDED*ispec;

  int ispec_irreg = d_irregular_element_number[ispec] - 1;

  // shared memory between all threads within this block
  __shared__ field scalar_field_displ[NGLL3];
  __shared__ field scalar_field_accel[NGLL3];

  int active = 0;

  // handles case when there is 1 extra block (due to rectangular grid)
  if (ispec < NSPEC_AB ){
    // acoustic elements only
    if (ispec_is_acoustic[ispec] ){
      active = 1;

      // copy field values
      iglob = d_ibool[ijk_ispec_padded] - 1;
      scalar_field_displ[ijk] = b_potential_acoustic[iglob];
      scalar_field_accel[ijk] = potential_acoustic[iglob];
    }
  }

  // synchronizes threads
  __syncthreads();

  if (active ){
    field accel_loc[3];
    field b_displ_loc[3];
    realw rhol;

    // gets material parameter
    rhol = rhostore[ijk_ispec_padded];

    // displacement vector from backward field
    compute_gradient_kernel(ijk,ispec,ispec_irreg,scalar_field_displ,b_displ_loc,
                            d_hprime_xx,
                            d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                            rhol,xix_regular,gravity);

    // acceleration vector
    compute_gradient_kernel(ijk,ispec,ispec_irreg,scalar_field_accel,accel_loc,
                            d_hprime_xx,
                            d_xix,d_xiy,d_xiz,d_etax,d_etay,d_etaz,d_gammax,d_gammay,d_gammaz,
                            rhol,xix_regular,gravity);

    // note: the gradients grad(..) above will have a 1/rho factor added in routine compute_gradient_kernel()
    //       (as in compute_gradient_in_acoustic() for CPU)
    //
    //       that is, b_displ_elm = 1/rho grad(phi)
    //       and      accel_elm   = 1/rho grad(phi^adj)
    //
    //       however, here we want to compute the contributions for the absolute kernel
    //         K_rho   = - int_0^T [ grad(phi^adj) * grad(phi) ] dt        (A-28)
    //       and thus we multiply the b_displ_elm and accel_elm by rho again to have the time step contribution
    //         b_displ_elem * accel_elm = {rho [1/rho grad(phi)]} * {rho [1/rho grad(phi^adj)]}
    //                                  = grad(phi) * grad(phi^adj) and times dt
    //
    // we could also just set rhol above to 1.0 and call compute_gradient_kernel(), but prefer here to make this more explicit.
    // in case the gradient routine changes, revisit this...
    accel_loc[0] = rhol * accel_loc[0];
    accel_loc[1] = rhol * accel_loc[1];
    accel_loc[2] = rhol * accel_loc[2];

    b_displ_loc[0] = rhol * b_displ_loc[0];
    b_displ_loc[1] = rhol * b_displ_loc[1];
    b_displ_loc[2] = rhol * b_displ_loc[2];

    // the sum function is here to enable summing over wavefields when NB_RUNS_ACOUSTIC_GPU > 1

    // density kernel
    // (multiplication with rho(x) factor will be done after the time loop)
    rho_ac_kl[ijk_ispec] += deltat * sum(accel_loc[0]*b_displ_loc[0] + accel_loc[1]*b_displ_loc[1] + accel_loc[2]*b_displ_loc[2]);

    // bulk modulus kernel
    // (multiplication with 1/kappa(x) factor will be done after the time loop)
    kappa_ac_kl[ijk_ispec] += deltat * sum(potential_acoustic[iglob] * b_potential_dot_dot_acoustic[iglob]);
  } // active
}


