#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!                          S p e c f e m 3 D
!                          -----------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/


__global__ void compute_add_sources_kernel(realw* accel,
                                           int* d_ibool,
                                           realw* sourcearrays,
                                           field* stf_pre_compute,
                                           int myrank,
                                           int* islice_selected_source,
                                           int* ispec_selected_source,
                                           int* ispec_is_elastic,
                                           int NSOURCES) {
  int i = threadIdx.x;
  int j = threadIdx.y;
  int k = threadIdx.z;

  int isource  = blockIdx.x + gridDim.x*blockIdx.y; // bx

  int ispec,iglob;
  field stf;

  if (isource < NSOURCES) { // when NSOURCES > 65535, but mod(nspec_top,2) > 0, we end up with an extra block.

    if (myrank == islice_selected_source[isource]) {

      ispec = ispec_selected_source[isource]-1;

      if (ispec_is_elastic[ispec]) {

        stf = stf_pre_compute[isource];
        iglob = d_ibool[INDEX4_PADDED(NGLLX,NGLLX,NGLLX,i,j,k,ispec)]-1;

        atomicAdd(&accel[iglob*3+0],sourcearrays[INDEX5(NSOURCES,NDIM,NGLLX,NGLLX,isource,0,i,j,k)]*stf);
        atomicAdd(&accel[iglob*3+1],sourcearrays[INDEX5(NSOURCES,NDIM,NGLLX,NGLLX,isource,1,i,j,k)]*stf);
        atomicAdd(&accel[iglob*3+2],sourcearrays[INDEX5(NSOURCES,NDIM,NGLLX,NGLLX,isource,2,i,j,k)]*stf);
      }
    }
  }

}


