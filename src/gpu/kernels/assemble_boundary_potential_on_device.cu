#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!                          S p e c f e m 3 D
!                          -----------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/


__global__ void assemble_boundary_potential_on_device(field* d_potential_dot_dot_acoustic,
                                                      field* d_send_potential_dot_dot_buffer,
                                                      const int num_interfaces_ext_mesh,
                                                      const int max_nibool_interfaces_ext_mesh,
                                                      const int* d_nibool_interfaces_ext_mesh,
                                                      const int* d_ibool_interfaces_ext_mesh) {

  int id = threadIdx.x + (blockIdx.x + blockIdx.y*gridDim.x)*blockDim.x;
  int ientry,iglob;

  for( int iinterface=0; iinterface < num_interfaces_ext_mesh; iinterface++) {
    if (id<d_nibool_interfaces_ext_mesh[iinterface]) {

      // entry in interface array
      ientry = id + max_nibool_interfaces_ext_mesh*iinterface;
      // global index in wavefield
      iglob = d_ibool_interfaces_ext_mesh[ientry] - 1;

      // for testing atomic operations against not atomic operations (0.1ms vs. 0.04 ms)
      // d_potential_dot_dot_acoustic[3*(d_ibool_interfaces_ext_mesh[id+max_nibool_interfaces_ext_mesh*iinterface]-1)] +=
      // d_send_potential_dot_dot_buffer[3*(id + max_nibool_interfaces_ext_mesh*iinterface)];

      atomicAdd(&d_potential_dot_dot_acoustic[iglob],d_send_potential_dot_dot_buffer[ientry]);
    }
  }
  // ! This step is done via previous function transfer_and_assemble...
  // ! do iinterface = 1, num_interfaces_ext_mesh
  // !   do ipoin = 1, nibool_interfaces_ext_mesh(iinterface)
  // !     array_val(:,ibool_interfaces_ext_mesh(ipoin,iinterface)) = &
  // !          array_val(:,ibool_interfaces_ext_mesh(ipoin,iinterface)) + buffer_recv_vector_ext_mesh(:,ipoin,iinterface)
  // !   enddo
  // ! enddo
}

