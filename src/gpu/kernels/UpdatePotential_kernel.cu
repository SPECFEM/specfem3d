#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!                          S p e c f e m 3 D
!                          -----------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/


__global__ void UpdatePotential_kernel(field* potential_acoustic,
                                       field* potential_dot_acoustic,
                                       field* potential_dot_dot_acoustic,
                                       int size,
                                       realw deltat,
                                       realw deltatsqover2,
                                       realw deltatover2) {

  int id = threadIdx.x + (blockIdx.x + blockIdx.y*gridDim.x)*blockDim.x;

  // because of block and grid sizing problems, there is a small
  // amount of buffer at the end of the calculation
  if (id < size) {
    field p_dot = potential_dot_acoustic[id];
    field p_dot_dot = potential_dot_dot_acoustic[id];

    potential_acoustic[id] += deltat*p_dot + deltatsqover2*p_dot_dot;

    potential_dot_acoustic[id] = p_dot + deltatover2*p_dot_dot;

    potential_dot_dot_acoustic[id] = Make_field(0.f);
  }

// -----------------
// total of: 6 FLOP per thread (without id calculation)
//
//           8 * 4 BYTE = 32 DRAM accesses per thread
//
// arithmetic intensity: 6 FLOP / 32 BYTES ~ 0.19 FLOP/BYTE
// -----------------
//
// nvprof: nvprof --metrics flops_sp ./xspecfem3D
//          -> 8199750 FLOPS (Single) floating-point operations for 1366625 threads
//                                    1366625 (NGLOB) -> 10677 * 128 active threads- 31 ghost threads
//          -> 6 FLOP per thread


}

