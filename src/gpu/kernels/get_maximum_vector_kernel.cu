#include "hip/hip_runtime.h"
/*
!=====================================================================
!
!               S p e c f e m 3 D  V e r s i o n  3 . 0
!               ---------------------------------------
!
!    Main historical authors: Dimitri Komatitsch and Jeroen Tromp
!                             CNRS, France
!                      and Princeton University, USA
!                (there are currently many more authors!)
!                          (c) October 2017
!
! This program is free software; you can redistribute it and/or modify
! it under the terms of the GNU General Public License as published by
! the Free Software Foundation; either version 3 of the License, or
! (at your option) any later version.
!
! This program is distributed in the hope that it will be useful,
! but WITHOUT ANY WARRANTY; without even the implied warranty of
! MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
! GNU General Public License for more details.
!
! You should have received a copy of the GNU General Public License along
! with this program; if not, write to the Free Software Foundation, Inc.,
! 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA.
!
!=====================================================================
*/


__global__ void get_maximum_vector_kernel(realw* array, int size, realw* d_max){

  // reduction example:
  __shared__ realw sdata[BLOCKSIZE_TRANSFER] ;

  // load shared mem
  unsigned int tid = threadIdx.x;
  unsigned int bx = blockIdx.y*gridDim.x+blockIdx.x;
  //unsigned int i = tid + bx*blockDim.x;
  unsigned int i = threadIdx.x + (blockIdx.x + blockIdx.y*gridDim.x)*blockDim.x;

  // loads values into shared memory: assume array is a vector array
  sdata[tid] = (i < size) ? (array[i*3]*array[i*3] + array[i*3+1]*array[i*3+1] + array[i*3+2]*array[i*3+2]) : 0.0 ;

  __syncthreads();

  // do reduction in shared mem
  for(unsigned int s=blockDim.x/2; s>0; s>>=1)
  {
    if (tid < s){
      // summation:
      //sdata[tid] += sdata[tid + s];
      // maximum:
      if (sdata[tid] < sdata[tid + s]) sdata[tid] = sdata[tid + s];
    }
    __syncthreads();
  }

  // write result for this block to global mem
  if (tid == 0) d_max[bx] = sdata[0];

}



