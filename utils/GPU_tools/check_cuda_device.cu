/*
**************************

check_cuda_device utility

**************************

this utility program will output GPU device information


for compilation, see the command-line examples given here:

- example without MPI support:

nvcc --gpu-architecture=sm_60 -o check_cuda_device check_cuda_device.cu
./check_cuda_device

- example with MPI support:

nvcc -arch=sm_60 -DWITH_MPI -I/usr/lib/openmpi/include -o check_cuda_device check_cuda_device.cu -lmpi -L/usr/lib/openmpi/lib
mpirun -np 2 ./check_cuda_device


*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>

#ifdef WITH_MPI
#include <mpi.h>
#endif

#include <sys/time.h>
#include <sys/resource.h>

/* ----------------------------------------------------------------------------------------------- */

void get_free_memory(double* free_db, double* used_db, double* total_db) {

  // gets memory usage in byte
  size_t free_byte ;
  size_t total_byte ;
  hipError_t cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;
  if ( hipSuccess != cuda_status ){
    printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );
    exit(EXIT_FAILURE);
  }

  *free_db = (double)free_byte ;
  *total_db = (double)total_byte ;
  *used_db = *total_db - *free_db ;
  return;
}

/* ----------------------------------------------------------------------------------------------- */

void exit_on_error(const char* info) {
  printf("\nERROR: %s\n",info);
  fflush(stdout);

  // stops program
#ifdef WITH_MPI
  MPI_Abort(MPI_COMM_WORLD,1);
#endif
  //free(info);
  exit(EXIT_FAILURE);
  return;
}

/* ----------------------------------------------------------------------------------------------- */

void exit_on_gpu_error(const char* kernel_name) {
  // sync and check to catch errors from previous async operations
#if CUDA_VERSION < 4000 || (defined (__CUDACC_VER_MAJOR__) && (__CUDACC_VER_MAJOR__ < 4))
  hipDeviceSynchronize();
#else
  hipDeviceSynchronize();
#endif

  hipError_t err = hipGetLastError();
  if (err != hipSuccess){
    printf("Error after %s: %s\n", kernel_name, hipGetErrorString(err));

    // releases previous contexts
#if CUDA_VERSION < 4000 || (defined (__CUDACC_VER_MAJOR__) && (__CUDACC_VER_MAJOR__ < 4))
    hipDeviceReset();
#else
    hipDeviceReset();
#endif

    // stops program
    //free(kernel_name);
#ifdef WITH_MPI
    MPI_Abort(MPI_COMM_WORLD,1);
#endif
    exit(EXIT_FAILURE);
  }
}



/* ----------------------------------------------------------------------------------------------- */

// GPU initialization

/* ----------------------------------------------------------------------------------------------- */


void initialize_cuda_device(int* myrank_f,int* ncuda_devices) {

  int device;
  int device_count;
  hipError_t err;

  // Gets rank number of MPI process
  int myrank = *myrank_f;

#ifdef WITH_MPI
  int sizeprocs;
  MPI_Comm_size(MPI_COMM_WORLD,&sizeprocs);
  MPI_Barrier(MPI_COMM_WORLD);
#endif

  /*
  // cuda initialization (needs -lcuda library)
  // note:   hipInit initializes the driver API.
  //             it is needed for any following CUDA driver API function call (format cuFUNCTION(..) )
  //             however, for the CUDA runtime API functions (format cudaFUNCTION(..) )
  //             the initialization is implicit, thus hipInit() here would not be needed...
  hipError_t status = hipInit(0);
  if ( hipSuccess != status ) exit_on_error("CUDA driver API device initialization failed\n");

  // returns a handle to the first cuda compute device
  hipDevice_t dev;
  status = hipDeviceGet(&dev, 0);
  if ( hipSuccess != status ) exit_on_error("CUDA device not found\n");

  // gets device properties
  int major,minor;
  status = hipDeviceComputeCapability(&major,&minor,dev);
  if ( hipSuccess != status ) exit_on_error("CUDA device information not found\n");

  // make sure that the device has compute capability >= 1.3
  if (major < 1){
    fprintf(stderr,"Compute capability major number should be at least 1, got: %d \nexiting...\n",major);
    exit_on_error("CUDA Compute capability major number should be at least 1\n");
  }
  if (major == 1 && minor < 3){
    fprintf(stderr,"Compute capability should be at least 1.3, got: %d.%d \nexiting...\n",major,minor);
    exit_on_error("CUDA Compute capability major number should be at least 1.3\n");
  }
   */

  // note: from here on we use the runtime API  ...

  // Gets number of GPU devices
  device_count = 0;
  hipGetDeviceCount(&device_count);
  // Do not check if command failed with `exit_on_gpu_error` since it calls cudaDevice()/ThreadSynchronize():
  // If multiple MPI tasks access multiple GPUs per node, they will try to synchronize
  // GPU 0 and depending on the order of the calls, an error will be raised
  // when setting the device number. If MPS is enabled, some GPUs will silently not be used.
  //
  // being verbose and catches error from first call to CUDA runtime function, without synchronize call
  err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr,"Error after hipGetDeviceCount: %s\n", hipGetErrorString(err));
    exit_on_error("\
CUDA runtime error: hipGetDeviceCount failed\n\n\
please check if driver and runtime libraries work together\n\n");
  }

  // returns device count to fortran
  if (device_count == 0) {
    exit_on_error("CUDA runtime error: there is no device supporting CUDA\n");
  }

  // output
#ifdef WITH_MPI
  // output infos for mpi ranks ordered
  for(int iproc = 0;iproc < sizeprocs; iproc++){
    if (iproc == myrank){
      printf("process %d found number of CUDA devices = %d\n",myrank,device_count);
      fflush(stdout);
    }
    // synchronizes mpi processes
    MPI_Barrier(MPI_COMM_WORLD);
  }
  sleep(1);
  if (myrank == 0){printf("\n\n");fflush(stdout);}
  // synchronizes mpi processes
  MPI_Barrier(MPI_COMM_WORLD);
#else
  printf("\nfound number of CUDA devices = %d\n",device_count);fflush(stdout);
#endif

  *ncuda_devices = device_count;

  // user error info
  const char* err_info = "\
Please check GPU settings on your node \n\
and/or check CUDA MPS setup to use a single GPU with multiple MPI processes,\n\
e.g., on titan enable environment CRAY_CUDA_MPS=1 to use a single GPU with multiple MPI processes\n\n";

  // Sets the active device
  // generalized for more GPUs per node
  // note: without previous context release, hipSetDevice will complain with the cuda error
  //         "setting the device when a process is active is not allowed"

  // releases previous contexts
#if CUDA_VERSION < 4000 || (defined (__CUDACC_VER_MAJOR__) && (__CUDACC_VER_MAJOR__ < 4))
  hipDeviceReset();
#else
  hipDeviceReset();
#endif

#ifdef WITH_MPI
  // check
  if (hipGetLastError() != hipSuccess){
    fprintf(stderr,"Error: %s\n", hipGetErrorString(hipGetLastError()));
    exit_on_error("CUDA runtime error: hipDeviceReset failed\n\n");
  }
  // synchronizes mpi processes
  MPI_Barrier(MPI_COMM_WORLD);
  // output infos for mpi ranks ordered
  for(int iproc = 0;iproc < sizeprocs; iproc++){
    if (iproc == myrank){
      printf("process %d: cuda device count = %d (would select device = %d)\n",myrank,device_count,myrank % device_count);
      fflush(stdout);
    }
    // synchronizes mpi processes
    MPI_Barrier(MPI_COMM_WORLD);
  }
  sleep(1);
  if (myrank == 0){printf("\n\n");fflush(stdout);}
  // synchronizes mpi processes
  MPI_Barrier(MPI_COMM_WORLD);
#endif

  // loops over all devices for displaying info
  for(int i=0;i < device_count; i++){

#ifdef WITH_MPI
    // output info
    MPI_Barrier(MPI_COMM_WORLD);
    if (myrank == 0){printf("cuda set device %d\n\n",i);fflush(stdout);}
    for(int iproc = 0;iproc < sizeprocs; iproc++){
      if (iproc == myrank){
        printf("process %d: hipSetDevice %d\n",myrank,i);fflush(stdout);
#endif

    // sets active device
    //device = myrank % device_count;
    device = i;
    err = hipSetDevice( device );
    if (err != hipSuccess) {
      fprintf(stderr,"Error hipSetDevice: %s\n", hipGetErrorString(err));
      if (err == cudaErrorDevicesUnavailable){ fprintf(stderr,"\n%s\n", err_info); }
      exit_on_error("CUDA runtime error: hipSetDevice failed\n\n");
    }

    // checks device execution
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
      fprintf(stderr,"Error hipDeviceSynchronize: %s\n", hipGetErrorString(err));
      exit_on_error("CUDA runtime error: hipDeviceSynchronize failed\n\n");
    }

#ifdef WITH_MPI
      } // if
      MPI_Barrier(MPI_COMM_WORLD);
    }//for
    // double check if CUDA context gets created on multiple processes
    MPI_Barrier(MPI_COMM_WORLD);
    for(int iproc = 0;iproc < sizeprocs; iproc++){
      if (iproc == myrank){
        printf("process %d: create context\n",myrank);
        fflush(stdout);
        // creates context
        err = hipFree(0);
        if (err != hipSuccess) {
          printf("Error hipFree: %s\n", hipGetErrorString(err));
          exit_on_error("CUDA runtime error: hipFree for context failed\n\n");
        }
      }
      // synchronizes mpi processes
      MPI_Barrier(MPI_COMM_WORLD);
    }
    sleep(1);
    if (myrank == 0){printf("\n\n");fflush(stdout);}
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    // double check
    exit_on_gpu_error("hipSetDevice has invalid device");

#ifdef WITH_MPI
    // check
    MPI_Barrier(MPI_COMM_WORLD);
    if (myrank == 0){printf("cuda get device %d\n\n",i);fflush(stdout);}
#endif

    // double check that device was  properly selected
    hipGetDevice(&device);

    err = hipGetLastError();
    if (err != hipSuccess) {
      printf("Error hipGetDevice: %s\n", hipGetErrorString(err));
      if (err == cudaErrorDevicesUnavailable){ printf("\n%s\n", err_info); }
      exit_on_error("CUDA runtime error: hipGetDevice failed\n\n");
    }

#ifdef WITH_MPI
    // output infos for mpi ranks ordered
    MPI_Barrier(MPI_COMM_WORLD);
    for(int iproc = 0;iproc < sizeprocs; iproc++){
      if (iproc == myrank){
        printf("device set/get: rank %d set %d get %d\n - return %s\n",myrank,i,device,hipGetErrorString(err));
        fflush(stdout);
      }
      // synchronizes mpi processes
      MPI_Barrier(MPI_COMM_WORLD);
    }
    sleep(1);
    if (myrank == 0){printf("\n\n");fflush(stdout);}
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    // checks device id
    //if( device != (myrank % device_count) ){
    if( device != i ){
       printf("error rank: %d devices: %d \n",myrank,device_count);
       printf("  hipSetDevice()=%d\n  hipGetDevice()=%d\n",myrank%device_count,device);
       exit_on_error("CUDA set/get device error: device id conflict \n");
    }

    // get device properties
    struct hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,device);

    exit_on_gpu_error("hipGetDeviceProperties failed");

    // exit if the machine has no CUDA-enabled device
    if (deviceProp.major == 9999 && deviceProp.minor == 9999){
      printf("No CUDA-enabled device found, exiting...\n\n");
      exit_on_error("CUDA runtime error: there is no CUDA-enabled device found\n");
    }

    // memory infos via hipMemGetInfo()
    double free_db,used_db,total_db;
    get_free_memory(&free_db,&used_db,&total_db);

    // ordering mpi output
#ifdef WITH_MPI
    // synchronizes mpi processes
    MPI_Barrier(MPI_COMM_WORLD);
    // output infos for mpi ranks ordered
    for(int iproc = 0;iproc < sizeprocs; iproc++){
      if (iproc == myrank){
        //printf("\n\nGPU device for rank: %d - total procs: %d\n\n",myrank,sizeprocs);
#endif

    // outputs device infos to file
    printf("\n\nGPU device id: %d\n\n",i);

    // display device properties
    printf("Device Name = %s\n\n",deviceProp.name);
    printf("memory:\n");
    printf("  totalGlobalMem (in MB, dividing by powers of 1024): %f\n",(unsigned long) deviceProp.totalGlobalMem / (1024.f * 1024.f));
    printf("  totalGlobalMem (in GB, dividing by powers of 1024): %f\n\n",(unsigned long) deviceProp.totalGlobalMem / (1024.f * 1024.f * 1024.f));
    printf("  totalGlobalMem (in MB, dividing by powers of 1000): %f\n",(unsigned long) deviceProp.totalGlobalMem / (1000.f * 1000.f));
    printf("  totalGlobalMem (in GB, dividing by powers of 1000): %f\n\n",(unsigned long) deviceProp.totalGlobalMem / (1000.f * 1000.f * 1000.f));
    printf("  sharedMemPerBlock (in bytes): %lu\n\n",(unsigned long) deviceProp.sharedMemPerBlock);
    printf("blocks:\n");
    printf("  Maximum number of registers per block: %d\n",deviceProp.regsPerBlock);
    printf("  Maximum number of threads per block: %d\n",deviceProp.maxThreadsPerBlock);
    printf("  Maximum size of each dimension of a block: %d x %d x %d\n",
            deviceProp.maxThreadsDim[0],deviceProp.maxThreadsDim[1],deviceProp.maxThreadsDim[2]);
    printf("  Maximum sizes of each dimension of a grid: %d x %d x %d\n\n",
            deviceProp.maxGridSize[0],deviceProp.maxGridSize[1],deviceProp.maxGridSize[2]);
    printf("features:\n");
    printf("  Compute capability of the device = %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("  multiProcessorCount: %d\n",deviceProp.multiProcessorCount);
    if(deviceProp.canMapHostMemory){
      printf("  canMapHostMemory: TRUE\n");
    }else{
      printf("  canMapHostMemory: FALSE\n");
    }
    if(deviceProp.deviceOverlap){
      printf("  deviceOverlap: TRUE\n");
    }else{
      printf("  deviceOverlap: FALSE\n");
    }
    printf("  Compute Mode: %d\n", deviceProp.computeMode);
    fflush(stdout);


    // outputs initial memory infos via hipMemGetInfo()
    printf("\n%d: GPU memory usage (dividing by powers of 1024): used = %f MB, free = %f MB, total = %f MB",myrank,
            used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
    printf("\n%d: GPU memory usage (dividing by powers of 1000): used = %f MB, free = %f MB, total = %f MB\n\n",myrank,
            used_db/1000.0/1000.0, free_db/1000.0/1000.0, total_db/1000.0/1000.0);

    // ordering mpi output
#ifdef WITH_MPI
      } //if
      // synchronizes mpi processes
      MPI_Barrier(MPI_COMM_WORLD);
    } //for
#endif

    // make sure that the device has compute capability >= 1.3
    if (deviceProp.major < 1){
      printf("Compute capability major number should be at least 1, exiting...\n\n");
      exit_on_error("CUDA Compute capability major number should be at least 1\n");
    }
    if (deviceProp.major == 1 && deviceProp.minor < 3){
      printf("Compute capability should be at least 1.3, exiting...\n");
      exit_on_error("CUDA Compute capability major number should be at least 1.3\n");
    }
    // we use pinned memory for asynchronous copy
    if( ! deviceProp.canMapHostMemory){
      printf("Device capability should allow to map host memory, exiting...\n");
      exit_on_error("CUDA Device capability canMapHostMemory should be TRUE\n");
    }

    // tests the device with a small memory allocation
    int size = 128;
    int* d_array;
    err = hipMalloc((void**)&d_array,size*sizeof(int));
    if (err != hipSuccess) {
      printf("Error testing memory allocation on device failed\n");
      printf("Error rank %d: hipMalloc failed: %s\n", myrank,hipGetErrorString(err));
      if (err == cudaErrorDevicesUnavailable){ printf("\n%s\n", err_info); }
      exit_on_error("CUDA runtime error: hipMalloc failed\n\n");
    }
    err = hipFree(d_array);
    if (err != hipSuccess) {
      printf("Error hipFree failed: %s\n", hipGetErrorString(err));
      if (err == cudaErrorDevicesUnavailable){ printf("\n%s\n", err_info); }
      exit_on_error("CUDA runtime error: hipFree failed\n\n");
    }

    // double check
    exit_on_gpu_error("cuda Malloc/Free test failed");

    // synchronizes GPU
#if CUDA_VERSION < 4000 || (defined (__CUDACC_VER_MAJOR__) && (__CUDACC_VER_MAJOR__ < 4))
    hipDeviceSynchronize();
#else
    hipDeviceSynchronize();
#endif

    // synchronizes mpi processes
#ifdef WITH_MPI
    // output infos for mpi ranks ordered
    MPI_Barrier(MPI_COMM_WORLD);
    for(int iproc = 0;iproc < sizeprocs; iproc++){
      if (iproc == myrank){
        printf("rank %d on device %d okay\n",myrank,device);
        fflush(stdout);
      }
      // synchronizes mpi processes
      MPI_Barrier(MPI_COMM_WORLD);
    }
    sleep(1);
    if (myrank == 0){printf("\n\n");fflush(stdout);}
    MPI_Barrier(MPI_COMM_WORLD);
#endif
  } // loop device_count

}


int main(int argc, char **argv)
{
  int myrank,ndevices;

  // initialize
#ifdef WITH_MPI
  int size;
  int rc;
  rc = MPI_Init(&argc,&argv);
  if (rc != MPI_SUCCESS) {
    printf ("Error starting MPI program. Terminating.\n");
    MPI_Abort(MPI_COMM_WORLD, rc);
  }
  MPI_Comm_size(MPI_COMM_WORLD,&size);
  MPI_Comm_rank(MPI_COMM_WORLD, &myrank);
  if( myrank == 0 ){ printf ("Number of MPI processes = %d\n\n",size);fflush(stdout); }
  MPI_Barrier(MPI_COMM_WORLD);
#else
  myrank = 0;
#endif

  ndevices = 0;

  // initializes cuda devices
  initialize_cuda_device(&myrank,&ndevices);

  // releases previous contexts
#if CUDA_VERSION < 4000 || (defined (__CUDACC_VER_MAJOR__) && (__CUDACC_VER_MAJOR__ < 4))
  hipDeviceReset();
#else
  hipDeviceReset();
#endif

  printf("number of total devices: %d\n\n",ndevices);

#ifdef WITH_MPI
  MPI_Finalize();
#endif
  return 0;
}

